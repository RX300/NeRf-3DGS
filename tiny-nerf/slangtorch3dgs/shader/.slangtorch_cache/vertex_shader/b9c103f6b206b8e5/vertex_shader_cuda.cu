#define SLANG_PRELUDE_EXPORT

#ifdef __CUDACC_RTC__
#define SLANG_CUDA_RTC 1
#else
#define SLANG_CUDA_RTC 0
#endif

#if SLANG_CUDA_RTC

#else


#include <hip/hip_runtime.h>
#include <cstdint>
#include <stdio.h>

#endif

// Define SLANG_CUDA_ENABLE_HALF to use the cuda_fp16 include to add half support.
// For this to work NVRTC needs to have the path to the CUDA SDK.
//
// As it stands the includes paths defined for Slang are passed down to NVRTC. Similarly defines
// defined for the Slang compile are passed down.

#ifdef SLANG_CUDA_ENABLE_HALF
// We don't want half2 operators, because it will implement comparison operators that return a
// bool(!). We want to generate those functions. Doing so means that we will have to define all
// the other half2 operators.
#define __CUDA_NO_HALF2_OPERATORS__
#include <cuda_fp16.h>
#endif

#ifdef SLANG_CUDA_ENABLE_OPTIX
#include <optix.h>
#endif

// Define slang offsetof implementation
#ifndef SLANG_OFFSET_OF
#define SLANG_OFFSET_OF(type, member) (size_t)((char*)&(((type*)0)->member) - (char*)0)
#endif

#ifndef SLANG_ALIGN_OF
#define SLANG_ALIGN_OF(type) __alignof__(type)
#endif

// Must be large enough to cause overflow and therefore infinity
#ifndef SLANG_INFINITY
#define SLANG_INFINITY ((float)(1e+300 * 1e+300))
#endif

// For now we'll disable any asserts in this prelude
#define SLANG_PRELUDE_ASSERT(x)

#ifndef SLANG_CUDA_WARP_SIZE
#define SLANG_CUDA_WARP_SIZE 32
#endif

#define SLANG_CUDA_WARP_MASK \
    (SLANG_CUDA_WARP_SIZE - 1) // Used for masking threadIdx.x to the warp lane index
#define SLANG_CUDA_WARP_BITMASK (~int(0))

//
#define SLANG_FORCE_INLINE inline

#define SLANG_CUDA_CALL __device__

#define SLANG_FORCE_INLINE inline
#define SLANG_INLINE inline


// Since we are using unsigned arithmatic care is need in this comparison.
// It is *assumed* that sizeInBytes >= elemSize. Which means (sizeInBytes >= elemSize) >= 0
// Which means only a single test is needed

// Asserts for bounds checking.
// It is assumed index/count are unsigned types.
#define SLANG_BOUND_ASSERT(index, count) SLANG_PRELUDE_ASSERT(index < count);
#define SLANG_BOUND_ASSERT_BYTE_ADDRESS(index, elemSize, sizeInBytes) \
    SLANG_PRELUDE_ASSERT(index <= (sizeInBytes - elemSize) && (index & 3) == 0);

// Macros to zero index if an access is out of range
#define SLANG_BOUND_ZERO_INDEX(index, count) index = (index < count) ? index : 0;
#define SLANG_BOUND_ZERO_INDEX_BYTE_ADDRESS(index, elemSize, sizeInBytes) \
    index = (index <= (sizeInBytes - elemSize)) ? index : 0;

// The 'FIX' macro define how the index is fixed. The default is to do nothing. If
// SLANG_ENABLE_BOUND_ZERO_INDEX the fix macro will zero the index, if out of range
#ifdef SLANG_ENABLE_BOUND_ZERO_INDEX
#define SLANG_BOUND_FIX(index, count) SLANG_BOUND_ZERO_INDEX(index, count)
#define SLANG_BOUND_FIX_BYTE_ADDRESS(index, elemSize, sizeInBytes) \
    SLANG_BOUND_ZERO_INDEX_BYTE_ADDRESS(index, elemSize, sizeInBytes)
#define SLANG_BOUND_FIX_FIXED_ARRAY(index, count) \
    SLANG_BOUND_ZERO_INDEX(index, count) SLANG_BOUND_ZERO_INDEX(index, count)
#else
#define SLANG_BOUND_FIX(index, count)
#define SLANG_BOUND_FIX_BYTE_ADDRESS(index, elemSize, sizeInBytes)
#define SLANG_BOUND_FIX_FIXED_ARRAY(index, count)
#endif

#ifndef SLANG_BOUND_CHECK
#define SLANG_BOUND_CHECK(index, count) \
    SLANG_BOUND_ASSERT(index, count) SLANG_BOUND_FIX(index, count)
#endif

#ifndef SLANG_BOUND_CHECK_BYTE_ADDRESS
#define SLANG_BOUND_CHECK_BYTE_ADDRESS(index, elemSize, sizeInBytes) \
    SLANG_BOUND_ASSERT_BYTE_ADDRESS(index, elemSize, sizeInBytes)    \
    SLANG_BOUND_FIX_BYTE_ADDRESS(index, elemSize, sizeInBytes)
#endif

#ifndef SLANG_BOUND_CHECK_FIXED_ARRAY
#define SLANG_BOUND_CHECK_FIXED_ARRAY(index, count) \
    SLANG_BOUND_ASSERT(index, count) SLANG_BOUND_FIX_FIXED_ARRAY(index, count)
#endif

// This macro handles how out-of-range surface coordinates are handled;
// I can equal
// cudaBoundaryModeClamp, in which case out-of-range coordinates are clamped to the valid range
// cudaBoundaryModeZero, in which case out-of-range reads return zero and out-of-range writes are
// ignored cudaBoundaryModeTrap, in which case out-of-range accesses cause the kernel execution to
// fail.

#ifndef SLANG_CUDA_BOUNDARY_MODE
#define SLANG_CUDA_BOUNDARY_MODE hipBoundaryModeZero

// Can be one of SLANG_CUDA_PTX_BOUNDARY_MODE. Only applies *PTX* emitted CUDA operations
// which currently is just RWTextureRW format writes
//
// .trap         causes an execution trap on out-of-bounds addresses
// .clamp        stores data at the nearest surface location (sized appropriately)
// .zero         drops stores to out-of-bounds addresses

#define SLANG_PTX_BOUNDARY_MODE "zero"
#endif

struct TypeInfo
{
    size_t typeSize;
};

template<typename T, size_t SIZE>
struct FixedArray
{
    SLANG_CUDA_CALL const T& operator[](size_t index) const
    {
        SLANG_BOUND_CHECK_FIXED_ARRAY(index, SIZE);
        return m_data[index];
    }
    SLANG_CUDA_CALL T& operator[](size_t index)
    {
        SLANG_BOUND_CHECK_FIXED_ARRAY(index, SIZE);
        return m_data[index];
    }

    T m_data[SIZE];
};

// An array that has no specified size, becomes a 'Array'. This stores the size so it can
// potentially do bounds checking.
template<typename T>
struct Array
{
    SLANG_CUDA_CALL const T& operator[](size_t index) const
    {
        SLANG_BOUND_CHECK(index, count);
        return data[index];
    }
    SLANG_CUDA_CALL T& operator[](size_t index)
    {
        SLANG_BOUND_CHECK(index, count);
        return data[index];
    }

    T* data;
    size_t count;
};

// Typically defined in cuda.h, but we can't ship/rely on that, so just define here
typedef unsigned long long hipTextureObject_t;
typedef unsigned long long hipSurfaceObject_t;

// On CUDA sampler state is actually bound up with the texture object. We have a SamplerState type,
// backed as a pointer, to simplify code generation, with the downside that such a binding will take
// up uniform space, even though it will have no effect.
// TODO(JS): Consider ways to strip use of variables of this type so have no binding,
struct SamplerStateUnused;
typedef SamplerStateUnused* SamplerState;


// TODO(JS): Not clear yet if this can be handled on CUDA, by just ignoring.
// For now, just map to the index type.
typedef size_t NonUniformResourceIndex;

// Code generator will generate the specific type
template<typename T, int ROWS, int COLS>
struct Matrix;

typedef int1 bool1;
typedef int2 bool2;
typedef int3 bool3;
typedef int4 bool4;

#if SLANG_CUDA_RTC

typedef signed char int8_t;
typedef short int16_t;
typedef int int32_t;
typedef long long int64_t;

typedef unsigned char uint8_t;
typedef unsigned short uint16_t;
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;

#endif

typedef long long longlong;
typedef unsigned long long ulonglong;

typedef unsigned char uchar;
typedef unsigned short ushort;
typedef unsigned int uint;

union Union32
{
    uint32_t u;
    int32_t i;
    float f;
};

union Union64
{
    uint64_t u;
    int64_t i;
    double d;
};

template<typename T>
SLANG_FORCE_INLINE SLANG_CUDA_CALL float make_float(T val)
{
    return (float)val;
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL float _slang_fmod(float x, float y)
{
    return ::fmodf(x, y);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double _slang_fmod(double x, double y)
{
    return ::fmod(x, y);
}

#if SLANG_CUDA_ENABLE_HALF

// Add the other vector half types
struct __half1
{
    __half x;
};
struct __align__(4) __half3
{
    __half x, y, z;
};
struct __align__(4) __half4
{
    __half x, y, z, w;
};
#endif

#define SLANG_VECTOR_GET_ELEMENT(T)                                                   \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T _slang_vector_get_element(T##1 x, int index) \
    {                                                                                 \
        return ((T*)(&x))[index];                                                     \
    }                                                                                 \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T _slang_vector_get_element(T##2 x, int index) \
    {                                                                                 \
        return ((T*)(&x))[index];                                                     \
    }                                                                                 \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T _slang_vector_get_element(T##3 x, int index) \
    {                                                                                 \
        return ((T*)(&x))[index];                                                     \
    }                                                                                 \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T _slang_vector_get_element(T##4 x, int index) \
    {                                                                                 \
        return ((T*)(&x))[index];                                                     \
    }
SLANG_VECTOR_GET_ELEMENT(int)
SLANG_VECTOR_GET_ELEMENT(uint)
SLANG_VECTOR_GET_ELEMENT(short)
SLANG_VECTOR_GET_ELEMENT(ushort)
SLANG_VECTOR_GET_ELEMENT(char)
SLANG_VECTOR_GET_ELEMENT(uchar)
SLANG_VECTOR_GET_ELEMENT(longlong)
SLANG_VECTOR_GET_ELEMENT(ulonglong)
SLANG_VECTOR_GET_ELEMENT(float)
SLANG_VECTOR_GET_ELEMENT(double)

#define SLANG_VECTOR_GET_ELEMENT_PTR(T)                                                      \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T* _slang_vector_get_element_ptr(T##1 * x, int index) \
    {                                                                                        \
        return ((T*)(x)) + index;                                                            \
    }                                                                                        \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T* _slang_vector_get_element_ptr(T##2 * x, int index) \
    {                                                                                        \
        return ((T*)(x)) + index;                                                            \
    }                                                                                        \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T* _slang_vector_get_element_ptr(T##3 * x, int index) \
    {                                                                                        \
        return ((T*)(x)) + index;                                                            \
    }                                                                                        \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T* _slang_vector_get_element_ptr(T##4 * x, int index) \
    {                                                                                        \
        return ((T*)(x)) + index;                                                            \
    }
SLANG_VECTOR_GET_ELEMENT_PTR(int)
SLANG_VECTOR_GET_ELEMENT_PTR(uint)
SLANG_VECTOR_GET_ELEMENT_PTR(short)
SLANG_VECTOR_GET_ELEMENT_PTR(ushort)
SLANG_VECTOR_GET_ELEMENT_PTR(char)
SLANG_VECTOR_GET_ELEMENT_PTR(uchar)
SLANG_VECTOR_GET_ELEMENT_PTR(longlong)
SLANG_VECTOR_GET_ELEMENT_PTR(ulonglong)
SLANG_VECTOR_GET_ELEMENT_PTR(float)
SLANG_VECTOR_GET_ELEMENT_PTR(double)

#if SLANG_CUDA_ENABLE_HALF
SLANG_VECTOR_GET_ELEMENT(__half)
SLANG_VECTOR_GET_ELEMENT_PTR(__half)
#endif

#define SLANG_CUDA_VECTOR_BINARY_OP(T, n, op)                                                 \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##n operator op(T##n thisVal, T##n other)             \
    {                                                                                         \
        T##n result;                                                                          \
        for (int i = 0; i < n; i++)                                                           \
            *_slang_vector_get_element_ptr(&result, i) =                                      \
                _slang_vector_get_element(thisVal, i) op _slang_vector_get_element(other, i); \
        return result;                                                                        \
    }
#define SLANG_CUDA_VECTOR_BINARY_COMPARE_OP(T, n, op)                                \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL bool##n operator op(T##n thisVal, T##n other) \
    {                                                                                \
        bool##n result;                                                              \
        for (int i = 0; i < n; i++)                                                  \
            *_slang_vector_get_element_ptr(&result, i) =                             \
                (int)(_slang_vector_get_element(thisVal, i)                          \
                          op _slang_vector_get_element(other, i));                   \
        return result;                                                               \
    }
#define SLANG_CUDA_VECTOR_UNARY_OP(T, n, op)                                                       \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##n operator op(T##n thisVal)                              \
    {                                                                                              \
        T##n result;                                                                               \
        for (int i = 0; i < n; i++)                                                                \
            *_slang_vector_get_element_ptr(&result, i) = op _slang_vector_get_element(thisVal, i); \
        return result;                                                                             \
    }

#define SLANG_CUDA_VECTOR_INT_OP(T, n)            \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, +)          \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, -)          \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, *)          \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, /)          \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, %)          \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, ^)          \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, &)          \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, |)          \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, &&)         \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, ||)         \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, >>)         \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, <<)         \
    SLANG_CUDA_VECTOR_BINARY_COMPARE_OP(T, n, >)  \
    SLANG_CUDA_VECTOR_BINARY_COMPARE_OP(T, n, <)  \
    SLANG_CUDA_VECTOR_BINARY_COMPARE_OP(T, n, >=) \
    SLANG_CUDA_VECTOR_BINARY_COMPARE_OP(T, n, <=) \
    SLANG_CUDA_VECTOR_BINARY_COMPARE_OP(T, n, ==) \
    SLANG_CUDA_VECTOR_BINARY_COMPARE_OP(T, n, !=) \
    SLANG_CUDA_VECTOR_UNARY_OP(T, n, !)           \
    SLANG_CUDA_VECTOR_UNARY_OP(T, n, -)           \
    SLANG_CUDA_VECTOR_UNARY_OP(T, n, ~)

#define SLANG_CUDA_VECTOR_INT_OPS(T) \
    SLANG_CUDA_VECTOR_INT_OP(T, 2)   \
    SLANG_CUDA_VECTOR_INT_OP(T, 3)   \
    SLANG_CUDA_VECTOR_INT_OP(T, 4)

SLANG_CUDA_VECTOR_INT_OPS(int)
SLANG_CUDA_VECTOR_INT_OPS(uint)
SLANG_CUDA_VECTOR_INT_OPS(ushort)
SLANG_CUDA_VECTOR_INT_OPS(short)
SLANG_CUDA_VECTOR_INT_OPS(char)
SLANG_CUDA_VECTOR_INT_OPS(uchar)
SLANG_CUDA_VECTOR_INT_OPS(longlong)
SLANG_CUDA_VECTOR_INT_OPS(ulonglong)

#define SLANG_CUDA_VECTOR_FLOAT_OP(T, n)          \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, +)          \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, -)          \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, *)          \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, /)          \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, &&)         \
    SLANG_CUDA_VECTOR_BINARY_OP(T, n, ||)         \
    SLANG_CUDA_VECTOR_BINARY_COMPARE_OP(T, n, >)  \
    SLANG_CUDA_VECTOR_BINARY_COMPARE_OP(T, n, <)  \
    SLANG_CUDA_VECTOR_BINARY_COMPARE_OP(T, n, >=) \
    SLANG_CUDA_VECTOR_BINARY_COMPARE_OP(T, n, <=) \
    SLANG_CUDA_VECTOR_BINARY_COMPARE_OP(T, n, ==) \
    SLANG_CUDA_VECTOR_BINARY_COMPARE_OP(T, n, !=) \
    SLANG_CUDA_VECTOR_UNARY_OP(T, n, -)
#define SLANG_CUDA_VECTOR_FLOAT_OPS(T) \
    SLANG_CUDA_VECTOR_FLOAT_OP(T, 2)   \
    SLANG_CUDA_VECTOR_FLOAT_OP(T, 3)   \
    SLANG_CUDA_VECTOR_FLOAT_OP(T, 4)

SLANG_CUDA_VECTOR_FLOAT_OPS(float)
SLANG_CUDA_VECTOR_FLOAT_OPS(double)
#if SLANG_CUDA_ENABLE_HALF
SLANG_CUDA_VECTOR_FLOAT_OPS(__half)
#endif
#define SLANG_CUDA_FLOAT_VECTOR_MOD_IMPL(T, n)                                             \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##n operator%(const T##n& left, const T##n& right) \
    {                                                                                      \
        T##n result;                                                                       \
        for (int i = 0; i < n; i++)                                                        \
            *_slang_vector_get_element_ptr(&result, i) = _slang_fmod(                      \
                _slang_vector_get_element(left, i),                                        \
                _slang_vector_get_element(right, i));                                      \
        return result;                                                                     \
    }
#define SLANG_CUDA_FLOAT_VECTOR_MOD(T)     \
    SLANG_CUDA_FLOAT_VECTOR_MOD_IMPL(T, 2) \
    SLANG_CUDA_FLOAT_VECTOR_MOD_IMPL(T, 3) \
    SLANG_CUDA_FLOAT_VECTOR_MOD_IMPL(T, 4)

SLANG_CUDA_FLOAT_VECTOR_MOD(float)
SLANG_CUDA_FLOAT_VECTOR_MOD(double)

#if SLANG_CUDA_RTC || SLANG_CUDA_ENABLE_HALF
#define SLANG_MAKE_VECTOR(T)                                                \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##2 make_##T##2(T x, T y)           \
    {                                                                       \
        return T##2 {x, y};                                                 \
    }                                                                       \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##3 make_##T##3(T x, T y, T z)      \
    {                                                                       \
        return T##3 {x, y, z};                                              \
    }                                                                       \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##4 make_##T##4(T x, T y, T z, T w) \
    {                                                                       \
        return T##4 {x, y, z, w};                                           \
    }
#endif

#if SLANG_CUDA_RTC
SLANG_MAKE_VECTOR(int)
SLANG_MAKE_VECTOR(uint)
SLANG_MAKE_VECTOR(short)
SLANG_MAKE_VECTOR(ushort)
SLANG_MAKE_VECTOR(char)
SLANG_MAKE_VECTOR(uchar)
SLANG_MAKE_VECTOR(float)
SLANG_MAKE_VECTOR(double)
SLANG_MAKE_VECTOR(longlong)
SLANG_MAKE_VECTOR(ulonglong)
#endif

#if SLANG_CUDA_ENABLE_HALF
SLANG_MAKE_VECTOR(__half)
#endif

SLANG_FORCE_INLINE SLANG_CUDA_CALL bool1 make_bool1(bool x)
{
    return bool1{x};
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL bool2 make_bool2(bool x, bool y)
{
    return bool2{x, y};
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL bool3 make_bool3(bool x, bool y, bool z)
{
    return bool3{x, y, z};
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL bool4 make_bool4(bool x, bool y, bool z, bool w)
{
    return bool4{x, y, z, w};
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL bool2 make_bool2(bool x)
{
    return bool2{x, x};
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL bool3 make_bool3(bool x)
{
    return bool3{x, x, x};
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL bool4 make_bool4(bool x)
{
    return bool4{x, x, x, x};
}

#if SLANG_CUDA_RTC
#define SLANG_MAKE_VECTOR_FROM_SCALAR(T)                     \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##1 make_##T##1(T x) \
    {                                                        \
        return T##1 {x};                                     \
    }                                                        \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##2 make_##T##2(T x) \
    {                                                        \
        return make_##T##2(x, x);                            \
    }                                                        \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##3 make_##T##3(T x) \
    {                                                        \
        return make_##T##3(x, x, x);                         \
    }                                                        \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##4 make_##T##4(T x) \
    {                                                        \
        return make_##T##4(x, x, x, x);                      \
    }
#else
#define SLANG_MAKE_VECTOR_FROM_SCALAR(T)                     \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##2 make_##T##2(T x) \
    {                                                        \
        return make_##T##2(x, x);                            \
    }                                                        \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##3 make_##T##3(T x) \
    {                                                        \
        return make_##T##3(x, x, x);                         \
    }                                                        \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##4 make_##T##4(T x) \
    {                                                        \
        return make_##T##4(x, x, x, x);                      \
    }
#endif
SLANG_MAKE_VECTOR_FROM_SCALAR(int)
SLANG_MAKE_VECTOR_FROM_SCALAR(uint)
SLANG_MAKE_VECTOR_FROM_SCALAR(short)
SLANG_MAKE_VECTOR_FROM_SCALAR(ushort)
SLANG_MAKE_VECTOR_FROM_SCALAR(char)
SLANG_MAKE_VECTOR_FROM_SCALAR(uchar)
SLANG_MAKE_VECTOR_FROM_SCALAR(longlong)
SLANG_MAKE_VECTOR_FROM_SCALAR(ulonglong)
SLANG_MAKE_VECTOR_FROM_SCALAR(float)
SLANG_MAKE_VECTOR_FROM_SCALAR(double)
#if SLANG_CUDA_ENABLE_HALF
SLANG_MAKE_VECTOR_FROM_SCALAR(__half)
#if !SLANG_CUDA_RTC
SLANG_FORCE_INLINE SLANG_CUDA_CALL __half1 make___half1(__half x)
{
    return __half1{x};
}
#endif
#endif

#define SLANG_CUDA_VECTOR_ATOMIC_BINARY_IMPL(Fn, T, N)                                            \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T##N Fn(T##N* address, T##N val)                           \
    {                                                                                             \
        T##N result;                                                                              \
        for (int i = 0; i < N; i++)                                                               \
            *_slang_vector_get_element_ptr(&result, i) =                                          \
                Fn(_slang_vector_get_element_ptr(address, i), _slang_vector_get_element(val, i)); \
        return result;                                                                            \
    }

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 900
SLANG_CUDA_VECTOR_ATOMIC_BINARY_IMPL(atomicAdd, float, 2)
SLANG_CUDA_VECTOR_ATOMIC_BINARY_IMPL(atomicAdd, float, 4)
#endif
SLANG_CUDA_VECTOR_ATOMIC_BINARY_IMPL(atomicAdd, float, 3)
SLANG_CUDA_VECTOR_ATOMIC_BINARY_IMPL(atomicAdd, int, 2)
SLANG_CUDA_VECTOR_ATOMIC_BINARY_IMPL(atomicAdd, int, 3)
SLANG_CUDA_VECTOR_ATOMIC_BINARY_IMPL(atomicAdd, int, 4)
SLANG_CUDA_VECTOR_ATOMIC_BINARY_IMPL(atomicAdd, uint, 2)
SLANG_CUDA_VECTOR_ATOMIC_BINARY_IMPL(atomicAdd, uint, 3)
SLANG_CUDA_VECTOR_ATOMIC_BINARY_IMPL(atomicAdd, uint, 4)
SLANG_CUDA_VECTOR_ATOMIC_BINARY_IMPL(atomicAdd, ulonglong, 2)
SLANG_CUDA_VECTOR_ATOMIC_BINARY_IMPL(atomicAdd, ulonglong, 3)
SLANG_CUDA_VECTOR_ATOMIC_BINARY_IMPL(atomicAdd, ulonglong, 4)

template<typename T, int n>
struct GetVectorTypeImpl
{
};

#define GET_VECTOR_TYPE_IMPL(T, n)                                     \
    template<>                                                         \
    struct GetVectorTypeImpl<T, n>                                     \
    {                                                                  \
        typedef T##n type;                                             \
        static SLANG_FORCE_INLINE SLANG_CUDA_CALL T##n fromScalar(T v) \
        {                                                              \
            return make_##T##n(v);                                     \
        }                                                              \
    };
#define GET_VECTOR_TYPE_IMPL_N(T) \
    GET_VECTOR_TYPE_IMPL(T, 1)    \
    GET_VECTOR_TYPE_IMPL(T, 2)    \
    GET_VECTOR_TYPE_IMPL(T, 3)    \
    GET_VECTOR_TYPE_IMPL(T, 4)

GET_VECTOR_TYPE_IMPL_N(int)
GET_VECTOR_TYPE_IMPL_N(uint)
GET_VECTOR_TYPE_IMPL_N(short)
GET_VECTOR_TYPE_IMPL_N(ushort)
GET_VECTOR_TYPE_IMPL_N(char)
GET_VECTOR_TYPE_IMPL_N(uchar)
GET_VECTOR_TYPE_IMPL_N(longlong)
GET_VECTOR_TYPE_IMPL_N(ulonglong)
GET_VECTOR_TYPE_IMPL_N(float)
GET_VECTOR_TYPE_IMPL_N(double)
#if SLANG_CUDA_ENABLE_HALF
GET_VECTOR_TYPE_IMPL_N(__half)
#endif
template<typename T, int n>
using Vector = typename GetVectorTypeImpl<T, n>::type;

template<typename T, int n, typename OtherT, int m>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Vector<T, n> _slang_vector_reshape(const Vector<OtherT, m> other)
{
    Vector<T, n> result;
    for (int i = 0; i < n; i++)
    {
        OtherT otherElement = T(0);
        if (i < m)
            otherElement = _slang_vector_get_element(other, i);
        *_slang_vector_get_element_ptr(&result, i) = (T)otherElement;
    }
    return result;
}

template<typename T, int ROWS, int COLS>
struct Matrix
{
    Vector<T, COLS> rows[ROWS];
    SLANG_FORCE_INLINE SLANG_CUDA_CALL Vector<T, COLS>& operator[](size_t index)
    {
        return rows[index];
    }
};


template<typename T, int ROWS, int COLS>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, ROWS, COLS> makeMatrix(T scalar)
{
    Matrix<T, ROWS, COLS> result;
    for (int i = 0; i < ROWS; i++)
        result.rows[i] = GetVectorTypeImpl<T, COLS>::fromScalar(scalar);
    return result;
}

template<typename T, int ROWS, int COLS>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, ROWS, COLS> makeMatrix(const Vector<T, COLS>& row0)
{
    Matrix<T, ROWS, COLS> result;
    result.rows[0] = row0;
    return result;
}

template<typename T, int ROWS, int COLS>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, ROWS, COLS> makeMatrix(
    const Vector<T, COLS>& row0,
    const Vector<T, COLS>& row1)
{
    Matrix<T, ROWS, COLS> result;
    result.rows[0] = row0;
    result.rows[1] = row1;
    return result;
}

template<typename T, int ROWS, int COLS>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, ROWS, COLS> makeMatrix(
    const Vector<T, COLS>& row0,
    const Vector<T, COLS>& row1,
    const Vector<T, COLS>& row2)
{
    Matrix<T, ROWS, COLS> result;
    result.rows[0] = row0;
    result.rows[1] = row1;
    result.rows[2] = row2;
    return result;
}

template<typename T, int ROWS, int COLS>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, ROWS, COLS> makeMatrix(
    const Vector<T, COLS>& row0,
    const Vector<T, COLS>& row1,
    const Vector<T, COLS>& row2,
    const Vector<T, COLS>& row3)
{
    Matrix<T, ROWS, COLS> result;
    result.rows[0] = row0;
    result.rows[1] = row1;
    result.rows[2] = row2;
    result.rows[3] = row3;
    return result;
}

template<typename T, int ROWS, int COLS, typename U, int otherRow, int otherCol>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, ROWS, COLS> makeMatrix(
    const Matrix<U, otherRow, otherCol>& other)
{
    Matrix<T, ROWS, COLS> result;
    int minRow = ROWS;
    int minCol = COLS;
    if (minRow > otherRow)
        minRow = otherRow;
    if (minCol > otherCol)
        minCol = otherCol;
    for (int i = 0; i < minRow; i++)
        for (int j = 0; j < minCol; j++)
            *_slang_vector_get_element_ptr(result.rows + i, j) =
                (T)_slang_vector_get_element(other.rows[i], j);
    return result;
}

template<typename T, int ROWS, int COLS>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, ROWS, COLS> makeMatrix(T v0, T v1, T v2, T v3)
{
    Matrix<T, ROWS, COLS> rs;
    rs.rows[0].x = v0;
    rs.rows[0].y = v1;
    rs.rows[1].x = v2;
    rs.rows[1].y = v3;
    return rs;
}

template<typename T, int ROWS, int COLS>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, ROWS, COLS> makeMatrix(
    T v0,
    T v1,
    T v2,
    T v3,
    T v4,
    T v5)
{
    Matrix<T, ROWS, COLS> rs;
    if (COLS == 3)
    {
        *_slang_vector_get_element_ptr(&rs.rows[0], 0) = v0;
        *_slang_vector_get_element_ptr(&rs.rows[0], 1) = v1;
        *_slang_vector_get_element_ptr(&rs.rows[0], 2) = v2;
        *_slang_vector_get_element_ptr(&rs.rows[1], 0) = v3;
        *_slang_vector_get_element_ptr(&rs.rows[1], 1) = v4;
        *_slang_vector_get_element_ptr(&rs.rows[1], 2) = v5;
    }
    else
    {
        rs.rows[0].x = v0;
        rs.rows[0].y = v1;
        rs.rows[1].x = v2;
        rs.rows[1].y = v3;
        rs.rows[2].x = v4;
        rs.rows[2].y = v5;
    }
    return rs;
}

template<typename T, int ROWS, int COLS>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, ROWS, COLS> makeMatrix(
    T v0,
    T v1,
    T v2,
    T v3,
    T v4,
    T v5,
    T v6,
    T v7)
{
    Matrix<T, ROWS, COLS> rs;
    if (COLS == 4)
    {
        *_slang_vector_get_element_ptr(&rs.rows[0], 0) = v0;
        *_slang_vector_get_element_ptr(&rs.rows[0], 1) = v1;
        *_slang_vector_get_element_ptr(&rs.rows[0], 2) = v2;
        *_slang_vector_get_element_ptr(&rs.rows[0], 3) = v3;
        *_slang_vector_get_element_ptr(&rs.rows[1], 0) = v4;
        *_slang_vector_get_element_ptr(&rs.rows[1], 1) = v5;
        *_slang_vector_get_element_ptr(&rs.rows[1], 2) = v6;
        *_slang_vector_get_element_ptr(&rs.rows[1], 3) = v7;
    }
    else
    {
        rs.rows[0].x = v0;
        rs.rows[0].y = v1;
        rs.rows[1].x = v2;
        rs.rows[1].y = v3;
        rs.rows[2].x = v4;
        rs.rows[2].y = v5;
        rs.rows[3].x = v6;
        rs.rows[3].y = v7;
    }
    return rs;
}

template<typename T, int ROWS, int COLS>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, ROWS, COLS> makeMatrix(
    T v0,
    T v1,
    T v2,
    T v3,
    T v4,
    T v5,
    T v6,
    T v7,
    T v8)
{
    Matrix<T, ROWS, COLS> rs;
    rs.rows[0].x = v0;
    rs.rows[0].y = v1;
    rs.rows[0].z = v2;
    rs.rows[1].x = v3;
    rs.rows[1].y = v4;
    rs.rows[1].z = v5;
    rs.rows[2].x = v6;
    rs.rows[2].y = v7;
    rs.rows[2].z = v8;
    return rs;
}

template<typename T, int ROWS, int COLS>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, ROWS, COLS> makeMatrix(
    T v0,
    T v1,
    T v2,
    T v3,
    T v4,
    T v5,
    T v6,
    T v7,
    T v8,
    T v9,
    T v10,
    T v11)
{
    Matrix<T, ROWS, COLS> rs;
    if (COLS == 4)
    {
        *_slang_vector_get_element_ptr(&rs.rows[0], 0) = v0;
        *_slang_vector_get_element_ptr(&rs.rows[0], 1) = v1;
        *_slang_vector_get_element_ptr(&rs.rows[0], 2) = v2;
        *_slang_vector_get_element_ptr(&rs.rows[0], 3) = v3;
        *_slang_vector_get_element_ptr(&rs.rows[1], 0) = v4;
        *_slang_vector_get_element_ptr(&rs.rows[1], 1) = v5;
        *_slang_vector_get_element_ptr(&rs.rows[1], 2) = v6;
        *_slang_vector_get_element_ptr(&rs.rows[1], 3) = v7;
        *_slang_vector_get_element_ptr(&rs.rows[2], 0) = v8;
        *_slang_vector_get_element_ptr(&rs.rows[2], 1) = v9;
        *_slang_vector_get_element_ptr(&rs.rows[2], 2) = v10;
        *_slang_vector_get_element_ptr(&rs.rows[2], 3) = v11;
    }
    else
    {
        rs.rows[0].x = v0;
        rs.rows[0].y = v1;
        rs.rows[0].z = v2;
        rs.rows[1].x = v3;
        rs.rows[1].y = v4;
        rs.rows[1].z = v5;
        rs.rows[2].x = v6;
        rs.rows[2].y = v7;
        rs.rows[2].z = v8;
        rs.rows[3].x = v9;
        rs.rows[3].y = v10;
        rs.rows[3].z = v11;
    }
    return rs;
}

template<typename T, int ROWS, int COLS>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, ROWS, COLS> makeMatrix(
    T v0,
    T v1,
    T v2,
    T v3,
    T v4,
    T v5,
    T v6,
    T v7,
    T v8,
    T v9,
    T v10,
    T v11,
    T v12,
    T v13,
    T v14,
    T v15)
{
    Matrix<T, ROWS, COLS> rs;
    rs.rows[0].x = v0;
    rs.rows[0].y = v1;
    rs.rows[0].z = v2;
    rs.rows[0].w = v3;
    rs.rows[1].x = v4;
    rs.rows[1].y = v5;
    rs.rows[1].z = v6;
    rs.rows[1].w = v7;
    rs.rows[2].x = v8;
    rs.rows[2].y = v9;
    rs.rows[2].z = v10;
    rs.rows[2].w = v11;
    rs.rows[3].x = v12;
    rs.rows[3].y = v13;
    rs.rows[3].z = v14;
    rs.rows[3].w = v15;
    return rs;
}

#define SLANG_MATRIX_BINARY_OP(T, op)                                   \
    template<int R, int C>                                              \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, R, C> operator op(     \
        const Matrix<T, R, C>& thisVal,                                 \
        const Matrix<T, R, C>& other)                                   \
    {                                                                   \
        Matrix<T, R, C> result;                                         \
        for (int i = 0; i < R; i++)                                     \
            for (int j = 0; j < C; j++)                                 \
                *_slang_vector_get_element_ptr(result.rows + i, j) =    \
                    _slang_vector_get_element(thisVal.rows[i], j)       \
                        op _slang_vector_get_element(other.rows[i], j); \
        return result;                                                  \
    }

#define SLANG_MATRIX_UNARY_OP(T, op)                                                               \
    template<int R, int C>                                                                         \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, R, C> operator op(const Matrix<T, R, C>& thisVal) \
    {                                                                                              \
        Matrix<T, R, C> result;                                                                    \
        for (int i = 0; i < R; i++)                                                                \
            for (int j = 0; j < C; j++)                                                            \
                *_slang_vector_get_element_ptr(result.rows + i, j) =                               \
                    op _slang_vector_get_element(thisVal.rows[i], j);                              \
        return result;                                                                             \
    }
#define SLANG_INT_MATRIX_OPS(T)   \
    SLANG_MATRIX_BINARY_OP(T, +)  \
    SLANG_MATRIX_BINARY_OP(T, -)  \
    SLANG_MATRIX_BINARY_OP(T, *)  \
    SLANG_MATRIX_BINARY_OP(T, /)  \
    SLANG_MATRIX_BINARY_OP(T, &)  \
    SLANG_MATRIX_BINARY_OP(T, |)  \
    SLANG_MATRIX_BINARY_OP(T, &&) \
    SLANG_MATRIX_BINARY_OP(T, ||) \
    SLANG_MATRIX_BINARY_OP(T, ^)  \
    SLANG_MATRIX_BINARY_OP(T, %)  \
    SLANG_MATRIX_UNARY_OP(T, !)   \
    SLANG_MATRIX_UNARY_OP(T, ~)
#define SLANG_FLOAT_MATRIX_OPS(T) \
    SLANG_MATRIX_BINARY_OP(T, +)  \
    SLANG_MATRIX_BINARY_OP(T, -)  \
    SLANG_MATRIX_BINARY_OP(T, *)  \
    SLANG_MATRIX_BINARY_OP(T, /)  \
    SLANG_MATRIX_UNARY_OP(T, -)
SLANG_INT_MATRIX_OPS(int)
SLANG_INT_MATRIX_OPS(uint)
SLANG_INT_MATRIX_OPS(short)
SLANG_INT_MATRIX_OPS(ushort)
SLANG_INT_MATRIX_OPS(char)
SLANG_INT_MATRIX_OPS(uchar)
SLANG_INT_MATRIX_OPS(longlong)
SLANG_INT_MATRIX_OPS(ulonglong)
SLANG_FLOAT_MATRIX_OPS(float)
SLANG_FLOAT_MATRIX_OPS(double)
#if SLANG_CUDA_ENABLE_HALF
SLANG_FLOAT_MATRIX_OPS(__half)
#endif
#define SLANG_MATRIX_INT_NEG_OP(T)                                                        \
    template<int R, int C>                                                                \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, R, C> operator-(Matrix<T, R, C> thisVal) \
    {                                                                                     \
        Matrix<T, R, C> result;                                                           \
        for (int i = 0; i < R; i++)                                                       \
            for (int j = 0; j < C; j++)                                                   \
                *_slang_vector_get_element_ptr(result.rows + i, j) =                      \
                    0 - _slang_vector_get_element(thisVal.rows[i], j);                    \
        return result;                                                                    \
    }
SLANG_MATRIX_INT_NEG_OP(int)
SLANG_MATRIX_INT_NEG_OP(uint)
SLANG_MATRIX_INT_NEG_OP(short)
SLANG_MATRIX_INT_NEG_OP(ushort)
SLANG_MATRIX_INT_NEG_OP(char)
SLANG_MATRIX_INT_NEG_OP(uchar)
SLANG_MATRIX_INT_NEG_OP(longlong)
SLANG_MATRIX_INT_NEG_OP(ulonglong)

#define SLANG_FLOAT_MATRIX_MOD(T)                                                 \
    template<int R, int C>                                                        \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<T, R, C> operator%(                 \
        Matrix<T, R, C> left,                                                     \
        Matrix<T, R, C> right)                                                    \
    {                                                                             \
        Matrix<T, R, C> result;                                                   \
        for (int i = 0; i < R; i++)                                               \
            for (int j = 0; j < C; j++)                                           \
                *_slang_vector_get_element_ptr(result.rows + i, j) = _slang_fmod( \
                    _slang_vector_get_element(left.rows[i], j),                   \
                    _slang_vector_get_element(right.rows[i], j));                 \
        return result;                                                            \
    }

SLANG_FLOAT_MATRIX_MOD(float)
SLANG_FLOAT_MATRIX_MOD(double)
#if SLANG_CUDA_ENABLE_HALF
template<int R, int C>
SLANG_FORCE_INLINE SLANG_CUDA_CALL Matrix<__half, R, C> operator%(
    Matrix<__half, R, C> left,
    Matrix<__half, R, C> right)
{
    Matrix<__half, R, C> result;
    for (int i = 0; i < R; i++)
        for (int j = 0; j < C; j++)
            *_slang_vector_get_element_ptr(result.rows + i, j) = __float2half(_slang_fmod(
                __half2float(_slang_vector_get_element(left.rows[i], j)),
                __half2float(_slang_vector_get_element(right.rows[i], j))));
    return result;
}
#endif
#undef SLANG_FLOAT_MATRIX_MOD
#undef SLANG_MATRIX_BINARY_OP
#undef SLANG_MATRIX_UNARY_OP
#undef SLANG_INT_MATRIX_OPS
#undef SLANG_FLOAT_MATRIX_OPS
#undef SLANG_MATRIX_INT_NEG_OP
#undef SLANG_FLOAT_MATRIX_MOD

#define SLANG_SELECT_IMPL(T, N)                                                                  \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL Vector<T, N> _slang_select(                               \
        bool##N condition,                                                                       \
        Vector<T, N> v0,                                                                         \
        Vector<T, N> v1)                                                                         \
    {                                                                                            \
        Vector<T, N> result;                                                                     \
        for (int i = 0; i < N; i++)                                                              \
        {                                                                                        \
            *_slang_vector_get_element_ptr(&result, i) = _slang_vector_get_element(condition, i) \
                                                             ? _slang_vector_get_element(v0, i)  \
                                                             : _slang_vector_get_element(v1, i); \
        }                                                                                        \
        return result;                                                                           \
    }
#define SLANG_SELECT_T(T)   \
    SLANG_SELECT_IMPL(T, 2) \
    SLANG_SELECT_IMPL(T, 3) \
    SLANG_SELECT_IMPL(T, 4)

SLANG_SELECT_T(int)
SLANG_SELECT_T(uint)
SLANG_SELECT_T(short)
SLANG_SELECT_T(ushort)
SLANG_SELECT_T(char)
SLANG_SELECT_T(uchar)
SLANG_SELECT_T(float)
SLANG_SELECT_T(double)

template<typename T>
SLANG_FORCE_INLINE SLANG_CUDA_CALL T _slang_select(bool condition, T v0, T v1)
{
    return condition ? v0 : v1;
}

//
// Half support
//

#if SLANG_CUDA_ENABLE_HALF
SLANG_SELECT_T(__half)

// Convenience functions ushort -> half

SLANG_FORCE_INLINE SLANG_CUDA_CALL __half2 __ushort_as_half(const ushort2& i)
{
    return __halves2half2(__ushort_as_half(i.x), __ushort_as_half(i.y));
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL __half3 __ushort_as_half(const ushort3& i)
{
    return __half3{__ushort_as_half(i.x), __ushort_as_half(i.y), __ushort_as_half(i.z)};
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL __half4 __ushort_as_half(const ushort4& i)
{
    return __half4{
        __ushort_as_half(i.x),
        __ushort_as_half(i.y),
        __ushort_as_half(i.z),
        __ushort_as_half(i.w)};
}

// Convenience functions half -> ushort

SLANG_FORCE_INLINE SLANG_CUDA_CALL ushort2 __half_as_ushort(const __half2& i)
{
    return make_ushort2(__half_as_ushort(i.x), __half_as_ushort(i.y));
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL ushort3 __half_as_ushort(const __half3& i)
{
    return make_ushort3(__half_as_ushort(i.x), __half_as_ushort(i.y), __half_as_ushort(i.z));
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL ushort4 __half_as_ushort(const __half4& i)
{
    return make_ushort4(
        __half_as_ushort(i.x),
        __half_as_ushort(i.y),
        __half_as_ushort(i.z),
        __half_as_ushort(i.w));
}

// This is a little bit of a hack. Fortunately CUDA has the definitions of the templated types in
// include/surface_indirect_functions.h
// Here we find the template definition requires a specialization of __nv_isurf_trait to allow
// a specialization of the surface write functions.
// This *isn't* a problem on the read functions as they don't have a return type that uses this
// mechanism

template<>
struct __nv_isurf_trait<__half>
{
    typedef void type;
};
template<>
struct __nv_isurf_trait<__half2>
{
    typedef void type;
};
template<>
struct __nv_isurf_trait<__half4>
{
    typedef void type;
};

#define SLANG_DROP_PARENS(...) __VA_ARGS__

#define SLANG_SURFACE_READ(FUNC_NAME, TYPE_ARGS, ARGS)                                             \
    template<>                                                                                     \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL __half FUNC_NAME<__half>(                                   \
        cudaSurfaceObject_t surfObj,                                                               \
        SLANG_DROP_PARENS TYPE_ARGS,                                                               \
        cudaSurfaceBoundaryMode boundaryMode)                                                      \
    {                                                                                              \
        return __ushort_as_half(FUNC_NAME<ushort>(surfObj, SLANG_DROP_PARENS ARGS, boundaryMode)); \
    }                                                                                              \
                                                                                                   \
    template<>                                                                                     \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL __half2 FUNC_NAME<__half2>(                                 \
        cudaSurfaceObject_t surfObj,                                                               \
        SLANG_DROP_PARENS TYPE_ARGS,                                                               \
        cudaSurfaceBoundaryMode boundaryMode)                                                      \
    {                                                                                              \
        return __ushort_as_half(                                                                   \
            FUNC_NAME<ushort2>(surfObj, SLANG_DROP_PARENS ARGS, boundaryMode));                    \
    }                                                                                              \
                                                                                                   \
    template<>                                                                                     \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL __half4 FUNC_NAME<__half4>(                                 \
        cudaSurfaceObject_t surfObj,                                                               \
        SLANG_DROP_PARENS TYPE_ARGS,                                                               \
        cudaSurfaceBoundaryMode boundaryMode)                                                      \
    {                                                                                              \
        return __ushort_as_half(                                                                   \
            FUNC_NAME<ushort4>(surfObj, SLANG_DROP_PARENS ARGS, boundaryMode));                    \
    }

SLANG_SURFACE_READ(surf1Dread, (int x), (x))
SLANG_SURFACE_READ(surf2Dread, (int x, int y), (x, y))
SLANG_SURFACE_READ(surf3Dread, (int x, int y, int z), (x, y, z))
SLANG_SURFACE_READ(surf1DLayeredread, (int x, int layer), (x, layer))
SLANG_SURFACE_READ(surf2DLayeredread, (int x, int y, int layer), (x, y, layer))
SLANG_SURFACE_READ(surfCubemapread, (int x, int y, int face), (x, y, face))
SLANG_SURFACE_READ(surfCubemapLayeredread, (int x, int y, int layerFace), (x, y, layerFace))

#define SLANG_SURFACE_WRITE(FUNC_NAME, TYPE_ARGS, ARGS)                                            \
    template<>                                                                                     \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL void FUNC_NAME<__half>(                                     \
        __half data,                                                                               \
        cudaSurfaceObject_t surfObj,                                                               \
        SLANG_DROP_PARENS TYPE_ARGS,                                                               \
        cudaSurfaceBoundaryMode boundaryMode)                                                      \
    {                                                                                              \
        FUNC_NAME<ushort>(__half_as_ushort(data), surfObj, SLANG_DROP_PARENS ARGS, boundaryMode);  \
    }                                                                                              \
                                                                                                   \
    template<>                                                                                     \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL void FUNC_NAME<__half2>(                                    \
        __half2 data,                                                                              \
        cudaSurfaceObject_t surfObj,                                                               \
        SLANG_DROP_PARENS TYPE_ARGS,                                                               \
        cudaSurfaceBoundaryMode boundaryMode)                                                      \
    {                                                                                              \
        FUNC_NAME<ushort2>(__half_as_ushort(data), surfObj, SLANG_DROP_PARENS ARGS, boundaryMode); \
    }                                                                                              \
                                                                                                   \
    template<>                                                                                     \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL void FUNC_NAME<__half4>(                                    \
        __half4 data,                                                                              \
        cudaSurfaceObject_t surfObj,                                                               \
        SLANG_DROP_PARENS TYPE_ARGS,                                                               \
        cudaSurfaceBoundaryMode boundaryMode)                                                      \
    {                                                                                              \
        FUNC_NAME<ushort4>(__half_as_ushort(data), surfObj, SLANG_DROP_PARENS ARGS, boundaryMode); \
    }

SLANG_SURFACE_WRITE(surf1Dwrite, (int x), (x))
SLANG_SURFACE_WRITE(surf2Dwrite, (int x, int y), (x, y))
SLANG_SURFACE_WRITE(surf3Dwrite, (int x, int y, int z), (x, y, z))
SLANG_SURFACE_WRITE(surf1DLayeredwrite, (int x, int layer), (x, layer))
SLANG_SURFACE_WRITE(surf2DLayeredwrite, (int x, int y, int layer), (x, y, layer))
SLANG_SURFACE_WRITE(surfCubemapwrite, (int x, int y, int face), (x, y, face))
SLANG_SURFACE_WRITE(surfCubemapLayeredwrite, (int x, int y, int layerFace), (x, y, layerFace))

// ! Hack to test out reading !!!
// Only works converting *from* half

// template <typename T>
// SLANG_FORCE_INLINE SLANG_CUDA_CALL T surf2Dread_convert(cudaSurfaceObject_t surfObj, int x, int
// y, cudaSurfaceBoundaryMode boundaryMode);

#define SLANG_SURFACE_READ_HALF_CONVERT(FUNC_NAME, TYPE_ARGS, ARGS)                              \
                                                                                                 \
    template<typename T>                                                                         \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL T FUNC_NAME##_convert(                                    \
        cudaSurfaceObject_t surfObj,                                                             \
        SLANG_DROP_PARENS TYPE_ARGS,                                                             \
        cudaSurfaceBoundaryMode boundaryMode);                                                   \
                                                                                                 \
    template<>                                                                                   \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL float FUNC_NAME##_convert<float>(                         \
        cudaSurfaceObject_t surfObj,                                                             \
        SLANG_DROP_PARENS TYPE_ARGS,                                                             \
        cudaSurfaceBoundaryMode boundaryMode)                                                    \
    {                                                                                            \
        return __ushort_as_half(                                                                 \
            FUNC_NAME<uint16_t>(surfObj, SLANG_DROP_PARENS ARGS, boundaryMode));                 \
    }                                                                                            \
                                                                                                 \
    template<>                                                                                   \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL float2 FUNC_NAME##_convert<float2>(                       \
        cudaSurfaceObject_t surfObj,                                                             \
        SLANG_DROP_PARENS TYPE_ARGS,                                                             \
        cudaSurfaceBoundaryMode boundaryMode)                                                    \
    {                                                                                            \
        const __half2 v =                                                                        \
            __ushort_as_half(FUNC_NAME<ushort2>(surfObj, SLANG_DROP_PARENS ARGS, boundaryMode)); \
        return float2{v.x, v.y};                                                                 \
    }                                                                                            \
                                                                                                 \
    template<>                                                                                   \
    SLANG_FORCE_INLINE SLANG_CUDA_CALL float4 FUNC_NAME##_convert<float4>(                       \
        cudaSurfaceObject_t surfObj,                                                             \
        SLANG_DROP_PARENS TYPE_ARGS,                                                             \
        cudaSurfaceBoundaryMode boundaryMode)                                                    \
    {                                                                                            \
        const __half4 v =                                                                        \
            __ushort_as_half(FUNC_NAME<ushort4>(surfObj, SLANG_DROP_PARENS ARGS, boundaryMode)); \
        return float4{v.x, v.y, v.z, v.w};                                                       \
    }

SLANG_SURFACE_READ_HALF_CONVERT(surf1Dread, (int x), (x))
SLANG_SURFACE_READ_HALF_CONVERT(surf2Dread, (int x, int y), (x, y))
SLANG_SURFACE_READ_HALF_CONVERT(surf3Dread, (int x, int y, int z), (x, y, z))

#endif

// Support for doing format conversion when writing to a surface/RWTexture

// NOTE! For normal surface access x values are *byte* addressed.
// For the _convert versions they are *not*. They don't need to be because sust.p does not require
// it.

template<typename T>
SLANG_FORCE_INLINE SLANG_CUDA_CALL void surf1Dwrite_convert(
    T,
    hipSurfaceObject_t surfObj,
    int x,
    hipSurfaceBoundaryMode boundaryMode);
template<typename T>
SLANG_FORCE_INLINE SLANG_CUDA_CALL void surf2Dwrite_convert(
    T,
    hipSurfaceObject_t surfObj,
    int x,
    int y,
    hipSurfaceBoundaryMode boundaryMode);
template<typename T>
SLANG_FORCE_INLINE SLANG_CUDA_CALL void surf3Dwrite_convert(
    T,
    hipSurfaceObject_t surfObj,
    int x,
    int y,
    int z,
    hipSurfaceBoundaryMode boundaryMode);

// https://docs.nvidia.com/cuda/inline-ptx-assembly/index.html
// https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#surface-instructions-sust

// Float

template<>
SLANG_FORCE_INLINE SLANG_CUDA_CALL void surf1Dwrite_convert<float>(
    float v,
    hipSurfaceObject_t surfObj,
    int x,
    hipSurfaceBoundaryMode boundaryMode)
{
    asm volatile(
        "{sust.p.1d.b32." SLANG_PTX_BOUNDARY_MODE " [%0, {%1}], {%2};}\n\t" ::"l"(surfObj),
        "r"(x),
        "f"(v));
}

template<>
SLANG_FORCE_INLINE SLANG_CUDA_CALL void surf2Dwrite_convert<float>(
    float v,
    hipSurfaceObject_t surfObj,
    int x,
    int y,
    hipSurfaceBoundaryMode boundaryMode)
{
    asm volatile(
        "{sust.p.2d.b32." SLANG_PTX_BOUNDARY_MODE " [%0, {%1,%2}], {%3};}\n\t" ::"l"(surfObj),
        "r"(x),
        "r"(y),
        "f"(v));
}

template<>
SLANG_FORCE_INLINE SLANG_CUDA_CALL void surf3Dwrite_convert<float>(
    float v,
    hipSurfaceObject_t surfObj,
    int x,
    int y,
    int z,
    hipSurfaceBoundaryMode boundaryMode)
{
    asm volatile(
        "{sust.p.2d.b32." SLANG_PTX_BOUNDARY_MODE " [%0, {%1,%2,%3}], {%4};}\n\t" ::"l"(surfObj),
        "r"(x),
        "r"(y),
        "r"(z),
        "f"(v));
}

// Float2

template<>
SLANG_FORCE_INLINE SLANG_CUDA_CALL void surf1Dwrite_convert<float2>(
    float2 v,
    hipSurfaceObject_t surfObj,
    int x,
    hipSurfaceBoundaryMode boundaryMode)
{
    const float vx = v.x, vy = v.y;
    asm volatile(
        "{sust.p.1d.b32." SLANG_PTX_BOUNDARY_MODE " [%0, {%1}], {%2,%3};}\n\t" ::"l"(surfObj),
        "r"(x),
        "f"(vx),
        "f"(vy));
}

template<>
SLANG_FORCE_INLINE SLANG_CUDA_CALL void surf2Dwrite_convert<float2>(
    float2 v,
    hipSurfaceObject_t surfObj,
    int x,
    int y,
    hipSurfaceBoundaryMode boundaryMode)
{
    const float vx = v.x, vy = v.y;
    asm volatile(
        "{sust.p.2d.b32." SLANG_PTX_BOUNDARY_MODE " [%0, {%1,%2}], {%3,%4};}\n\t" ::"l"(surfObj),
        "r"(x),
        "r"(y),
        "f"(vx),
        "f"(vy));
}

template<>
SLANG_FORCE_INLINE SLANG_CUDA_CALL void surf3Dwrite_convert<float2>(
    float2 v,
    hipSurfaceObject_t surfObj,
    int x,
    int y,
    int z,
    hipSurfaceBoundaryMode boundaryMode)
{
    const float vx = v.x, vy = v.y;
    asm volatile(
        "{sust.p.2d.b32." SLANG_PTX_BOUNDARY_MODE " [%0, {%1,%2,%3}], {%4,%5};}\n\t" ::"l"(surfObj),
        "r"(x),
        "r"(y),
        "r"(z),
        "f"(vx),
        "f"(vy));
}

// Float4
template<>
SLANG_FORCE_INLINE SLANG_CUDA_CALL void surf1Dwrite_convert<float4>(
    float4 v,
    hipSurfaceObject_t surfObj,
    int x,
    hipSurfaceBoundaryMode boundaryMode)
{
    const float vx = v.x, vy = v.y, vz = v.z, vw = v.w;
    asm volatile(
        "{sust.p.1d.b32." SLANG_PTX_BOUNDARY_MODE " [%0, {%1}], {%2,%3,%4,%5};}\n\t" ::"l"(surfObj),
        "r"(x),
        "f"(vx),
        "f"(vy),
        "f"(vz),
        "f"(vw));
}

template<>
SLANG_FORCE_INLINE SLANG_CUDA_CALL void surf2Dwrite_convert<float4>(
    float4 v,
    hipSurfaceObject_t surfObj,
    int x,
    int y,
    hipSurfaceBoundaryMode boundaryMode)
{
    const float vx = v.x, vy = v.y, vz = v.z, vw = v.w;
    asm volatile(
        "{sust.p.2d.b32." SLANG_PTX_BOUNDARY_MODE
        " [%0, {%1,%2}], {%3,%4,%5,%6};}\n\t" ::"l"(surfObj),
        "r"(x),
        "r"(y),
        "f"(vx),
        "f"(vy),
        "f"(vz),
        "f"(vw));
}

template<>
SLANG_FORCE_INLINE SLANG_CUDA_CALL void surf3Dwrite_convert<float4>(
    float4 v,
    hipSurfaceObject_t surfObj,
    int x,
    int y,
    int z,
    hipSurfaceBoundaryMode boundaryMode)
{
    const float vx = v.x, vy = v.y, vz = v.z, vw = v.w;
    asm volatile(
        "{sust.p.2d.b32." SLANG_PTX_BOUNDARY_MODE
        " [%0, {%1,%2,%3}], {%4,%5,%6,%7};}\n\t" ::"l"(surfObj),
        "r"(x),
        "r"(y),
        "r"(z),
        "f"(vx),
        "f"(vy),
        "f"(vz),
        "f"(vw));
}

// ----------------------------- F32 -----------------------------------------

// Unary
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_ceil(float f)
{
    return ::ceilf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_floor(float f)
{
    return ::floorf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_round(float f)
{
    return ::roundf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_sin(float f)
{
    return ::sinf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_cos(float f)
{
    return ::cosf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL void F32_sincos(float f, float* s, float* c)
{
    ::sincosf(f, s, c);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_tan(float f)
{
    return ::tanf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_asin(float f)
{
    return ::asinf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_acos(float f)
{
    return ::acosf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_atan(float f)
{
    return ::atanf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_sinh(float f)
{
    return ::sinhf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_cosh(float f)
{
    return ::coshf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_tanh(float f)
{
    return ::tanhf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_log2(float f)
{
    return ::log2f(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_log(float f)
{
    return ::logf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_log10(float f)
{
    return ::log10f(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_exp2(float f)
{
    return ::exp2f(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_exp(float f)
{
    return ::expf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_abs(float f)
{
    return ::fabsf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_trunc(float f)
{
    return ::truncf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_sqrt(float f)
{
    return ::sqrtf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_rsqrt(float f)
{
    return ::rsqrtf(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_sign(float f)
{
    return (f == 0.0f) ? f : ((f < 0.0f) ? -1.0f : 1.0f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_frac(float f)
{
    return f - F32_floor(f);
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL bool F32_isnan(float f)
{
    return isnan(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL bool F32_isfinite(float f)
{
    return isfinite(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL bool F32_isinf(float f)
{
    return isinf(f);
}

// Binary
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_min(float a, float b)
{
    return ::fminf(a, b);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_max(float a, float b)
{
    return ::fmaxf(a, b);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_pow(float a, float b)
{
    return ::powf(a, b);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_fmod(float a, float b)
{
    return ::fmodf(a, b);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_remainder(float a, float b)
{
    return ::remainderf(a, b);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_atan2(float a, float b)
{
    return float(::atan2(a, b));
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_frexp(float x, int* e)
{
    return frexpf(x, e);
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_modf(float x, float* ip)
{
    return ::modff(x, ip);
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL uint32_t F32_asuint(float f)
{
    Union32 u;
    u.f = f;
    return u.u;
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL int32_t F32_asint(float f)
{
    Union32 u;
    u.f = f;
    return u.i;
}

// Ternary
SLANG_FORCE_INLINE SLANG_CUDA_CALL float F32_fma(float a, float b, float c)
{
    return ::fmaf(a, b, c);
}


// ----------------------------- F64 -----------------------------------------

// Unary
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_ceil(double f)
{
    return ::ceil(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_floor(double f)
{
    return ::floor(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_round(double f)
{
    return ::round(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_sin(double f)
{
    return ::sin(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_cos(double f)
{
    return ::cos(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL void F64_sincos(double f, double* s, double* c)
{
    ::sincos(f, s, c);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_tan(double f)
{
    return ::tan(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_asin(double f)
{
    return ::asin(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_acos(double f)
{
    return ::acos(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_atan(double f)
{
    return ::atan(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_sinh(double f)
{
    return ::sinh(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_cosh(double f)
{
    return ::cosh(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_tanh(double f)
{
    return ::tanh(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_log2(double f)
{
    return ::log2(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_log(double f)
{
    return ::log(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_log10(float f)
{
    return ::log10(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_exp2(double f)
{
    return ::exp2(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_exp(double f)
{
    return ::exp(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_abs(double f)
{
    return ::fabs(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_trunc(double f)
{
    return ::trunc(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_sqrt(double f)
{
    return ::sqrt(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_rsqrt(double f)
{
    return ::rsqrt(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_sign(double f)
{
    return (f == 0.0) ? f : ((f < 0.0) ? -1.0 : 1.0);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_frac(double f)
{
    return f - F64_floor(f);
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL bool F64_isnan(double f)
{
    return isnan(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL bool F64_isfinite(double f)
{
    return isfinite(f);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL bool F64_isinf(double f)
{
    return isinf(f);
}

// Binary
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_min(double a, double b)
{
    return ::fmin(a, b);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_max(double a, double b)
{
    return ::fmax(a, b);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_pow(double a, double b)
{
    return ::pow(a, b);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_fmod(double a, double b)
{
    return ::fmod(a, b);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_remainder(double a, double b)
{
    return ::remainder(a, b);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_atan2(double a, double b)
{
    return ::atan2(a, b);
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_frexp(double x, int* e)
{
    return ::frexp(x, e);
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_modf(double x, double* ip)
{
    return ::modf(x, ip);
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL void F64_asuint(double d, uint32_t* low, uint32_t* hi)
{
    Union64 u;
    u.d = d;
    *low = uint32_t(u.u);
    *hi = uint32_t(u.u >> 32);
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL void F64_asint(double d, int32_t* low, int32_t* hi)
{
    Union64 u;
    u.d = d;
    *low = int32_t(u.u);
    *hi = int32_t(u.u >> 32);
}

// Ternary
SLANG_FORCE_INLINE SLANG_CUDA_CALL double F64_fma(double a, double b, double c)
{
    return ::fma(a, b, c);
}

// ----------------------------- I32 -----------------------------------------

// Unary
SLANG_FORCE_INLINE SLANG_CUDA_CALL int32_t I32_abs(int32_t f)
{
    return (f < 0) ? -f : f;
}

// Binary
SLANG_FORCE_INLINE SLANG_CUDA_CALL int32_t I32_min(int32_t a, int32_t b)
{
    return a < b ? a : b;
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL int32_t I32_max(int32_t a, int32_t b)
{
    return a > b ? a : b;
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL float I32_asfloat(int32_t x)
{
    Union32 u;
    u.i = x;
    return u.f;
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL uint32_t I32_asuint(int32_t x)
{
    return uint32_t(x);
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL double I32_asdouble(int32_t low, int32_t hi)
{
    Union64 u;
    u.u = (uint64_t(hi) << 32) | uint32_t(low);
    return u.d;
}

// ----------------------------- U32 -----------------------------------------

// Unary
SLANG_FORCE_INLINE SLANG_CUDA_CALL uint32_t U32_abs(uint32_t f)
{
    return f;
}

// Binary
SLANG_FORCE_INLINE SLANG_CUDA_CALL uint32_t U32_min(uint32_t a, uint32_t b)
{
    return a < b ? a : b;
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL uint32_t U32_max(uint32_t a, uint32_t b)
{
    return a > b ? a : b;
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL float U32_asfloat(uint32_t x)
{
    Union32 u;
    u.u = x;
    return u.f;
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL uint32_t U32_asint(int32_t x)
{
    return uint32_t(x);
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL double U32_asdouble(uint32_t low, uint32_t hi)
{
    Union64 u;
    u.u = (uint64_t(hi) << 32) | low;
    return u.d;
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL uint32_t U32_countbits(uint32_t v)
{
    // https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__INTRINSIC__INT.html#group__CUDA__MATH__INTRINSIC__INT_1g43c9c7d2b9ebf202ff1ef5769989be46
    return __popc(v);
}


// ----------------------------- I64 -----------------------------------------

SLANG_FORCE_INLINE SLANG_CUDA_CALL int64_t I64_abs(int64_t f)
{
    return (f < 0) ? -f : f;
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL int64_t I64_min(int64_t a, int64_t b)
{
    return a < b ? a : b;
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL int64_t I64_max(int64_t a, int64_t b)
{
    return a > b ? a : b;
}

// ----------------------------- U64 -----------------------------------------

SLANG_FORCE_INLINE SLANG_CUDA_CALL int64_t U64_abs(uint64_t f)
{
    return f;
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL int64_t U64_min(uint64_t a, uint64_t b)
{
    return a < b ? a : b;
}
SLANG_FORCE_INLINE SLANG_CUDA_CALL int64_t U64_max(uint64_t a, uint64_t b)
{
    return a > b ? a : b;
}

SLANG_FORCE_INLINE SLANG_CUDA_CALL uint32_t U64_countbits(uint64_t v)
{
    // https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__INTRINSIC__INT.html#group__CUDA__MATH__INTRINSIC__INT_1g43c9c7d2b9ebf202ff1ef5769989be46
    return __popcll(v);
}


// ----------------------------- ResourceType -----------------------------------------


// https://docs.microsoft.com/en-us/windows/win32/direct3dhlsl/sm5-object-structuredbuffer-getdimensions
// Missing  Load(_In_  int  Location, _Out_ uint Status);

template<typename T>
struct StructuredBuffer
{
    SLANG_CUDA_CALL const T& operator[](size_t index) const
    {
#ifndef SLANG_CUDA_STRUCTURED_BUFFER_NO_COUNT
        SLANG_BOUND_CHECK(index, count);
#endif
        return data[index];
    }

    SLANG_CUDA_CALL const T& Load(size_t index) const
    {
#ifndef SLANG_CUDA_STRUCTURED_BUFFER_NO_COUNT
        SLANG_BOUND_CHECK(index, count);
#endif
        return data[index];
    }

#ifndef SLANG_CUDA_STRUCTURED_BUFFER_NO_COUNT
    SLANG_CUDA_CALL void GetDimensions(uint32_t* outNumStructs, uint32_t* outStride)
    {
        *outNumStructs = uint32_t(count);
        *outStride = uint32_t(sizeof(T));
    }
#endif

    T* data;
#ifndef SLANG_CUDA_STRUCTURED_BUFFER_NO_COUNT
    size_t count;
#endif
};

template<typename T>
struct RWStructuredBuffer : StructuredBuffer<T>
{
    SLANG_CUDA_CALL T& operator[](size_t index) const
    {
#ifndef SLANG_CUDA_STRUCTURED_BUFFER_NO_COUNT
        SLANG_BOUND_CHECK(index, this->count);
#endif
        return this->data[index];
    }
};

// Missing  Load(_In_  int  Location, _Out_ uint Status);
struct ByteAddressBuffer
{
    SLANG_CUDA_CALL void GetDimensions(uint32_t* outDim) const { *outDim = uint32_t(sizeInBytes); }
    SLANG_CUDA_CALL uint32_t Load(size_t index) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, 4, sizeInBytes);
        return data[index >> 2];
    }
    SLANG_CUDA_CALL uint2 Load2(size_t index) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, 8, sizeInBytes);
        const size_t dataIdx = index >> 2;
        return uint2{data[dataIdx], data[dataIdx + 1]};
    }
    SLANG_CUDA_CALL uint3 Load3(size_t index) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, 12, sizeInBytes);
        const size_t dataIdx = index >> 2;
        return uint3{data[dataIdx], data[dataIdx + 1], data[dataIdx + 2]};
    }
    SLANG_CUDA_CALL uint4 Load4(size_t index) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, 16, sizeInBytes);
        const size_t dataIdx = index >> 2;
        return uint4{data[dataIdx], data[dataIdx + 1], data[dataIdx + 2], data[dataIdx + 3]};
    }
    template<typename T>
    SLANG_CUDA_CALL T Load(size_t index) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, sizeof(T), sizeInBytes);
        T data;
        memcpy(&data, ((const char*)this->data) + index, sizeof(T));
        return data;
    }
    template<typename T>
    SLANG_CUDA_CALL StructuredBuffer<T> asStructuredBuffer() const
    {
        StructuredBuffer<T> rs;
        rs.data = (T*)data;
        rs.count = sizeInBytes / sizeof(T);
        return rs;
    }
    const uint32_t* data;
    size_t sizeInBytes; //< Must be multiple of 4
};

// https://docs.microsoft.com/en-us/windows/win32/direct3dhlsl/sm5-object-rwbyteaddressbuffer
// Missing support for Atomic operations
// Missing support for Load with status
struct RWByteAddressBuffer
{
    SLANG_CUDA_CALL void GetDimensions(uint32_t* outDim) const { *outDim = uint32_t(sizeInBytes); }

    SLANG_CUDA_CALL uint32_t Load(size_t index) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, 4, sizeInBytes);
        return data[index >> 2];
    }
    SLANG_CUDA_CALL uint2 Load2(size_t index) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, 8, sizeInBytes);
        const size_t dataIdx = index >> 2;
        return uint2{data[dataIdx], data[dataIdx + 1]};
    }
    SLANG_CUDA_CALL uint3 Load3(size_t index) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, 12, sizeInBytes);
        const size_t dataIdx = index >> 2;
        return uint3{data[dataIdx], data[dataIdx + 1], data[dataIdx + 2]};
    }
    SLANG_CUDA_CALL uint4 Load4(size_t index) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, 16, sizeInBytes);
        const size_t dataIdx = index >> 2;
        return uint4{data[dataIdx], data[dataIdx + 1], data[dataIdx + 2], data[dataIdx + 3]};
    }
    template<typename T>
    SLANG_CUDA_CALL T Load(size_t index) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, sizeof(T), sizeInBytes);
        T data;
        memcpy(&data, ((const char*)this->data) + index, sizeof(T));
        return data;
    }

    SLANG_CUDA_CALL void Store(size_t index, uint32_t v) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, 4, sizeInBytes);
        data[index >> 2] = v;
    }
    SLANG_CUDA_CALL void Store2(size_t index, uint2 v) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, 8, sizeInBytes);
        const size_t dataIdx = index >> 2;
        data[dataIdx + 0] = v.x;
        data[dataIdx + 1] = v.y;
    }
    SLANG_CUDA_CALL void Store3(size_t index, uint3 v) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, 12, sizeInBytes);
        const size_t dataIdx = index >> 2;
        data[dataIdx + 0] = v.x;
        data[dataIdx + 1] = v.y;
        data[dataIdx + 2] = v.z;
    }
    SLANG_CUDA_CALL void Store4(size_t index, uint4 v) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, 16, sizeInBytes);
        const size_t dataIdx = index >> 2;
        data[dataIdx + 0] = v.x;
        data[dataIdx + 1] = v.y;
        data[dataIdx + 2] = v.z;
        data[dataIdx + 3] = v.w;
    }
    template<typename T>
    SLANG_CUDA_CALL void Store(size_t index, T const& value) const
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, sizeof(T), sizeInBytes);
        memcpy((char*)data + index, &value, sizeof(T));
    }

    /// Can be used in the core module to gain access
    template<typename T>
    SLANG_CUDA_CALL T* _getPtrAt(size_t index)
    {
        SLANG_BOUND_CHECK_BYTE_ADDRESS(index, sizeof(T), sizeInBytes);
        return (T*)(((char*)data) + index);
    }
    template<typename T>
    SLANG_CUDA_CALL RWStructuredBuffer<T> asStructuredBuffer() const
    {
        RWStructuredBuffer<T> rs;
        rs.data = (T*)data;
        rs.count = sizeInBytes / sizeof(T);
        return rs;
    }
    uint32_t* data;
    size_t sizeInBytes; //< Must be multiple of 4
};


// ---------------------- Wave --------------------------------------

// TODO(JS): It appears that cuda does not have a simple way to get a lane index.
//
// Another approach could be...
// laneId = ((threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x) &
// SLANG_CUDA_WARP_MASK If that is really true another way to do this, would be for code generator
// to add this function with the [numthreads] baked in.
//
// For now I'll just assume you have a launch that makes the following correct if the kernel uses
// WaveGetLaneIndex()
#ifndef SLANG_USE_ASM_LANE_ID
__forceinline__ __device__ uint32_t _getLaneId()
{
    // If the launch is (or I guess some multiple of the warp size)
    // we try this mechanism, which is apparently faster.
    return threadIdx.x & SLANG_CUDA_WARP_MASK;
}
#else
__forceinline__ __device__ uint32_t _getLaneId()
{
    // https://stackoverflow.com/questions/44337309/whats-the-most-efficient-way-to-calculate-the-warp-id-lane-id-in-a-1-d-grid#
    // This mechanism is not the fastest way to do it, and that is why the other mechanism
    // is the default. But the other mechanism relies on a launch that makes the assumption
    // true.
    unsigned ret;
    asm volatile("mov.u32 %0, %laneid;" : "=r"(ret));
    return ret;
}
#endif

typedef int WarpMask;

// It appears that the __activemask() cannot always be used because
// threads need to be converged.
//
// For CUDA the article claims mask has to be used carefully
// https://devblogs.nvidia.com/using-cuda-warp-level-primitives/
// With the Warp intrinsics there is no mask, and it's just the 'active lanes'.
// __activemask() though does not require there is convergence, so that doesn't work.
//
// '__ballot_sync' produces a convergance.
//
// From the CUDA docs:
// ```For __all_sync, __any_sync, and __ballot_sync, a mask must be passed that specifies the
// threads participating in the call. A bit, representing the thread's lane ID, must be set for each
// participating thread to ensure they are properly converged before the intrinsic is executed by
// the hardware. All active threads named in mask must execute the same intrinsic with the same
// mask, or the result is undefined.```
//
// Currently there isn't a mechanism to correctly get the mask without it being passed through.
// Doing so will most likely require some changes to slang code generation to track masks, for now
// then we use _getActiveMask.

// Return mask of all the lanes less than the current lane
__forceinline__ __device__ WarpMask _getLaneLtMask()
{
    return (int(1) << _getLaneId()) - 1;
}

// TODO(JS):
// THIS IS NOT CORRECT! That determining the appropriate active mask requires appropriate
// mask tracking.
__forceinline__ __device__ WarpMask _getActiveMask()
{
    return __ballot_sync(__activemask(), true);
}

// Return a mask suitable for the 'MultiPrefix' style functions
__forceinline__ __device__ WarpMask _getMultiPrefixMask(int mask)
{
    return mask;
}

// Note! Note will return true if mask is 0, but thats okay, because there must be one
// lane active to execute anything
__inline__ __device__ bool _waveIsSingleLane(WarpMask mask)
{
    return (mask & (mask - 1)) == 0;
}

// Returns the power of 2 size of run of set bits. Returns 0 if not a suitable run.
// Examples:
// 0b00000000'00000000'00000000'11111111 -> 8
// 0b11111111'11111111'11111111'11111111 -> 32
// 0b00000000'00000000'00000000'00011111 -> 0 (since 5 is not a power of 2)
// 0b00000000'00000000'00000000'11110000 -> 0 (since the run of bits does not start at the LSB)
// 0b00000000'00000000'00000000'00100111 -> 0 (since it is not a single contiguous run)
__inline__ __device__ int _waveCalcPow2Offset(WarpMask mask)
{
    // This should be the most common case, so fast path it
    if (mask == SLANG_CUDA_WARP_BITMASK)
    {
        return SLANG_CUDA_WARP_SIZE;
    }
    // Is it a contiguous run of bits?
    if ((mask & (mask + 1)) == 0)
    {
        // const int offsetSize = __ffs(mask + 1) - 1;
        const int offset = 32 - __clz(mask);
        // Is it a power of 2 size
        if ((offset & (offset - 1)) == 0)
        {
            return offset;
        }
    }
    return 0;
}

__inline__ __device__ bool _waveIsFirstLane()
{
    const WarpMask mask = __activemask();
    // We special case bit 0, as that most warps are expected to be fully active.

    // mask & -mask, isolates the lowest set bit.
    // return (mask & 1 ) || ((mask & -mask) == (1 << _getLaneId()));

    // This mechanism is most similar to what was in an nVidia post, so assume it is prefered.
    return (mask & 1) || ((__ffs(mask) - 1) == _getLaneId());
}

template<typename T>
struct WaveOpOr
{
    __inline__ __device__ static T getInitial(T a) { return 0; }
    __inline__ __device__ static T doOp(T a, T b) { return a | b; }
};

template<typename T>
struct WaveOpAnd
{
    __inline__ __device__ static T getInitial(T a) { return ~T(0); }
    __inline__ __device__ static T doOp(T a, T b) { return a & b; }
};

template<typename T>
struct WaveOpXor
{
    __inline__ __device__ static T getInitial(T a) { return 0; }
    __inline__ __device__ static T doOp(T a, T b) { return a ^ b; }
    __inline__ __device__ static T doInverse(T a, T b) { return a ^ b; }
};

template<typename T>
struct WaveOpAdd
{
    __inline__ __device__ static T getInitial(T a) { return 0; }
    __inline__ __device__ static T doOp(T a, T b) { return a + b; }
    __inline__ __device__ static T doInverse(T a, T b) { return a - b; }
};

template<typename T>
struct WaveOpMul
{
    __inline__ __device__ static T getInitial(T a) { return T(1); }
    __inline__ __device__ static T doOp(T a, T b) { return a * b; }
    // Using this inverse for int is probably undesirable - because in general it requires T to have
    // more precision There is also a performance aspect to it, where divides are generally
    // significantly slower
    __inline__ __device__ static T doInverse(T a, T b) { return a / b; }
};

template<typename T>
struct WaveOpMax
{
    __inline__ __device__ static T getInitial(T a) { return a; }
    __inline__ __device__ static T doOp(T a, T b) { return a > b ? a : b; }
};

template<typename T>
struct WaveOpMin
{
    __inline__ __device__ static T getInitial(T a) { return a; }
    __inline__ __device__ static T doOp(T a, T b) { return a < b ? a : b; }
};

template<typename T>
struct ElementTypeTrait;

// Scalar
template<>
struct ElementTypeTrait<int>
{
    typedef int Type;
};
template<>
struct ElementTypeTrait<uint>
{
    typedef uint Type;
};
template<>
struct ElementTypeTrait<float>
{
    typedef float Type;
};
template<>
struct ElementTypeTrait<double>
{
    typedef double Type;
};
template<>
struct ElementTypeTrait<uint64_t>
{
    typedef uint64_t Type;
};
template<>
struct ElementTypeTrait<int64_t>
{
    typedef int64_t Type;
};

// Vector
template<>
struct ElementTypeTrait<int1>
{
    typedef int Type;
};
template<>
struct ElementTypeTrait<int2>
{
    typedef int Type;
};
template<>
struct ElementTypeTrait<int3>
{
    typedef int Type;
};
template<>
struct ElementTypeTrait<int4>
{
    typedef int Type;
};

template<>
struct ElementTypeTrait<uint1>
{
    typedef uint Type;
};
template<>
struct ElementTypeTrait<uint2>
{
    typedef uint Type;
};
template<>
struct ElementTypeTrait<uint3>
{
    typedef uint Type;
};
template<>
struct ElementTypeTrait<uint4>
{
    typedef uint Type;
};

template<>
struct ElementTypeTrait<float1>
{
    typedef float Type;
};
template<>
struct ElementTypeTrait<float2>
{
    typedef float Type;
};
template<>
struct ElementTypeTrait<float3>
{
    typedef float Type;
};
template<>
struct ElementTypeTrait<float4>
{
    typedef float Type;
};

template<>
struct ElementTypeTrait<double1>
{
    typedef double Type;
};
template<>
struct ElementTypeTrait<double2>
{
    typedef double Type;
};
template<>
struct ElementTypeTrait<double3>
{
    typedef double Type;
};
template<>
struct ElementTypeTrait<double4>
{
    typedef double Type;
};

// Matrix
template<typename T, int ROWS, int COLS>
struct ElementTypeTrait<Matrix<T, ROWS, COLS>>
{
    typedef T Type;
};

// Scalar
template<typename INTF, typename T>
__device__ T _waveReduceScalar(WarpMask mask, T val)
{
    const int offsetSize = _waveCalcPow2Offset(mask);
    if (offsetSize > 0)
    {
        // Fast path O(log2(activeLanes))
        for (int offset = offsetSize >> 1; offset > 0; offset >>= 1)
        {
            val = INTF::doOp(val, __shfl_xor_sync(mask, val, offset));
        }
    }
    else if (!_waveIsSingleLane(mask))
    {
        T result = INTF::getInitial(val);
        int remaining = mask;
        while (remaining)
        {
            const int laneBit = remaining & -remaining;
            // Get the sourceLane
            const int srcLane = __ffs(laneBit) - 1;
            // Broadcast (can also broadcast to self)
            result = INTF::doOp(result, __shfl_sync(mask, val, srcLane));
            remaining &= ~laneBit;
        }
        return result;
    }
    return val;
}


// Multiple values
template<typename INTF, typename T, size_t COUNT>
__device__ void _waveReduceMultiple(WarpMask mask, T* val)
{
    const int offsetSize = _waveCalcPow2Offset(mask);
    if (offsetSize > 0)
    {
        // Fast path O(log2(activeLanes))
        for (int offset = offsetSize >> 1; offset > 0; offset >>= 1)
        {
            for (size_t i = 0; i < COUNT; ++i)
            {
                val[i] = INTF::doOp(val[i], __shfl_xor_sync(mask, val[i], offset));
            }
        }
    }
    else if (!_waveIsSingleLane(mask))
    {
        // Copy the original
        T originalVal[COUNT];
        for (size_t i = 0; i < COUNT; ++i)
        {
            const T v = val[i];
            originalVal[i] = v;
            val[i] = INTF::getInitial(v);
        }

        int remaining = mask;
        while (remaining)
        {
            const int laneBit = remaining & -remaining;
            // Get the sourceLane
            const int srcLane = __ffs(laneBit) - 1;
            // Broadcast (can also broadcast to self)
            for (size_t i = 0; i < COUNT; ++i)
            {
                val[i] = INTF::doOp(val[i], __shfl_sync(mask, originalVal[i], srcLane));
            }
            remaining &= ~laneBit;
        }
    }
}

template<typename INTF, typename T>
__device__ void _waveReduceMultiple(WarpMask mask, T* val)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    _waveReduceMultiple<INTF, ElemType, sizeof(T) / sizeof(ElemType)>(mask, (ElemType*)val);
}

template<typename T>
__inline__ __device__ T _waveOr(WarpMask mask, T val)
{
    return _waveReduceScalar<WaveOpOr<T>, T>(mask, val);
}

template<typename T>
__inline__ __device__ T _waveAnd(WarpMask mask, T val)
{
    return _waveReduceScalar<WaveOpAnd<T>, T>(mask, val);
}

template<typename T>
__inline__ __device__ T _waveXor(WarpMask mask, T val)
{
    return _waveReduceScalar<WaveOpXor<T>, T>(mask, val);
}

template<typename T>
__inline__ __device__ T _waveProduct(WarpMask mask, T val)
{
    return _waveReduceScalar<WaveOpMul<T>, T>(mask, val);
}

template<typename T>
__inline__ __device__ T _waveSum(WarpMask mask, T val)
{
    return _waveReduceScalar<WaveOpAdd<T>, T>(mask, val);
}

template<typename T>
__inline__ __device__ T _waveMin(WarpMask mask, T val)
{
    return _waveReduceScalar<WaveOpMin<T>, T>(mask, val);
}

template<typename T>
__inline__ __device__ T _waveMax(WarpMask mask, T val)
{
    return _waveReduceScalar<WaveOpMax<T>, T>(mask, val);
}

// Fast-path specializations when CUDA warp reduce operators are available
#if __CUDA_ARCH__ >= 800 // 8.x or higher
template<>
__inline__ __device__ unsigned _waveOr<unsigned>(WarpMask mask, unsigned val)
{
    return __reduce_or_sync(mask, val);
}

template<>
__inline__ __device__ unsigned _waveAnd<unsigned>(WarpMask mask, unsigned val)
{
    return __reduce_and_sync(mask, val);
}

template<>
__inline__ __device__ unsigned _waveXor<unsigned>(WarpMask mask, unsigned val)
{
    return __reduce_xor_sync(mask, val);
}

template<>
__inline__ __device__ unsigned _waveSum<unsigned>(WarpMask mask, unsigned val)
{
    return __reduce_add_sync(mask, val);
}

template<>
__inline__ __device__ int _waveSum<int>(WarpMask mask, int val)
{
    return __reduce_add_sync(mask, val);
}

template<>
__inline__ __device__ unsigned _waveMin<unsigned>(WarpMask mask, unsigned val)
{
    return __reduce_min_sync(mask, val);
}

template<>
__inline__ __device__ int _waveMin<int>(WarpMask mask, int val)
{
    return __reduce_min_sync(mask, val);
}

template<>
__inline__ __device__ unsigned _waveMax<unsigned>(WarpMask mask, unsigned val)
{
    return __reduce_max_sync(mask, val);
}

template<>
__inline__ __device__ int _waveMax<int>(WarpMask mask, int val)
{
    return __reduce_max_sync(mask, val);
}
#endif


// Multiple

template<typename T>
__inline__ __device__ T _waveOrMultiple(WarpMask mask, T val)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    _waveReduceMultiple<WaveOpOr<ElemType>>(mask, &val);
    return val;
}

template<typename T>
__inline__ __device__ T _waveAndMultiple(WarpMask mask, T val)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    _waveReduceMultiple<WaveOpAnd<ElemType>>(mask, &val);
    return val;
}

template<typename T>
__inline__ __device__ T _waveXorMultiple(WarpMask mask, T val)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    _waveReduceMultiple<WaveOpXor<ElemType>>(mask, &val);
    return val;
}

template<typename T>
__inline__ __device__ T _waveProductMultiple(WarpMask mask, T val)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    _waveReduceMultiple<WaveOpMul<ElemType>>(mask, &val);
    return val;
}

template<typename T>
__inline__ __device__ T _waveSumMultiple(WarpMask mask, T val)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    _waveReduceMultiple<WaveOpAdd<ElemType>>(mask, &val);
    return val;
}

template<typename T>
__inline__ __device__ T _waveMinMultiple(WarpMask mask, T val)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    _waveReduceMultiple<WaveOpMin<ElemType>>(mask, &val);
    return val;
}

template<typename T>
__inline__ __device__ T _waveMaxMultiple(WarpMask mask, T val)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    _waveReduceMultiple<WaveOpMax<ElemType>>(mask, &val);
    return val;
}


template<typename T>
__inline__ __device__ bool _waveAllEqual(WarpMask mask, T val)
{
    int pred;
    __match_all_sync(mask, val, &pred);
    return pred != 0;
}

template<typename T>
__inline__ __device__ bool _waveAllEqualMultiple(WarpMask mask, T inVal)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    const size_t count = sizeof(T) / sizeof(ElemType);
    int pred;
    const ElemType* src = (const ElemType*)&inVal;
    for (size_t i = 0; i < count; ++i)
    {
        __match_all_sync(mask, src[i], &pred);
        if (pred == 0)
        {
            return false;
        }
    }
    return true;
}

template<typename T>
__inline__ __device__ T _waveReadFirst(WarpMask mask, T val)
{
    const int lowestLaneId = __ffs(mask) - 1;
    return __shfl_sync(mask, val, lowestLaneId);
}

template<typename T>
__inline__ __device__ T _waveReadFirstMultiple(WarpMask mask, T inVal)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    const size_t count = sizeof(T) / sizeof(ElemType);
    T outVal;
    const ElemType* src = (const ElemType*)&inVal;
    ElemType* dst = (ElemType*)&outVal;
    const int lowestLaneId = __ffs(mask) - 1;
    for (size_t i = 0; i < count; ++i)
    {
        dst[i] = __shfl_sync(mask, src[i], lowestLaneId);
    }
    return outVal;
}

template<typename T>
__inline__ __device__ T _waveShuffleMultiple(WarpMask mask, T inVal, int lane)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    const size_t count = sizeof(T) / sizeof(ElemType);
    T outVal;
    const ElemType* src = (const ElemType*)&inVal;
    ElemType* dst = (ElemType*)&outVal;
    for (size_t i = 0; i < count; ++i)
    {
        dst[i] = __shfl_sync(mask, src[i], lane);
    }
    return outVal;
}

// Scalar

// Invertable means that when we get to the end of the reduce, we can remove val (to make
// exclusive), using the inverse of the op.
template<typename INTF, typename T>
__device__ T _wavePrefixInvertableScalar(WarpMask mask, T val)
{
    const int offsetSize = _waveCalcPow2Offset(mask);

    const int laneId = _getLaneId();
    T result;
    if (offsetSize > 0)
    {
        // Sum is calculated inclusive of this lanes value
        result = val;
        for (int i = 1; i < offsetSize; i += i)
        {
            const T readVal = __shfl_up_sync(mask, result, i, offsetSize);
            if (laneId >= i)
            {
                result = INTF::doOp(result, readVal);
            }
        }
        // Remove val from the result, by applyin inverse
        result = INTF::doInverse(result, val);
    }
    else
    {
        result = INTF::getInitial(val);
        if (!_waveIsSingleLane(mask))
        {
            int remaining = mask;
            while (remaining)
            {
                const int laneBit = remaining & -remaining;
                // Get the sourceLane
                const int srcLane = __ffs(laneBit) - 1;
                // Broadcast (can also broadcast to self)
                const T readValue = __shfl_sync(mask, val, srcLane);
                // Only accumulate if srcLane is less than this lane
                if (srcLane < laneId)
                {
                    result = INTF::doOp(result, readValue);
                }
                remaining &= ~laneBit;
            }
        }
    }
    return result;
}


// This implementation separately tracks the value to be propogated, and the value
// that is the final result
template<typename INTF, typename T>
__device__ T _wavePrefixScalar(WarpMask mask, T val)
{
    const int offsetSize = _waveCalcPow2Offset(mask);

    const int laneId = _getLaneId();
    T result = INTF::getInitial(val);
    if (offsetSize > 0)
    {
        // For transmitted value we will do it inclusively with this lanes value
        // For the result we do not include the lanes value. This means an extra multiply for each
        // iteration but means we don't need to have a divide at the end and also removes overflow
        // issues in that scenario.
        for (int i = 1; i < offsetSize; i += i)
        {
            const T readVal = __shfl_up_sync(mask, val, i, offsetSize);
            if (laneId >= i)
            {
                result = INTF::doOp(result, readVal);
                val = INTF::doOp(val, readVal);
            }
        }
    }
    else
    {
        if (!_waveIsSingleLane(mask))
        {
            int remaining = mask;
            while (remaining)
            {
                const int laneBit = remaining & -remaining;
                // Get the sourceLane
                const int srcLane = __ffs(laneBit) - 1;
                // Broadcast (can also broadcast to self)
                const T readValue = __shfl_sync(mask, val, srcLane);
                // Only accumulate if srcLane is less than this lane
                if (srcLane < laneId)
                {
                    result = INTF::doOp(result, readValue);
                }
                remaining &= ~laneBit;
            }
        }
    }
    return result;
}


template<typename INTF, typename T, size_t COUNT>
__device__ T _waveOpCopy(T* dst, const T* src)
{
    for (size_t j = 0; j < COUNT; ++j)
    {
        dst[j] = src[j];
    }
}


template<typename INTF, typename T, size_t COUNT>
__device__ T _waveOpDoInverse(T* inOut, const T* val)
{
    for (size_t j = 0; j < COUNT; ++j)
    {
        inOut[j] = INTF::doInverse(inOut[j], val[j]);
    }
}

template<typename INTF, typename T, size_t COUNT>
__device__ T _waveOpSetInitial(T* out, const T* val)
{
    for (size_t j = 0; j < COUNT; ++j)
    {
        out[j] = INTF::getInitial(val[j]);
    }
}

template<typename INTF, typename T, size_t COUNT>
__device__ T _wavePrefixInvertableMultiple(WarpMask mask, T* val)
{
    const int offsetSize = _waveCalcPow2Offset(mask);

    const int laneId = _getLaneId();
    T originalVal[COUNT];
    _waveOpCopy<INTF, T, COUNT>(originalVal, val);

    if (offsetSize > 0)
    {
        // Sum is calculated inclusive of this lanes value
        for (int i = 1; i < offsetSize; i += i)
        {
            // TODO(JS): Note that here I don't split the laneId outside so it's only tested once.
            // This may be better but it would also mean that there would be shfl between lanes
            // that are on different (albeit identical) instructions. So this seems more likely to
            // work as expected with everything in lock step.
            for (size_t j = 0; j < COUNT; ++j)
            {
                const T readVal = __shfl_up_sync(mask, val[j], i, offsetSize);
                if (laneId >= i)
                {
                    val[j] = INTF::doOp(val[j], readVal);
                }
            }
        }
        // Remove originalVal from the result, by applyin inverse
        _waveOpDoInverse<INTF, T, COUNT>(val, originalVal);
    }
    else
    {
        _waveOpSetInitial<INTF, T, COUNT>(val, val);
        if (!_waveIsSingleLane(mask))
        {
            int remaining = mask;
            while (remaining)
            {
                const int laneBit = remaining & -remaining;
                // Get the sourceLane
                const int srcLane = __ffs(laneBit) - 1;

                for (size_t j = 0; j < COUNT; ++j)
                {
                    // Broadcast (can also broadcast to self)
                    const T readValue = __shfl_sync(mask, originalVal[j], srcLane);
                    // Only accumulate if srcLane is less than this lane
                    if (srcLane < laneId)
                    {
                        val[j] = INTF::doOp(val[j], readValue);
                    }
                    remaining &= ~laneBit;
                }
            }
        }
    }
}

template<typename INTF, typename T, size_t COUNT>
__device__ T _wavePrefixMultiple(WarpMask mask, T* val)
{
    const int offsetSize = _waveCalcPow2Offset(mask);

    const int laneId = _getLaneId();

    T work[COUNT];
    _waveOpCopy<INTF, T, COUNT>(work, val);
    _waveOpSetInitial<INTF, T, COUNT>(val, val);

    if (offsetSize > 0)
    {
        // For transmitted value we will do it inclusively with this lanes value
        // For the result we do not include the lanes value. This means an extra op for each
        // iteration but means we don't need to have a divide at the end and also removes overflow
        // issues in that scenario.
        for (int i = 1; i < offsetSize; i += i)
        {
            for (size_t j = 0; j < COUNT; ++j)
            {
                const T readVal = __shfl_up_sync(mask, work[j], i, offsetSize);
                if (laneId >= i)
                {
                    work[j] = INTF::doOp(work[j], readVal);
                    val[j] = INTF::doOp(val[j], readVal);
                }
            }
        }
    }
    else
    {
        if (!_waveIsSingleLane(mask))
        {
            int remaining = mask;
            while (remaining)
            {
                const int laneBit = remaining & -remaining;
                // Get the sourceLane
                const int srcLane = __ffs(laneBit) - 1;

                for (size_t j = 0; j < COUNT; ++j)
                {
                    // Broadcast (can also broadcast to self)
                    const T readValue = __shfl_sync(mask, work[j], srcLane);
                    // Only accumulate if srcLane is less than this lane
                    if (srcLane < laneId)
                    {
                        val[j] = INTF::doOp(val[j], readValue);
                    }
                }
                remaining &= ~laneBit;
            }
        }
    }
}

template<typename T>
__inline__ __device__ T _wavePrefixProduct(WarpMask mask, T val)
{
    return _wavePrefixScalar<WaveOpMul<T>, T>(mask, val);
}

template<typename T>
__inline__ __device__ T _wavePrefixSum(WarpMask mask, T val)
{
    return _wavePrefixInvertableScalar<WaveOpAdd<T>, T>(mask, val);
}

template<typename T>
__inline__ __device__ T _wavePrefixXor(WarpMask mask, T val)
{
    return _wavePrefixInvertableScalar<WaveOpXor<T>, T>(mask, val);
}

template<typename T>
__inline__ __device__ T _wavePrefixOr(WarpMask mask, T val)
{
    return _wavePrefixScalar<WaveOpOr<T>, T>(mask, val);
}

template<typename T>
__inline__ __device__ T _wavePrefixAnd(WarpMask mask, T val)
{
    return _wavePrefixScalar<WaveOpAnd<T>, T>(mask, val);
}


template<typename T>
__inline__ __device__ T _wavePrefixProductMultiple(WarpMask mask, T val)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    _wavePrefixInvertableMultiple<WaveOpMul<ElemType>, ElemType, sizeof(T) / sizeof(ElemType)>(
        mask,
        (ElemType*)&val);
    return val;
}

template<typename T>
__inline__ __device__ T _wavePrefixSumMultiple(WarpMask mask, T val)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    _wavePrefixInvertableMultiple<WaveOpAdd<ElemType>, ElemType, sizeof(T) / sizeof(ElemType)>(
        mask,
        (ElemType*)&val);
    return val;
}

template<typename T>
__inline__ __device__ T _wavePrefixXorMultiple(WarpMask mask, T val)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    _wavePrefixInvertableMultiple<WaveOpXor<ElemType>, ElemType, sizeof(T) / sizeof(ElemType)>(
        mask,
        (ElemType*)&val);
    return val;
}

template<typename T>
__inline__ __device__ T _wavePrefixOrMultiple(WarpMask mask, T val)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    _wavePrefixMultiple<WaveOpOr<ElemType>, ElemType, sizeof(T) / sizeof(ElemType)>(
        mask,
        (ElemType*)&val);
    return val;
}

template<typename T>
__inline__ __device__ T _wavePrefixAndMultiple(WarpMask mask, T val)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    _wavePrefixMultiple<WaveOpAnd<ElemType>, ElemType, sizeof(T) / sizeof(ElemType)>(
        mask,
        (ElemType*)&val);
    return val;
}

template<typename T>
__inline__ __device__ uint4 _waveMatchScalar(WarpMask mask, T val)
{
    int pred;
    return make_uint4(__match_all_sync(mask, val, &pred), 0, 0, 0);
}

template<typename T>
__inline__ __device__ uint4 _waveMatchMultiple(WarpMask mask, const T& inVal)
{
    typedef typename ElementTypeTrait<T>::Type ElemType;
    const size_t count = sizeof(T) / sizeof(ElemType);
    int pred;
    const ElemType* src = (const ElemType*)&inVal;
    uint matchBits = 0xffffffff;
    for (size_t i = 0; i < count && matchBits; ++i)
    {
        matchBits = matchBits & __match_all_sync(mask, src[i], &pred);
    }
    return make_uint4(matchBits, 0, 0, 0);
}

__device__ uint getAt(dim3 a, int b)
{
    SLANG_PRELUDE_ASSERT(b >= 0 && b < 3);
    return (&a.x)[b];
}
__device__ uint3 operator*(uint3 a, dim3 b)
{
    uint3 r;
    r.x = a.x * b.x;
    r.y = a.y * b.y;
    r.z = a.z * b.z;
    return r;
}

template<typename TResult, typename TInput>
__inline__ __device__ TResult slang_bit_cast(TInput val)
{
    return *(TResult*)(&val);
}

/* !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! */


/* Type that defines the uniform entry point params. The actual content of this type is dependent on
the entry point parameters, and can be found via reflection or defined such that it matches the
shader appropriately.
*/
struct UniformEntryPointParams;
struct UniformState;

// ---------------------- OptiX Ray Payload --------------------------------------
#ifdef SLANG_CUDA_ENABLE_OPTIX
struct RayDesc
{
    float3 Origin;
    float TMin;
    float3 Direction;
    float TMax;
};

static __forceinline__ __device__ void* unpackOptiXRayPayloadPointer(uint32_t i0, uint32_t i1)
{
    const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__ void packOptiXRayPayloadPointer(
    void* ptr,
    uint32_t& i0,
    uint32_t& i1)
{
    const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

static __forceinline__ __device__ void* getOptiXRayPayloadPtr()
{
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return unpackOptiXRayPayloadPointer(u0, u1);
}

template<typename T>
__forceinline__ __device__ void* traceOptiXRay(
    OptixTraversableHandle AccelerationStructure,
    uint32_t RayFlags,
    uint32_t InstanceInclusionMask,
    uint32_t RayContributionToHitGroupIndex,
    uint32_t MultiplierForGeometryContributionToHitGroupIndex,
    uint32_t MissShaderIndex,
    RayDesc Ray,
    T* Payload)
{
    uint32_t r0, r1;
    packOptiXRayPayloadPointer((void*)Payload, r0, r1);
    optixTrace(
        AccelerationStructure,
        Ray.Origin,
        Ray.Direction,
        Ray.TMin,
        Ray.TMax,
        0.f, /* Time for motion blur, currently unsupported in slang */
        InstanceInclusionMask,
        RayFlags,
        RayContributionToHitGroupIndex,
        MultiplierForGeometryContributionToHitGroupIndex,
        MissShaderIndex,
        r0,
        r1);
}

#endif

static const int kSlangTorchTensorMaxDim = 5;

// TensorView
struct TensorView
{
    uint8_t* data;
    uint32_t strides[kSlangTorchTensorMaxDim];
    uint32_t sizes[kSlangTorchTensorMaxDim];
    uint32_t dimensionCount;

    template<typename T>
    __device__ T* data_ptr()
    {
        return reinterpret_cast<T*>(data);
    }

    template<typename T>
    __device__ T* data_ptr_at(uint32_t index)
    {
        uint64_t offset = strides[0] * index;
        return reinterpret_cast<T*>(data + offset);
    }

    template<typename T>
    __device__ T* data_ptr_at(uint2 index)
    {
        uint64_t offset = strides[0] * index.x + strides[1] * index.y;
        return reinterpret_cast<T*>(data + offset);
    }

    template<typename T>
    __device__ T* data_ptr_at(uint3 index)
    {
        uint64_t offset = strides[0] * index.x + strides[1] * index.y + strides[2] * index.z;
        return reinterpret_cast<T*>(data + offset);
    }

    template<typename T>
    __device__ T* data_ptr_at(uint4 index)
    {
        uint64_t offset = strides[0] * index.x + strides[1] * index.y + strides[2] * index.z +
                          strides[3] * index.w;
        return reinterpret_cast<T*>(data + offset);
    }

    template<typename T, unsigned int N>
    __device__ T* data_ptr_at(uint index[N])
    {
        uint64_t offset = 0;
        for (unsigned int i = 0; i < N; ++i)
        {
            offset += strides[i] * index[i];
        }
        return reinterpret_cast<T*>(data + offset);
    }

    template<typename T>
    __device__ T& load(uint32_t x)
    {
        return *reinterpret_cast<T*>(data + strides[0] * x);
    }
    template<typename T>
    __device__ T& load(uint32_t x, uint32_t y)
    {
        return *reinterpret_cast<T*>(data + strides[0] * x + strides[1] * y);
    }
    template<typename T>
    __device__ T& load(uint2 index)
    {
        return *reinterpret_cast<T*>(data + strides[0] * index.x + strides[1] * index.y);
    }
    template<typename T>
    __device__ T& load(uint32_t x, uint32_t y, uint32_t z)
    {
        return *reinterpret_cast<T*>(data + strides[0] * x + strides[1] * y + strides[2] * z);
    }
    template<typename T>
    __device__ T& load(uint3 index)
    {
        return *reinterpret_cast<T*>(
            data + strides[0] * index.x + strides[1] * index.y + strides[2] * index.z);
    }
    template<typename T>
    __device__ T& load(uint32_t x, uint32_t y, uint32_t z, uint32_t w)
    {
        return *reinterpret_cast<T*>(
            data + strides[0] * x + strides[1] * y + strides[2] * z + strides[3] * w);
    }
    template<typename T>
    __device__ T& load(uint4 index)
    {
        return *reinterpret_cast<T*>(
            data + strides[0] * index.x + strides[1] * index.y + strides[2] * index.z +
            strides[3] * index.w);
    }
    template<typename T>
    __device__ T& load(uint32_t i0, uint32_t i1, uint32_t i2, uint32_t i3, uint32_t i4)
    {
        return *reinterpret_cast<T*>(
            data + strides[0] * i0 + strides[1] * i1 + strides[2] * i2 + strides[3] * i3 +
            strides[4] * i4);
    }

    // Generic version of load
    template<typename T, unsigned int N>
    __device__ T& load(uint index[N])
    {
        uint64_t offset = 0;
        for (unsigned int i = 0; i < N; ++i)
        {
            offset += strides[i] * index[i];
        }
        return *reinterpret_cast<T*>(data + offset);
    }

    template<typename T>
    __device__ void store(uint32_t x, T val)
    {
        *reinterpret_cast<T*>(data + strides[0] * x) = val;
    }
    template<typename T>
    __device__ void store(uint32_t x, uint32_t y, T val)
    {
        *reinterpret_cast<T*>(data + strides[0] * x + strides[1] * y) = val;
    }
    template<typename T>
    __device__ void store(uint2 index, T val)
    {
        *reinterpret_cast<T*>(data + strides[0] * index.x + strides[1] * index.y) = val;
    }
    template<typename T>
    __device__ void store(uint32_t x, uint32_t y, uint32_t z, T val)
    {
        *reinterpret_cast<T*>(data + strides[0] * x + strides[1] * y + strides[2] * z) = val;
    }
    template<typename T>
    __device__ void store(uint3 index, T val)
    {
        *reinterpret_cast<T*>(
            data + strides[0] * index.x + strides[1] * index.y + strides[2] * index.z) = val;
    }
    template<typename T>
    __device__ void store(uint32_t x, uint32_t y, uint32_t z, uint32_t w, T val)
    {
        *reinterpret_cast<T*>(
            data + strides[0] * x + strides[1] * y + strides[2] * z + strides[3] * w) = val;
    }
    template<typename T>
    __device__ void store(uint4 index, T val)
    {
        *reinterpret_cast<T*>(
            data + strides[0] * index.x + strides[1] * index.y + strides[2] * index.z +
            strides[3] * index.w) = val;
    }
    template<typename T>
    __device__ void store(uint32_t i0, uint32_t i1, uint32_t i2, uint32_t i3, uint32_t i4, T val)
    {
        *reinterpret_cast<T*>(
            data + strides[0] * i0 + strides[1] * i1 + strides[2] * i2 + strides[3] * i3 +
            strides[4] * i4) = val;
    }

    // Generic version
    template<typename T, unsigned int N>
    __device__ void store(uint index[N], T val)
    {
        uint64_t offset = 0;
        for (unsigned int i = 0; i < N; ++i)
        {
            offset += strides[i] * index[i];
        }
        *reinterpret_cast<T*>(data + offset) = val;
    }
};


#line 694 "diff.meta.slang"
struct AtomicAdd_0
{
    TensorView diff_0;
};


#line 707
__device__ float AtomicAdd_load_forward_0(AtomicAdd_0 this_0, uint2  i_0)
{
    float _S1 = ((this_0.diff_0).load<float>((i_0)));

#line 709
    return _S1;
}


#line 707
__device__ float AtomicAdd_load_forward_1(AtomicAdd_0 this_1, uint3  i_1)
{
    float _S2 = ((this_1.diff_0).load<float>((i_1)));

#line 709
    return _S2;
}


#line 720
__device__ void AtomicAdd_load_backward_0(AtomicAdd_0 this_2, uint2  i_2, float dOut_0)
{
    float oldVal_0;
    *((&oldVal_0)) = atomicAdd((this_2.diff_0).data_ptr_at<float>((i_2)), (dOut_0));
    return;
}


#line 720
__device__ void AtomicAdd_load_backward_1(AtomicAdd_0 this_3, uint3  i_3, float dOut_1)
{
    float oldVal_1;
    *((&oldVal_1)) = atomicAdd((this_3.diff_0).data_ptr_at<float>((i_3)), (dOut_1));
    return;
}


#line 790
__device__ void AtomicAdd_storeOnce_forward_0(AtomicAdd_0 this_4, uint2  i_4, float dx_0)
{
    (this_4.diff_0).store<float>((i_4), (dx_0));
    return;
}


#line 790
__device__ void AtomicAdd_storeOnce_forward_1(AtomicAdd_0 this_5, uint3  i_5, float dx_1)
{
    (this_5.diff_0).store<float>((i_5), (dx_1));
    return;
}


#line 802
__device__ float AtomicAdd_storeOnce_backward_0(AtomicAdd_0 this_6, uint2  i_6)
{
    float _S3 = ((this_6.diff_0).load<float>((i_6)));

#line 804
    return _S3;
}


#line 802
__device__ float AtomicAdd_storeOnce_backward_1(AtomicAdd_0 this_7, uint3  i_7)
{
    float _S4 = ((this_7.diff_0).load<float>((i_7)));

#line 804
    return _S4;
}


#line 78 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
struct Camera_Differential_0
{
    Matrix<float, 4, 4>  world_view_transform_0;
    Matrix<float, 4, 4>  proj_mat_0;
    float3  position_0;
    float fovy_0;
    float fovx_0;
};


#line 1 "token paste"
__device__ Camera_Differential_0 Camera_x24_syn_dzero_0()
{

#line 1
    Camera_Differential_0 result_0;

#line 1805 "core.meta.slang"
    Matrix<float, 4, 4>  _S5 = makeMatrix<float, 4, 4> (0.0f);

#line 1805
    (&result_0)->world_view_transform_0 = _S5;

#line 1805
    (&result_0)->proj_mat_0 = _S5;

#line 1805
    (&result_0)->position_0 = make_float3 (0.0f);

#line 1805
    (&result_0)->fovy_0 = 0.0f;

#line 1805
    (&result_0)->fovx_0 = 0.0f;

#line 1805
    return result_0;
}


#line 1805
__device__ Camera_Differential_0 Camera_x24_syn_dadd_0(Camera_Differential_0 SLANG_anonymous_0_0, Camera_Differential_0 SLANG_anonymous_1_0)
{

#line 1805
    Camera_Differential_0 result_1;

#line 1805
    (&result_1)->world_view_transform_0 = SLANG_anonymous_0_0.world_view_transform_0 + SLANG_anonymous_1_0.world_view_transform_0;

#line 1805
    (&result_1)->proj_mat_0 = SLANG_anonymous_0_0.proj_mat_0 + SLANG_anonymous_1_0.proj_mat_0;

#line 1805
    (&result_1)->position_0 = SLANG_anonymous_0_0.position_0 + SLANG_anonymous_1_0.position_0;

#line 1805
    (&result_1)->fovy_0 = SLANG_anonymous_0_0.fovy_0 + SLANG_anonymous_1_0.fovy_0;

#line 1805
    (&result_1)->fovx_0 = SLANG_anonymous_0_0.fovx_0 + SLANG_anonymous_1_0.fovx_0;

#line 1805
    return result_1;
}


#line 34 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/spherical_harmonics.slang"
struct SpherHarmCoeffs_0
{
    float3  coeff0_0;
    float3  coeff1_0;
    float3  coeff2_0;
    float3  coeff3_0;
    float3  coeff4_0;
    float3  coeff5_0;
    float3  coeff6_0;
    float3  coeff7_0;
    float3  coeff8_0;
    float3  coeff9_0;
    float3  coeff10_0;
    float3  coeff11_0;
    float3  coeff12_0;
    float3  coeff13_0;
    float3  coeff14_0;
    float3  coeff15_0;
};


#line 34
__device__ SpherHarmCoeffs_0 SpherHarmCoeffs_x24_syn_dzero_0()
{

#line 34
    SpherHarmCoeffs_0 result_2;

#line 1751 "core.meta.slang"
    float3  _S6 = make_float3 (0.0f);

#line 1751
    (&result_2)->coeff0_0 = _S6;

#line 1751
    (&result_2)->coeff1_0 = _S6;

#line 1751
    (&result_2)->coeff2_0 = _S6;

#line 1751
    (&result_2)->coeff3_0 = _S6;

#line 1751
    (&result_2)->coeff4_0 = _S6;

#line 1751
    (&result_2)->coeff5_0 = _S6;

#line 1751
    (&result_2)->coeff6_0 = _S6;

#line 1751
    (&result_2)->coeff7_0 = _S6;

#line 1751
    (&result_2)->coeff8_0 = _S6;

#line 1751
    (&result_2)->coeff9_0 = _S6;

#line 1751
    (&result_2)->coeff10_0 = _S6;

#line 1751
    (&result_2)->coeff11_0 = _S6;

#line 1751
    (&result_2)->coeff12_0 = _S6;

#line 1751
    (&result_2)->coeff13_0 = _S6;

#line 1751
    (&result_2)->coeff14_0 = _S6;

#line 1751
    (&result_2)->coeff15_0 = _S6;

#line 1751
    return result_2;
}


#line 1751
__device__ SpherHarmCoeffs_0 SpherHarmCoeffs_x24_syn_dadd_0(SpherHarmCoeffs_0 SLANG_anonymous_0_1, SpherHarmCoeffs_0 SLANG_anonymous_1_1)
{

#line 1751
    SpherHarmCoeffs_0 result_3;

#line 1751
    (&result_3)->coeff0_0 = SLANG_anonymous_0_1.coeff0_0 + SLANG_anonymous_1_1.coeff0_0;

#line 1751
    (&result_3)->coeff1_0 = SLANG_anonymous_0_1.coeff1_0 + SLANG_anonymous_1_1.coeff1_0;

#line 1751
    (&result_3)->coeff2_0 = SLANG_anonymous_0_1.coeff2_0 + SLANG_anonymous_1_1.coeff2_0;

#line 1751
    (&result_3)->coeff3_0 = SLANG_anonymous_0_1.coeff3_0 + SLANG_anonymous_1_1.coeff3_0;

#line 1751
    (&result_3)->coeff4_0 = SLANG_anonymous_0_1.coeff4_0 + SLANG_anonymous_1_1.coeff4_0;

#line 1751
    (&result_3)->coeff5_0 = SLANG_anonymous_0_1.coeff5_0 + SLANG_anonymous_1_1.coeff5_0;

#line 1751
    (&result_3)->coeff6_0 = SLANG_anonymous_0_1.coeff6_0 + SLANG_anonymous_1_1.coeff6_0;

#line 1751
    (&result_3)->coeff7_0 = SLANG_anonymous_0_1.coeff7_0 + SLANG_anonymous_1_1.coeff7_0;

#line 1751
    (&result_3)->coeff8_0 = SLANG_anonymous_0_1.coeff8_0 + SLANG_anonymous_1_1.coeff8_0;

#line 1751
    (&result_3)->coeff9_0 = SLANG_anonymous_0_1.coeff9_0 + SLANG_anonymous_1_1.coeff9_0;

#line 1751
    (&result_3)->coeff10_0 = SLANG_anonymous_0_1.coeff10_0 + SLANG_anonymous_1_1.coeff10_0;

#line 1751
    (&result_3)->coeff11_0 = SLANG_anonymous_0_1.coeff11_0 + SLANG_anonymous_1_1.coeff11_0;

#line 1751
    (&result_3)->coeff12_0 = SLANG_anonymous_0_1.coeff12_0 + SLANG_anonymous_1_1.coeff12_0;

#line 1751
    (&result_3)->coeff13_0 = SLANG_anonymous_0_1.coeff13_0 + SLANG_anonymous_1_1.coeff13_0;

#line 1751
    (&result_3)->coeff14_0 = SLANG_anonymous_0_1.coeff14_0 + SLANG_anonymous_1_1.coeff14_0;

#line 1751
    (&result_3)->coeff15_0 = SLANG_anonymous_0_1.coeff15_0 + SLANG_anonymous_1_1.coeff15_0;

#line 1751
    return result_3;
}


#line 161 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
struct Gaussian_3D_0
{
    float3  xyz_ws_0;
    SpherHarmCoeffs_0 sh_coeffs_0;
    float4  rotations_0;
    float3  scales_0;
};


#line 1352 "diff.meta.slang"
__device__ Gaussian_3D_0 Gaussian_3D_x24_syn_dzero_0()
{

#line 1352
    Gaussian_3D_0 result_4;

#line 1751 "core.meta.slang"
    float3  _S7 = make_float3 (0.0f);

#line 1751
    (&result_4)->xyz_ws_0 = _S7;

#line 1751
    (&result_4)->sh_coeffs_0 = SpherHarmCoeffs_x24_syn_dzero_0();

#line 1751
    (&result_4)->rotations_0 = make_float4 (0.0f);

#line 1751
    (&result_4)->scales_0 = _S7;

#line 1751
    return result_4;
}


#line 1751
__device__ Gaussian_3D_0 Gaussian_3D_x24_syn_dadd_0(Gaussian_3D_0 SLANG_anonymous_0_2, Gaussian_3D_0 SLANG_anonymous_1_2)
{

#line 1751
    Gaussian_3D_0 result_5;

#line 1751
    (&result_5)->xyz_ws_0 = SLANG_anonymous_0_2.xyz_ws_0 + SLANG_anonymous_1_2.xyz_ws_0;

#line 1751
    (&result_5)->sh_coeffs_0 = SpherHarmCoeffs_x24_syn_dadd_0(SLANG_anonymous_0_2.sh_coeffs_0, SLANG_anonymous_1_2.sh_coeffs_0);

#line 1751
    (&result_5)->rotations_0 = SLANG_anonymous_0_2.rotations_0 + SLANG_anonymous_1_2.rotations_0;

#line 1751
    (&result_5)->scales_0 = SLANG_anonymous_0_2.scales_0 + SLANG_anonymous_1_2.scales_0;

#line 1751
    return result_5;
}


#line 186 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
struct Splat_2D_Vertex_0
{
    float3  xyz_vs_0;
    float3  rgb_0;
    Matrix<float, 2, 2>  cov_vs_0;
};


#line 186
__device__ Splat_2D_Vertex_0 Splat_2D_Vertex_x24_syn_dzero_0()
{

#line 186
    Splat_2D_Vertex_0 result_6;

#line 1751 "core.meta.slang"
    float3  _S8 = make_float3 (0.0f);

#line 1751
    (&result_6)->xyz_vs_0 = _S8;

#line 1751
    (&result_6)->rgb_0 = _S8;

#line 1751
    (&result_6)->cov_vs_0 = makeMatrix<float, 2, 2> (0.0f);

#line 1751
    return result_6;
}


#line 1751
__device__ Splat_2D_Vertex_0 Splat_2D_Vertex_x24_syn_dadd_0(Splat_2D_Vertex_0 SLANG_anonymous_0_3, Splat_2D_Vertex_0 SLANG_anonymous_1_3)
{

#line 1751
    Splat_2D_Vertex_0 result_7;

#line 1751
    (&result_7)->xyz_vs_0 = SLANG_anonymous_0_3.xyz_vs_0 + SLANG_anonymous_1_3.xyz_vs_0;

#line 1751
    (&result_7)->rgb_0 = SLANG_anonymous_0_3.rgb_0 + SLANG_anonymous_1_3.rgb_0;

#line 1751
    (&result_7)->cov_vs_0 = SLANG_anonymous_0_3.cov_vs_0 + SLANG_anonymous_1_3.cov_vs_0;

#line 1751
    return result_7;
}


#line 809 "diff.meta.slang"
struct DiffTensorView_0
{
    TensorView primal_0;
    AtomicAdd_0 diff_1;
};


#line 814
__device__ uint DiffTensorView_size_0(DiffTensorView_0 this_8, uint i_8)
{
    uint _S9 = ((this_8.primal_0).sizes[(i_8)]);

#line 816
    return _S9;
}


#line 78 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
struct Camera_0
{
    Matrix<float, 4, 4>  world_view_transform_1;
    Matrix<float, 4, 4>  proj_mat_1;
    float3  position_1;
    float fovy_1;
    float fovx_1;
    int H_0;
    int W_0;
};

__device__ Camera_0 load_camera_0(TensorView world_view_transform_t_0, TensorView proj_mat_t_0, TensorView position_t_0, float fovy_2, float fovx_2, uint H_1, uint W_1)
{

#line 90
    float _S10 = ((world_view_transform_t_0).load<float>((0U), (0U)));

#line 90
    float _S11 = ((world_view_transform_t_0).load<float>((0U), (1U)));

#line 90
    float _S12 = ((world_view_transform_t_0).load<float>((0U), (2U)));

#line 90
    float _S13 = ((world_view_transform_t_0).load<float>((0U), (3U)));

#line 90
    float _S14 = ((world_view_transform_t_0).load<float>((1U), (0U)));

#line 90
    float _S15 = ((world_view_transform_t_0).load<float>((1U), (1U)));

#line 90
    float _S16 = ((world_view_transform_t_0).load<float>((1U), (2U)));

#line 90
    float _S17 = ((world_view_transform_t_0).load<float>((1U), (3U)));

#line 90
    float _S18 = ((world_view_transform_t_0).load<float>((2U), (0U)));

#line 90
    float _S19 = ((world_view_transform_t_0).load<float>((2U), (1U)));

#line 90
    float _S20 = ((world_view_transform_t_0).load<float>((2U), (2U)));

#line 90
    float _S21 = ((world_view_transform_t_0).load<float>((2U), (3U)));

#line 90
    float _S22 = ((world_view_transform_t_0).load<float>((3U), (0U)));

#line 90
    float _S23 = ((world_view_transform_t_0).load<float>((3U), (1U)));

#line 90
    float _S24 = ((world_view_transform_t_0).load<float>((3U), (2U)));

#line 90
    float _S25 = ((world_view_transform_t_0).load<float>((3U), (3U)));

#line 90
    Matrix<float, 4, 4>  world_view_transform_2 = makeMatrix<float, 4, 4> (_S10, _S11, _S12, _S13, _S14, _S15, _S16, _S17, _S18, _S19, _S20, _S21, _S22, _S23, _S24, _S25);

#line 95
    float _S26 = ((proj_mat_t_0).load<float>((0U), (0U)));

#line 95
    float _S27 = ((proj_mat_t_0).load<float>((0U), (1U)));

#line 95
    float _S28 = ((proj_mat_t_0).load<float>((0U), (2U)));

#line 95
    float _S29 = ((proj_mat_t_0).load<float>((0U), (3U)));

#line 95
    float _S30 = ((proj_mat_t_0).load<float>((1U), (0U)));

#line 95
    float _S31 = ((proj_mat_t_0).load<float>((1U), (1U)));

#line 95
    float _S32 = ((proj_mat_t_0).load<float>((1U), (2U)));

#line 95
    float _S33 = ((proj_mat_t_0).load<float>((1U), (3U)));

#line 95
    float _S34 = ((proj_mat_t_0).load<float>((2U), (0U)));

#line 95
    float _S35 = ((proj_mat_t_0).load<float>((2U), (1U)));

#line 95
    float _S36 = ((proj_mat_t_0).load<float>((2U), (2U)));

#line 95
    float _S37 = ((proj_mat_t_0).load<float>((2U), (3U)));

#line 95
    float _S38 = ((proj_mat_t_0).load<float>((3U), (0U)));

#line 95
    float _S39 = ((proj_mat_t_0).load<float>((3U), (1U)));

#line 95
    float _S40 = ((proj_mat_t_0).load<float>((3U), (2U)));

#line 95
    float _S41 = ((proj_mat_t_0).load<float>((3U), (3U)));

#line 95
    Matrix<float, 4, 4>  proj_mat_2 = makeMatrix<float, 4, 4> (_S26, _S27, _S28, _S29, _S30, _S31, _S32, _S33, _S34, _S35, _S36, _S37, _S38, _S39, _S40, _S41);



    float _S42 = ((position_t_0).load<float>((0U)));

#line 99
    float _S43 = ((position_t_0).load<float>((1U)));

#line 99
    float _S44 = ((position_t_0).load<float>((2U)));

    Camera_0 _S45 = { world_view_transform_2, proj_mat_2, make_float3 (_S42, _S43, _S44), fovy_2, fovx_2, int(H_1), int(W_1) };

#line 101
    return _S45;
}


#line 850 "diff.meta.slang"
__device__ float DiffTensorView_load_0(DiffTensorView_0 this_9, uint2  i_9)
{

#line 850
    float _S46 = ((this_9.primal_0).load<float>((i_9)));

#line 850
    return _S46;
}


#line 850
__device__ float DiffTensorView_load_1(DiffTensorView_0 this_10, uint3  i_10)
{

#line 850
    float _S47 = ((this_10.primal_0).load<float>((i_10)));

#line 850
    return _S47;
}


#line 26 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
__device__ float3  read_t3_float3_0(uint idx_0, DiffTensorView_0 t3_0)
{
    return make_float3 (DiffTensorView_load_0(t3_0, make_uint2 (idx_0, 0U)), DiffTensorView_load_0(t3_0, make_uint2 (idx_0, 1U)), DiffTensorView_load_0(t3_0, make_uint2 (idx_0, 2U)));
}


#line 62 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/spherical_harmonics.slang"
__device__ SpherHarmCoeffs_0 read_spherical_harmonics_coeffs_0(uint g_idx_0, DiffTensorView_0 sh_coeffs_1, uint active_sh_0)
{
    SpherHarmCoeffs_0 g_sh_coeffs_0;
    (&g_sh_coeffs_0)->coeff0_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 0U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 0U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 0U, 2U)));

    if(active_sh_0 > 0U)
    {

#line 68
        (&g_sh_coeffs_0)->coeff1_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 1U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 1U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 1U, 2U)));
        (&g_sh_coeffs_0)->coeff2_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 2U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 2U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 2U, 2U)));
        (&g_sh_coeffs_0)->coeff3_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 3U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 3U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 3U, 2U)));

        if(active_sh_0 > 1U)
        {

#line 73
            (&g_sh_coeffs_0)->coeff4_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 4U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 4U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 4U, 2U)));
            (&g_sh_coeffs_0)->coeff5_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 5U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 5U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 5U, 2U)));
            (&g_sh_coeffs_0)->coeff6_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 6U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 6U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 6U, 2U)));
            (&g_sh_coeffs_0)->coeff7_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 7U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 7U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 7U, 2U)));
            (&g_sh_coeffs_0)->coeff8_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 8U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 8U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 8U, 2U)));

            if(active_sh_0 > 2U)
            {

#line 80
                (&g_sh_coeffs_0)->coeff9_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 9U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 9U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 9U, 2U)));
                (&g_sh_coeffs_0)->coeff10_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 10U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 10U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 10U, 2U)));
                (&g_sh_coeffs_0)->coeff11_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 11U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 11U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 11U, 2U)));
                (&g_sh_coeffs_0)->coeff12_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 12U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 12U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 12U, 2U)));
                (&g_sh_coeffs_0)->coeff13_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 13U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 13U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 13U, 2U)));
                (&g_sh_coeffs_0)->coeff14_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 14U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 14U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 14U, 2U)));
                (&g_sh_coeffs_0)->coeff15_0 = make_float3 (DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 15U, 0U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 15U, 1U)), DiffTensorView_load_1(sh_coeffs_1, make_uint3 (g_idx_0, 15U, 2U)));

#line 79
            }

#line 72
        }

#line 67
    }

#line 90
    return g_sh_coeffs_0;
}


#line 34 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
__device__ float4  read_t4_float4_0(uint idx_1, DiffTensorView_0 t4_0)
{
    return make_float4 (DiffTensorView_load_0(t4_0, make_uint2 (idx_1, 0U)), DiffTensorView_load_0(t4_0, make_uint2 (idx_1, 1U)), DiffTensorView_load_0(t4_0, make_uint2 (idx_1, 2U)), DiffTensorView_load_0(t4_0, make_uint2 (idx_1, 3U)));
}


#line 170
__device__ Gaussian_3D_0 load_gaussian_0(int g_idx_1, DiffTensorView_0 xyz_ws_1, DiffTensorView_0 sh_coeffs_2, DiffTensorView_0 rotations_1, DiffTensorView_0 scales_1, uint active_sh_1)
{

#line 177
    uint _S48 = uint(g_idx_1);

#line 182
    Gaussian_3D_0 _S49 = { read_t3_float3_0(_S48, xyz_ws_1), read_spherical_harmonics_coeffs_0(_S48, sh_coeffs_2, active_sh_1), read_t4_float4_0(_S48, rotations_1), read_t3_float3_0(_S48, scales_1) };

#line 182
    return _S49;
}


#line 182
struct DiffPair_matrixx3Cfloatx2C4x2C4x3E_0
{
    Matrix<float, 4, 4>  primal_1;
    Matrix<float, 4, 4>  differential_0;
};


#line 1386 "diff.meta.slang"
__device__ void mul_0(DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 * left_0, DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 * right_0, Matrix<float, 4, 4>  dOut_2)
{
    Matrix<float, 4, 4>  left_d_result_0;

#line 1393
    *&(((&left_d_result_0)->rows + (int(0)))->x) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(0)))->y) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(0)))->z) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(0)))->w) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(1)))->x) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(1)))->y) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(1)))->z) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(1)))->w) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(2)))->x) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(2)))->y) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(2)))->z) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(2)))->w) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(3)))->x) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(3)))->y) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(3)))->z) = 0.0f;

#line 1393
    *&(((&left_d_result_0)->rows + (int(3)))->w) = 0.0f;

    Matrix<float, 4, 4>  right_d_result_0;

#line 1400
    *&(((&right_d_result_0)->rows + (int(0)))->x) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(0)))->y) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(0)))->z) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(0)))->w) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(1)))->x) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(1)))->y) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(1)))->z) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(1)))->w) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(2)))->x) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(2)))->y) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(2)))->z) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(2)))->w) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(3)))->x) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(3)))->y) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(3)))->z) = 0.0f;

#line 1400
    *&(((&right_d_result_0)->rows + (int(3)))->w) = 0.0f;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->x) = *&(((&left_d_result_0)->rows + (int(0)))->x) + (*right_0).primal_1.rows[int(0)].x * dOut_2.rows[int(0)].x;
    *&(((&right_d_result_0)->rows + (int(0)))->x) = *&(((&right_d_result_0)->rows + (int(0)))->x) + (*left_0).primal_1.rows[int(0)].x * dOut_2.rows[int(0)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->y) = *&(((&left_d_result_0)->rows + (int(0)))->y) + (*right_0).primal_1.rows[int(1)].x * dOut_2.rows[int(0)].x;
    *&(((&right_d_result_0)->rows + (int(1)))->x) = *&(((&right_d_result_0)->rows + (int(1)))->x) + (*left_0).primal_1.rows[int(0)].y * dOut_2.rows[int(0)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->z) = *&(((&left_d_result_0)->rows + (int(0)))->z) + (*right_0).primal_1.rows[int(2)].x * dOut_2.rows[int(0)].x;
    *&(((&right_d_result_0)->rows + (int(2)))->x) = *&(((&right_d_result_0)->rows + (int(2)))->x) + (*left_0).primal_1.rows[int(0)].z * dOut_2.rows[int(0)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->w) = *&(((&left_d_result_0)->rows + (int(0)))->w) + (*right_0).primal_1.rows[int(3)].x * dOut_2.rows[int(0)].x;
    *&(((&right_d_result_0)->rows + (int(3)))->x) = *&(((&right_d_result_0)->rows + (int(3)))->x) + (*left_0).primal_1.rows[int(0)].w * dOut_2.rows[int(0)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->x) = *&(((&left_d_result_0)->rows + (int(0)))->x) + (*right_0).primal_1.rows[int(0)].y * dOut_2.rows[int(0)].y;
    *&(((&right_d_result_0)->rows + (int(0)))->y) = *&(((&right_d_result_0)->rows + (int(0)))->y) + (*left_0).primal_1.rows[int(0)].x * dOut_2.rows[int(0)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->y) = *&(((&left_d_result_0)->rows + (int(0)))->y) + (*right_0).primal_1.rows[int(1)].y * dOut_2.rows[int(0)].y;
    *&(((&right_d_result_0)->rows + (int(1)))->y) = *&(((&right_d_result_0)->rows + (int(1)))->y) + (*left_0).primal_1.rows[int(0)].y * dOut_2.rows[int(0)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->z) = *&(((&left_d_result_0)->rows + (int(0)))->z) + (*right_0).primal_1.rows[int(2)].y * dOut_2.rows[int(0)].y;
    *&(((&right_d_result_0)->rows + (int(2)))->y) = *&(((&right_d_result_0)->rows + (int(2)))->y) + (*left_0).primal_1.rows[int(0)].z * dOut_2.rows[int(0)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->w) = *&(((&left_d_result_0)->rows + (int(0)))->w) + (*right_0).primal_1.rows[int(3)].y * dOut_2.rows[int(0)].y;
    *&(((&right_d_result_0)->rows + (int(3)))->y) = *&(((&right_d_result_0)->rows + (int(3)))->y) + (*left_0).primal_1.rows[int(0)].w * dOut_2.rows[int(0)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->x) = *&(((&left_d_result_0)->rows + (int(0)))->x) + (*right_0).primal_1.rows[int(0)].z * dOut_2.rows[int(0)].z;
    *&(((&right_d_result_0)->rows + (int(0)))->z) = *&(((&right_d_result_0)->rows + (int(0)))->z) + (*left_0).primal_1.rows[int(0)].x * dOut_2.rows[int(0)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->y) = *&(((&left_d_result_0)->rows + (int(0)))->y) + (*right_0).primal_1.rows[int(1)].z * dOut_2.rows[int(0)].z;
    *&(((&right_d_result_0)->rows + (int(1)))->z) = *&(((&right_d_result_0)->rows + (int(1)))->z) + (*left_0).primal_1.rows[int(0)].y * dOut_2.rows[int(0)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->z) = *&(((&left_d_result_0)->rows + (int(0)))->z) + (*right_0).primal_1.rows[int(2)].z * dOut_2.rows[int(0)].z;
    *&(((&right_d_result_0)->rows + (int(2)))->z) = *&(((&right_d_result_0)->rows + (int(2)))->z) + (*left_0).primal_1.rows[int(0)].z * dOut_2.rows[int(0)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->w) = *&(((&left_d_result_0)->rows + (int(0)))->w) + (*right_0).primal_1.rows[int(3)].z * dOut_2.rows[int(0)].z;
    *&(((&right_d_result_0)->rows + (int(3)))->z) = *&(((&right_d_result_0)->rows + (int(3)))->z) + (*left_0).primal_1.rows[int(0)].w * dOut_2.rows[int(0)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->x) = *&(((&left_d_result_0)->rows + (int(0)))->x) + (*right_0).primal_1.rows[int(0)].w * dOut_2.rows[int(0)].w;
    *&(((&right_d_result_0)->rows + (int(0)))->w) = *&(((&right_d_result_0)->rows + (int(0)))->w) + (*left_0).primal_1.rows[int(0)].x * dOut_2.rows[int(0)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->y) = *&(((&left_d_result_0)->rows + (int(0)))->y) + (*right_0).primal_1.rows[int(1)].w * dOut_2.rows[int(0)].w;
    *&(((&right_d_result_0)->rows + (int(1)))->w) = *&(((&right_d_result_0)->rows + (int(1)))->w) + (*left_0).primal_1.rows[int(0)].y * dOut_2.rows[int(0)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->z) = *&(((&left_d_result_0)->rows + (int(0)))->z) + (*right_0).primal_1.rows[int(2)].w * dOut_2.rows[int(0)].w;
    *&(((&right_d_result_0)->rows + (int(2)))->w) = *&(((&right_d_result_0)->rows + (int(2)))->w) + (*left_0).primal_1.rows[int(0)].z * dOut_2.rows[int(0)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(0)))->w) = *&(((&left_d_result_0)->rows + (int(0)))->w) + (*right_0).primal_1.rows[int(3)].w * dOut_2.rows[int(0)].w;
    *&(((&right_d_result_0)->rows + (int(3)))->w) = *&(((&right_d_result_0)->rows + (int(3)))->w) + (*left_0).primal_1.rows[int(0)].w * dOut_2.rows[int(0)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->x) = *&(((&left_d_result_0)->rows + (int(1)))->x) + (*right_0).primal_1.rows[int(0)].x * dOut_2.rows[int(1)].x;
    *&(((&right_d_result_0)->rows + (int(0)))->x) = *&(((&right_d_result_0)->rows + (int(0)))->x) + (*left_0).primal_1.rows[int(1)].x * dOut_2.rows[int(1)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->y) = *&(((&left_d_result_0)->rows + (int(1)))->y) + (*right_0).primal_1.rows[int(1)].x * dOut_2.rows[int(1)].x;
    *&(((&right_d_result_0)->rows + (int(1)))->x) = *&(((&right_d_result_0)->rows + (int(1)))->x) + (*left_0).primal_1.rows[int(1)].y * dOut_2.rows[int(1)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->z) = *&(((&left_d_result_0)->rows + (int(1)))->z) + (*right_0).primal_1.rows[int(2)].x * dOut_2.rows[int(1)].x;
    *&(((&right_d_result_0)->rows + (int(2)))->x) = *&(((&right_d_result_0)->rows + (int(2)))->x) + (*left_0).primal_1.rows[int(1)].z * dOut_2.rows[int(1)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->w) = *&(((&left_d_result_0)->rows + (int(1)))->w) + (*right_0).primal_1.rows[int(3)].x * dOut_2.rows[int(1)].x;
    *&(((&right_d_result_0)->rows + (int(3)))->x) = *&(((&right_d_result_0)->rows + (int(3)))->x) + (*left_0).primal_1.rows[int(1)].w * dOut_2.rows[int(1)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->x) = *&(((&left_d_result_0)->rows + (int(1)))->x) + (*right_0).primal_1.rows[int(0)].y * dOut_2.rows[int(1)].y;
    *&(((&right_d_result_0)->rows + (int(0)))->y) = *&(((&right_d_result_0)->rows + (int(0)))->y) + (*left_0).primal_1.rows[int(1)].x * dOut_2.rows[int(1)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->y) = *&(((&left_d_result_0)->rows + (int(1)))->y) + (*right_0).primal_1.rows[int(1)].y * dOut_2.rows[int(1)].y;
    *&(((&right_d_result_0)->rows + (int(1)))->y) = *&(((&right_d_result_0)->rows + (int(1)))->y) + (*left_0).primal_1.rows[int(1)].y * dOut_2.rows[int(1)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->z) = *&(((&left_d_result_0)->rows + (int(1)))->z) + (*right_0).primal_1.rows[int(2)].y * dOut_2.rows[int(1)].y;
    *&(((&right_d_result_0)->rows + (int(2)))->y) = *&(((&right_d_result_0)->rows + (int(2)))->y) + (*left_0).primal_1.rows[int(1)].z * dOut_2.rows[int(1)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->w) = *&(((&left_d_result_0)->rows + (int(1)))->w) + (*right_0).primal_1.rows[int(3)].y * dOut_2.rows[int(1)].y;
    *&(((&right_d_result_0)->rows + (int(3)))->y) = *&(((&right_d_result_0)->rows + (int(3)))->y) + (*left_0).primal_1.rows[int(1)].w * dOut_2.rows[int(1)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->x) = *&(((&left_d_result_0)->rows + (int(1)))->x) + (*right_0).primal_1.rows[int(0)].z * dOut_2.rows[int(1)].z;
    *&(((&right_d_result_0)->rows + (int(0)))->z) = *&(((&right_d_result_0)->rows + (int(0)))->z) + (*left_0).primal_1.rows[int(1)].x * dOut_2.rows[int(1)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->y) = *&(((&left_d_result_0)->rows + (int(1)))->y) + (*right_0).primal_1.rows[int(1)].z * dOut_2.rows[int(1)].z;
    *&(((&right_d_result_0)->rows + (int(1)))->z) = *&(((&right_d_result_0)->rows + (int(1)))->z) + (*left_0).primal_1.rows[int(1)].y * dOut_2.rows[int(1)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->z) = *&(((&left_d_result_0)->rows + (int(1)))->z) + (*right_0).primal_1.rows[int(2)].z * dOut_2.rows[int(1)].z;
    *&(((&right_d_result_0)->rows + (int(2)))->z) = *&(((&right_d_result_0)->rows + (int(2)))->z) + (*left_0).primal_1.rows[int(1)].z * dOut_2.rows[int(1)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->w) = *&(((&left_d_result_0)->rows + (int(1)))->w) + (*right_0).primal_1.rows[int(3)].z * dOut_2.rows[int(1)].z;
    *&(((&right_d_result_0)->rows + (int(3)))->z) = *&(((&right_d_result_0)->rows + (int(3)))->z) + (*left_0).primal_1.rows[int(1)].w * dOut_2.rows[int(1)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->x) = *&(((&left_d_result_0)->rows + (int(1)))->x) + (*right_0).primal_1.rows[int(0)].w * dOut_2.rows[int(1)].w;
    *&(((&right_d_result_0)->rows + (int(0)))->w) = *&(((&right_d_result_0)->rows + (int(0)))->w) + (*left_0).primal_1.rows[int(1)].x * dOut_2.rows[int(1)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->y) = *&(((&left_d_result_0)->rows + (int(1)))->y) + (*right_0).primal_1.rows[int(1)].w * dOut_2.rows[int(1)].w;
    *&(((&right_d_result_0)->rows + (int(1)))->w) = *&(((&right_d_result_0)->rows + (int(1)))->w) + (*left_0).primal_1.rows[int(1)].y * dOut_2.rows[int(1)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->z) = *&(((&left_d_result_0)->rows + (int(1)))->z) + (*right_0).primal_1.rows[int(2)].w * dOut_2.rows[int(1)].w;
    *&(((&right_d_result_0)->rows + (int(2)))->w) = *&(((&right_d_result_0)->rows + (int(2)))->w) + (*left_0).primal_1.rows[int(1)].z * dOut_2.rows[int(1)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(1)))->w) = *&(((&left_d_result_0)->rows + (int(1)))->w) + (*right_0).primal_1.rows[int(3)].w * dOut_2.rows[int(1)].w;
    *&(((&right_d_result_0)->rows + (int(3)))->w) = *&(((&right_d_result_0)->rows + (int(3)))->w) + (*left_0).primal_1.rows[int(1)].w * dOut_2.rows[int(1)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->x) = *&(((&left_d_result_0)->rows + (int(2)))->x) + (*right_0).primal_1.rows[int(0)].x * dOut_2.rows[int(2)].x;
    *&(((&right_d_result_0)->rows + (int(0)))->x) = *&(((&right_d_result_0)->rows + (int(0)))->x) + (*left_0).primal_1.rows[int(2)].x * dOut_2.rows[int(2)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->y) = *&(((&left_d_result_0)->rows + (int(2)))->y) + (*right_0).primal_1.rows[int(1)].x * dOut_2.rows[int(2)].x;
    *&(((&right_d_result_0)->rows + (int(1)))->x) = *&(((&right_d_result_0)->rows + (int(1)))->x) + (*left_0).primal_1.rows[int(2)].y * dOut_2.rows[int(2)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->z) = *&(((&left_d_result_0)->rows + (int(2)))->z) + (*right_0).primal_1.rows[int(2)].x * dOut_2.rows[int(2)].x;
    *&(((&right_d_result_0)->rows + (int(2)))->x) = *&(((&right_d_result_0)->rows + (int(2)))->x) + (*left_0).primal_1.rows[int(2)].z * dOut_2.rows[int(2)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->w) = *&(((&left_d_result_0)->rows + (int(2)))->w) + (*right_0).primal_1.rows[int(3)].x * dOut_2.rows[int(2)].x;
    *&(((&right_d_result_0)->rows + (int(3)))->x) = *&(((&right_d_result_0)->rows + (int(3)))->x) + (*left_0).primal_1.rows[int(2)].w * dOut_2.rows[int(2)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->x) = *&(((&left_d_result_0)->rows + (int(2)))->x) + (*right_0).primal_1.rows[int(0)].y * dOut_2.rows[int(2)].y;
    *&(((&right_d_result_0)->rows + (int(0)))->y) = *&(((&right_d_result_0)->rows + (int(0)))->y) + (*left_0).primal_1.rows[int(2)].x * dOut_2.rows[int(2)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->y) = *&(((&left_d_result_0)->rows + (int(2)))->y) + (*right_0).primal_1.rows[int(1)].y * dOut_2.rows[int(2)].y;
    *&(((&right_d_result_0)->rows + (int(1)))->y) = *&(((&right_d_result_0)->rows + (int(1)))->y) + (*left_0).primal_1.rows[int(2)].y * dOut_2.rows[int(2)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->z) = *&(((&left_d_result_0)->rows + (int(2)))->z) + (*right_0).primal_1.rows[int(2)].y * dOut_2.rows[int(2)].y;
    *&(((&right_d_result_0)->rows + (int(2)))->y) = *&(((&right_d_result_0)->rows + (int(2)))->y) + (*left_0).primal_1.rows[int(2)].z * dOut_2.rows[int(2)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->w) = *&(((&left_d_result_0)->rows + (int(2)))->w) + (*right_0).primal_1.rows[int(3)].y * dOut_2.rows[int(2)].y;
    *&(((&right_d_result_0)->rows + (int(3)))->y) = *&(((&right_d_result_0)->rows + (int(3)))->y) + (*left_0).primal_1.rows[int(2)].w * dOut_2.rows[int(2)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->x) = *&(((&left_d_result_0)->rows + (int(2)))->x) + (*right_0).primal_1.rows[int(0)].z * dOut_2.rows[int(2)].z;
    *&(((&right_d_result_0)->rows + (int(0)))->z) = *&(((&right_d_result_0)->rows + (int(0)))->z) + (*left_0).primal_1.rows[int(2)].x * dOut_2.rows[int(2)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->y) = *&(((&left_d_result_0)->rows + (int(2)))->y) + (*right_0).primal_1.rows[int(1)].z * dOut_2.rows[int(2)].z;
    *&(((&right_d_result_0)->rows + (int(1)))->z) = *&(((&right_d_result_0)->rows + (int(1)))->z) + (*left_0).primal_1.rows[int(2)].y * dOut_2.rows[int(2)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->z) = *&(((&left_d_result_0)->rows + (int(2)))->z) + (*right_0).primal_1.rows[int(2)].z * dOut_2.rows[int(2)].z;
    *&(((&right_d_result_0)->rows + (int(2)))->z) = *&(((&right_d_result_0)->rows + (int(2)))->z) + (*left_0).primal_1.rows[int(2)].z * dOut_2.rows[int(2)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->w) = *&(((&left_d_result_0)->rows + (int(2)))->w) + (*right_0).primal_1.rows[int(3)].z * dOut_2.rows[int(2)].z;
    *&(((&right_d_result_0)->rows + (int(3)))->z) = *&(((&right_d_result_0)->rows + (int(3)))->z) + (*left_0).primal_1.rows[int(2)].w * dOut_2.rows[int(2)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->x) = *&(((&left_d_result_0)->rows + (int(2)))->x) + (*right_0).primal_1.rows[int(0)].w * dOut_2.rows[int(2)].w;
    *&(((&right_d_result_0)->rows + (int(0)))->w) = *&(((&right_d_result_0)->rows + (int(0)))->w) + (*left_0).primal_1.rows[int(2)].x * dOut_2.rows[int(2)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->y) = *&(((&left_d_result_0)->rows + (int(2)))->y) + (*right_0).primal_1.rows[int(1)].w * dOut_2.rows[int(2)].w;
    *&(((&right_d_result_0)->rows + (int(1)))->w) = *&(((&right_d_result_0)->rows + (int(1)))->w) + (*left_0).primal_1.rows[int(2)].y * dOut_2.rows[int(2)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->z) = *&(((&left_d_result_0)->rows + (int(2)))->z) + (*right_0).primal_1.rows[int(2)].w * dOut_2.rows[int(2)].w;
    *&(((&right_d_result_0)->rows + (int(2)))->w) = *&(((&right_d_result_0)->rows + (int(2)))->w) + (*left_0).primal_1.rows[int(2)].z * dOut_2.rows[int(2)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(2)))->w) = *&(((&left_d_result_0)->rows + (int(2)))->w) + (*right_0).primal_1.rows[int(3)].w * dOut_2.rows[int(2)].w;
    *&(((&right_d_result_0)->rows + (int(3)))->w) = *&(((&right_d_result_0)->rows + (int(3)))->w) + (*left_0).primal_1.rows[int(2)].w * dOut_2.rows[int(2)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->x) = *&(((&left_d_result_0)->rows + (int(3)))->x) + (*right_0).primal_1.rows[int(0)].x * dOut_2.rows[int(3)].x;
    *&(((&right_d_result_0)->rows + (int(0)))->x) = *&(((&right_d_result_0)->rows + (int(0)))->x) + (*left_0).primal_1.rows[int(3)].x * dOut_2.rows[int(3)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->y) = *&(((&left_d_result_0)->rows + (int(3)))->y) + (*right_0).primal_1.rows[int(1)].x * dOut_2.rows[int(3)].x;
    *&(((&right_d_result_0)->rows + (int(1)))->x) = *&(((&right_d_result_0)->rows + (int(1)))->x) + (*left_0).primal_1.rows[int(3)].y * dOut_2.rows[int(3)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->z) = *&(((&left_d_result_0)->rows + (int(3)))->z) + (*right_0).primal_1.rows[int(2)].x * dOut_2.rows[int(3)].x;
    *&(((&right_d_result_0)->rows + (int(2)))->x) = *&(((&right_d_result_0)->rows + (int(2)))->x) + (*left_0).primal_1.rows[int(3)].z * dOut_2.rows[int(3)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->w) = *&(((&left_d_result_0)->rows + (int(3)))->w) + (*right_0).primal_1.rows[int(3)].x * dOut_2.rows[int(3)].x;
    *&(((&right_d_result_0)->rows + (int(3)))->x) = *&(((&right_d_result_0)->rows + (int(3)))->x) + (*left_0).primal_1.rows[int(3)].w * dOut_2.rows[int(3)].x;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->x) = *&(((&left_d_result_0)->rows + (int(3)))->x) + (*right_0).primal_1.rows[int(0)].y * dOut_2.rows[int(3)].y;
    *&(((&right_d_result_0)->rows + (int(0)))->y) = *&(((&right_d_result_0)->rows + (int(0)))->y) + (*left_0).primal_1.rows[int(3)].x * dOut_2.rows[int(3)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->y) = *&(((&left_d_result_0)->rows + (int(3)))->y) + (*right_0).primal_1.rows[int(1)].y * dOut_2.rows[int(3)].y;
    *&(((&right_d_result_0)->rows + (int(1)))->y) = *&(((&right_d_result_0)->rows + (int(1)))->y) + (*left_0).primal_1.rows[int(3)].y * dOut_2.rows[int(3)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->z) = *&(((&left_d_result_0)->rows + (int(3)))->z) + (*right_0).primal_1.rows[int(2)].y * dOut_2.rows[int(3)].y;
    *&(((&right_d_result_0)->rows + (int(2)))->y) = *&(((&right_d_result_0)->rows + (int(2)))->y) + (*left_0).primal_1.rows[int(3)].z * dOut_2.rows[int(3)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->w) = *&(((&left_d_result_0)->rows + (int(3)))->w) + (*right_0).primal_1.rows[int(3)].y * dOut_2.rows[int(3)].y;
    *&(((&right_d_result_0)->rows + (int(3)))->y) = *&(((&right_d_result_0)->rows + (int(3)))->y) + (*left_0).primal_1.rows[int(3)].w * dOut_2.rows[int(3)].y;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->x) = *&(((&left_d_result_0)->rows + (int(3)))->x) + (*right_0).primal_1.rows[int(0)].z * dOut_2.rows[int(3)].z;
    *&(((&right_d_result_0)->rows + (int(0)))->z) = *&(((&right_d_result_0)->rows + (int(0)))->z) + (*left_0).primal_1.rows[int(3)].x * dOut_2.rows[int(3)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->y) = *&(((&left_d_result_0)->rows + (int(3)))->y) + (*right_0).primal_1.rows[int(1)].z * dOut_2.rows[int(3)].z;
    *&(((&right_d_result_0)->rows + (int(1)))->z) = *&(((&right_d_result_0)->rows + (int(1)))->z) + (*left_0).primal_1.rows[int(3)].y * dOut_2.rows[int(3)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->z) = *&(((&left_d_result_0)->rows + (int(3)))->z) + (*right_0).primal_1.rows[int(2)].z * dOut_2.rows[int(3)].z;
    *&(((&right_d_result_0)->rows + (int(2)))->z) = *&(((&right_d_result_0)->rows + (int(2)))->z) + (*left_0).primal_1.rows[int(3)].z * dOut_2.rows[int(3)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->w) = *&(((&left_d_result_0)->rows + (int(3)))->w) + (*right_0).primal_1.rows[int(3)].z * dOut_2.rows[int(3)].z;
    *&(((&right_d_result_0)->rows + (int(3)))->z) = *&(((&right_d_result_0)->rows + (int(3)))->z) + (*left_0).primal_1.rows[int(3)].w * dOut_2.rows[int(3)].z;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->x) = *&(((&left_d_result_0)->rows + (int(3)))->x) + (*right_0).primal_1.rows[int(0)].w * dOut_2.rows[int(3)].w;
    *&(((&right_d_result_0)->rows + (int(0)))->w) = *&(((&right_d_result_0)->rows + (int(0)))->w) + (*left_0).primal_1.rows[int(3)].x * dOut_2.rows[int(3)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->y) = *&(((&left_d_result_0)->rows + (int(3)))->y) + (*right_0).primal_1.rows[int(1)].w * dOut_2.rows[int(3)].w;
    *&(((&right_d_result_0)->rows + (int(1)))->w) = *&(((&right_d_result_0)->rows + (int(1)))->w) + (*left_0).primal_1.rows[int(3)].y * dOut_2.rows[int(3)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->z) = *&(((&left_d_result_0)->rows + (int(3)))->z) + (*right_0).primal_1.rows[int(2)].w * dOut_2.rows[int(3)].w;
    *&(((&right_d_result_0)->rows + (int(2)))->w) = *&(((&right_d_result_0)->rows + (int(2)))->w) + (*left_0).primal_1.rows[int(3)].z * dOut_2.rows[int(3)].w;

#line 1411
    *&(((&left_d_result_0)->rows + (int(3)))->w) = *&(((&left_d_result_0)->rows + (int(3)))->w) + (*right_0).primal_1.rows[int(3)].w * dOut_2.rows[int(3)].w;
    *&(((&right_d_result_0)->rows + (int(3)))->w) = *&(((&right_d_result_0)->rows + (int(3)))->w) + (*left_0).primal_1.rows[int(3)].w * dOut_2.rows[int(3)].w;

#line 1412
    left_0->primal_1 = (*left_0).primal_1;

#line 1412
    left_0->differential_0 = left_d_result_0;

#line 1412
    right_0->primal_1 = (*right_0).primal_1;

#line 1412
    right_0->differential_0 = right_d_result_0;

#line 1418
    return;
}


#line 1418
struct DiffPair_matrixx3Cfloatx2C3x2C3x3E_0
{
    Matrix<float, 3, 3>  primal_1;
    Matrix<float, 3, 3>  differential_0;
};


#line 1386
__device__ void mul_1(DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * left_1, DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * right_1, Matrix<float, 3, 3>  dOut_3)
{
    Matrix<float, 3, 3>  left_d_result_1;

#line 1393
    *&(((&left_d_result_1)->rows + (int(0)))->x) = 0.0f;

#line 1393
    *&(((&left_d_result_1)->rows + (int(0)))->y) = 0.0f;

#line 1393
    *&(((&left_d_result_1)->rows + (int(0)))->z) = 0.0f;

#line 1393
    *&(((&left_d_result_1)->rows + (int(1)))->x) = 0.0f;

#line 1393
    *&(((&left_d_result_1)->rows + (int(1)))->y) = 0.0f;

#line 1393
    *&(((&left_d_result_1)->rows + (int(1)))->z) = 0.0f;

#line 1393
    *&(((&left_d_result_1)->rows + (int(2)))->x) = 0.0f;

#line 1393
    *&(((&left_d_result_1)->rows + (int(2)))->y) = 0.0f;

#line 1393
    *&(((&left_d_result_1)->rows + (int(2)))->z) = 0.0f;

    Matrix<float, 3, 3>  right_d_result_1;

#line 1400
    *&(((&right_d_result_1)->rows + (int(0)))->x) = 0.0f;

#line 1400
    *&(((&right_d_result_1)->rows + (int(0)))->y) = 0.0f;

#line 1400
    *&(((&right_d_result_1)->rows + (int(0)))->z) = 0.0f;

#line 1400
    *&(((&right_d_result_1)->rows + (int(1)))->x) = 0.0f;

#line 1400
    *&(((&right_d_result_1)->rows + (int(1)))->y) = 0.0f;

#line 1400
    *&(((&right_d_result_1)->rows + (int(1)))->z) = 0.0f;

#line 1400
    *&(((&right_d_result_1)->rows + (int(2)))->x) = 0.0f;

#line 1400
    *&(((&right_d_result_1)->rows + (int(2)))->y) = 0.0f;

#line 1400
    *&(((&right_d_result_1)->rows + (int(2)))->z) = 0.0f;

#line 1411
    *&(((&left_d_result_1)->rows + (int(0)))->x) = *&(((&left_d_result_1)->rows + (int(0)))->x) + (*right_1).primal_1.rows[int(0)].x * dOut_3.rows[int(0)].x;
    *&(((&right_d_result_1)->rows + (int(0)))->x) = *&(((&right_d_result_1)->rows + (int(0)))->x) + (*left_1).primal_1.rows[int(0)].x * dOut_3.rows[int(0)].x;

#line 1411
    *&(((&left_d_result_1)->rows + (int(0)))->y) = *&(((&left_d_result_1)->rows + (int(0)))->y) + (*right_1).primal_1.rows[int(1)].x * dOut_3.rows[int(0)].x;
    *&(((&right_d_result_1)->rows + (int(1)))->x) = *&(((&right_d_result_1)->rows + (int(1)))->x) + (*left_1).primal_1.rows[int(0)].y * dOut_3.rows[int(0)].x;

#line 1411
    *&(((&left_d_result_1)->rows + (int(0)))->z) = *&(((&left_d_result_1)->rows + (int(0)))->z) + (*right_1).primal_1.rows[int(2)].x * dOut_3.rows[int(0)].x;
    *&(((&right_d_result_1)->rows + (int(2)))->x) = *&(((&right_d_result_1)->rows + (int(2)))->x) + (*left_1).primal_1.rows[int(0)].z * dOut_3.rows[int(0)].x;

#line 1411
    *&(((&left_d_result_1)->rows + (int(0)))->x) = *&(((&left_d_result_1)->rows + (int(0)))->x) + (*right_1).primal_1.rows[int(0)].y * dOut_3.rows[int(0)].y;
    *&(((&right_d_result_1)->rows + (int(0)))->y) = *&(((&right_d_result_1)->rows + (int(0)))->y) + (*left_1).primal_1.rows[int(0)].x * dOut_3.rows[int(0)].y;

#line 1411
    *&(((&left_d_result_1)->rows + (int(0)))->y) = *&(((&left_d_result_1)->rows + (int(0)))->y) + (*right_1).primal_1.rows[int(1)].y * dOut_3.rows[int(0)].y;
    *&(((&right_d_result_1)->rows + (int(1)))->y) = *&(((&right_d_result_1)->rows + (int(1)))->y) + (*left_1).primal_1.rows[int(0)].y * dOut_3.rows[int(0)].y;

#line 1411
    *&(((&left_d_result_1)->rows + (int(0)))->z) = *&(((&left_d_result_1)->rows + (int(0)))->z) + (*right_1).primal_1.rows[int(2)].y * dOut_3.rows[int(0)].y;
    *&(((&right_d_result_1)->rows + (int(2)))->y) = *&(((&right_d_result_1)->rows + (int(2)))->y) + (*left_1).primal_1.rows[int(0)].z * dOut_3.rows[int(0)].y;

#line 1411
    *&(((&left_d_result_1)->rows + (int(0)))->x) = *&(((&left_d_result_1)->rows + (int(0)))->x) + (*right_1).primal_1.rows[int(0)].z * dOut_3.rows[int(0)].z;
    *&(((&right_d_result_1)->rows + (int(0)))->z) = *&(((&right_d_result_1)->rows + (int(0)))->z) + (*left_1).primal_1.rows[int(0)].x * dOut_3.rows[int(0)].z;

#line 1411
    *&(((&left_d_result_1)->rows + (int(0)))->y) = *&(((&left_d_result_1)->rows + (int(0)))->y) + (*right_1).primal_1.rows[int(1)].z * dOut_3.rows[int(0)].z;
    *&(((&right_d_result_1)->rows + (int(1)))->z) = *&(((&right_d_result_1)->rows + (int(1)))->z) + (*left_1).primal_1.rows[int(0)].y * dOut_3.rows[int(0)].z;

#line 1411
    *&(((&left_d_result_1)->rows + (int(0)))->z) = *&(((&left_d_result_1)->rows + (int(0)))->z) + (*right_1).primal_1.rows[int(2)].z * dOut_3.rows[int(0)].z;
    *&(((&right_d_result_1)->rows + (int(2)))->z) = *&(((&right_d_result_1)->rows + (int(2)))->z) + (*left_1).primal_1.rows[int(0)].z * dOut_3.rows[int(0)].z;

#line 1411
    *&(((&left_d_result_1)->rows + (int(1)))->x) = *&(((&left_d_result_1)->rows + (int(1)))->x) + (*right_1).primal_1.rows[int(0)].x * dOut_3.rows[int(1)].x;
    *&(((&right_d_result_1)->rows + (int(0)))->x) = *&(((&right_d_result_1)->rows + (int(0)))->x) + (*left_1).primal_1.rows[int(1)].x * dOut_3.rows[int(1)].x;

#line 1411
    *&(((&left_d_result_1)->rows + (int(1)))->y) = *&(((&left_d_result_1)->rows + (int(1)))->y) + (*right_1).primal_1.rows[int(1)].x * dOut_3.rows[int(1)].x;
    *&(((&right_d_result_1)->rows + (int(1)))->x) = *&(((&right_d_result_1)->rows + (int(1)))->x) + (*left_1).primal_1.rows[int(1)].y * dOut_3.rows[int(1)].x;

#line 1411
    *&(((&left_d_result_1)->rows + (int(1)))->z) = *&(((&left_d_result_1)->rows + (int(1)))->z) + (*right_1).primal_1.rows[int(2)].x * dOut_3.rows[int(1)].x;
    *&(((&right_d_result_1)->rows + (int(2)))->x) = *&(((&right_d_result_1)->rows + (int(2)))->x) + (*left_1).primal_1.rows[int(1)].z * dOut_3.rows[int(1)].x;

#line 1411
    *&(((&left_d_result_1)->rows + (int(1)))->x) = *&(((&left_d_result_1)->rows + (int(1)))->x) + (*right_1).primal_1.rows[int(0)].y * dOut_3.rows[int(1)].y;
    *&(((&right_d_result_1)->rows + (int(0)))->y) = *&(((&right_d_result_1)->rows + (int(0)))->y) + (*left_1).primal_1.rows[int(1)].x * dOut_3.rows[int(1)].y;

#line 1411
    *&(((&left_d_result_1)->rows + (int(1)))->y) = *&(((&left_d_result_1)->rows + (int(1)))->y) + (*right_1).primal_1.rows[int(1)].y * dOut_3.rows[int(1)].y;
    *&(((&right_d_result_1)->rows + (int(1)))->y) = *&(((&right_d_result_1)->rows + (int(1)))->y) + (*left_1).primal_1.rows[int(1)].y * dOut_3.rows[int(1)].y;

#line 1411
    *&(((&left_d_result_1)->rows + (int(1)))->z) = *&(((&left_d_result_1)->rows + (int(1)))->z) + (*right_1).primal_1.rows[int(2)].y * dOut_3.rows[int(1)].y;
    *&(((&right_d_result_1)->rows + (int(2)))->y) = *&(((&right_d_result_1)->rows + (int(2)))->y) + (*left_1).primal_1.rows[int(1)].z * dOut_3.rows[int(1)].y;

#line 1411
    *&(((&left_d_result_1)->rows + (int(1)))->x) = *&(((&left_d_result_1)->rows + (int(1)))->x) + (*right_1).primal_1.rows[int(0)].z * dOut_3.rows[int(1)].z;
    *&(((&right_d_result_1)->rows + (int(0)))->z) = *&(((&right_d_result_1)->rows + (int(0)))->z) + (*left_1).primal_1.rows[int(1)].x * dOut_3.rows[int(1)].z;

#line 1411
    *&(((&left_d_result_1)->rows + (int(1)))->y) = *&(((&left_d_result_1)->rows + (int(1)))->y) + (*right_1).primal_1.rows[int(1)].z * dOut_3.rows[int(1)].z;
    *&(((&right_d_result_1)->rows + (int(1)))->z) = *&(((&right_d_result_1)->rows + (int(1)))->z) + (*left_1).primal_1.rows[int(1)].y * dOut_3.rows[int(1)].z;

#line 1411
    *&(((&left_d_result_1)->rows + (int(1)))->z) = *&(((&left_d_result_1)->rows + (int(1)))->z) + (*right_1).primal_1.rows[int(2)].z * dOut_3.rows[int(1)].z;
    *&(((&right_d_result_1)->rows + (int(2)))->z) = *&(((&right_d_result_1)->rows + (int(2)))->z) + (*left_1).primal_1.rows[int(1)].z * dOut_3.rows[int(1)].z;

#line 1411
    *&(((&left_d_result_1)->rows + (int(2)))->x) = *&(((&left_d_result_1)->rows + (int(2)))->x) + (*right_1).primal_1.rows[int(0)].x * dOut_3.rows[int(2)].x;
    *&(((&right_d_result_1)->rows + (int(0)))->x) = *&(((&right_d_result_1)->rows + (int(0)))->x) + (*left_1).primal_1.rows[int(2)].x * dOut_3.rows[int(2)].x;

#line 1411
    *&(((&left_d_result_1)->rows + (int(2)))->y) = *&(((&left_d_result_1)->rows + (int(2)))->y) + (*right_1).primal_1.rows[int(1)].x * dOut_3.rows[int(2)].x;
    *&(((&right_d_result_1)->rows + (int(1)))->x) = *&(((&right_d_result_1)->rows + (int(1)))->x) + (*left_1).primal_1.rows[int(2)].y * dOut_3.rows[int(2)].x;

#line 1411
    *&(((&left_d_result_1)->rows + (int(2)))->z) = *&(((&left_d_result_1)->rows + (int(2)))->z) + (*right_1).primal_1.rows[int(2)].x * dOut_3.rows[int(2)].x;
    *&(((&right_d_result_1)->rows + (int(2)))->x) = *&(((&right_d_result_1)->rows + (int(2)))->x) + (*left_1).primal_1.rows[int(2)].z * dOut_3.rows[int(2)].x;

#line 1411
    *&(((&left_d_result_1)->rows + (int(2)))->x) = *&(((&left_d_result_1)->rows + (int(2)))->x) + (*right_1).primal_1.rows[int(0)].y * dOut_3.rows[int(2)].y;
    *&(((&right_d_result_1)->rows + (int(0)))->y) = *&(((&right_d_result_1)->rows + (int(0)))->y) + (*left_1).primal_1.rows[int(2)].x * dOut_3.rows[int(2)].y;

#line 1411
    *&(((&left_d_result_1)->rows + (int(2)))->y) = *&(((&left_d_result_1)->rows + (int(2)))->y) + (*right_1).primal_1.rows[int(1)].y * dOut_3.rows[int(2)].y;
    *&(((&right_d_result_1)->rows + (int(1)))->y) = *&(((&right_d_result_1)->rows + (int(1)))->y) + (*left_1).primal_1.rows[int(2)].y * dOut_3.rows[int(2)].y;

#line 1411
    *&(((&left_d_result_1)->rows + (int(2)))->z) = *&(((&left_d_result_1)->rows + (int(2)))->z) + (*right_1).primal_1.rows[int(2)].y * dOut_3.rows[int(2)].y;
    *&(((&right_d_result_1)->rows + (int(2)))->y) = *&(((&right_d_result_1)->rows + (int(2)))->y) + (*left_1).primal_1.rows[int(2)].z * dOut_3.rows[int(2)].y;

#line 1411
    *&(((&left_d_result_1)->rows + (int(2)))->x) = *&(((&left_d_result_1)->rows + (int(2)))->x) + (*right_1).primal_1.rows[int(0)].z * dOut_3.rows[int(2)].z;
    *&(((&right_d_result_1)->rows + (int(0)))->z) = *&(((&right_d_result_1)->rows + (int(0)))->z) + (*left_1).primal_1.rows[int(2)].x * dOut_3.rows[int(2)].z;

#line 1411
    *&(((&left_d_result_1)->rows + (int(2)))->y) = *&(((&left_d_result_1)->rows + (int(2)))->y) + (*right_1).primal_1.rows[int(1)].z * dOut_3.rows[int(2)].z;
    *&(((&right_d_result_1)->rows + (int(1)))->z) = *&(((&right_d_result_1)->rows + (int(1)))->z) + (*left_1).primal_1.rows[int(2)].y * dOut_3.rows[int(2)].z;

#line 1411
    *&(((&left_d_result_1)->rows + (int(2)))->z) = *&(((&left_d_result_1)->rows + (int(2)))->z) + (*right_1).primal_1.rows[int(2)].z * dOut_3.rows[int(2)].z;
    *&(((&right_d_result_1)->rows + (int(2)))->z) = *&(((&right_d_result_1)->rows + (int(2)))->z) + (*left_1).primal_1.rows[int(2)].z * dOut_3.rows[int(2)].z;

#line 1412
    left_1->primal_1 = (*left_1).primal_1;

#line 1412
    left_1->differential_0 = left_d_result_1;

#line 1412
    right_1->primal_1 = (*right_1).primal_1;

#line 1412
    right_1->differential_0 = right_d_result_1;

#line 1418
    return;
}


#line 11019 "hlsl.meta.slang"
__device__ Matrix<float, 4, 4>  mul_2(Matrix<float, 4, 4>  left_2, Matrix<float, 4, 4>  right_2)
{

#line 11031
    Matrix<float, 4, 4>  result_8;

#line 11031
    int r_0 = int(0);
    for(;;)
    {

#line 11032
        if(r_0 < int(4))
        {
        }
        else
        {

#line 11032
            break;
        }

#line 11032
        int _S50 = r_0;

#line 11032
        int c_0 = int(0);
        for(;;)
        {

#line 11033
            if(c_0 < int(4))
            {
            }
            else
            {

#line 11033
                break;
            }

#line 11033
            int i_11 = int(0);

#line 11033
            float sum_0 = 0.0f;


            for(;;)
            {

#line 11036
                if(i_11 < int(4))
                {
                }
                else
                {

#line 11036
                    break;
                }
                float sum_1 = sum_0 + _slang_vector_get_element(left_2.rows[_S50], i_11) * _slang_vector_get_element(right_2.rows[i_11], c_0);

#line 11036
                i_11 = i_11 + int(1);

#line 11036
                sum_0 = sum_1;

#line 11036
            }



            *_slang_vector_get_element_ptr(((&result_8)->rows + (r_0)), c_0) = sum_0;

#line 11033
            c_0 = c_0 + int(1);

#line 11033
        }

#line 11032
        r_0 = r_0 + int(1);

#line 11032
    }

#line 11042
    return result_8;
}


#line 11019
__device__ Matrix<float, 3, 3>  mul_3(Matrix<float, 3, 3>  left_3, Matrix<float, 3, 3>  right_3)
{

#line 11031
    Matrix<float, 3, 3>  result_9;

#line 11031
    int r_1 = int(0);
    for(;;)
    {

#line 11032
        if(r_1 < int(3))
        {
        }
        else
        {

#line 11032
            break;
        }

#line 11032
        int _S51 = r_1;

#line 11032
        int c_1 = int(0);
        for(;;)
        {

#line 11033
            if(c_1 < int(3))
            {
            }
            else
            {

#line 11033
                break;
            }

#line 11033
            int i_12 = int(0);

#line 11033
            float sum_2 = 0.0f;


            for(;;)
            {

#line 11036
                if(i_12 < int(3))
                {
                }
                else
                {

#line 11036
                    break;
                }
                float sum_3 = sum_2 + _slang_vector_get_element(left_3.rows[_S51], i_12) * _slang_vector_get_element(right_3.rows[i_12], c_1);

#line 11036
                i_12 = i_12 + int(1);

#line 11036
                sum_2 = sum_3;

#line 11036
            }



            *_slang_vector_get_element_ptr(((&result_9)->rows + (r_1)), c_1) = sum_2;

#line 11033
            c_1 = c_1 + int(1);

#line 11033
        }

#line 11032
        r_1 = r_1 + int(1);

#line 11032
    }

#line 11042
    return result_9;
}


#line 11042
struct DiffPair_vectorx3Cfloatx2C4x3E_0
{
    float4  primal_1;
    float4  differential_0;
};


#line 1349 "diff.meta.slang"
__device__ void _d_mul_0(DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 * left_4, DiffPair_vectorx3Cfloatx2C4x3E_0 * right_4, float4  dOut_4)
{

    float4  right_d_result_2;

#line 1352
    float _S52 = (*left_4).primal_1.rows[int(0)].x * dOut_4.x;

#line 1351
    Matrix<float, 4, 4>  left_d_result_2;

#line 1361
    *&(((&left_d_result_2)->rows + (int(0)))->x) = (*right_4).primal_1.x * dOut_4.x;

#line 1360
    float sum_4 = _S52 + (*left_4).primal_1.rows[int(1)].x * dOut_4.y;
    *&(((&left_d_result_2)->rows + (int(1)))->x) = (*right_4).primal_1.x * dOut_4.y;

#line 1360
    float sum_5 = sum_4 + (*left_4).primal_1.rows[int(2)].x * dOut_4.z;
    *&(((&left_d_result_2)->rows + (int(2)))->x) = (*right_4).primal_1.x * dOut_4.z;

#line 1360
    float sum_6 = sum_5 + (*left_4).primal_1.rows[int(3)].x * dOut_4.w;
    *&(((&left_d_result_2)->rows + (int(3)))->x) = (*right_4).primal_1.x * dOut_4.w;

    *&((&right_d_result_2)->x) = sum_6;

#line 1363
    float _S53 = (*left_4).primal_1.rows[int(0)].y * dOut_4.x;

#line 1361
    *&(((&left_d_result_2)->rows + (int(0)))->y) = (*right_4).primal_1.y * dOut_4.x;

#line 1360
    float sum_7 = _S53 + (*left_4).primal_1.rows[int(1)].y * dOut_4.y;
    *&(((&left_d_result_2)->rows + (int(1)))->y) = (*right_4).primal_1.y * dOut_4.y;

#line 1360
    float sum_8 = sum_7 + (*left_4).primal_1.rows[int(2)].y * dOut_4.z;
    *&(((&left_d_result_2)->rows + (int(2)))->y) = (*right_4).primal_1.y * dOut_4.z;

#line 1360
    float sum_9 = sum_8 + (*left_4).primal_1.rows[int(3)].y * dOut_4.w;
    *&(((&left_d_result_2)->rows + (int(3)))->y) = (*right_4).primal_1.y * dOut_4.w;

    *&((&right_d_result_2)->y) = sum_9;

#line 1363
    float _S54 = (*left_4).primal_1.rows[int(0)].z * dOut_4.x;

#line 1361
    *&(((&left_d_result_2)->rows + (int(0)))->z) = (*right_4).primal_1.z * dOut_4.x;

#line 1360
    float sum_10 = _S54 + (*left_4).primal_1.rows[int(1)].z * dOut_4.y;
    *&(((&left_d_result_2)->rows + (int(1)))->z) = (*right_4).primal_1.z * dOut_4.y;

#line 1360
    float sum_11 = sum_10 + (*left_4).primal_1.rows[int(2)].z * dOut_4.z;
    *&(((&left_d_result_2)->rows + (int(2)))->z) = (*right_4).primal_1.z * dOut_4.z;

#line 1360
    float sum_12 = sum_11 + (*left_4).primal_1.rows[int(3)].z * dOut_4.w;
    *&(((&left_d_result_2)->rows + (int(3)))->z) = (*right_4).primal_1.z * dOut_4.w;

    *&((&right_d_result_2)->z) = sum_12;

#line 1363
    float _S55 = (*left_4).primal_1.rows[int(0)].w * dOut_4.x;

#line 1361
    *&(((&left_d_result_2)->rows + (int(0)))->w) = (*right_4).primal_1.w * dOut_4.x;

#line 1360
    float sum_13 = _S55 + (*left_4).primal_1.rows[int(1)].w * dOut_4.y;
    *&(((&left_d_result_2)->rows + (int(1)))->w) = (*right_4).primal_1.w * dOut_4.y;

#line 1360
    float sum_14 = sum_13 + (*left_4).primal_1.rows[int(2)].w * dOut_4.z;
    *&(((&left_d_result_2)->rows + (int(2)))->w) = (*right_4).primal_1.w * dOut_4.z;

#line 1360
    float sum_15 = sum_14 + (*left_4).primal_1.rows[int(3)].w * dOut_4.w;
    *&(((&left_d_result_2)->rows + (int(3)))->w) = (*right_4).primal_1.w * dOut_4.w;

    *&((&right_d_result_2)->w) = sum_15;

#line 1363
    left_4->primal_1 = (*left_4).primal_1;

#line 1363
    left_4->differential_0 = left_d_result_2;

#line 1363
    right_4->primal_1 = (*right_4).primal_1;

#line 1363
    right_4->differential_0 = right_d_result_2;



    return;
}


#line 10938 "hlsl.meta.slang"
__device__ float4  mul_4(Matrix<float, 4, 4>  left_5, float4  right_5)
{

#line 10950
    float4  result_10;

#line 10950
    int i_13 = int(0);
    for(;;)
    {

#line 10951
        if(i_13 < int(4))
        {
        }
        else
        {

#line 10951
            break;
        }

#line 10951
        int _S56 = i_13;

#line 10951
        int j_0 = int(0);

#line 10951
        float sum_16 = 0.0f;


        for(;;)
        {

#line 10954
            if(j_0 < int(4))
            {
            }
            else
            {

#line 10954
                break;
            }
            float sum_17 = sum_16 + _slang_vector_get_element(left_5.rows[_S56], j_0) * _slang_vector_get_element(right_5, j_0);

#line 10954
            j_0 = j_0 + int(1);

#line 10954
            sum_16 = sum_17;

#line 10954
        }



        *_slang_vector_get_element_ptr(&result_10, i_13) = sum_16;

#line 10951
        i_13 = i_13 + int(1);

#line 10951
    }

#line 10960
    return result_10;
}


#line 105 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
__device__ float3  geom_transform_points_0(float3  point_0, Matrix<float, 4, 4>  transf_matrix_0)
{
    float4  p_out_0 = mul_4(transf_matrix_0, make_float4 (point_0.x, point_0.y, point_0.z, 1.0f));
    return float3 {p_out_0.x, p_out_0.y, p_out_0.z} / make_float3 (p_out_0.w + 1.00000001168609742e-07f);
}


__device__ float3  geom_transform_points2_0(float3  point_1, Matrix<float, 4, 4>  transf_matrix_1)
{

    return float3 {mul_4(transf_matrix_1, make_float4 (point_1.x, point_1.y, point_1.z, 1.0f)).x, mul_4(transf_matrix_1, make_float4 (point_1.x, point_1.y, point_1.z, 1.0f)).y, mul_4(transf_matrix_1, make_float4 (point_1.x, point_1.y, point_1.z, 1.0f)).z};
}


__device__ float3  project_point_0(float3  point_2, Camera_0 cam_0)
{

#line 120
    float3  proj_point_0 = geom_transform_points_0(point_2, mul_2(cam_0.proj_mat_1, cam_0.world_view_transform_1));

    *&((&proj_point_0)->z) = geom_transform_points2_0(point_2, cam_0.world_view_transform_1).z;
    return proj_point_0;
}


#line 123
struct DiffPair_float_0
{
    float primal_1;
    float differential_0;
};


#line 1935 "diff.meta.slang"
__device__ void _d_max_0(DiffPair_float_0 * dpx_0, DiffPair_float_0 * dpy_0, float dOut_5)
{
    DiffPair_float_0 _S57 = *dpx_0;

#line 1937
    float _S58;

#line 1937
    if((*dpx_0).primal_1 > (*dpy_0).primal_1)
    {

#line 1937
        _S58 = dOut_5;

#line 1937
    }
    else
    {

#line 1937
        _S58 = 0.0f;

#line 1937
    }

#line 1937
    dpx_0->primal_1 = _S57.primal_1;

#line 1937
    dpx_0->differential_0 = _S58;
    DiffPair_float_0 _S59 = *dpy_0;

#line 1938
    if((*dpy_0).primal_1 > _S57.primal_1)
    {

#line 1938
        _S58 = dOut_5;

#line 1938
    }
    else
    {

#line 1938
        _S58 = 0.0f;

#line 1938
    }

#line 1938
    dpy_0->primal_1 = _S59.primal_1;

#line 1938
    dpy_0->differential_0 = _S58;
    return;
}


#line 1923
__device__ DiffPair_float_0 _d_max_1(DiffPair_float_0 dpx_1, DiffPair_float_0 dpy_1)
{

    float _S60 = (F32_max((dpx_1.primal_1), (dpy_1.primal_1)));

#line 1926
    float _S61;
    if(dpx_1.primal_1 > dpy_1.primal_1)
    {

#line 1927
        _S61 = dpx_1.differential_0;

#line 1927
    }
    else
    {

#line 1927
        _S61 = dpy_1.differential_0;

#line 1927
    }

#line 1927
    DiffPair_float_0 _S62 = { _S60, _S61 };

#line 1925
    return _S62;
}


#line 1 "token paste"
__device__ void _d_sqrt_0(DiffPair_float_0 * dpx_2, float dOut_6)
{

#line 1719 "diff.meta.slang"
    float _S63 = 0.5f / (F32_sqrt(((F32_max((1.00000001168609742e-07f), ((*dpx_2).primal_1)))))) * dOut_6;

#line 1719
    dpx_2->primal_1 = (*dpx_2).primal_1;

#line 1719
    dpx_2->differential_0 = _S63;



    return;
}


#line 1 "token paste"
__device__ DiffPair_float_0 _d_sqrt_1(DiffPair_float_0 dpx_3)
{

#line 1689 "diff.meta.slang"
    DiffPair_float_0 _S64 = { (F32_sqrt((dpx_3.primal_1))), 0.5f / (F32_sqrt(((F32_max((1.00000001168609742e-07f), (dpx_3.primal_1)))))) * dpx_3.differential_0 };


    return _S64;
}


#line 7891 "hlsl.meta.slang"
__device__ float dot_0(float3  x_0, float3  y_0)
{

#line 7891
    int i_14 = int(0);

#line 7891
    float result_11 = 0.0f;

#line 7904
    for(;;)
    {

#line 7904
        if(i_14 < int(3))
        {
        }
        else
        {

#line 7904
            break;
        }

#line 7905
        float result_12 = result_11 + _slang_vector_get_element(x_0, i_14) * _slang_vector_get_element(y_0, i_14);

#line 7904
        i_14 = i_14 + int(1);

#line 7904
        result_11 = result_12;

#line 7904
    }

    return result_11;
}


#line 9729
__device__ float length_0(float3  x_1)
{

#line 9741
    return (F32_sqrt((dot_0(x_1, x_1))));
}


#line 11211
__device__ float3  normalize_0(float3  x_2)
{

#line 11223
    return x_2 / make_float3 (length_0(x_2));
}


#line 11223
struct DiffPair_vectorx3Cfloatx2C3x3E_0
{
    float3  primal_1;
    float3  differential_0;
};


#line 1 "token paste"
__device__ void _d_max_vector_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * dpx_4, DiffPair_vectorx3Cfloatx2C3x3E_0 * dpy_2, float3  dOut_7)
{

#line 1558 "diff.meta.slang"
    DiffPair_float_0 left_dp_0;

#line 1558
    (&left_dp_0)->primal_1 = (*dpx_4).primal_1.x;

#line 1558
    (&left_dp_0)->differential_0 = 0.0f;
    DiffPair_float_0 right_dp_0;

#line 1559
    (&right_dp_0)->primal_1 = (*dpy_2).primal_1.x;

#line 1559
    (&right_dp_0)->differential_0 = 0.0f;
    _d_max_0(&left_dp_0, &right_dp_0, dOut_7.x);

#line 1555
    float3  left_d_result_3;

#line 1561
    *&((&left_d_result_3)->x) = left_dp_0.differential_0;

#line 1555
    float3  right_d_result_3;

#line 1562
    *&((&right_d_result_3)->x) = right_dp_0.differential_0;

#line 1558
    DiffPair_float_0 left_dp_1;

#line 1558
    (&left_dp_1)->primal_1 = (*dpx_4).primal_1.y;

#line 1558
    (&left_dp_1)->differential_0 = 0.0f;
    DiffPair_float_0 right_dp_1;

#line 1559
    (&right_dp_1)->primal_1 = (*dpy_2).primal_1.y;

#line 1559
    (&right_dp_1)->differential_0 = 0.0f;
    _d_max_0(&left_dp_1, &right_dp_1, dOut_7.y);
    *&((&left_d_result_3)->y) = left_dp_1.differential_0;
    *&((&right_d_result_3)->y) = right_dp_1.differential_0;

#line 1558
    DiffPair_float_0 left_dp_2;

#line 1558
    (&left_dp_2)->primal_1 = (*dpx_4).primal_1.z;

#line 1558
    (&left_dp_2)->differential_0 = 0.0f;
    DiffPair_float_0 right_dp_2;

#line 1559
    (&right_dp_2)->primal_1 = (*dpy_2).primal_1.z;

#line 1559
    (&right_dp_2)->differential_0 = 0.0f;
    _d_max_0(&left_dp_2, &right_dp_2, dOut_7.z);
    *&((&left_d_result_3)->z) = left_dp_2.differential_0;
    *&((&right_d_result_3)->z) = right_dp_2.differential_0;

#line 1562
    dpx_4->primal_1 = (*dpx_4).primal_1;

#line 1562
    dpx_4->differential_0 = left_d_result_3;

#line 1562
    dpy_2->primal_1 = (*dpy_2).primal_1;

#line 1562
    dpy_2->differential_0 = right_d_result_3;



    return;
}


#line 1 "token paste"
__device__ DiffPair_vectorx3Cfloatx2C3x3E_0 _d_max_vector_1(DiffPair_vectorx3Cfloatx2C3x3E_0 dpx_5, DiffPair_vectorx3Cfloatx2C3x3E_0 dpy_3)
{

#line 1514 "diff.meta.slang"
    DiffPair_float_0 _S65 = { dpx_5.primal_1.x, dpx_5.differential_0.x };

#line 1514
    DiffPair_float_0 _S66 = { dpy_3.primal_1.x, dpy_3.differential_0.x };

#line 1520
    DiffPair_float_0 dp_elem_0 = _d_max_1(_S65, _S66);

#line 1516
    float3  result_13;

#line 1523
    *&((&result_13)->x) = dp_elem_0.primal_1;

#line 1517
    float3  d_result_0;

#line 1524
    *&((&d_result_0)->x) = dp_elem_0.differential_0;

#line 1524
    DiffPair_float_0 _S67 = { dpx_5.primal_1.y, dpx_5.differential_0.y };

#line 1524
    DiffPair_float_0 _S68 = { dpy_3.primal_1.y, dpy_3.differential_0.y };

#line 1520
    DiffPair_float_0 dp_elem_1 = _d_max_1(_S67, _S68);


    *&((&result_13)->y) = dp_elem_1.primal_1;
    *&((&d_result_0)->y) = dp_elem_1.differential_0;

#line 1524
    DiffPair_float_0 _S69 = { dpx_5.primal_1.z, dpx_5.differential_0.z };

#line 1524
    DiffPair_float_0 _S70 = { dpy_3.primal_1.z, dpy_3.differential_0.z };

#line 1520
    DiffPair_float_0 dp_elem_2 = _d_max_1(_S69, _S70);


    *&((&result_13)->z) = dp_elem_2.primal_1;
    *&((&d_result_0)->z) = dp_elem_2.differential_0;

#line 1524
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S71 = { result_13, d_result_0 };

    return _S71;
}


#line 10224 "hlsl.meta.slang"
__device__ float3  max_0(float3  x_3, float3  y_1)
{

#line 5510
    float3  result_14;

#line 5510
    int i_15 = int(0);

#line 5510
    for(;;)
    {

#line 5510
        if(i_15 < int(3))
        {
        }
        else
        {

#line 5510
            break;
        }

#line 5510
        *_slang_vector_get_element_ptr(&result_14, i_15) = (F32_max((_slang_vector_get_element(x_3, i_15)), (_slang_vector_get_element(y_1, i_15))));

#line 5510
        i_15 = i_15 + int(1);

#line 5510
    }

#line 5510
    return result_14;
}


#line 94 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/spherical_harmonics.slang"
__device__ float3  compute_color_from_sh_coeffs_0(SpherHarmCoeffs_0 sh_0, float3  g_xyz_ws_0, float3  cam_pos_0, uint active_sh_2)
{
    float3  dir_0 = normalize_0(g_xyz_ws_0 - cam_pos_0);

    float3  rgb_1 = make_float3 (0.282094806432724f) * sh_0.coeff0_0;

#line 98
    float3  rgb_2;
    if(active_sh_2 > 0U)
    {

#line 100
        float _S72 = dir_0.y;

#line 100
        float _S73 = dir_0.z;

#line 100
        float _S74 = dir_0.x;

#line 100
        float3  rgb_3 = rgb_1 - make_float3 (0.48860251903533936f * _S72) * sh_0.coeff1_0 + make_float3 (0.48860251903533936f * _S73) * sh_0.coeff2_0 - make_float3 (0.48860251903533936f * _S74) * sh_0.coeff3_0;
        if(active_sh_2 > 1U)
        {
            float xx_0 = _S74 * _S74;

#line 103
            float yy_0 = _S72 * _S72;

#line 103
            float zz_0 = _S73 * _S73;
            float xy_0 = _S74 * _S72;



            float _S75 = 2.0f * zz_0;

            float _S76 = xx_0 - yy_0;

#line 109
            float3  rgb_4 = rgb_3 + make_float3 (1.09254848957061768f * xy_0) * sh_0.coeff4_0 + make_float3 (-1.09254848957061768f * (_S72 * _S73)) * sh_0.coeff5_0 + make_float3 (0.31539157032966614f * (_S75 - xx_0 - yy_0)) * sh_0.coeff6_0 + make_float3 (-1.09254848957061768f * (_S74 * _S73)) * sh_0.coeff7_0 + make_float3 (0.54627424478530884f * _S76) * sh_0.coeff8_0;


            if(active_sh_2 > 2U)
            {

                float _S77 = 3.0f * xx_0;

                float _S78 = 4.0f * zz_0 - xx_0 - yy_0;
                float _S79 = 3.0f * yy_0;

#line 118
                rgb_2 = rgb_4 + make_float3 (-0.59004360437393188f * _S72 * (_S77 - yy_0)) * sh_0.coeff9_0 + make_float3 (2.89061141014099121f * xy_0 * _S73) * sh_0.coeff10_0 + make_float3 (-0.4570457935333252f * _S72 * _S78) * sh_0.coeff11_0 + make_float3 (0.37317633628845215f * _S73 * (_S75 - _S77 - _S79)) * sh_0.coeff12_0 + make_float3 (-0.4570457935333252f * _S74 * _S78) * sh_0.coeff13_0 + make_float3 (1.44530570507049561f * _S73 * _S76) * sh_0.coeff14_0 + make_float3 (-0.59004360437393188f * _S74 * (xx_0 - _S79)) * sh_0.coeff15_0;

#line 112
            }
            else
            {

#line 112
                rgb_2 = rgb_4;

#line 112
            }

#line 101
        }
        else
        {

#line 101
            rgb_2 = rgb_3;

#line 101
        }

#line 99
    }
    else
    {

#line 99
        rgb_2 = rgb_1;

#line 99
    }

#line 128
    return max_0(rgb_2 + make_float3 (0.5f), make_float3 (0.0f));
}


#line 12514 "hlsl.meta.slang"
__device__ Matrix<float, 3, 3>  transpose_0(Matrix<float, 3, 3>  x_4)
{

#line 12525
    Matrix<float, 3, 3>  result_15;

#line 12525
    int r_2 = int(0);
    for(;;)
    {

#line 12526
        if(r_2 < int(3))
        {
        }
        else
        {

#line 12526
            break;
        }

#line 12526
        int c_2 = int(0);
        for(;;)
        {

#line 12527
            if(c_2 < int(3))
            {
            }
            else
            {

#line 12527
                break;
            }

#line 12528
            *_slang_vector_get_element_ptr(((&result_15)->rows + (r_2)), c_2) = _slang_vector_get_element(x_4.rows[c_2], r_2);

#line 12527
            c_2 = c_2 + int(1);

#line 12527
        }

#line 12526
        r_2 = r_2 + int(1);

#line 12526
    }


    return result_15;
}


#line 280 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
__device__ Matrix<float, 3, 3>  get_covariance_from_quat_scales_0(float4  q_0, float3  s_0)
{

#line 280
    float y_2 = q_0.z;



    float _S80 = y_2 * y_2;

#line 284
    float _S81 = q_0.w * q_0.w;

#line 284
    float _S82 = q_0.y * q_0.z;

#line 284
    float _S83 = q_0.x * q_0.w;

#line 284
    float _S84 = q_0.y * q_0.w;

#line 284
    float _S85 = q_0.x * q_0.z;
    float _S86 = q_0.y * q_0.y;

#line 285
    float _S87 = q_0.z * q_0.w;

#line 285
    float _S88 = q_0.x * q_0.y;

#line 292
    Matrix<float, 3, 3>  L_0 = mul_3(makeMatrix<float, 3, 3> (1.0f - 2.0f * (_S80 + _S81), 2.0f * (_S82 - _S83), 2.0f * (_S84 + _S85), 2.0f * (_S82 + _S83), 1.0f - 2.0f * (_S86 + _S81), 2.0f * (_S87 - _S88), 2.0f * (_S84 - _S85), 2.0f * (_S87 + _S88), 1.0f - 2.0f * (_S86 + _S80)), makeMatrix<float, 3, 3> (s_0.x, 0.0f, 0.0f, 0.0f, s_0.y, 0.0f, 0.0f, 0.0f, s_0.z));

    return mul_3(L_0, transpose_0(L_0));
}


#line 1 "token paste"
__device__ void _d_tan_0(DiffPair_float_0 * dpx_6, float dOut_8)
{

#line 1719 "diff.meta.slang"
    float _S89 = 1.0f / ((F32_cos(((*dpx_6).primal_1))) * (F32_cos(((*dpx_6).primal_1)))) * dOut_8;

#line 1719
    dpx_6->primal_1 = (*dpx_6).primal_1;

#line 1719
    dpx_6->differential_0 = _S89;



    return;
}


#line 1 "token paste"
__device__ DiffPair_float_0 _d_tan_1(DiffPair_float_0 dpx_7)
{

#line 1805 "diff.meta.slang"
    float _S90 = (F32_cos((dpx_7.primal_1)));

#line 1805
    DiffPair_float_0 _S91 = { (F32_tan((dpx_7.primal_1))), 1.0f / (_S90 * _S90) * dpx_7.differential_0 };

#line 1692
    return _S91;
}


#line 1960
__device__ void _d_min_0(DiffPair_float_0 * dpx_8, DiffPair_float_0 * dpy_4, float dOut_9)
{
    DiffPair_float_0 _S92 = *dpx_8;

#line 1962
    float _S93;

#line 1962
    if((*dpx_8).primal_1 < (*dpy_4).primal_1)
    {

#line 1962
        _S93 = dOut_9;

#line 1962
    }
    else
    {

#line 1962
        _S93 = 0.0f;

#line 1962
    }

#line 1962
    dpx_8->primal_1 = _S92.primal_1;

#line 1962
    dpx_8->differential_0 = _S93;
    DiffPair_float_0 _S94 = *dpy_4;

#line 1963
    if((*dpy_4).primal_1 < _S92.primal_1)
    {

#line 1963
        _S93 = dOut_9;

#line 1963
    }
    else
    {

#line 1963
        _S93 = 0.0f;

#line 1963
    }

#line 1963
    dpy_4->primal_1 = _S94.primal_1;

#line 1963
    dpy_4->differential_0 = _S93;
    return;
}


#line 1948
__device__ DiffPair_float_0 _d_min_1(DiffPair_float_0 dpx_9, DiffPair_float_0 dpy_5)
{

    float _S95 = (F32_min((dpx_9.primal_1), (dpy_5.primal_1)));

#line 1951
    float _S96;
    if(dpx_9.primal_1 < dpy_5.primal_1)
    {

#line 1952
        _S96 = dpx_9.differential_0;

#line 1952
    }
    else
    {

#line 1952
        _S96 = dpy_5.differential_0;

#line 1952
    }

#line 1952
    DiffPair_float_0 _S97 = { _S95, _S96 };

#line 1950
    return _S97;
}


#line 127 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
__device__ Matrix<float, 3, 3>  compute_jacobian_0(float3  xyz_ws_2, Camera_0 cam_1)
{

#line 128
    float tan_half_fovx_0 = (F32_tan((cam_1.fovx_1 / 2.0f)));
    float tan_half_fovy_0 = (F32_tan((cam_1.fovy_1 / 2.0f)));
    float h_x_0 = float(cam_1.W_0) / (2.0f * tan_half_fovx_0);
    float h_y_0 = float(cam_1.H_0) / (2.0f * tan_half_fovy_0);

    float3  _S98 = geom_transform_points_0(xyz_ws_2, cam_1.world_view_transform_1);

#line 133
    float3  t_0 = _S98;


    float limx_0 = 1.29999995231628418f * tan_half_fovx_0;
    float limy_0 = 1.29999995231628418f * tan_half_fovy_0;
    float _S99 = _S98.z;
    float tytz_0 = _S98.y / _S99;
    *&((&t_0)->x) = (F32_min((limx_0), ((F32_max((- limx_0), (_S98.x / _S99)))))) * _S99;
    *&((&t_0)->y) = (F32_min((limy_0), ((F32_max((- limy_0), (tytz_0)))))) * t_0.z;

#line 147
    return makeMatrix<float, 3, 3> (h_x_0 / t_0.z, 0.0f, - (h_x_0 * t_0.x) / (t_0.z * t_0.z), 0.0f, h_y_0 / t_0.z, - (h_y_0 * t_0.y) / (t_0.z * t_0.z), 0.0f, 0.0f, 0.0f);
}


__device__ Matrix<float, 2, 2>  covariance_3d_to_2d_0(Camera_0 cam_2, float3  xyz_ws_3, Matrix<float, 3, 3>  cov_ws_0)
{

#line 151
    Matrix<float, 3, 3>  _S100 = makeMatrix<float, 3, 3> (float3 {cam_2.world_view_transform_1.rows[int(0)].x, cam_2.world_view_transform_1.rows[int(0)].y, cam_2.world_view_transform_1.rows[int(0)].z}, float3 {cam_2.world_view_transform_1.rows[int(1)].x, cam_2.world_view_transform_1.rows[int(1)].y, cam_2.world_view_transform_1.rows[int(1)].z}, float3 {cam_2.world_view_transform_1.rows[int(2)].x, cam_2.world_view_transform_1.rows[int(2)].y, cam_2.world_view_transform_1.rows[int(2)].z});

    Matrix<float, 3, 3>  J_0 = compute_jacobian_0(xyz_ws_3, cam_2);
    Matrix<float, 3, 3>  cov_vs_1 = mul_3(J_0, mul_3(_S100, mul_3(cov_ws_0, mul_3(transpose_0(_S100), transpose_0(J_0)))));
    *&(((&cov_vs_1)->rows + (int(0)))->x) = *&(((&cov_vs_1)->rows + (int(0)))->x) + 0.30000001192092896f;
    *&(((&cov_vs_1)->rows + (int(1)))->y) = *&(((&cov_vs_1)->rows + (int(1)))->y) + 0.30000001192092896f;

    return makeMatrix<float, 2, 2> (float2 {cov_vs_1.rows[int(0)].x, cov_vs_1.rows[int(0)].y}, float2 {cov_vs_1.rows[int(1)].x, cov_vs_1.rows[int(1)].y});
}


#line 222
__device__ Splat_2D_Vertex_0 project_gaussian_to_camera_0(Gaussian_3D_0 g_0, Camera_0 cam_3, uint active_sh_3)
{

#line 223
    float3  xyz_vs_1 = project_point_0(g_0.xyz_ws_0, cam_3);
    if(xyz_vs_1.z <= 0.20000000298023224f)
    {

#line 225
        float3  _S101 = make_float3 (0.0f);

#line 225
        Splat_2D_Vertex_0 _S102 = { _S101, _S101, makeMatrix<float, 2, 2> (0.0f) };

#line 225
        return _S102;
    }

#line 231
    Splat_2D_Vertex_0 _S103 = { xyz_vs_1, compute_color_from_sh_coeffs_0(g_0.sh_coeffs_0, g_0.xyz_ws_0, cam_3.position_1, active_sh_3), covariance_3d_to_2d_0(cam_3, g_0.xyz_ws_0, get_covariance_from_quat_scales_0(g_0.rotations_0, g_0.scales_0)) };

#line 231
    return _S103;
}


#line 203
__device__ float compute_det_0(Matrix<float, 2, 2>  M_0)
{

#line 204
    return M_0.rows[int(0)].x * M_0.rows[int(1)].y - M_0.rows[int(0)].y * M_0.rows[int(1)].x;
}


#line 193
__device__ float splat_radius_0(Matrix<float, 2, 2>  cov_vs_2, float det_0)
{

#line 194
    float mid_0 = 0.5f * (cov_vs_2.rows[int(0)].x + cov_vs_2.rows[int(1)].y);
    float _S104 = (F32_sqrt(((F32_max((0.10000000149011612f), (mid_0 * mid_0 - det_0))))));



    return (F32_ceil((3.0f * (F32_sqrt(((F32_max((mid_0 + _S104), (mid_0 - _S104)))))))));
}


#line 61
__device__ float ndc2pix_0(float v_0, int S_0)
{
    return ((v_0 + 1.0f) * float(S_0) - 1.0f) * 0.5f;
}


#line 72
__device__ float clip_0(float val_0, float min_val_0, float max_val_0)
{
    return (F32_max((min_val_0), ((F32_min((max_val_0), (val_0))))));
}


#line 4 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/vertex_shader.slang"
struct rectangle_0
{
    int min_x_0;
    int min_y_0;
    int max_x_0;
    int max_y_0;
};

__device__ rectangle_0 get_rectangle_tile_space_0(float2  ndc_xy_0, float radius_0, uint grid_height_0, uint grid_width_0, uint tile_height_0, uint tile_width_0)
{

#line 20
    rectangle_0 rect_tile_space_0;

    float _S105 = ndc_xy_0.x;

#line 22
    float _S106 = float(tile_width_0);

#line 22
    float _S107 = float(grid_width_0);

#line 22
    (&rect_tile_space_0)->min_x_0 = int((F32_floor((clip_0((_S105 - radius_0) / _S106, 0.0f, _S107)))));
    float _S108 = ndc_xy_0.y;

#line 23
    float _S109 = float(tile_height_0);

#line 23
    float _S110 = float(grid_height_0);

#line 23
    (&rect_tile_space_0)->min_y_0 = int((F32_floor((clip_0((_S108 - radius_0) / _S109, 0.0f, _S110)))));
    (&rect_tile_space_0)->max_x_0 = int((F32_ceil((clip_0((_S105 + radius_0) / _S106, 0.0f, _S107)))));
    (&rect_tile_space_0)->max_y_0 = int((F32_ceil((clip_0((_S108 + radius_0) / _S109, 0.0f, _S110)))));

#line 31
    return rect_tile_space_0;
}


#line 1035 "diff.meta.slang"
__device__ void DiffTensorView_storeOnce_forward_0(DiffTensorView_0 this_11, uint2  x_5, DiffPair_float_0 dpval_0)
{
    (this_11.primal_0).store<float>((x_5), (dpval_0.primal_1));
    AtomicAdd_storeOnce_forward_0(this_11.diff_1, x_5, dpval_0.differential_0);
    return;
}


#line 1035
__device__ void DiffTensorView_storeOnce_forward_1(DiffTensorView_0 this_12, uint3  x_6, DiffPair_float_0 dpval_1)
{
    (this_12.primal_0).store<float>((x_6), (dpval_1.primal_1));
    AtomicAdd_storeOnce_forward_1(this_12.diff_1, x_6, dpval_1.differential_0);
    return;
}


#line 1026
__device__ void DiffTensorView_storeOnce_0(DiffTensorView_0 this_13, uint2  x_7, float val_1)
{

#line 1026
    (this_13.primal_0).store<float>((x_7), (val_1));

#line 1026
    return;
}


#line 1026
__device__ void DiffTensorView_storeOnce_1(DiffTensorView_0 this_14, uint3  x_8, float val_2)
{

#line 1026
    (this_14.primal_0).store<float>((x_8), (val_2));

#line 1026
    return;
}


#line 37 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/vertex_shader.slang"
struct s_bwd_prop_vertex_shader_Intermediates_0
{
    Camera_0 _S111;
    Gaussian_3D_0 _S112;
};


#line 65
__device__ float3  s_primal_ctx_read_t3_float3_0(uint idx_2, DiffTensorView_0 t3_1)
{

#line 26 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
    return make_float3 (DiffTensorView_load_0(t3_1, make_uint2 (idx_2, 0U)), DiffTensorView_load_0(t3_1, make_uint2 (idx_2, 1U)), DiffTensorView_load_0(t3_1, make_uint2 (idx_2, 2U)));
}


#line 26
__device__ SpherHarmCoeffs_0 s_primal_ctx_read_spherical_harmonics_coeffs_0(uint g_idx_2, DiffTensorView_0 sh_coeffs_3, uint active_sh_4)
{

#line 64 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/spherical_harmonics.slang"
    float3  _S113 = make_float3 (0.0f);
    float3  _S114 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 0U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 0U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 0U, 2U)));

#line 65
    SpherHarmCoeffs_0 g_sh_coeffs_1;

    if(active_sh_4 > 0U)
    {

#line 68
        float3  _S115 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 1U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 1U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 1U, 2U)));
        float3  _S116 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 2U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 2U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 2U, 2U)));
        float3  _S117 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 3U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 3U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 3U, 2U)));

        if(active_sh_4 > 1U)
        {

#line 73
            float3  _S118 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 4U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 4U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 4U, 2U)));
            float3  _S119 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 5U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 5U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 5U, 2U)));
            float3  _S120 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 6U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 6U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 6U, 2U)));
            float3  _S121 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 7U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 7U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 7U, 2U)));
            float3  _S122 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 8U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 8U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 8U, 2U)));

            if(active_sh_4 > 2U)
            {

#line 80
                float3  _S123 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 9U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 9U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 9U, 2U)));
                float3  _S124 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 10U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 10U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 10U, 2U)));
                float3  _S125 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 11U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 11U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 11U, 2U)));
                float3  _S126 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 12U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 12U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 12U, 2U)));
                float3  _S127 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 13U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 13U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 13U, 2U)));
                float3  _S128 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 14U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 14U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 14U, 2U)));
                float3  _S129 = make_float3 (DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 15U, 0U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 15U, 1U)), DiffTensorView_load_1(sh_coeffs_3, make_uint3 (g_idx_2, 15U, 2U)));

#line 86
                (&g_sh_coeffs_1)->coeff0_0 = _S114;

#line 86
                (&g_sh_coeffs_1)->coeff1_0 = _S115;

#line 86
                (&g_sh_coeffs_1)->coeff2_0 = _S116;

#line 86
                (&g_sh_coeffs_1)->coeff3_0 = _S117;

#line 86
                (&g_sh_coeffs_1)->coeff4_0 = _S118;

#line 86
                (&g_sh_coeffs_1)->coeff5_0 = _S119;

#line 86
                (&g_sh_coeffs_1)->coeff6_0 = _S120;

#line 86
                (&g_sh_coeffs_1)->coeff7_0 = _S121;

#line 86
                (&g_sh_coeffs_1)->coeff8_0 = _S122;

#line 86
                (&g_sh_coeffs_1)->coeff9_0 = _S123;

#line 86
                (&g_sh_coeffs_1)->coeff10_0 = _S124;

#line 86
                (&g_sh_coeffs_1)->coeff11_0 = _S125;

#line 86
                (&g_sh_coeffs_1)->coeff12_0 = _S126;

#line 86
                (&g_sh_coeffs_1)->coeff13_0 = _S127;

#line 86
                (&g_sh_coeffs_1)->coeff14_0 = _S128;

#line 86
                (&g_sh_coeffs_1)->coeff15_0 = _S129;

#line 79
            }
            else
            {

#line 79
                (&g_sh_coeffs_1)->coeff0_0 = _S114;

#line 79
                (&g_sh_coeffs_1)->coeff1_0 = _S115;

#line 79
                (&g_sh_coeffs_1)->coeff2_0 = _S116;

#line 79
                (&g_sh_coeffs_1)->coeff3_0 = _S117;

#line 79
                (&g_sh_coeffs_1)->coeff4_0 = _S118;

#line 79
                (&g_sh_coeffs_1)->coeff5_0 = _S119;

#line 79
                (&g_sh_coeffs_1)->coeff6_0 = _S120;

#line 79
                (&g_sh_coeffs_1)->coeff7_0 = _S121;

#line 79
                (&g_sh_coeffs_1)->coeff8_0 = _S122;

#line 79
                (&g_sh_coeffs_1)->coeff9_0 = _S113;

#line 79
                (&g_sh_coeffs_1)->coeff10_0 = _S113;

#line 79
                (&g_sh_coeffs_1)->coeff11_0 = _S113;

#line 79
                (&g_sh_coeffs_1)->coeff12_0 = _S113;

#line 79
                (&g_sh_coeffs_1)->coeff13_0 = _S113;

#line 79
                (&g_sh_coeffs_1)->coeff14_0 = _S113;

#line 79
                (&g_sh_coeffs_1)->coeff15_0 = _S113;

#line 79
            }

#line 72
        }
        else
        {

#line 72
            (&g_sh_coeffs_1)->coeff0_0 = _S114;

#line 72
            (&g_sh_coeffs_1)->coeff1_0 = _S115;

#line 72
            (&g_sh_coeffs_1)->coeff2_0 = _S116;

#line 72
            (&g_sh_coeffs_1)->coeff3_0 = _S117;

#line 72
            (&g_sh_coeffs_1)->coeff4_0 = _S113;

#line 72
            (&g_sh_coeffs_1)->coeff5_0 = _S113;

#line 72
            (&g_sh_coeffs_1)->coeff6_0 = _S113;

#line 72
            (&g_sh_coeffs_1)->coeff7_0 = _S113;

#line 72
            (&g_sh_coeffs_1)->coeff8_0 = _S113;

#line 72
            (&g_sh_coeffs_1)->coeff9_0 = _S113;

#line 72
            (&g_sh_coeffs_1)->coeff10_0 = _S113;

#line 72
            (&g_sh_coeffs_1)->coeff11_0 = _S113;

#line 72
            (&g_sh_coeffs_1)->coeff12_0 = _S113;

#line 72
            (&g_sh_coeffs_1)->coeff13_0 = _S113;

#line 72
            (&g_sh_coeffs_1)->coeff14_0 = _S113;

#line 72
            (&g_sh_coeffs_1)->coeff15_0 = _S113;

#line 72
        }

#line 67
    }
    else
    {

#line 67
        (&g_sh_coeffs_1)->coeff0_0 = _S114;

#line 67
        (&g_sh_coeffs_1)->coeff1_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff2_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff3_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff4_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff5_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff6_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff7_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff8_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff9_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff10_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff11_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff12_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff13_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff14_0 = _S113;

#line 67
        (&g_sh_coeffs_1)->coeff15_0 = _S113;

#line 67
    }

#line 67
    return g_sh_coeffs_1;
}


#line 67
__device__ float4  s_primal_ctx_read_t4_float4_0(uint idx_3, DiffTensorView_0 t4_1)
{

#line 34 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
    return make_float4 (DiffTensorView_load_0(t4_1, make_uint2 (idx_3, 0U)), DiffTensorView_load_0(t4_1, make_uint2 (idx_3, 1U)), DiffTensorView_load_0(t4_1, make_uint2 (idx_3, 2U)), DiffTensorView_load_0(t4_1, make_uint2 (idx_3, 3U)));
}


#line 34
__device__ Gaussian_3D_0 s_primal_ctx_load_gaussian_0(int g_idx_3, DiffTensorView_0 xyz_ws_4, DiffTensorView_0 sh_coeffs_4, DiffTensorView_0 rotations_2, DiffTensorView_0 scales_2, uint active_sh_5)
{

#line 177
    uint _S130 = uint(g_idx_3);

#line 182
    Gaussian_3D_0 _S131 = { s_primal_ctx_read_t3_float3_0(_S130, xyz_ws_4), s_primal_ctx_read_spherical_harmonics_coeffs_0(_S130, sh_coeffs_4, active_sh_5), s_primal_ctx_read_t4_float4_0(_S130, rotations_2), s_primal_ctx_read_t3_float3_0(_S130, scales_2) };

#line 182
    return _S131;
}


#line 182
__device__ Matrix<float, 4, 4>  s_primal_ctx_mul_0(Matrix<float, 4, 4>  _S132, Matrix<float, 4, 4>  _S133)
{

#line 182
    return mul_2(_S132, _S133);
}


#line 182
__device__ float4  s_primal_ctx_mul_1(Matrix<float, 4, 4>  _S134, float4  _S135)
{

#line 182
    return mul_4(_S134, _S135);
}


#line 182
__device__ float3  s_primal_ctx_geom_transform_points_0(float3  dppoint_0, Matrix<float, 4, 4>  dptransf_matrix_0)
{

#line 105
    float4  _S136 = s_primal_ctx_mul_1(dptransf_matrix_0, make_float4 (dppoint_0.x, dppoint_0.y, dppoint_0.z, 1.0f));

#line 105
    return float3 {_S136.x, _S136.y, _S136.z} / make_float3 (_S136.w + 1.00000001168609742e-07f);
}


#line 105
__device__ float3  s_primal_ctx_geom_transform_points2_0(float3  dppoint_1, Matrix<float, 4, 4>  dptransf_matrix_1)
{

#line 112
    return float3 {s_primal_ctx_mul_1(dptransf_matrix_1, make_float4 (dppoint_1.x, dppoint_1.y, dppoint_1.z, 1.0f)).x, s_primal_ctx_mul_1(dptransf_matrix_1, make_float4 (dppoint_1.x, dppoint_1.y, dppoint_1.z, 1.0f)).y, s_primal_ctx_mul_1(dptransf_matrix_1, make_float4 (dppoint_1.x, dppoint_1.y, dppoint_1.z, 1.0f)).z};
}


#line 112
__device__ float3  s_primal_ctx_project_point_0(float3  dppoint_2, Camera_0 dpcam_0)
{

#line 122
    float _S137 = s_primal_ctx_geom_transform_points2_0(dppoint_2, dpcam_0.world_view_transform_1).z;

#line 122
    float3  _S138 = s_primal_ctx_geom_transform_points_0(dppoint_2, s_primal_ctx_mul_0(dpcam_0.proj_mat_1, dpcam_0.world_view_transform_1));

#line 122
    *&((&_S138)->z) = _S137;

#line 122
    return _S138;
}


#line 122
__device__ float3  s_primal_ctx_max_0(float3  _S139, float3  _S140)
{

#line 122
    return max_0(_S139, _S140);
}


#line 122
__device__ float3  s_primal_ctx_compute_color_from_sh_coeffs_0(SpherHarmCoeffs_0 dpsh_0, float3  dpg_xyz_ws_0, float3  dpcam_pos_0, uint active_sh_6)
{

#line 96 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/spherical_harmonics.slang"
    float3  _S141 = normalize_0(dpg_xyz_ws_0 - dpcam_pos_0);

    float3  rgb_5 = make_float3 (0.282094806432724f) * dpsh_0.coeff0_0;

#line 98
    float3  rgb_6;
    if(active_sh_6 > 0U)
    {

#line 100
        float _S142 = _S141.y;

#line 100
        float _S143 = _S141.z;

#line 100
        float _S144 = _S141.x;

#line 100
        float3  rgb_7 = rgb_5 - make_float3 (0.48860251903533936f * _S142) * dpsh_0.coeff1_0 + make_float3 (0.48860251903533936f * _S143) * dpsh_0.coeff2_0 - make_float3 (0.48860251903533936f * _S144) * dpsh_0.coeff3_0;
        if(active_sh_6 > 1U)
        {
            float xx_1 = _S144 * _S144;

#line 103
            float yy_1 = _S142 * _S142;

#line 103
            float zz_1 = _S143 * _S143;
            float xy_1 = _S144 * _S142;



            float _S145 = 2.0f * zz_1;

            float _S146 = xx_1 - yy_1;

#line 109
            float3  rgb_8 = rgb_7 + make_float3 (1.09254848957061768f * xy_1) * dpsh_0.coeff4_0 + make_float3 (-1.09254848957061768f * (_S142 * _S143)) * dpsh_0.coeff5_0 + make_float3 (0.31539157032966614f * (_S145 - xx_1 - yy_1)) * dpsh_0.coeff6_0 + make_float3 (-1.09254848957061768f * (_S144 * _S143)) * dpsh_0.coeff7_0 + make_float3 (0.54627424478530884f * _S146) * dpsh_0.coeff8_0;


            if(active_sh_6 > 2U)
            {

                float _S147 = 3.0f * xx_1;

                float _S148 = 4.0f * zz_1 - xx_1 - yy_1;
                float _S149 = 3.0f * yy_1;

#line 118
                rgb_6 = rgb_8 + make_float3 (-0.59004360437393188f * _S142 * (_S147 - yy_1)) * dpsh_0.coeff9_0 + make_float3 (2.89061141014099121f * xy_1 * _S143) * dpsh_0.coeff10_0 + make_float3 (-0.4570457935333252f * _S142 * _S148) * dpsh_0.coeff11_0 + make_float3 (0.37317633628845215f * _S143 * (_S145 - _S147 - _S149)) * dpsh_0.coeff12_0 + make_float3 (-0.4570457935333252f * _S144 * _S148) * dpsh_0.coeff13_0 + make_float3 (1.44530570507049561f * _S143 * _S146) * dpsh_0.coeff14_0 + make_float3 (-0.59004360437393188f * _S144 * (xx_1 - _S149)) * dpsh_0.coeff15_0;

#line 112
            }
            else
            {

#line 112
                rgb_6 = rgb_8;

#line 112
            }

#line 101
        }
        else
        {

#line 101
            rgb_6 = rgb_7;

#line 101
        }

#line 99
    }
    else
    {

#line 99
        rgb_6 = rgb_5;

#line 99
    }

#line 99
    return s_primal_ctx_max_0(rgb_6 + make_float3 (0.5f), make_float3 (0.0f));
}


#line 99
__device__ Matrix<float, 3, 3>  s_primal_ctx_mul_2(Matrix<float, 3, 3>  _S150, Matrix<float, 3, 3>  _S151)
{

#line 99
    return mul_3(_S150, _S151);
}


#line 99
__device__ Matrix<float, 3, 3>  s_primal_ctx_get_covariance_from_quat_scales_0(float4  dpq_0, float3  dps_0)
{

#line 280 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
    float _S152 = dpq_0.z;



    float _S153 = _S152 * _S152;

#line 284
    float _S154 = dpq_0.w * dpq_0.w;

#line 284
    float _S155 = dpq_0.y * dpq_0.z;

#line 284
    float _S156 = dpq_0.x * dpq_0.w;

#line 284
    float _S157 = dpq_0.y * dpq_0.w;

#line 284
    float _S158 = dpq_0.x * dpq_0.z;
    float _S159 = dpq_0.y * dpq_0.y;

#line 285
    float _S160 = dpq_0.z * dpq_0.w;

#line 285
    float _S161 = dpq_0.x * dpq_0.y;

#line 285
    Matrix<float, 3, 3>  _S162 = s_primal_ctx_mul_2(makeMatrix<float, 3, 3> (1.0f - 2.0f * (_S153 + _S154), 2.0f * (_S155 - _S156), 2.0f * (_S157 + _S158), 2.0f * (_S155 + _S156), 1.0f - 2.0f * (_S159 + _S154), 2.0f * (_S160 - _S161), 2.0f * (_S157 - _S158), 2.0f * (_S160 + _S161), 1.0f - 2.0f * (_S159 + _S153)), makeMatrix<float, 3, 3> (dps_0.x, 0.0f, 0.0f, 0.0f, dps_0.y, 0.0f, 0.0f, 0.0f, dps_0.z));

#line 285
    return s_primal_ctx_mul_2(_S162, transpose_0(_S162));
}


#line 285
__device__ float s_primal_ctx_tan_0(float _S163)
{

#line 285
    return (F32_tan((_S163)));
}


#line 285
__device__ float s_primal_ctx_max_1(float _S164, float _S165)
{

#line 285
    return (F32_max((_S164), (_S165)));
}


#line 285
__device__ float s_primal_ctx_min_0(float _S166, float _S167)
{

#line 285
    return (F32_min((_S166), (_S167)));
}


#line 285
__device__ Matrix<float, 3, 3>  s_primal_ctx_compute_jacobian_0(float3  dpxyz_ws_0, Camera_0 dpcam_1)
{

#line 127
    float _S168 = s_primal_ctx_tan_0(dpcam_1.fovx_1 / 2.0f);

#line 127
    float _S169 = s_primal_ctx_tan_0(dpcam_1.fovy_1 / 2.0f);


    float h_x_1 = float(dpcam_1.W_0) / (2.0f * _S168);
    float h_y_1 = float(dpcam_1.H_0) / (2.0f * _S169);

#line 131
    float3  _S170 = s_primal_ctx_geom_transform_points_0(dpxyz_ws_0, dpcam_1.world_view_transform_1);

#line 136
    float limx_1 = 1.29999995231628418f * _S168;
    float limy_1 = 1.29999995231628418f * _S169;
    float _S171 = _S170.z;
    float tytz_1 = _S170.y / _S171;
    float _S172 = s_primal_ctx_min_0(limx_1, s_primal_ctx_max_1(- limx_1, _S170.x / _S171)) * _S171;

#line 140
    float3  _S173 = _S170;

#line 140
    *&((&_S173)->x) = _S172;

#line 140
    *&((&_S173)->y) = s_primal_ctx_min_0(limy_1, s_primal_ctx_max_1(- limy_1, tytz_1)) * _S173.z;


    float _S174 = _S173.z;

#line 143
    float _S175 = _S174 * _S174;

#line 143
    return makeMatrix<float, 3, 3> (h_x_1 / _S174, 0.0f, - (h_x_1 * _S173.x) / _S175, 0.0f, h_y_1 / _S174, - (h_y_1 * _S173.y) / _S175, 0.0f, 0.0f, 0.0f);
}


#line 143
__device__ Matrix<float, 2, 2>  s_primal_ctx_covariance_3d_to_2d_0(Camera_0 dpcam_2, float3  dpxyz_ws_1, Matrix<float, 3, 3>  dpcov_ws_0)
{

#line 151
    Matrix<float, 3, 3>  _S176 = makeMatrix<float, 3, 3> (float3 {dpcam_2.world_view_transform_1.rows[int(0)].x, dpcam_2.world_view_transform_1.rows[int(0)].y, dpcam_2.world_view_transform_1.rows[int(0)].z}, float3 {dpcam_2.world_view_transform_1.rows[int(1)].x, dpcam_2.world_view_transform_1.rows[int(1)].y, dpcam_2.world_view_transform_1.rows[int(1)].z}, float3 {dpcam_2.world_view_transform_1.rows[int(2)].x, dpcam_2.world_view_transform_1.rows[int(2)].y, dpcam_2.world_view_transform_1.rows[int(2)].z});

#line 151
    Matrix<float, 3, 3>  _S177 = s_primal_ctx_compute_jacobian_0(dpxyz_ws_1, dpcam_2);

#line 151
    Matrix<float, 3, 3>  _S178 = s_primal_ctx_mul_2(_S177, s_primal_ctx_mul_2(_S176, s_primal_ctx_mul_2(dpcov_ws_0, s_primal_ctx_mul_2(transpose_0(_S176), transpose_0(_S177)))));



    float _S179 = _S178.rows[int(0)].x + 0.30000001192092896f;

#line 155
    Matrix<float, 3, 3>  _S180 = _S178;

#line 155
    *&(((&_S180)->rows + (int(0)))->x) = _S179;

#line 155
    *&(((&_S180)->rows + (int(1)))->y) = _S178.rows[int(1)].y + 0.30000001192092896f;

#line 155
    return makeMatrix<float, 2, 2> (float2 {_S180.rows[int(0)].x, _S180.rows[int(0)].y}, float2 {_S180.rows[int(1)].x, _S180.rows[int(1)].y});
}


#line 155
__device__ Splat_2D_Vertex_0 s_primal_ctx_project_gaussian_to_camera_0(Gaussian_3D_0 dpg_0, Camera_0 dpcam_3, uint active_sh_7)
{

#line 222
    float3  _S181 = s_primal_ctx_project_point_0(dpg_0.xyz_ws_0, dpcam_3);

    bool _S182 = _S181.z <= 0.20000000298023224f;

#line 224
    Splat_2D_Vertex_0 _S183;

#line 224
    if(_S182)
    {

#line 225
        float3  _S184 = make_float3 (0.0f);

#line 225
        Matrix<float, 2, 2>  _S185 = makeMatrix<float, 2, 2> (0.0f);

#line 225
        (&_S183)->xyz_vs_0 = _S184;

#line 225
        (&_S183)->rgb_0 = _S184;

#line 225
        (&_S183)->cov_vs_0 = _S185;

#line 225
    }

#line 225
    bool _S186 = !_S182;

#line 225
    if(_S186)
    {

#line 225
        float3  _S187 = s_primal_ctx_compute_color_from_sh_coeffs_0(dpg_0.sh_coeffs_0, dpg_0.xyz_ws_0, dpcam_3.position_1, active_sh_7);

#line 225
        Matrix<float, 2, 2>  _S188 = s_primal_ctx_covariance_3d_to_2d_0(dpcam_3, dpg_0.xyz_ws_0, s_primal_ctx_get_covariance_from_quat_scales_0(dpg_0.rotations_0, dpg_0.scales_0));

#line 225
        (&_S183)->xyz_vs_0 = _S181;

#line 225
        (&_S183)->rgb_0 = _S187;

#line 225
        (&_S183)->cov_vs_0 = _S188;

#line 225
    }

#line 225
    return _S183;
}


#line 225
__device__ float s_primal_ctx_compute_det_0(Matrix<float, 2, 2>  dpM_0)
{

#line 203
    return dpM_0.rows[int(0)].x * dpM_0.rows[int(1)].y - dpM_0.rows[int(0)].y * dpM_0.rows[int(1)].x;
}


#line 203
__device__ float s_primal_ctx_ndc2pix_0(float dpv_0, int S_1)
{

#line 61
    return ((dpv_0 + 1.0f) * float(S_1) - 1.0f) * 0.5f;
}


#line 61
__device__ void s_primal_ctx_vertex_shader_0(DiffTensorView_0 xyz_ws_5, DiffTensorView_0 sh_coeffs_5, DiffTensorView_0 rotations_3, DiffTensorView_0 scales_3, uint active_sh_8, TensorView world_view_transform_3, TensorView proj_mat_3, TensorView cam_pos_1, TensorView out_tiles_touched_0, TensorView out_rect_tile_space_0, TensorView out_radii_0, DiffTensorView_0 out_xyz_vs_0, DiffTensorView_0 out_inv_cov_vs_0, DiffTensorView_0 out_rgb_0, float fovy_3, float fovx_3, uint image_height_0, uint image_width_0, uint grid_height_1, uint grid_width_1, uint tile_height_1, uint tile_width_1, s_bwd_prop_vertex_shader_Intermediates_0 * _s_diff_ctx_0)
{

#line 58 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/vertex_shader.slang"
    Matrix<float, 4, 4>  _S189 = makeMatrix<float, 4, 4> (0.0f);

#line 58
    float3  _S190 = make_float3 (0.0f);

#line 58
    Camera_0 _S191 = { _S189, _S189, _S190, 0.0f, 0.0f, int(0), int(0) };

#line 58
    SpherHarmCoeffs_0 _S192 = { _S190, _S190, _S190, _S190, _S190, _S190, _S190, _S190, _S190, _S190, _S190, _S190, _S190, _S190, _S190, _S190 };

#line 58
    float4  _S193 = make_float4 (0.0f);

#line 58
    Gaussian_3D_0 _S194 = { _S190, _S192, _S193, _S190 };

#line 58
    _s_diff_ctx_0->_S111 = _S191;

#line 58
    _s_diff_ctx_0->_S112 = _S194;

#line 58
    (&_s_diff_ctx_0->_S111)->world_view_transform_1 = _S189;

#line 58
    (&_s_diff_ctx_0->_S111)->proj_mat_1 = _S189;

#line 58
    (&_s_diff_ctx_0->_S111)->position_1 = _S190;

#line 58
    (&_s_diff_ctx_0->_S111)->fovy_1 = 0.0f;

#line 58
    (&_s_diff_ctx_0->_S111)->fovx_1 = 0.0f;

#line 58
    (&_s_diff_ctx_0->_S111)->H_0 = int(0);

#line 58
    (&_s_diff_ctx_0->_S111)->W_0 = int(0);

#line 58
    (&_s_diff_ctx_0->_S112)->xyz_ws_0 = _S190;

#line 58
    (&_s_diff_ctx_0->_S112)->sh_coeffs_0 = _S192;

#line 58
    (&_s_diff_ctx_0->_S112)->rotations_0 = _S193;

#line 58
    (&_s_diff_ctx_0->_S112)->scales_0 = _S190;

    uint g_idx_4 = ((blockIdx)).x * ((blockDim)).x + ((threadIdx)).x;

#line 60
    bool _S195 = !(g_idx_4 >= DiffTensorView_size_0(xyz_ws_5, 0U));

#line 60
    if(_S195)
    {



        Camera_0 cam_4 = load_camera_0(world_view_transform_3, proj_mat_3, cam_pos_1, fovy_3, fovx_3, image_height_0, image_width_0);

#line 65
        _s_diff_ctx_0->_S111 = cam_4;

#line 65
        Gaussian_3D_0 _S196 = s_primal_ctx_load_gaussian_0(int(g_idx_4), xyz_ws_5, sh_coeffs_5, rotations_3, scales_3, active_sh_8);
        _s_diff_ctx_0->_S112 = _S196;

#line 66
        Splat_2D_Vertex_0 _S197 = s_primal_ctx_project_gaussian_to_camera_0(_S196, cam_4, active_sh_8);

        float _S198 = _S197.xyz_vs_0.z;

#line 68
        bool _bflag_0;

#line 68
        if(_S198 <= 0.20000000298023224f)
        {

#line 68
            _bflag_0 = false;

#line 68
        }
        else
        {

#line 68
            _bflag_0 = _S195;

#line 68
        }

#line 68
        if(_bflag_0)
        {

#line 68
            float _S199 = s_primal_ctx_compute_det_0(_S197.cov_vs_0);

#line 74
            if(_S199 == 0.0f)
            {

#line 74
                _bflag_0 = false;

#line 74
            }

#line 74
            if(_bflag_0)
            {
                float radius_1 = splat_radius_0(_S197.cov_vs_0, _S199);


                float _S200 = _S197.xyz_vs_0.x;

#line 79
                float _S201 = _S197.xyz_vs_0.y;
                rectangle_0 rect_tile_space_1 = get_rectangle_tile_space_0(make_float2 (s_primal_ctx_ndc2pix_0(_S200, int(image_width_0)), s_primal_ctx_ndc2pix_0(_S201, int(image_height_0))), radius_1, grid_height_1, grid_width_1, tile_height_1, tile_width_1);

                int n_tiles_0 = (rect_tile_space_1.max_x_0 - rect_tile_space_1.min_x_0) * (rect_tile_space_1.max_y_0 - rect_tile_space_1.min_y_0);

                if(n_tiles_0 == int(0))
                {

#line 84
                    _bflag_0 = false;

#line 84
                }

#line 84
                if(_bflag_0)
                {


                    Matrix<float, 2, 2>  g_inv_cov_vs_0 = makeMatrix<float, 2, 2> (_S197.cov_vs_0.rows[int(1)].y, - _S197.cov_vs_0.rows[int(0)].y, - _S197.cov_vs_0.rows[int(1)].x, _S197.cov_vs_0.rows[int(0)].x) / makeMatrix<float, 2, 2> (_S199);

                    (out_radii_0).store<int>((g_idx_4), (int(uint(radius_1))));
                    (out_tiles_touched_0).store<int>((g_idx_4), (n_tiles_0));
                    uint2  _S202 = make_uint2 (g_idx_4, 0U);

#line 92
                    (out_rect_tile_space_0).store<int>((g_idx_4), (0U), (rect_tile_space_1.min_x_0));
                    uint2  _S203 = make_uint2 (g_idx_4, 1U);

#line 93
                    (out_rect_tile_space_0).store<int>((g_idx_4), (1U), (rect_tile_space_1.min_y_0));
                    uint2  _S204 = make_uint2 (g_idx_4, 2U);

#line 94
                    (out_rect_tile_space_0).store<int>((g_idx_4), (2U), (rect_tile_space_1.max_x_0));
                    (out_rect_tile_space_0).store<int>((g_idx_4), (3U), (rect_tile_space_1.max_y_0));

                    DiffTensorView_storeOnce_0(out_xyz_vs_0, _S202, _S200);
                    DiffTensorView_storeOnce_0(out_xyz_vs_0, _S203, _S201);
                    DiffTensorView_storeOnce_0(out_xyz_vs_0, _S204, _S198);
                    DiffTensorView_storeOnce_1(out_inv_cov_vs_0, make_uint3 (g_idx_4, 0U, 0U), g_inv_cov_vs_0.rows[int(0)].x);
                    DiffTensorView_storeOnce_1(out_inv_cov_vs_0, make_uint3 (g_idx_4, 0U, 1U), g_inv_cov_vs_0.rows[int(0)].y);
                    DiffTensorView_storeOnce_1(out_inv_cov_vs_0, make_uint3 (g_idx_4, 1U, 0U), g_inv_cov_vs_0.rows[int(1)].x);
                    DiffTensorView_storeOnce_1(out_inv_cov_vs_0, make_uint3 (g_idx_4, 1U, 1U), g_inv_cov_vs_0.rows[int(1)].y);
                    DiffTensorView_storeOnce_0(out_rgb_0, _S202, _S197.rgb_0.x);
                    DiffTensorView_storeOnce_0(out_rgb_0, _S203, _S197.rgb_0.y);
                    DiffTensorView_storeOnce_0(out_rgb_0, _S204, _S197.rgb_0.z);

#line 106
                }

#line 106
            }

#line 106
        }

#line 106
    }

#line 106
    return;
}


#line 61 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
__device__ void s_bwd_prop_ndc2pix_0(DiffPair_float_0 * dpv_1, int S_2, float _s_dOut_0)
{
    float _S205 = float(S_2) * (0.5f * _s_dOut_0);

#line 63
    dpv_1->primal_1 = (*dpv_1).primal_1;

#line 63
    dpv_1->differential_0 = _S205;

#line 61
    return;
}


#line 61
struct DiffPair_matrixx3Cfloatx2C2x2C2x3E_0
{
    Matrix<float, 2, 2>  primal_1;
    Matrix<float, 2, 2>  differential_0;
};


#line 203
__device__ void s_bwd_prop_compute_det_0(DiffPair_matrixx3Cfloatx2C2x2C2x3E_0 * dpM_1, float _s_dOut_1)
{

#line 204
    float _S206 = - _s_dOut_1;

#line 204
    float _S207 = (*dpM_1).primal_1.rows[int(0)].y * _S206;

#line 204
    float _S208 = (*dpM_1).primal_1.rows[int(1)].x * _S206;

#line 204
    float _S209 = (*dpM_1).primal_1.rows[int(0)].x * _s_dOut_1;

#line 204
    float _S210 = (*dpM_1).primal_1.rows[int(1)].y * _s_dOut_1;

#line 1751 "core.meta.slang"
    float2  _S211 = make_float2 (0.0f);

#line 1751
    float2  _S212 = _S211;

#line 1751
    *&((&_S212)->x) = _S207;

#line 1751
    *&((&_S212)->y) = _S209;

#line 1751
    float2  _S213 = _S211;

#line 1751
    *&((&_S213)->y) = _S208;

#line 1751
    *&((&_S213)->x) = _S210;

#line 1751
    Matrix<float, 2, 2>  _S214 = makeMatrix<float, 2, 2> (0.0f);

#line 1751
    _S214[int(1)] = _S212;

#line 1751
    _S214[int(0)] = _S213;

#line 1751
    dpM_1->primal_1 = (*dpM_1).primal_1;

#line 1751
    dpM_1->differential_0 = _S214;

#line 203 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
    return;
}


#line 203
struct DiffPair_Gaussian_3D_0
{
    Gaussian_3D_0 primal_1;
    Gaussian_3D_0 differential_0;
};


#line 67 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/vertex_shader.slang"
struct DiffPair_Camera_0
{
    Camera_0 primal_1;
    Camera_Differential_0 differential_0;
};


#line 229 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
__device__ void s_bwd_prop_mul_0(DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * _S215, DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * _S216, Matrix<float, 3, 3>  _S217)
{

#line 229
    mul_1(_S215, _S216, _S217);

#line 229
    return;
}


#line 229
__device__ void s_bwd_prop_min_0(DiffPair_float_0 * _S218, DiffPair_float_0 * _S219, float _S220)
{

#line 229
    _d_min_0(_S218, _S219, _S220);

#line 229
    return;
}


#line 229
__device__ void s_bwd_prop_max_0(DiffPair_float_0 * _S221, DiffPair_float_0 * _S222, float _S223)
{

#line 229
    _d_max_0(_S221, _S222, _S223);

#line 229
    return;
}


#line 228
__device__ void s_bwd_prop_mul_1(DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 * _S224, DiffPair_vectorx3Cfloatx2C4x3E_0 * _S225, float4  _S226)
{

#line 228
    _d_mul_0(_S224, _S225, _S226);

#line 228
    return;
}


#line 105
__device__ void s_bwd_prop_geom_transform_points_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * dppoint_3, DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 * dptransf_matrix_2, float3  _s_dOut_2)
{
    float4  _S227 = make_float4 ((*dppoint_3).primal_1.x, (*dppoint_3).primal_1.y, (*dppoint_3).primal_1.z, 1.0f);

#line 107
    float4  _S228 = s_primal_ctx_mul_1((*dptransf_matrix_2).primal_1, _S227);
    float _S229 = _S228.w + 1.00000001168609742e-07f;

#line 108
    float3  _S230 = _s_dOut_2 / make_float3 (_S229 * _S229);

#line 108
    float3  _S231 = float3 {_S228.x, _S228.y, _S228.z} * - _S230;

#line 108
    float3  _S232 = make_float3 (_S229) * _S230;

#line 107
    float4  _S233 = make_float4 (_S232.x, _S232.y, _S232.z, _S231.x + _S231.y + _S231.z);

#line 107
    Matrix<float, 4, 4>  _S234 = makeMatrix<float, 4, 4> (0.0f);

#line 107
    DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 _S235;

#line 107
    (&_S235)->primal_1 = (*dptransf_matrix_2).primal_1;

#line 107
    (&_S235)->differential_0 = _S234;

#line 107
    float4  _S236 = make_float4 (0.0f);

#line 107
    DiffPair_vectorx3Cfloatx2C4x3E_0 _S237;

#line 107
    (&_S237)->primal_1 = _S227;

#line 107
    (&_S237)->differential_0 = _S236;

#line 107
    s_bwd_prop_mul_1(&_S235, &_S237, _S233);

#line 107
    float3  _S238 = float3 {_S237.differential_0.x, _S237.differential_0.y, _S237.differential_0.z};

#line 107
    dptransf_matrix_2->primal_1 = (*dptransf_matrix_2).primal_1;

#line 107
    dptransf_matrix_2->differential_0 = _S235.differential_0;

#line 107
    dppoint_3->primal_1 = (*dppoint_3).primal_1;

#line 107
    dppoint_3->differential_0 = _S238;

#line 105
    return;
}


#line 105
__device__ void s_bwd_prop_tan_0(DiffPair_float_0 * _S239, float _S240)
{

#line 105
    _d_tan_0(_S239, _S240);

#line 105
    return;
}


#line 127
__device__ void s_bwd_prop_compute_jacobian_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * dpxyz_ws_2, DiffPair_Camera_0 * dpcam_4, Matrix<float, 3, 3>  s_diff_J_T_0)
{

#line 128
    float _S241 = (*dpcam_4).primal_1.fovx_1 / 2.0f;

#line 128
    float _S242 = s_primal_ctx_tan_0(_S241);
    float _S243 = (*dpcam_4).primal_1.fovy_1 / 2.0f;

#line 129
    float _S244 = s_primal_ctx_tan_0(_S243);
    float _S245 = float((*dpcam_4).primal_1.W_0);

#line 130
    float _S246 = 2.0f * _S242;

#line 130
    float h_x_2 = _S245 / _S246;

#line 130
    float _S247 = _S246 * _S246;
    float _S248 = float((*dpcam_4).primal_1.H_0);

#line 131
    float _S249 = 2.0f * _S244;

#line 131
    float h_y_2 = _S248 / _S249;

#line 131
    float _S250 = _S249 * _S249;

#line 131
    float3  _S251 = s_primal_ctx_geom_transform_points_0((*dpxyz_ws_2).primal_1, (*dpcam_4).primal_1.world_view_transform_1);

#line 136
    float limx_2 = 1.29999995231628418f * _S242;
    float limy_2 = 1.29999995231628418f * _S244;
    float _S252 = _S251.x;

#line 138
    float _S253 = _S251.z;

#line 138
    float txtz_0 = _S252 / _S253;

#line 138
    float _S254 = _S253 * _S253;
    float _S255 = _S251.y;

#line 139
    float tytz_2 = _S255 / _S253;
    float _S256 = - limx_2;

#line 140
    float _S257 = s_primal_ctx_max_1(_S256, txtz_0);

#line 140
    float _S258 = s_primal_ctx_min_0(limx_2, _S257);

#line 140
    float _S259 = _S258 * _S253;

#line 140
    float3  _S260 = _S251;

#line 140
    *&((&_S260)->x) = _S259;
    float _S261 = - limy_2;

#line 141
    float _S262 = s_primal_ctx_max_1(_S261, tytz_2);

#line 141
    float _S263 = s_primal_ctx_min_0(limy_2, _S262);

#line 141
    float _S264 = _S260.z;

#line 141
    *&((&_S260)->y) = _S263 * _S264;

    float _S265 = _S260.z;

#line 143
    float _S266 = _S265 * _S265;

#line 143
    float _S267 = _S260.x;

#line 143
    float _S268 = _S266 * _S266;
    float _S269 = _S260.y;

#line 144
    float _S270 = s_diff_J_T_0.rows[int(1)].z / _S268;

#line 144
    float _S271 = - (_S266 * _S270);

#line 144
    float _S272 = h_y_2 * _S271;

#line 144
    float _S273 = _S269 * _S271;

#line 144
    float _S274 = s_diff_J_T_0.rows[int(1)].y / _S266;

#line 144
    float _S275 = _S265 * _S274;

#line 143
    float _S276 = s_diff_J_T_0.rows[int(0)].z / _S268;

#line 143
    float _S277 = _S265 * (- (h_y_2 * _S269) * - _S270 + - (h_x_2 * _S267) * - _S276);

#line 143
    float _S278 = - (_S266 * _S276);

#line 143
    float _S279 = _S267 * _S278;

#line 143
    float _S280 = s_diff_J_T_0.rows[int(0)].x / _S266;

#line 143
    float _S281 = _S265 * _S280;

#line 143
    _S260 = make_float3 (h_x_2 * _S278, _S272, h_y_2 * - _S274 + _S277 + _S277 + h_x_2 * - _S280);

#line 143
    *&((&_S260)->y) = 0.0f;

#line 141
    float _S282 = _S263 * _S272;

#line 141
    float _S283 = _S264 * _S272;

#line 141
    DiffPair_float_0 _S284;

#line 141
    (&_S284)->primal_1 = limy_2;

#line 141
    (&_S284)->differential_0 = 0.0f;

#line 141
    DiffPair_float_0 _S285;

#line 141
    (&_S285)->primal_1 = _S262;

#line 141
    (&_S285)->differential_0 = 0.0f;

#line 141
    s_bwd_prop_min_0(&_S284, &_S285, _S283);

#line 141
    DiffPair_float_0 _S286;

#line 141
    (&_S286)->primal_1 = _S261;

#line 141
    (&_S286)->differential_0 = 0.0f;

#line 141
    DiffPair_float_0 _S287;

#line 141
    (&_S287)->primal_1 = tytz_2;

#line 141
    (&_S287)->differential_0 = 0.0f;

#line 141
    s_bwd_prop_max_0(&_S286, &_S287, _S285.differential_0);

#line 141
    float _S288 = - _S286.differential_0;

#line 140
    float3  _S289 = _S260 + make_float3 (0.0f, 0.0f, _S282);

#line 140
    _S260 = _S289;

#line 140
    *&((&_S260)->x) = 0.0f;

#line 140
    float _S290 = _S258 * _S289.x;

#line 140
    float _S291 = _S253 * _S289.x;

#line 140
    DiffPair_float_0 _S292;

#line 140
    (&_S292)->primal_1 = limx_2;

#line 140
    (&_S292)->differential_0 = 0.0f;

#line 140
    DiffPair_float_0 _S293;

#line 140
    (&_S293)->primal_1 = _S257;

#line 140
    (&_S293)->differential_0 = 0.0f;

#line 140
    s_bwd_prop_min_0(&_S292, &_S293, _S291);

#line 140
    DiffPair_float_0 _S294;

#line 140
    (&_S294)->primal_1 = _S256;

#line 140
    (&_S294)->differential_0 = 0.0f;

#line 140
    DiffPair_float_0 _S295;

#line 140
    (&_S295)->primal_1 = txtz_0;

#line 140
    (&_S295)->differential_0 = 0.0f;

#line 140
    s_bwd_prop_max_0(&_S294, &_S295, _S293.differential_0);

#line 139
    float _S296 = _S287.differential_0 / _S254;

#line 138
    float _S297 = _S295.differential_0 / _S254;

#line 137
    float _S298 = 1.29999995231628418f * (_S284.differential_0 + _S288);

#line 136
    float _S299 = 1.29999995231628418f * (_S292.differential_0 + - _S294.differential_0);

#line 133
    float3  _S300 = _S260 + make_float3 (_S253 * _S297, _S253 * _S296, _S290 + _S255 * - _S296 + _S252 * - _S297);

#line 133
    float3  _S301 = make_float3 (0.0f);

#line 133
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S302;

#line 133
    (&_S302)->primal_1 = (*dpxyz_ws_2).primal_1;

#line 133
    (&_S302)->differential_0 = _S301;

#line 133
    Matrix<float, 4, 4>  _S303 = makeMatrix<float, 4, 4> (0.0f);

#line 133
    DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 _S304;

#line 133
    (&_S304)->primal_1 = (*dpcam_4).primal_1.world_view_transform_1;

#line 133
    (&_S304)->differential_0 = _S303;

#line 133
    s_bwd_prop_geom_transform_points_0(&_S302, &_S304, _S300);

#line 130
    float _S305 = 2.0f * (_S245 * - ((_S279 + _S281) / _S247));

#line 129
    float _S306 = _S298 + 2.0f * (_S248 * - ((_S273 + _S275) / _S250));

#line 129
    DiffPair_float_0 _S307;

#line 129
    (&_S307)->primal_1 = _S243;

#line 129
    (&_S307)->differential_0 = 0.0f;

#line 129
    s_bwd_prop_tan_0(&_S307, _S306);

#line 129
    float _S308 = 0.5f * _S307.differential_0;

#line 128
    float _S309 = _S299 + _S305;

#line 128
    DiffPair_float_0 _S310;

#line 128
    (&_S310)->primal_1 = _S241;

#line 128
    (&_S310)->differential_0 = 0.0f;

#line 128
    s_bwd_prop_tan_0(&_S310, _S309);

#line 128
    float _S311 = 0.5f * _S310.differential_0;

#line 128
    Camera_Differential_0 _S312 = Camera_x24_syn_dzero_0();

#line 128
    (&_S312)->world_view_transform_0 = _S304.differential_0;

#line 128
    (&_S312)->fovy_0 = _S308;

#line 128
    (&_S312)->fovx_0 = _S311;

#line 128
    dpcam_4->primal_1 = (*dpcam_4).primal_1;

#line 128
    dpcam_4->differential_0 = _S312;

#line 128
    dpxyz_ws_2->primal_1 = (*dpxyz_ws_2).primal_1;

#line 128
    dpxyz_ws_2->differential_0 = _S302.differential_0;

#line 127
    return;
}


#line 151
__device__ void s_bwd_prop_covariance_3d_to_2d_0(DiffPair_Camera_0 * dpcam_5, DiffPair_vectorx3Cfloatx2C3x3E_0 * dpxyz_ws_3, DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 * dpcov_ws_1, Matrix<float, 2, 2>  _s_dOut_3)
{

#line 151
    Matrix<float, 3, 3>  _S313 = makeMatrix<float, 3, 3> (float3 {(*dpcam_5).primal_1.world_view_transform_1.rows[int(0)].x, (*dpcam_5).primal_1.world_view_transform_1.rows[int(0)].y, (*dpcam_5).primal_1.world_view_transform_1.rows[int(0)].z}, float3 {(*dpcam_5).primal_1.world_view_transform_1.rows[int(1)].x, (*dpcam_5).primal_1.world_view_transform_1.rows[int(1)].y, (*dpcam_5).primal_1.world_view_transform_1.rows[int(1)].z}, float3 {(*dpcam_5).primal_1.world_view_transform_1.rows[int(2)].x, (*dpcam_5).primal_1.world_view_transform_1.rows[int(2)].y, (*dpcam_5).primal_1.world_view_transform_1.rows[int(2)].z});

#line 151
    Matrix<float, 3, 3>  _S314 = s_primal_ctx_compute_jacobian_0((*dpxyz_ws_3).primal_1, (*dpcam_5).primal_1);


    Matrix<float, 3, 3>  _S315 = transpose_0(_S313);

#line 154
    Matrix<float, 3, 3>  _S316 = transpose_0(_S314);

#line 154
    Matrix<float, 3, 3>  _S317 = s_primal_ctx_mul_2(_S315, _S316);

#line 154
    Matrix<float, 3, 3>  _S318 = s_primal_ctx_mul_2((*dpcov_ws_1).primal_1, _S317);

#line 154
    Matrix<float, 3, 3>  _S319 = s_primal_ctx_mul_2(_S313, _S318);

#line 154
    float3  _S320 = make_float3 (_s_dOut_3.rows[int(1)].x, _s_dOut_3.rows[int(1)].y, 0.0f);

#line 154
    float3  _S321 = make_float3 (_s_dOut_3.rows[int(0)].x, _s_dOut_3.rows[int(0)].y, 0.0f);

    Matrix<float, 3, 3>  _S322 = makeMatrix<float, 3, 3> (0.0f);

#line 156
    Matrix<float, 3, 3>  _S323 = _S322;

#line 156
    _S323[int(1)] = _S320;

#line 156
    _S323[int(0)] = _S321;

#line 156
    Matrix<float, 3, 3>  _S324 = _S323;

#line 156
    *&(((&_S324)->rows + (int(1)))->y) = 0.0f;

#line 1751 "core.meta.slang"
    float3  _S325 = make_float3 (0.0f);

#line 1751
    float3  _S326 = _S325;

#line 1751
    *&((&_S326)->y) = _S323.rows[int(1)].y;

#line 1751
    *&(((&_S324)->rows + (int(0)))->x) = 0.0f;

#line 155 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
    float3  _S327 = _S325;

#line 155
    *&((&_S327)->x) = _S323.rows[int(0)].x;

#line 154
    Matrix<float, 3, 3>  _S328 = _S322;

#line 154
    _S328[int(1)] = _S326;

#line 154
    _S328[int(0)] = _S327;

#line 154
    Matrix<float, 3, 3>  _S329 = _S324 + _S328;

#line 154
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S330;

#line 154
    (&_S330)->primal_1 = _S314;

#line 154
    (&_S330)->differential_0 = _S322;

#line 154
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S331;

#line 154
    (&_S331)->primal_1 = _S319;

#line 154
    (&_S331)->differential_0 = _S322;

#line 154
    s_bwd_prop_mul_0(&_S330, &_S331, _S329);

#line 154
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S332;

#line 154
    (&_S332)->primal_1 = _S313;

#line 154
    (&_S332)->differential_0 = _S322;

#line 154
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S333;

#line 154
    (&_S333)->primal_1 = _S318;

#line 154
    (&_S333)->differential_0 = _S322;

#line 154
    s_bwd_prop_mul_0(&_S332, &_S333, _S331.differential_0);

#line 154
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S334;

#line 154
    (&_S334)->primal_1 = (*dpcov_ws_1).primal_1;

#line 154
    (&_S334)->differential_0 = _S322;

#line 154
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S335;

#line 154
    (&_S335)->primal_1 = _S317;

#line 154
    (&_S335)->differential_0 = _S322;

#line 154
    s_bwd_prop_mul_0(&_S334, &_S335, _S333.differential_0);

#line 154
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S336;

#line 154
    (&_S336)->primal_1 = _S315;

#line 154
    (&_S336)->differential_0 = _S322;

#line 154
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S337;

#line 154
    (&_S337)->primal_1 = _S316;

#line 154
    (&_S337)->differential_0 = _S322;

#line 154
    s_bwd_prop_mul_0(&_S336, &_S337, _S335.differential_0);

#line 154
    Matrix<float, 3, 3>  _S338 = transpose_0(_S336.differential_0);

#line 153
    Matrix<float, 3, 3>  _S339 = _S330.differential_0 + transpose_0(_S337.differential_0);

#line 153
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S340;

#line 153
    (&_S340)->primal_1 = (*dpxyz_ws_3).primal_1;

#line 153
    (&_S340)->differential_0 = _S325;

#line 153
    Camera_Differential_0 _S341 = Camera_x24_syn_dzero_0();

#line 153
    DiffPair_Camera_0 _S342;

#line 153
    (&_S342)->primal_1 = (*dpcam_5).primal_1;

#line 153
    (&_S342)->differential_0 = _S341;

#line 153
    s_bwd_prop_compute_jacobian_0(&_S340, &_S342, _S339);

#line 153
    Matrix<float, 3, 3>  _S343 = _S332.differential_0 + _S338;

#line 153
    float4  _S344 = make_float4 (_S343.rows[int(2)].x, _S343.rows[int(2)].y, _S343.rows[int(2)].z, 0.0f);

#line 153
    float4  _S345 = make_float4 (_S343.rows[int(1)].x, _S343.rows[int(1)].y, _S343.rows[int(1)].z, 0.0f);

#line 153
    float4  _S346 = make_float4 (_S343.rows[int(0)].x, _S343.rows[int(0)].y, _S343.rows[int(0)].z, 0.0f);

#line 153
    Matrix<float, 4, 4>  _S347 = makeMatrix<float, 4, 4> (0.0f);

#line 153
    _S347[int(2)] = _S344;

#line 153
    _S347[int(1)] = _S345;

#line 153
    _S347[int(0)] = _S346;

#line 153
    dpcov_ws_1->primal_1 = (*dpcov_ws_1).primal_1;

#line 153
    dpcov_ws_1->differential_0 = _S334.differential_0;

#line 153
    dpxyz_ws_3->primal_1 = (*dpxyz_ws_3).primal_1;

#line 153
    dpxyz_ws_3->differential_0 = _S340.differential_0;

#line 153
    Camera_Differential_0 _S348 = _S341;

#line 153
    (&_S348)->world_view_transform_0 = _S347;

#line 153
    Camera_Differential_0 _S349 = Camera_x24_syn_dadd_0(_S342.differential_0, _S348);

#line 153
    dpcam_5->primal_1 = (*dpcam_5).primal_1;

#line 153
    dpcam_5->differential_0 = _S349;

#line 151
    return;
}


#line 280
__device__ void s_bwd_prop_get_covariance_from_quat_scales_0(DiffPair_vectorx3Cfloatx2C4x3E_0 * dpq_1, DiffPair_vectorx3Cfloatx2C3x3E_0 * dps_1, Matrix<float, 3, 3>  _s_dOut_4)
{

#line 280
    float _S350 = (*dpq_1).primal_1.z;



    float _S351 = _S350 * _S350;

#line 284
    float _S352 = (*dpq_1).primal_1.w * (*dpq_1).primal_1.w;

#line 284
    float _S353 = (*dpq_1).primal_1.y * (*dpq_1).primal_1.z;

#line 284
    float _S354 = (*dpq_1).primal_1.x * (*dpq_1).primal_1.w;

#line 284
    float _S355 = (*dpq_1).primal_1.y * (*dpq_1).primal_1.w;

#line 284
    float _S356 = (*dpq_1).primal_1.x * (*dpq_1).primal_1.z;
    float _S357 = (*dpq_1).primal_1.y * (*dpq_1).primal_1.y;

#line 285
    float _S358 = (*dpq_1).primal_1.z * (*dpq_1).primal_1.w;

#line 285
    float _S359 = (*dpq_1).primal_1.x * (*dpq_1).primal_1.y;

#line 283
    Matrix<float, 3, 3>  rotation_matrix_0 = makeMatrix<float, 3, 3> (1.0f - 2.0f * (_S351 + _S352), 2.0f * (_S353 - _S354), 2.0f * (_S355 + _S356), 2.0f * (_S353 + _S354), 1.0f - 2.0f * (_S357 + _S352), 2.0f * (_S358 - _S359), 2.0f * (_S355 - _S356), 2.0f * (_S358 + _S359), 1.0f - 2.0f * (_S357 + _S351));

#line 288
    Matrix<float, 3, 3>  scales_matrix_0 = makeMatrix<float, 3, 3> ((*dps_1).primal_1.x, 0.0f, 0.0f, 0.0f, (*dps_1).primal_1.y, 0.0f, 0.0f, 0.0f, (*dps_1).primal_1.z);

#line 288
    Matrix<float, 3, 3>  _S360 = s_primal_ctx_mul_2(rotation_matrix_0, scales_matrix_0);

#line 294
    Matrix<float, 3, 3>  _S361 = transpose_0(_S360);

#line 294
    Matrix<float, 3, 3>  _S362 = makeMatrix<float, 3, 3> (0.0f);

#line 294
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S363;

#line 294
    (&_S363)->primal_1 = _S360;

#line 294
    (&_S363)->differential_0 = _S362;

#line 294
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S364;

#line 294
    (&_S364)->primal_1 = _S361;

#line 294
    (&_S364)->differential_0 = _S362;

#line 294
    s_bwd_prop_mul_0(&_S363, &_S364, _s_dOut_4);

#line 292
    Matrix<float, 3, 3>  _S365 = _S363.differential_0 + transpose_0(_S364.differential_0);

#line 292
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S366;

#line 292
    (&_S366)->primal_1 = rotation_matrix_0;

#line 292
    (&_S366)->differential_0 = _S362;

#line 292
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S367;

#line 292
    (&_S367)->primal_1 = scales_matrix_0;

#line 292
    (&_S367)->differential_0 = _S362;

#line 292
    s_bwd_prop_mul_0(&_S366, &_S367, _S365);

#line 286
    float _S368 = 2.0f * - _S366.differential_0.rows[int(2)].z;

#line 286
    float _S369 = 2.0f * _S366.differential_0.rows[int(2)].y;

#line 286
    float _S370 = 2.0f * _S366.differential_0.rows[int(2)].x;

#line 285
    float _S371 = 2.0f * _S366.differential_0.rows[int(1)].z;

#line 285
    float _S372 = _S369 + - _S371;

#line 285
    float _S373 = _S369 + _S371;

#line 285
    float _S374 = 2.0f * - _S366.differential_0.rows[int(1)].y;

#line 285
    float _S375 = (*dpq_1).primal_1.y * (_S368 + _S374);

#line 285
    float _S376 = 2.0f * _S366.differential_0.rows[int(1)].x;

#line 284
    float _S377 = 2.0f * _S366.differential_0.rows[int(0)].z;

#line 284
    float _S378 = - _S370 + _S377;

#line 284
    float _S379 = _S370 + _S377;

#line 284
    float _S380 = 2.0f * _S366.differential_0.rows[int(0)].y;

#line 284
    float _S381 = _S376 + - _S380;

#line 284
    float _S382 = _S376 + _S380;

#line 284
    float _S383 = 2.0f * - _S366.differential_0.rows[int(0)].x;

#line 284
    float _S384 = (*dpq_1).primal_1.w * (_S374 + _S383);

#line 284
    float _S385 = (*dpq_1).primal_1.z * (_S368 + _S383);

#line 958 "core.meta.slang"
    float _S386 = (*dpq_1).primal_1.z * _S373 + (*dpq_1).primal_1.y * _S379 + (*dpq_1).primal_1.x * _S381 + _S384 + _S384;

#line 958
    float _S387 = (*dpq_1).primal_1.w * _S373 + (*dpq_1).primal_1.x * _S378 + (*dpq_1).primal_1.y * _S382 + _S385 + _S385;

#line 958
    float _S388 = (*dpq_1).primal_1.x * _S372 + _S375 + _S375 + (*dpq_1).primal_1.w * _S379 + (*dpq_1).primal_1.z * _S382;

#line 958
    float _S389 = (*dpq_1).primal_1.y * _S372 + (*dpq_1).primal_1.z * _S378 + (*dpq_1).primal_1.w * _S381;

#line 958
    float3  _S390 = make_float3 (0.0f);

#line 958
    *&((&_S390)->z) = _S367.differential_0.rows[int(2)].z;

#line 958
    *&((&_S390)->y) = _S367.differential_0.rows[int(1)].y;

#line 958
    *&((&_S390)->x) = _S367.differential_0.rows[int(0)].x;

#line 958
    dps_1->primal_1 = (*dps_1).primal_1;

#line 958
    dps_1->differential_0 = _S390;

#line 958
    float4  _S391 = make_float4 (0.0f);

#line 958
    *&((&_S391)->w) = _S386;

#line 958
    *&((&_S391)->z) = _S387;

#line 958
    *&((&_S391)->y) = _S388;

#line 958
    *&((&_S391)->x) = _S389;

#line 958
    dpq_1->primal_1 = (*dpq_1).primal_1;

#line 958
    dpq_1->differential_0 = _S391;

#line 280 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
    return;
}


#line 280
struct DiffPair_SpherHarmCoeffs_0
{
    SpherHarmCoeffs_0 primal_1;
    SpherHarmCoeffs_0 differential_0;
};


#line 227
__device__ void s_bwd_prop_max_1(DiffPair_vectorx3Cfloatx2C3x3E_0 * _S392, DiffPair_vectorx3Cfloatx2C3x3E_0 * _S393, float3  _S394)
{

#line 227
    _d_max_vector_0(_S392, _S393, _S394);

#line 227
    return;
}


#line 2117 "diff.meta.slang"
__device__ void s_bwd_prop_sqrt_0(DiffPair_float_0 * _S395, float _S396)
{

#line 2117
    _d_sqrt_0(_S395, _S396);

#line 2117
    return;
}


#line 2092
__device__ void s_bwd_prop_length_impl_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * dpx_10, float _s_dOut_5)
{

#line 2092
    float _S397 = (*dpx_10).primal_1.x;

#line 2092
    float _S398 = (*dpx_10).primal_1.y;

#line 2092
    float _S399 = (*dpx_10).primal_1.z;

#line 2099
    DiffPair_float_0 _S400;

#line 2099
    (&_S400)->primal_1 = _S397 * _S397 + _S398 * _S398 + _S399 * _S399;

#line 2099
    (&_S400)->differential_0 = 0.0f;

#line 2099
    s_bwd_prop_sqrt_0(&_S400, _s_dOut_5);

#line 2099
    float _S401 = (*dpx_10).primal_1.z * _S400.differential_0;

#line 958 "core.meta.slang"
    float _S402 = _S401 + _S401;

#line 958
    float _S403 = (*dpx_10).primal_1.y * _S400.differential_0;

#line 958
    float _S404 = _S403 + _S403;

#line 958
    float _S405 = (*dpx_10).primal_1.x * _S400.differential_0;

#line 958
    float _S406 = _S405 + _S405;

#line 958
    float3  _S407 = make_float3 (0.0f);

#line 958
    *&((&_S407)->z) = _S402;

#line 958
    *&((&_S407)->y) = _S404;

#line 958
    *&((&_S407)->x) = _S406;

#line 958
    dpx_10->primal_1 = (*dpx_10).primal_1;

#line 958
    dpx_10->differential_0 = _S407;

#line 2092 "diff.meta.slang"
    return;
}


#line 2092
__device__ void s_bwd_length_impl_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * _S408, float _S409)
{

#line 2092
    s_bwd_prop_length_impl_0(_S408, _S409);

#line 2092
    return;
}


#line 2154
__device__ void s_bwd_prop_normalize_impl_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * dpx_11, float3  _s_dOut_6)
{
    float _S410 = length_0((*dpx_11).primal_1);
    float3  _S411 = (*dpx_11).primal_1 * _s_dOut_6;

#line 2157
    float3  _S412 = make_float3 (1.0f / _S410) * _s_dOut_6;

#line 2157
    float _S413 = - ((_S411.x + _S411.y + _S411.z) / (_S410 * _S410));

#line 2156
    float3  _S414 = make_float3 (0.0f);

#line 2156
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S415;

#line 2156
    (&_S415)->primal_1 = (*dpx_11).primal_1;

#line 2156
    (&_S415)->differential_0 = _S414;

#line 2156
    s_bwd_length_impl_0(&_S415, _S413);

#line 2156
    float3  _S416 = _S412 + _S415.differential_0;

#line 2156
    dpx_11->primal_1 = (*dpx_11).primal_1;

#line 2156
    dpx_11->differential_0 = _S416;

#line 2154
    return;
}


#line 2154
__device__ void s_bwd_normalize_impl_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * _S417, float3  _S418)
{

#line 2154
    s_bwd_prop_normalize_impl_0(_S417, _S418);

#line 2154
    return;
}


#line 94 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/spherical_harmonics.slang"
__device__ void s_bwd_prop_compute_color_from_sh_coeffs_0(DiffPair_SpherHarmCoeffs_0 * dpsh_1, DiffPair_vectorx3Cfloatx2C3x3E_0 * dpg_xyz_ws_1, DiffPair_vectorx3Cfloatx2C3x3E_0 * dpcam_pos_1, uint active_sh_9, float3  _s_dOut_7)
{

#line 94
    DiffPair_SpherHarmCoeffs_0 _S419 = *dpsh_1;

#line 100
    float3  _S420 = make_float3 (0.0f);

#line 95
    float3  dir_1 = (*dpg_xyz_ws_1).primal_1 - (*dpcam_pos_1).primal_1;
    float3  _S421 = normalize_0(dir_1);

    float3  rgb_9 = make_float3 (0.282094806432724f) * (*dpsh_1).primal_1.coeff0_0;
    bool _S422 = active_sh_9 > 0U;

#line 99
    float3  rgb_10;

#line 99
    float3  _S423;

#line 99
    float3  _S424;

#line 99
    float3  _S425;

#line 99
    float3  _S426;

#line 99
    float3  _S427;

#line 99
    float3  _S428;

#line 99
    float3  _S429;

#line 99
    float3  _S430;

#line 99
    float3  _S431;

#line 99
    float3  _S432;

#line 99
    float3  _S433;

#line 99
    float3  _S434;

#line 99
    float3  _S435;

#line 99
    float3  _S436;

#line 99
    float3  _S437;

#line 99
    float3  _S438;

#line 99
    float3  _S439;

#line 99
    float3  _S440;

#line 99
    float3  _S441;

#line 99
    float3  _S442;

#line 99
    float3  _S443;

#line 99
    float3  _S444;

#line 99
    float3  _S445;

#line 99
    float3  _S446;

#line 99
    float3  _S447;

#line 99
    float3  _S448;

#line 99
    float3  _S449;

#line 99
    float3  _S450;

#line 99
    float3  _S451;

#line 99
    float3  _S452;

#line 99
    float _S453;

#line 99
    float _S454;

#line 99
    float _S455;

#line 99
    float _S456;

#line 99
    float _S457;

#line 99
    float _S458;

#line 99
    float _S459;

#line 99
    float _S460;

#line 99
    float _S461;

#line 99
    float _S462;

#line 99
    float _S463;

#line 99
    float _S464;

#line 99
    float _S465;

#line 99
    float _S466;

#line 99
    float _S467;

#line 99
    bool _S468;

#line 99
    bool _S469;

#line 99
    if(_S422)
    {

#line 100
        float _S470 = _S421.y;

#line 100
        float _S471 = 0.48860251903533936f * _S470;

#line 100
        float3  _S472 = make_float3 (_S471);

#line 100
        float _S473 = _S421.z;

#line 100
        float _S474 = 0.48860251903533936f * _S473;

#line 100
        float3  _S475 = make_float3 (_S474);

#line 100
        float _S476 = _S421.x;

#line 100
        float _S477 = 0.48860251903533936f * _S476;

#line 100
        float3  _S478 = make_float3 (_S477);

#line 100
        float3  rgb_11 = rgb_9 - make_float3 (_S471) * _S419.primal_1.coeff1_0 + make_float3 (_S474) * _S419.primal_1.coeff2_0 - make_float3 (_S477) * _S419.primal_1.coeff3_0;
        bool _S479 = active_sh_9 > 1U;

#line 101
        if(_S479)
        {
            float xx_2 = _S476 * _S476;

#line 103
            float yy_2 = _S470 * _S470;

#line 103
            float zz_2 = _S473 * _S473;
            float xy_2 = _S476 * _S470;

            float _S480 = 1.09254848957061768f * xy_2;

#line 106
            float3  _S481 = make_float3 (_S480);
            float _S482 = -1.09254848957061768f * (_S470 * _S473);

#line 107
            float3  _S483 = make_float3 (_S482);
            float _S484 = 2.0f * zz_2;

#line 108
            float _S485 = 0.31539157032966614f * (_S484 - xx_2 - yy_2);

#line 108
            float3  _S486 = make_float3 (_S485);
            float _S487 = -1.09254848957061768f * (_S476 * _S473);

#line 109
            float3  _S488 = make_float3 (_S487);
            float _S489 = xx_2 - yy_2;

#line 110
            float _S490 = 0.54627424478530884f * _S489;

#line 110
            float3  _S491 = make_float3 (_S490);

#line 109
            float3  rgb_12 = rgb_11 + make_float3 (_S480) * _S419.primal_1.coeff4_0 + make_float3 (_S482) * _S419.primal_1.coeff5_0 + make_float3 (_S485) * _S419.primal_1.coeff6_0 + make_float3 (_S487) * _S419.primal_1.coeff7_0 + make_float3 (_S490) * _S419.primal_1.coeff8_0;


            bool _S492 = active_sh_9 > 2U;

#line 112
            if(_S492)
            {

                float _S493 = -0.59004360437393188f * _S470;

#line 115
                float _S494 = 3.0f * xx_2;

#line 115
                float _S495 = _S494 - yy_2;

#line 115
                float _S496 = _S493 * _S495;

#line 115
                float3  _S497 = make_float3 (_S496);
                float _S498 = 2.89061141014099121f * xy_2;

#line 116
                float _S499 = _S498 * _S473;

#line 116
                float3  _S500 = make_float3 (_S499);
                float _S501 = -0.4570457935333252f * _S470;

#line 117
                float _S502 = 4.0f * zz_2 - xx_2 - yy_2;

#line 117
                float _S503 = _S501 * _S502;

#line 117
                float3  _S504 = make_float3 (_S503);
                float _S505 = 0.37317633628845215f * _S473;

#line 118
                float _S506 = 3.0f * yy_2;

#line 118
                float _S507 = _S484 - _S494 - _S506;

#line 118
                float _S508 = _S505 * _S507;

#line 118
                float3  _S509 = make_float3 (_S508);
                float _S510 = -0.4570457935333252f * _S476;

#line 119
                float _S511 = _S510 * _S502;

#line 119
                float3  _S512 = make_float3 (_S511);
                float _S513 = 1.44530570507049561f * _S473;

#line 120
                float _S514 = _S513 * _S489;

#line 120
                float3  _S515 = make_float3 (_S514);
                float _S516 = -0.59004360437393188f * _S476;

#line 121
                float _S517 = xx_2 - _S506;

#line 121
                float _S518 = _S516 * _S517;

#line 121
                float3  _S519 = make_float3 (_S518);

#line 121
                rgb_10 = rgb_12 + make_float3 (_S496) * _S419.primal_1.coeff9_0 + make_float3 (_S499) * _S419.primal_1.coeff10_0 + make_float3 (_S503) * _S419.primal_1.coeff11_0 + make_float3 (_S508) * _S419.primal_1.coeff12_0 + make_float3 (_S511) * _S419.primal_1.coeff13_0 + make_float3 (_S514) * _S419.primal_1.coeff14_0 + make_float3 (_S518) * _S419.primal_1.coeff15_0;

#line 121
                _S423 = _S519;

#line 121
                _S424 = _S419.primal_1.coeff15_0;

#line 121
                _S453 = _S516;

#line 121
                _S454 = _S517;

#line 121
                _S425 = _S515;

#line 121
                _S426 = _S419.primal_1.coeff14_0;

#line 121
                _S455 = _S513;

#line 121
                _S427 = _S512;

#line 121
                _S428 = _S419.primal_1.coeff13_0;

#line 121
                _S456 = _S510;

#line 121
                _S457 = _S502;

#line 121
                _S429 = _S509;

#line 121
                _S430 = _S419.primal_1.coeff12_0;

#line 121
                _S458 = _S505;

#line 121
                _S459 = _S507;

#line 121
                _S431 = _S504;

#line 121
                _S432 = _S419.primal_1.coeff11_0;

#line 121
                _S460 = _S501;

#line 121
                _S433 = _S500;

#line 121
                _S434 = _S419.primal_1.coeff10_0;

#line 121
                _S461 = _S498;

#line 121
                _S435 = _S497;

#line 121
                _S436 = _S419.primal_1.coeff9_0;

#line 121
                _S462 = _S493;

#line 121
                _S463 = _S495;

#line 121
            }
            else
            {

#line 121
                rgb_10 = rgb_12;

#line 121
                _S423 = _S420;

#line 121
                _S424 = _S420;

#line 121
                _S453 = 0.0f;

#line 121
                _S454 = 0.0f;

#line 121
                _S425 = _S420;

#line 121
                _S426 = _S420;

#line 121
                _S455 = 0.0f;

#line 121
                _S427 = _S420;

#line 121
                _S428 = _S420;

#line 121
                _S456 = 0.0f;

#line 121
                _S457 = 0.0f;

#line 121
                _S429 = _S420;

#line 121
                _S430 = _S420;

#line 121
                _S458 = 0.0f;

#line 121
                _S459 = 0.0f;

#line 121
                _S431 = _S420;

#line 121
                _S432 = _S420;

#line 121
                _S460 = 0.0f;

#line 121
                _S433 = _S420;

#line 121
                _S434 = _S420;

#line 121
                _S461 = 0.0f;

#line 121
                _S435 = _S420;

#line 121
                _S436 = _S420;

#line 121
                _S462 = 0.0f;

#line 121
                _S463 = 0.0f;

#line 121
            }

#line 119
            float _S520 = _S456;

#line 117
            float _S521 = _S457;
            float _S522 = _S458;

#line 118
            float _S523 = _S459;

#line 117
            float _S524 = _S460;

#line 116
            float _S525 = _S461;

#line 115
            float _S526 = _S462;

#line 115
            float _S527 = _S463;

#line 115
            _S468 = _S492;

#line 115
            _S456 = _S489;

#line 115
            _S457 = _S520;

#line 115
            _S458 = _S521;

#line 115
            _S459 = _S522;

#line 115
            _S460 = _S523;

#line 115
            _S461 = _S524;

#line 115
            _S462 = _S525;

#line 115
            _S463 = _S526;

#line 115
            _S464 = _S527;

#line 115
            _S437 = _S491;

#line 115
            _S438 = _S419.primal_1.coeff8_0;

#line 115
            _S439 = _S488;

#line 115
            _S440 = _S419.primal_1.coeff7_0;

#line 115
            _S441 = _S486;

#line 115
            _S442 = _S419.primal_1.coeff6_0;

#line 115
            _S443 = _S483;

#line 115
            _S444 = _S419.primal_1.coeff5_0;

#line 115
            _S445 = _S481;

#line 115
            _S446 = _S419.primal_1.coeff4_0;

#line 115
        }
        else
        {

#line 115
            rgb_10 = rgb_11;

#line 115
            _S468 = false;

#line 115
            _S423 = _S420;

#line 115
            _S424 = _S420;

#line 115
            _S453 = 0.0f;

#line 115
            _S454 = 0.0f;

#line 115
            _S425 = _S420;

#line 115
            _S426 = _S420;

#line 115
            _S455 = 0.0f;

#line 115
            _S456 = 0.0f;

#line 115
            _S427 = _S420;

#line 115
            _S428 = _S420;

#line 115
            _S457 = 0.0f;

#line 115
            _S458 = 0.0f;

#line 115
            _S429 = _S420;

#line 115
            _S430 = _S420;

#line 115
            _S459 = 0.0f;

#line 115
            _S460 = 0.0f;

#line 115
            _S431 = _S420;

#line 115
            _S432 = _S420;

#line 115
            _S461 = 0.0f;

#line 115
            _S433 = _S420;

#line 115
            _S434 = _S420;

#line 115
            _S462 = 0.0f;

#line 115
            _S435 = _S420;

#line 115
            _S436 = _S420;

#line 115
            _S463 = 0.0f;

#line 115
            _S464 = 0.0f;

#line 115
            _S437 = _S420;

#line 115
            _S438 = _S420;

#line 115
            _S439 = _S420;

#line 115
            _S440 = _S420;

#line 115
            _S441 = _S420;

#line 115
            _S442 = _S420;

#line 115
            _S443 = _S420;

#line 115
            _S444 = _S420;

#line 115
            _S445 = _S420;

#line 115
            _S446 = _S420;

#line 115
        }

#line 112
        bool _S528 = _S468;


        float _S529 = _S463;

#line 115
        float _S530 = _S464;

#line 115
        _S468 = _S479;

#line 115
        _S469 = _S528;

#line 115
        _S463 = _S473;

#line 115
        _S464 = _S529;

#line 115
        _S465 = _S530;

#line 115
        _S466 = _S476;

#line 115
        _S467 = _S470;

#line 115
        _S447 = _S478;

#line 115
        _S448 = _S419.primal_1.coeff3_0;

#line 115
        _S449 = _S475;

#line 115
        _S450 = _S419.primal_1.coeff2_0;

#line 115
        _S451 = _S472;

#line 115
        _S452 = _S419.primal_1.coeff1_0;

#line 115
    }
    else
    {

#line 115
        rgb_10 = rgb_9;

#line 115
        _S468 = false;

#line 115
        _S469 = false;

#line 115
        _S423 = _S420;

#line 115
        _S424 = _S420;

#line 115
        _S453 = 0.0f;

#line 115
        _S454 = 0.0f;

#line 115
        _S425 = _S420;

#line 115
        _S426 = _S420;

#line 115
        _S455 = 0.0f;

#line 115
        _S456 = 0.0f;

#line 115
        _S427 = _S420;

#line 115
        _S428 = _S420;

#line 115
        _S457 = 0.0f;

#line 115
        _S458 = 0.0f;

#line 115
        _S429 = _S420;

#line 115
        _S430 = _S420;

#line 115
        _S459 = 0.0f;

#line 115
        _S460 = 0.0f;

#line 115
        _S431 = _S420;

#line 115
        _S432 = _S420;

#line 115
        _S461 = 0.0f;

#line 115
        _S433 = _S420;

#line 115
        _S434 = _S420;

#line 115
        _S462 = 0.0f;

#line 115
        _S463 = 0.0f;

#line 115
        _S435 = _S420;

#line 115
        _S436 = _S420;

#line 115
        _S464 = 0.0f;

#line 115
        _S465 = 0.0f;

#line 115
        _S437 = _S420;

#line 115
        _S438 = _S420;

#line 115
        _S439 = _S420;

#line 115
        _S440 = _S420;

#line 115
        _S441 = _S420;

#line 115
        _S442 = _S420;

#line 115
        _S443 = _S420;

#line 115
        _S444 = _S420;

#line 115
        _S445 = _S420;

#line 115
        _S446 = _S420;

#line 115
        _S466 = 0.0f;

#line 115
        _S467 = 0.0f;

#line 115
        _S447 = _S420;

#line 115
        _S448 = _S420;

#line 115
        _S449 = _S420;

#line 115
        _S450 = _S420;

#line 115
        _S451 = _S420;

#line 115
        _S452 = _S420;

#line 115
    }

#line 126
    float3  rgb_13 = rgb_10 + make_float3 (0.5f);

    float3  _S531 = make_float3 (0.0f);

#line 128
    SpherHarmCoeffs_0 _S532 = SpherHarmCoeffs_x24_syn_dzero_0();

#line 128
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S533;

#line 128
    (&_S533)->primal_1 = rgb_13;

#line 128
    (&_S533)->differential_0 = _S420;

#line 128
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S534;

#line 128
    (&_S534)->primal_1 = _S531;

#line 128
    (&_S534)->differential_0 = _S420;

#line 128
    s_bwd_prop_max_1(&_S533, &_S534, _s_dOut_7);

#line 128
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S535 = _S533;

#line 128
    SpherHarmCoeffs_0 _S536;

#line 128
    if(_S422)
    {

#line 128
        if(_S468)
        {

#line 128
            if(_S469)
            {

#line 121
                float3  _S537 = _S423 * _S535.differential_0;

#line 121
                float3  _S538 = _S424 * _S535.differential_0;

#line 121
                float _S539 = _S538.x + _S538.y + _S538.z;

#line 121
                float _S540 = _S453 * _S539;

#line 120
                float3  _S541 = _S425 * _S535.differential_0;

#line 120
                float3  _S542 = _S426 * _S535.differential_0;

#line 120
                float _S543 = _S542.x + _S542.y + _S542.z;

#line 120
                float _S544 = _S455 * _S543;

#line 120
                float _S545 = 1.44530570507049561f * (_S456 * _S543);

#line 119
                float3  _S546 = _S427 * _S535.differential_0;

#line 119
                float3  _S547 = _S428 * _S535.differential_0;

#line 119
                float _S548 = _S547.x + _S547.y + _S547.z;

#line 118
                float3  _S549 = _S429 * _S535.differential_0;

#line 118
                float3  _S550 = _S430 * _S535.differential_0;

#line 118
                float _S551 = _S550.x + _S550.y + _S550.z;

#line 118
                float _S552 = _S459 * _S551;

#line 118
                float _S553 = - _S552;

#line 118
                float _S554 = 3.0f * (- _S540 + _S553);

#line 118
                float _S555 = 0.37317633628845215f * (_S460 * _S551);

#line 117
                float3  _S556 = _S431 * _S535.differential_0;

#line 117
                float3  _S557 = _S432 * _S535.differential_0;

#line 117
                float _S558 = _S557.x + _S557.y + _S557.z;

#line 117
                float _S559 = _S457 * _S548 + _S461 * _S558;

#line 117
                float _S560 = - _S559;

#line 117
                float _S561 = 4.0f * _S559;

#line 117
                float _S562 = -0.4570457935333252f * (_S458 * _S558);

#line 116
                float3  _S563 = _S433 * _S535.differential_0;

#line 116
                float3  _S564 = _S434 * _S535.differential_0;

#line 116
                float _S565 = _S564.x + _S564.y + _S564.z;

#line 116
                float _S566 = _S462 * _S565;

#line 116
                float _S567 = 2.89061141014099121f * (_S463 * _S565);

#line 115
                float3  _S568 = _S435 * _S535.differential_0;

#line 115
                float3  _S569 = _S436 * _S535.differential_0;

#line 115
                float _S570 = _S569.x + _S569.y + _S569.z;

#line 115
                float _S571 = _S464 * _S570;

#line 115
                float _S572 = - _S571;

#line 115
                float _S573 = 3.0f * (_S553 + _S571);

#line 115
                float _S574 = -0.59004360437393188f * (_S465 * _S570);

#line 100
                float _S575 = -0.59004360437393188f * (_S454 * _S539) + -0.4570457935333252f * (_S458 * _S548);

#line 100
                SpherHarmCoeffs_0 _S576 = _S532;

#line 100
                (&_S576)->coeff15_0 = _S537;

#line 100
                (&_S576)->coeff14_0 = _S541;

#line 100
                (&_S576)->coeff13_0 = _S546;

#line 100
                (&_S576)->coeff12_0 = _S549;

#line 100
                (&_S576)->coeff11_0 = _S556;

#line 100
                (&_S576)->coeff10_0 = _S563;

#line 100
                (&_S576)->coeff9_0 = _S568;

#line 100
                SpherHarmCoeffs_0 _S577 = SpherHarmCoeffs_x24_syn_dadd_0(_S532, _S576);


                float _S578 = _S554 + _S560 + _S572;

#line 103
                float _S579 = _S540 + _S560 + _S573;

#line 100
                float _S580 = _S545 + _S555 + _S566;

#line 100
                float _S581 = _S562 + _S574;

#line 100
                _S453 = _S544;

#line 100
                _S454 = _S552;

#line 100
                _S455 = _S567;

#line 100
                _S456 = _S561;

#line 100
                _S457 = _S578;

#line 100
                _S458 = _S579;

#line 100
                _S536 = _S577;

#line 100
                _S459 = _S575;

#line 100
                _S460 = _S581;

#line 100
                _S461 = _S580;

#line 100
            }
            else
            {

#line 100
                _S453 = 0.0f;

#line 100
                _S454 = 0.0f;

#line 100
                _S455 = 0.0f;

#line 100
                _S456 = 0.0f;

#line 100
                _S457 = 0.0f;

#line 100
                _S458 = 0.0f;

#line 100
                _S536 = _S532;

#line 100
                _S459 = 0.0f;

#line 100
                _S460 = 0.0f;

#line 100
                _S461 = 0.0f;

#line 100
            }

#line 110
            float3  _S582 = _S437 * _S535.differential_0;

#line 110
            float3  _S583 = _S438 * _S535.differential_0;

#line 110
            float _S584 = 0.54627424478530884f * (_S583.x + _S583.y + _S583.z) + _S453;

#line 109
            float3  _S585 = _S439 * _S535.differential_0;

#line 109
            float3  _S586 = _S440 * _S535.differential_0;

#line 109
            float s_diff_xz_T_0 = -1.09254848957061768f * (_S586.x + _S586.y + _S586.z);

#line 108
            float3  _S587 = _S441 * _S535.differential_0;

#line 108
            float3  _S588 = _S442 * _S535.differential_0;

#line 108
            float _S589 = 0.31539157032966614f * (_S588.x + _S588.y + _S588.z);

#line 108
            float _S590 = - _S589;

#line 107
            float3  _S591 = _S443 * _S535.differential_0;

#line 107
            float3  _S592 = _S444 * _S535.differential_0;

#line 107
            float s_diff_yz_T_0 = -1.09254848957061768f * (_S592.x + _S592.y + _S592.z);

#line 106
            float3  _S593 = _S445 * _S535.differential_0;

#line 106
            float3  _S594 = _S446 * _S535.differential_0;

#line 104
            float _S595 = _S466 * s_diff_xz_T_0;

#line 104
            float _S596 = _S463 * s_diff_xz_T_0;

#line 104
            float _S597 = _S467 * s_diff_yz_T_0;

#line 104
            float _S598 = _S463 * s_diff_yz_T_0;

#line 104
            float _S599 = 1.09254848957061768f * (_S594.x + _S594.y + _S594.z) + _S455;

#line 104
            float _S600 = _S466 * _S599;

#line 104
            float _S601 = _S467 * _S599;

#line 103
            float _S602 = 2.0f * (_S589 + _S454) + _S456;

#line 103
            float _S603 = _S463 * _S602;

#line 103
            float _S604 = _S463 * _S602;

#line 103
            float _S605 = - _S584 + _S590 + _S457;

#line 103
            float _S606 = _S467 * _S605;

#line 103
            float _S607 = _S467 * _S605;

#line 103
            float _S608 = _S584 + _S590 + _S458;

#line 103
            float _S609 = _S466 * _S608;

#line 103
            float _S610 = _S466 * _S608;

#line 103
            SpherHarmCoeffs_0 _S611 = _S532;

#line 103
            (&_S611)->coeff8_0 = _S582;

#line 103
            (&_S611)->coeff7_0 = _S585;

#line 103
            (&_S611)->coeff6_0 = _S587;

#line 103
            (&_S611)->coeff5_0 = _S591;

#line 103
            (&_S611)->coeff4_0 = _S593;

#line 103
            SpherHarmCoeffs_0 _S612 = SpherHarmCoeffs_x24_syn_dadd_0(_S536, _S611);

#line 100
            float _S613 = _S598 + _S600 + _S606 + _S607 + _S460;

#line 100
            float _S614 = _S595 + _S597 + _S603 + _S604 + _S461;

#line 100
            _S453 = _S596 + _S601 + _S609 + _S610 + _S459;

#line 100
            _S454 = _S614;

#line 100
            _S455 = _S613;

#line 100
            _S536 = _S612;

#line 100
        }
        else
        {

#line 100
            _S453 = 0.0f;

#line 100
            _S454 = 0.0f;

#line 100
            _S455 = 0.0f;

#line 100
            _S536 = _S532;

#line 100
        }

#line 100
        float3  _S615 = - _S535.differential_0;

#line 100
        float3  _S616 = _S447 * _S615;

#line 100
        float3  _S617 = _S448 * _S615;

#line 100
        float3  _S618 = _S449 * _S535.differential_0;

#line 100
        float3  _S619 = _S450 * _S535.differential_0;

#line 100
        float3  _S620 = _S451 * _S615;

#line 100
        float3  _S621 = _S452 * _S615;

#line 96
        float3  _S622 = make_float3 (0.48860251903533936f * (_S617.x + _S617.y + _S617.z) + _S453, 0.48860251903533936f * (_S621.x + _S621.y + _S621.z) + _S455, 0.48860251903533936f * (_S619.x + _S619.y + _S619.z) + _S454);

#line 96
        SpherHarmCoeffs_0 _S623 = _S532;

#line 96
        (&_S623)->coeff3_0 = _S616;

#line 96
        (&_S623)->coeff2_0 = _S618;

#line 96
        (&_S623)->coeff1_0 = _S620;

#line 96
        SpherHarmCoeffs_0 _S624 = SpherHarmCoeffs_x24_syn_dadd_0(_S536, _S623);

#line 96
        rgb_10 = _S622;

#line 96
        _S536 = _S624;

#line 96
    }
    else
    {

#line 96
        rgb_10 = _S420;

#line 96
        _S536 = _S532;

#line 96
    }

    float3  _S625 = make_float3 (0.282094806432724f) * _S535.differential_0;

#line 96
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S626;

#line 96
    (&_S626)->primal_1 = dir_1;

#line 96
    (&_S626)->differential_0 = _S420;

#line 96
    s_bwd_normalize_impl_0(&_S626, rgb_10);

#line 95
    float3  _S627 = - _S626.differential_0;

#line 95
    dpcam_pos_1->primal_1 = (*dpcam_pos_1).primal_1;

#line 95
    dpcam_pos_1->differential_0 = _S627;

#line 95
    dpg_xyz_ws_1->primal_1 = (*dpg_xyz_ws_1).primal_1;

#line 95
    dpg_xyz_ws_1->differential_0 = _S626.differential_0;

#line 95
    SpherHarmCoeffs_0 _S628 = _S532;

#line 95
    (&_S628)->coeff0_0 = _S625;

#line 95
    SpherHarmCoeffs_0 _S629 = SpherHarmCoeffs_x24_syn_dadd_0(_S536, _S628);

#line 95
    dpsh_1->primal_1 = (*dpsh_1).primal_1;

#line 95
    dpsh_1->differential_0 = _S629;

#line 94
    return;
}


#line 112 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
__device__ void s_bwd_prop_geom_transform_points2_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * dppoint_4, DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 * dptransf_matrix_3, float3  _s_dOut_8)
{
    float4  _S630 = make_float4 ((*dppoint_4).primal_1.x, (*dppoint_4).primal_1.y, (*dppoint_4).primal_1.z, 1.0f);

#line 114
    float4  _S631 = make_float4 (_s_dOut_8.x, _s_dOut_8.y, _s_dOut_8.z, 0.0f);

#line 114
    Matrix<float, 4, 4>  _S632 = makeMatrix<float, 4, 4> (0.0f);

#line 114
    DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 _S633;

#line 114
    (&_S633)->primal_1 = (*dptransf_matrix_3).primal_1;

#line 114
    (&_S633)->differential_0 = _S632;

#line 114
    float4  _S634 = make_float4 (0.0f);

#line 114
    DiffPair_vectorx3Cfloatx2C4x3E_0 _S635;

#line 114
    (&_S635)->primal_1 = _S630;

#line 114
    (&_S635)->differential_0 = _S634;

#line 114
    s_bwd_prop_mul_1(&_S633, &_S635, _S631);

#line 114
    float3  _S636 = float3 {_S635.differential_0.x, _S635.differential_0.y, _S635.differential_0.z};

#line 114
    dptransf_matrix_3->primal_1 = (*dptransf_matrix_3).primal_1;

#line 114
    dptransf_matrix_3->differential_0 = _S633.differential_0;

#line 114
    dppoint_4->primal_1 = (*dppoint_4).primal_1;

#line 114
    dppoint_4->differential_0 = _S636;

#line 112
    return;
}


#line 112
__device__ void s_bwd_prop_mul_2(DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 * _S637, DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 * _S638, Matrix<float, 4, 4>  _S639)
{

#line 112
    mul_0(_S637, _S638, _S639);

#line 112
    return;
}


#line 119
__device__ void s_bwd_prop_project_point_0(DiffPair_vectorx3Cfloatx2C3x3E_0 * dppoint_5, DiffPair_Camera_0 * dpcam_6, float3  _s_dOut_9)
{

#line 119
    Matrix<float, 4, 4>  _S640 = s_primal_ctx_mul_0((*dpcam_6).primal_1.proj_mat_1, (*dpcam_6).primal_1.world_view_transform_1);

#line 119
    float3  _S641 = _s_dOut_9;

#line 119
    *&((&_S641)->z) = 0.0f;

    float3  _S642 = make_float3 (0.0f, 0.0f, _s_dOut_9.z);

#line 121
    float3  _S643 = make_float3 (0.0f);

#line 121
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S644;

#line 121
    (&_S644)->primal_1 = (*dppoint_5).primal_1;

#line 121
    (&_S644)->differential_0 = _S643;

#line 121
    Matrix<float, 4, 4>  _S645 = makeMatrix<float, 4, 4> (0.0f);

#line 121
    DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 _S646;

#line 121
    (&_S646)->primal_1 = (*dpcam_6).primal_1.world_view_transform_1;

#line 121
    (&_S646)->differential_0 = _S645;

#line 121
    s_bwd_prop_geom_transform_points2_0(&_S644, &_S646, _S642);

#line 120
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S647;

#line 120
    (&_S647)->primal_1 = (*dppoint_5).primal_1;

#line 120
    (&_S647)->differential_0 = _S643;

#line 120
    DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 _S648;

#line 120
    (&_S648)->primal_1 = _S640;

#line 120
    (&_S648)->differential_0 = _S645;

#line 120
    s_bwd_prop_geom_transform_points_0(&_S647, &_S648, _S641);

#line 120
    DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 _S649;

#line 120
    (&_S649)->primal_1 = (*dpcam_6).primal_1.proj_mat_1;

#line 120
    (&_S649)->differential_0 = _S645;

#line 120
    DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 _S650;

#line 120
    (&_S650)->primal_1 = (*dpcam_6).primal_1.world_view_transform_1;

#line 120
    (&_S650)->differential_0 = _S645;

#line 120
    s_bwd_prop_mul_2(&_S649, &_S650, _S648.differential_0);

#line 120
    Matrix<float, 4, 4>  _S651 = _S646.differential_0 + _S650.differential_0;

#line 120
    Camera_Differential_0 _S652 = Camera_x24_syn_dzero_0();

#line 120
    (&_S652)->world_view_transform_0 = _S651;

#line 120
    (&_S652)->proj_mat_0 = _S649.differential_0;

#line 120
    dpcam_6->primal_1 = (*dpcam_6).primal_1;

#line 120
    dpcam_6->differential_0 = _S652;

#line 120
    float3  _S653 = _S644.differential_0 + _S647.differential_0;

#line 120
    dppoint_5->primal_1 = (*dppoint_5).primal_1;

#line 120
    dppoint_5->differential_0 = _S653;

#line 119
    return;
}


#line 222
__device__ void s_bwd_prop_project_gaussian_to_camera_0(DiffPair_Gaussian_3D_0 * dpg_1, DiffPair_Camera_0 * dpcam_7, uint active_sh_10, Splat_2D_Vertex_0 _s_dOut_10)
{

#line 222
    DiffPair_Gaussian_3D_0 _S654 = *dpg_1;

#line 222
    DiffPair_Camera_0 _S655 = *dpcam_7;

#line 222
    float3  _S656 = make_float3 (0.0f);

#line 222
    float4  _S657 = make_float4 (0.0f);

#line 228
    Matrix<float, 3, 3>  _S658 = makeMatrix<float, 3, 3> (0.0f);

#line 228
    bool _S659 = !(s_primal_ctx_project_point_0((*dpg_1).primal_1.xyz_ws_0, (*dpcam_7).primal_1).z <= 0.20000000298023224f);

#line 228
    Matrix<float, 3, 3>  _S660;

#line 228
    float4  _S661;

#line 228
    float3  _S662;

#line 228
    float3  _S663;

#line 228
    SpherHarmCoeffs_0 _S664;

#line 228
    if(_S659)
    {

#line 228
        _S660 = s_primal_ctx_get_covariance_from_quat_scales_0(_S654.primal_1.rotations_0, _S654.primal_1.scales_0);

#line 228
        _S661 = _S654.primal_1.rotations_0;

#line 228
        _S662 = _S654.primal_1.scales_0;

#line 228
        _S664 = _S654.primal_1.sh_coeffs_0;

#line 228
        _S663 = _S655.primal_1.position_1;

#line 228
    }
    else
    {

#line 228
        _S660 = _S658;

#line 228
        _S661 = _S657;

#line 228
        _S662 = _S656;

#line 228
        (&_S664)->coeff0_0 = _S656;

#line 228
        (&_S664)->coeff1_0 = _S656;

#line 228
        (&_S664)->coeff2_0 = _S656;

#line 228
        (&_S664)->coeff3_0 = _S656;

#line 228
        (&_S664)->coeff4_0 = _S656;

#line 228
        (&_S664)->coeff5_0 = _S656;

#line 228
        (&_S664)->coeff6_0 = _S656;

#line 228
        (&_S664)->coeff7_0 = _S656;

#line 228
        (&_S664)->coeff8_0 = _S656;

#line 228
        (&_S664)->coeff9_0 = _S656;

#line 228
        (&_S664)->coeff10_0 = _S656;

#line 228
        (&_S664)->coeff11_0 = _S656;

#line 228
        (&_S664)->coeff12_0 = _S656;

#line 228
        (&_S664)->coeff13_0 = _S656;

#line 228
        (&_S664)->coeff14_0 = _S656;

#line 228
        (&_S664)->coeff15_0 = _S656;

#line 228
        _S663 = _S656;

#line 228
    }

#line 228
    Camera_Differential_0 _S665 = Camera_x24_syn_dzero_0();

#line 228
    Gaussian_3D_0 _S666 = Gaussian_3D_x24_syn_dzero_0();

#line 228
    Splat_2D_Vertex_0 _S667 = Splat_2D_Vertex_x24_syn_dadd_0(_s_dOut_10, Splat_2D_Vertex_x24_syn_dzero_0());

#line 228
    Camera_Differential_0 _S668;

#line 228
    Gaussian_3D_0 _S669;

#line 228
    if(_S659)
    {

#line 229
        DiffPair_Camera_0 _S670;

#line 229
        (&_S670)->primal_1 = _S655.primal_1;

#line 229
        (&_S670)->differential_0 = _S665;

#line 229
        DiffPair_vectorx3Cfloatx2C3x3E_0 _S671;

#line 229
        (&_S671)->primal_1 = _S654.primal_1.xyz_ws_0;

#line 229
        (&_S671)->differential_0 = _S656;

#line 229
        DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S672;

#line 229
        (&_S672)->primal_1 = _S660;

#line 229
        (&_S672)->differential_0 = _S658;

#line 229
        s_bwd_prop_covariance_3d_to_2d_0(&_S670, &_S671, &_S672, _S667.cov_vs_0);

#line 228
        DiffPair_vectorx3Cfloatx2C4x3E_0 _S673;

#line 228
        (&_S673)->primal_1 = _S661;

#line 228
        (&_S673)->differential_0 = _S657;

#line 228
        DiffPair_vectorx3Cfloatx2C3x3E_0 _S674;

#line 228
        (&_S674)->primal_1 = _S662;

#line 228
        (&_S674)->differential_0 = _S656;

#line 228
        s_bwd_prop_get_covariance_from_quat_scales_0(&_S673, &_S674, _S672.differential_0);

#line 227
        SpherHarmCoeffs_0 _S675 = SpherHarmCoeffs_x24_syn_dzero_0();

#line 227
        DiffPair_SpherHarmCoeffs_0 _S676;

#line 227
        (&_S676)->primal_1 = _S664;

#line 227
        (&_S676)->differential_0 = _S675;

#line 227
        DiffPair_vectorx3Cfloatx2C3x3E_0 _S677;

#line 227
        (&_S677)->primal_1 = _S654.primal_1.xyz_ws_0;

#line 227
        (&_S677)->differential_0 = _S656;

#line 227
        DiffPair_vectorx3Cfloatx2C3x3E_0 _S678;

#line 227
        (&_S678)->primal_1 = _S663;

#line 227
        (&_S678)->differential_0 = _S656;

#line 227
        s_bwd_prop_compute_color_from_sh_coeffs_0(&_S676, &_S677, &_S678, active_sh_10, _S667.rgb_0);

#line 227
        Gaussian_3D_0 _S679 = _S666;

#line 227
        (&_S679)->scales_0 = _S674.differential_0;

#line 227
        (&_S679)->rotations_0 = _S673.differential_0;

#line 227
        (&_S679)->sh_coeffs_0 = _S676.differential_0;

#line 227
        Gaussian_3D_0 _S680 = Gaussian_3D_x24_syn_dadd_0(_S666, _S679);

#line 227
        float3  _S681 = _S671.differential_0 + _S677.differential_0;

#line 227
        Camera_Differential_0 _S682 = Camera_x24_syn_dadd_0(_S670.differential_0, _S665);

#line 227
        Camera_Differential_0 _S683 = _S665;

#line 227
        (&_S683)->position_0 = _S678.differential_0;

#line 227
        Camera_Differential_0 _S684 = Camera_x24_syn_dadd_0(_S682, _S683);

#line 227
        _S662 = _S667.xyz_vs_0;

#line 227
        _S663 = _S681;

#line 227
        _S668 = _S684;

#line 227
        _S669 = _S680;

#line 227
    }
    else
    {

#line 227
        _S662 = _S656;

#line 227
        _S663 = _S656;

#line 227
        _S668 = _S665;

#line 227
        _S669 = _S666;

#line 227
    }

#line 223
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S685;

#line 223
    (&_S685)->primal_1 = _S654.primal_1.xyz_ws_0;

#line 223
    (&_S685)->differential_0 = _S656;

#line 223
    DiffPair_Camera_0 _S686;

#line 223
    (&_S686)->primal_1 = _S655.primal_1;

#line 223
    (&_S686)->differential_0 = _S665;

#line 223
    s_bwd_prop_project_point_0(&_S685, &_S686, _S662);

#line 223
    float3  _S687 = _S685.differential_0 + _S663;

#line 223
    Camera_Differential_0 _S688 = Camera_x24_syn_dadd_0(_S686.differential_0, _S668);

#line 223
    dpcam_7->primal_1 = (*dpcam_7).primal_1;

#line 223
    dpcam_7->differential_0 = _S688;

#line 223
    Gaussian_3D_0 _S689 = _S666;

#line 223
    (&_S689)->xyz_ws_0 = _S687;

#line 223
    Gaussian_3D_0 _S690 = Gaussian_3D_x24_syn_dadd_0(_S669, _S689);

#line 223
    dpg_1->primal_1 = (*dpg_1).primal_1;

#line 223
    dpg_1->differential_0 = _S690;

#line 222
    return;
}


#line 26
__device__ void s_bwd_prop_read_t3_float3_0(uint idx_4, DiffTensorView_0 t3_2, float3  _s_dOut_11)
{
    uint2  _S691 = make_uint2 (idx_4, 0U);
    uint2  _S692 = make_uint2 (idx_4, 1U);

#line 28
    AtomicAdd_load_backward_0(t3_2.diff_1, make_uint2 (idx_4, 2U), _s_dOut_11.z);

#line 28
    AtomicAdd_load_backward_0(t3_2.diff_1, _S692, _s_dOut_11.y);

#line 28
    AtomicAdd_load_backward_0(t3_2.diff_1, _S691, _s_dOut_11.x);

#line 26
    return;
}


#line 34
__device__ void s_bwd_prop_read_t4_float4_0(uint idx_5, DiffTensorView_0 t4_2, float4  _s_dOut_12)
{
    uint2  _S693 = make_uint2 (idx_5, 0U);
    uint2  _S694 = make_uint2 (idx_5, 1U);
    uint2  _S695 = make_uint2 (idx_5, 2U);

#line 36
    AtomicAdd_load_backward_0(t4_2.diff_1, make_uint2 (idx_5, 3U), _s_dOut_12.w);

#line 36
    AtomicAdd_load_backward_0(t4_2.diff_1, _S695, _s_dOut_12.z);

#line 36
    AtomicAdd_load_backward_0(t4_2.diff_1, _S694, _s_dOut_12.y);

#line 36
    AtomicAdd_load_backward_0(t4_2.diff_1, _S693, _s_dOut_12.x);

#line 34
    return;
}


#line 62 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/spherical_harmonics.slang"
__device__ void s_bwd_prop_read_spherical_harmonics_coeffs_0(uint g_idx_5, DiffTensorView_0 sh_coeffs_6, uint active_sh_11, SpherHarmCoeffs_0 _s_dOut_13)
{

#line 68
    uint3  _S696 = make_uint3 (0U);

#line 65
    uint3  _S697 = make_uint3 (g_idx_5, 0U, 0U);

#line 65
    uint3  _S698 = make_uint3 (g_idx_5, 0U, 1U);

#line 65
    uint3  _S699 = make_uint3 (g_idx_5, 0U, 2U);

    bool _S700 = active_sh_11 > 0U;

#line 67
    uint3  _S701;

#line 67
    uint3  _S702;

#line 67
    uint3  _S703;

#line 67
    uint3  _S704;

#line 67
    uint3  _S705;

#line 67
    uint3  _S706;

#line 67
    uint3  _S707;

#line 67
    uint3  _S708;

#line 67
    uint3  _S709;

#line 67
    uint3  _S710;

#line 67
    uint3  _S711;

#line 67
    uint3  _S712;

#line 67
    uint3  _S713;

#line 67
    uint3  _S714;

#line 67
    uint3  _S715;

#line 67
    uint3  _S716;

#line 67
    uint3  _S717;

#line 67
    uint3  _S718;

#line 67
    uint3  _S719;

#line 67
    uint3  _S720;

#line 67
    uint3  _S721;

#line 67
    uint3  _S722;

#line 67
    uint3  _S723;

#line 67
    uint3  _S724;

#line 67
    uint3  _S725;

#line 67
    uint3  _S726;

#line 67
    uint3  _S727;

#line 67
    uint3  _S728;

#line 67
    uint3  _S729;

#line 67
    uint3  _S730;

#line 67
    uint3  _S731;

#line 67
    uint3  _S732;

#line 67
    uint3  _S733;

#line 67
    uint3  _S734;

#line 67
    uint3  _S735;

#line 67
    uint3  _S736;

#line 67
    uint3  _S737;

#line 67
    uint3  _S738;

#line 67
    uint3  _S739;

#line 67
    uint3  _S740;

#line 67
    uint3  _S741;

#line 67
    uint3  _S742;

#line 67
    uint3  _S743;

#line 67
    uint3  _S744;

#line 67
    uint3  _S745;

#line 67
    bool _S746;

#line 67
    bool _S747;

#line 67
    if(_S700)
    {

#line 68
        uint3  _S748 = make_uint3 (g_idx_5, 1U, 0U);

#line 68
        uint3  _S749 = make_uint3 (g_idx_5, 1U, 1U);

#line 68
        uint3  _S750 = make_uint3 (g_idx_5, 1U, 2U);
        uint3  _S751 = make_uint3 (g_idx_5, 2U, 0U);

#line 69
        uint3  _S752 = make_uint3 (g_idx_5, 2U, 1U);

#line 69
        uint3  _S753 = make_uint3 (g_idx_5, 2U, 2U);
        uint3  _S754 = make_uint3 (g_idx_5, 3U, 0U);

#line 70
        uint3  _S755 = make_uint3 (g_idx_5, 3U, 1U);

#line 70
        uint3  _S756 = make_uint3 (g_idx_5, 3U, 2U);

        bool _S757 = active_sh_11 > 1U;

#line 72
        if(_S757)
        {

#line 73
            uint3  _S758 = make_uint3 (g_idx_5, 4U, 0U);

#line 73
            uint3  _S759 = make_uint3 (g_idx_5, 4U, 1U);

#line 73
            uint3  _S760 = make_uint3 (g_idx_5, 4U, 2U);
            uint3  _S761 = make_uint3 (g_idx_5, 5U, 0U);

#line 74
            uint3  _S762 = make_uint3 (g_idx_5, 5U, 1U);

#line 74
            uint3  _S763 = make_uint3 (g_idx_5, 5U, 2U);
            uint3  _S764 = make_uint3 (g_idx_5, 6U, 0U);

#line 75
            uint3  _S765 = make_uint3 (g_idx_5, 6U, 1U);

#line 75
            uint3  _S766 = make_uint3 (g_idx_5, 6U, 2U);
            uint3  _S767 = make_uint3 (g_idx_5, 7U, 0U);

#line 76
            uint3  _S768 = make_uint3 (g_idx_5, 7U, 1U);

#line 76
            uint3  _S769 = make_uint3 (g_idx_5, 7U, 2U);
            uint3  _S770 = make_uint3 (g_idx_5, 8U, 0U);

#line 77
            uint3  _S771 = make_uint3 (g_idx_5, 8U, 1U);

#line 77
            uint3  _S772 = make_uint3 (g_idx_5, 8U, 2U);

            bool _S773 = active_sh_11 > 2U;

#line 79
            if(_S773)
            {

#line 80
                uint3  _S774 = make_uint3 (g_idx_5, 9U, 0U);

#line 80
                uint3  _S775 = make_uint3 (g_idx_5, 9U, 1U);

#line 80
                uint3  _S776 = make_uint3 (g_idx_5, 9U, 2U);
                uint3  _S777 = make_uint3 (g_idx_5, 10U, 0U);

#line 81
                uint3  _S778 = make_uint3 (g_idx_5, 10U, 1U);

#line 81
                uint3  _S779 = make_uint3 (g_idx_5, 10U, 2U);
                uint3  _S780 = make_uint3 (g_idx_5, 11U, 0U);

#line 82
                uint3  _S781 = make_uint3 (g_idx_5, 11U, 1U);

#line 82
                uint3  _S782 = make_uint3 (g_idx_5, 11U, 2U);
                uint3  _S783 = make_uint3 (g_idx_5, 12U, 0U);

#line 83
                uint3  _S784 = make_uint3 (g_idx_5, 12U, 1U);

#line 83
                uint3  _S785 = make_uint3 (g_idx_5, 12U, 2U);
                uint3  _S786 = make_uint3 (g_idx_5, 13U, 0U);

#line 84
                uint3  _S787 = make_uint3 (g_idx_5, 13U, 1U);

#line 84
                uint3  _S788 = make_uint3 (g_idx_5, 13U, 2U);
                uint3  _S789 = make_uint3 (g_idx_5, 14U, 0U);

#line 85
                uint3  _S790 = make_uint3 (g_idx_5, 14U, 1U);

#line 85
                uint3  _S791 = make_uint3 (g_idx_5, 14U, 2U);
                uint3  _S792 = make_uint3 (g_idx_5, 15U, 0U);

#line 86
                uint3  _S793 = make_uint3 (g_idx_5, 15U, 1U);

#line 86
                _S701 = make_uint3 (g_idx_5, 15U, 2U);

#line 86
                _S702 = _S793;

#line 86
                _S703 = _S792;

#line 86
                _S704 = _S791;

#line 86
                _S705 = _S790;

#line 86
                _S706 = _S789;

#line 86
                _S707 = _S788;

#line 86
                _S708 = _S787;

#line 86
                _S709 = _S786;

#line 86
                _S710 = _S785;

#line 86
                _S711 = _S784;

#line 86
                _S712 = _S783;

#line 86
                _S713 = _S782;

#line 86
                _S714 = _S781;

#line 86
                _S715 = _S780;

#line 86
                _S716 = _S779;

#line 86
                _S717 = _S778;

#line 86
                _S718 = _S777;

#line 86
                _S719 = _S776;

#line 86
                _S720 = _S775;

#line 86
                _S721 = _S774;

#line 86
            }
            else
            {

#line 86
                _S701 = _S696;

#line 86
                _S702 = _S696;

#line 86
                _S703 = _S696;

#line 86
                _S704 = _S696;

#line 86
                _S705 = _S696;

#line 86
                _S706 = _S696;

#line 86
                _S707 = _S696;

#line 86
                _S708 = _S696;

#line 86
                _S709 = _S696;

#line 86
                _S710 = _S696;

#line 86
                _S711 = _S696;

#line 86
                _S712 = _S696;

#line 86
                _S713 = _S696;

#line 86
                _S714 = _S696;

#line 86
                _S715 = _S696;

#line 86
                _S716 = _S696;

#line 86
                _S717 = _S696;

#line 86
                _S718 = _S696;

#line 86
                _S719 = _S696;

#line 86
                _S720 = _S696;

#line 86
                _S721 = _S696;

#line 86
            }

#line 86
            _S746 = _S773;

#line 86
            _S722 = _S772;

#line 86
            _S723 = _S771;

#line 86
            _S724 = _S770;

#line 86
            _S725 = _S769;

#line 86
            _S726 = _S768;

#line 86
            _S727 = _S767;

#line 86
            _S728 = _S766;

#line 86
            _S729 = _S765;

#line 86
            _S730 = _S764;

#line 86
            _S731 = _S763;

#line 86
            _S732 = _S762;

#line 86
            _S733 = _S761;

#line 86
            _S734 = _S760;

#line 86
            _S735 = _S759;

#line 86
            _S736 = _S758;

#line 86
        }
        else
        {

#line 86
            _S746 = false;

#line 86
            _S701 = _S696;

#line 86
            _S702 = _S696;

#line 86
            _S703 = _S696;

#line 86
            _S704 = _S696;

#line 86
            _S705 = _S696;

#line 86
            _S706 = _S696;

#line 86
            _S707 = _S696;

#line 86
            _S708 = _S696;

#line 86
            _S709 = _S696;

#line 86
            _S710 = _S696;

#line 86
            _S711 = _S696;

#line 86
            _S712 = _S696;

#line 86
            _S713 = _S696;

#line 86
            _S714 = _S696;

#line 86
            _S715 = _S696;

#line 86
            _S716 = _S696;

#line 86
            _S717 = _S696;

#line 86
            _S718 = _S696;

#line 86
            _S719 = _S696;

#line 86
            _S720 = _S696;

#line 86
            _S721 = _S696;

#line 86
            _S722 = _S696;

#line 86
            _S723 = _S696;

#line 86
            _S724 = _S696;

#line 86
            _S725 = _S696;

#line 86
            _S726 = _S696;

#line 86
            _S727 = _S696;

#line 86
            _S728 = _S696;

#line 86
            _S729 = _S696;

#line 86
            _S730 = _S696;

#line 86
            _S731 = _S696;

#line 86
            _S732 = _S696;

#line 86
            _S733 = _S696;

#line 86
            _S734 = _S696;

#line 86
            _S735 = _S696;

#line 86
            _S736 = _S696;

#line 86
        }

#line 79
        bool _S794 = _S746;

#line 79
        _S746 = _S757;

#line 79
        _S747 = _S794;

#line 79
        _S737 = _S756;

#line 79
        _S738 = _S755;

#line 79
        _S739 = _S754;

#line 79
        _S740 = _S753;

#line 79
        _S741 = _S752;

#line 79
        _S742 = _S751;

#line 79
        _S743 = _S750;

#line 79
        _S744 = _S749;

#line 79
        _S745 = _S748;

#line 79
    }
    else
    {

#line 79
        _S746 = false;

#line 79
        _S747 = false;

#line 79
        _S701 = _S696;

#line 79
        _S702 = _S696;

#line 79
        _S703 = _S696;

#line 79
        _S704 = _S696;

#line 79
        _S705 = _S696;

#line 79
        _S706 = _S696;

#line 79
        _S707 = _S696;

#line 79
        _S708 = _S696;

#line 79
        _S709 = _S696;

#line 79
        _S710 = _S696;

#line 79
        _S711 = _S696;

#line 79
        _S712 = _S696;

#line 79
        _S713 = _S696;

#line 79
        _S714 = _S696;

#line 79
        _S715 = _S696;

#line 79
        _S716 = _S696;

#line 79
        _S717 = _S696;

#line 79
        _S718 = _S696;

#line 79
        _S719 = _S696;

#line 79
        _S720 = _S696;

#line 79
        _S721 = _S696;

#line 79
        _S722 = _S696;

#line 79
        _S723 = _S696;

#line 79
        _S724 = _S696;

#line 79
        _S725 = _S696;

#line 79
        _S726 = _S696;

#line 79
        _S727 = _S696;

#line 79
        _S728 = _S696;

#line 79
        _S729 = _S696;

#line 79
        _S730 = _S696;

#line 79
        _S731 = _S696;

#line 79
        _S732 = _S696;

#line 79
        _S733 = _S696;

#line 79
        _S734 = _S696;

#line 79
        _S735 = _S696;

#line 79
        _S736 = _S696;

#line 79
        _S737 = _S696;

#line 79
        _S738 = _S696;

#line 79
        _S739 = _S696;

#line 79
        _S740 = _S696;

#line 79
        _S741 = _S696;

#line 79
        _S742 = _S696;

#line 79
        _S743 = _S696;

#line 79
        _S744 = _S696;

#line 79
        _S745 = _S696;

#line 79
    }

#line 79
    SpherHarmCoeffs_0 _S795 = SpherHarmCoeffs_x24_syn_dzero_0();

#line 77
    float3  _S796 = make_float3 (0.0f);

#line 77
    SpherHarmCoeffs_0 _S797;

#line 77
    float3  _S798;

#line 77
    if(_S700)
    {

#line 77
        float3  _S799;

#line 77
        float3  _S800;

#line 77
        float3  _S801;

#line 77
        if(_S746)
        {

#line 77
            float3  _S802;

#line 77
            float3  _S803;

#line 77
            float3  _S804;

#line 77
            float3  _S805;

#line 77
            float3  _S806;

#line 77
            if(_S747)
            {

#line 86
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S701, _s_dOut_13.coeff15_0.z);

#line 86
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S702, _s_dOut_13.coeff15_0.y);

#line 86
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S703, _s_dOut_13.coeff15_0.x);

#line 85
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S704, _s_dOut_13.coeff14_0.z);

#line 85
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S705, _s_dOut_13.coeff14_0.y);

#line 85
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S706, _s_dOut_13.coeff14_0.x);

#line 84
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S707, _s_dOut_13.coeff13_0.z);

#line 84
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S708, _s_dOut_13.coeff13_0.y);

#line 84
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S709, _s_dOut_13.coeff13_0.x);

#line 83
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S710, _s_dOut_13.coeff12_0.z);

#line 83
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S711, _s_dOut_13.coeff12_0.y);

#line 83
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S712, _s_dOut_13.coeff12_0.x);

#line 82
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S713, _s_dOut_13.coeff11_0.z);

#line 82
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S714, _s_dOut_13.coeff11_0.y);

#line 82
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S715, _s_dOut_13.coeff11_0.x);

#line 81
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S716, _s_dOut_13.coeff10_0.z);

#line 81
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S717, _s_dOut_13.coeff10_0.y);

#line 81
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S718, _s_dOut_13.coeff10_0.x);

#line 80
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S719, _s_dOut_13.coeff9_0.z);

#line 80
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S720, _s_dOut_13.coeff9_0.y);

#line 80
                AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S721, _s_dOut_13.coeff9_0.x);

#line 80
                _S797 = _S795;

#line 80
                _S798 = _s_dOut_13.coeff8_0;

#line 80
                _S799 = _s_dOut_13.coeff7_0;

#line 80
                _S800 = _s_dOut_13.coeff6_0;

#line 80
                _S801 = _s_dOut_13.coeff5_0;

#line 80
                _S802 = _s_dOut_13.coeff4_0;

#line 80
                _S803 = _s_dOut_13.coeff0_0;

#line 80
                _S804 = _s_dOut_13.coeff1_0;

#line 80
                _S805 = _s_dOut_13.coeff2_0;

#line 80
                _S806 = _s_dOut_13.coeff3_0;

#line 80
            }
            else
            {

#line 80
                _S797 = SpherHarmCoeffs_x24_syn_dadd_0(_s_dOut_13, _S795);

#line 80
                _S798 = _S796;

#line 80
                _S799 = _S796;

#line 80
                _S800 = _S796;

#line 80
                _S801 = _S796;

#line 80
                _S802 = _S796;

#line 80
                _S803 = _S796;

#line 80
                _S804 = _S796;

#line 80
                _S805 = _S796;

#line 80
                _S806 = _S796;

#line 80
            }

#line 77
            float3  _S807 = _S797.coeff8_0 + _S798;

#line 77
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S722, _S807.z);

#line 77
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S723, _S807.y);

#line 77
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S724, _S807.x);

#line 76
            float3  _S808 = _S797.coeff7_0 + _S799;

#line 76
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S725, _S808.z);

#line 76
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S726, _S808.y);

#line 76
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S727, _S808.x);

#line 75
            float3  _S809 = _S797.coeff6_0 + _S800;

#line 75
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S728, _S809.z);

#line 75
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S729, _S809.y);

#line 75
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S730, _S809.x);

#line 74
            float3  _S810 = _S797.coeff5_0 + _S801;

#line 74
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S731, _S810.z);

#line 74
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S732, _S810.y);

#line 74
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S733, _S810.x);

#line 73
            float3  _S811 = _S797.coeff4_0 + _S802;

#line 73
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S734, _S811.z);

#line 73
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S735, _S811.y);

#line 73
            AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S736, _S811.x);

#line 65
            float3  _S812 = _S797.coeff0_0 + _S803;


            float3  _S813 = _S797.coeff1_0 + _S804;
            float3  _S814 = _S797.coeff2_0 + _S805;
            float3  _S815 = _S797.coeff3_0 + _S806;

#line 70
            _S797 = _S795;

#line 70
            _S798 = _S815;

#line 70
            _S799 = _S814;

#line 70
            _S800 = _S813;

#line 70
            _S801 = _S812;

#line 70
        }
        else
        {

#line 70
            _S797 = SpherHarmCoeffs_x24_syn_dadd_0(_s_dOut_13, _S795);

#line 70
            _S798 = _S796;

#line 70
            _S799 = _S796;

#line 70
            _S800 = _S796;

#line 70
            _S801 = _S796;

#line 70
        }

#line 70
        float3  _S816 = _S797.coeff3_0 + _S798;

#line 70
        AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S737, _S816.z);

#line 70
        AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S738, _S816.y);

#line 70
        AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S739, _S816.x);

#line 69
        float3  _S817 = _S797.coeff2_0 + _S799;

#line 69
        AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S740, _S817.z);

#line 69
        AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S741, _S817.y);

#line 69
        AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S742, _S817.x);

#line 68
        float3  _S818 = _S797.coeff1_0 + _S800;

#line 68
        AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S743, _S818.z);

#line 68
        AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S744, _S818.y);

#line 68
        AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S745, _S818.x);

#line 65
        float3  _S819 = _S797.coeff0_0 + _S801;

#line 65
        _S797 = _S795;

#line 65
        _S798 = _S819;

#line 65
    }
    else
    {

#line 65
        _S797 = SpherHarmCoeffs_x24_syn_dadd_0(_s_dOut_13, _S795);

#line 65
        _S798 = _S796;

#line 65
    }

#line 65
    float3  _S820 = _S797.coeff0_0 + _S798;

#line 65
    AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S699, _S820.z);

#line 65
    AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S698, _S820.y);

#line 65
    AtomicAdd_load_backward_1(sh_coeffs_6.diff_1, _S697, _S820.x);

#line 62
    return;
}


#line 170 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
__device__ void s_bwd_prop_load_gaussian_0(int g_idx_6, DiffTensorView_0 xyz_ws_6, DiffTensorView_0 sh_coeffs_7, DiffTensorView_0 rotations_4, DiffTensorView_0 scales_4, uint active_sh_12, Gaussian_3D_0 _s_dOut_14)
{

#line 177
    uint _S821 = uint(g_idx_6);

#line 177
    s_bwd_prop_read_t3_float3_0(_S821, scales_4, _s_dOut_14.scales_0);

#line 177
    s_bwd_prop_read_t4_float4_0(_S821, rotations_4, _s_dOut_14.rotations_0);

#line 177
    s_bwd_prop_read_spherical_harmonics_coeffs_0(_S821, sh_coeffs_7, active_sh_12, _s_dOut_14.sh_coeffs_0);

#line 177
    s_bwd_prop_read_t3_float3_0(_S821, xyz_ws_6, _s_dOut_14.xyz_ws_0);

#line 170
    return;
}


#line 37 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/vertex_shader.slang"
__device__ void s_bwd_prop_vertex_shader_0(DiffTensorView_0 xyz_ws_7, DiffTensorView_0 sh_coeffs_8, DiffTensorView_0 rotations_5, DiffTensorView_0 scales_5, uint active_sh_13, TensorView world_view_transform_4, TensorView proj_mat_4, TensorView cam_pos_2, TensorView out_tiles_touched_1, TensorView out_rect_tile_space_1, TensorView out_radii_1, DiffTensorView_0 out_xyz_vs_1, DiffTensorView_0 out_inv_cov_vs_1, DiffTensorView_0 out_rgb_1, float fovy_4, float fovx_4, uint image_height_1, uint image_width_1, uint grid_height_2, uint grid_width_2, uint tile_height_2, uint tile_width_2, s_bwd_prop_vertex_shader_Intermediates_0 _s_diff_ctx_1)
{

#line 37
    Matrix<float, 2, 2>  _S822 = makeMatrix<float, 2, 2> (0.0f);

#line 92
    uint2  _S823 = make_uint2 (0U);

#line 100
    uint3  _S824 = make_uint3 (0U);

#line 60
    uint g_idx_7 = ((blockIdx)).x * ((blockDim)).x + ((threadIdx)).x;

#line 60
    bool _S825 = !(g_idx_7 >= DiffTensorView_size_0(xyz_ws_7, 0U));

#line 60
    bool _bflag_1;

#line 60
    bool _bflag_2;

#line 60
    bool _bflag_3;

#line 60
    uint2  _S826;

#line 60
    uint2  _S827;

#line 60
    uint2  _S828;

#line 60
    uint3  _S829;

#line 60
    uint3  _S830;

#line 60
    uint3  _S831;

#line 60
    uint3  _S832;

#line 60
    Matrix<float, 2, 2>  _S833;

#line 60
    Matrix<float, 2, 2>  _S834;

#line 60
    Matrix<float, 2, 2>  _S835;

#line 60
    Matrix<float, 2, 2>  _S836;

#line 60
    float _S837;

#line 60
    float _S838;

#line 60
    int _S839;

#line 60
    int _S840;

#line 60
    int _S841;

#line 60
    if(_S825)
    {

#line 66
        int _S842 = int(g_idx_7);

#line 66
        Splat_2D_Vertex_0 _S843 = s_primal_ctx_project_gaussian_to_camera_0(_s_diff_ctx_1._S112, _s_diff_ctx_1._S111, active_sh_13);

        if(_S843.xyz_vs_0.z <= 0.20000000298023224f)
        {

#line 68
            _bflag_1 = false;

#line 68
        }
        else
        {

#line 68
            _bflag_1 = _S825;

#line 68
        }

#line 68
        if(_bflag_1)
        {

#line 68
            float _S844 = s_primal_ctx_compute_det_0(_S843.cov_vs_0);

#line 88
            Matrix<float, 2, 2>  _S845 = makeMatrix<float, 2, 2> (_S844);

#line 74
            if(_S844 == 0.0f)
            {

#line 74
                _bflag_2 = false;

#line 74
            }
            else
            {

#line 74
                _bflag_2 = _bflag_1;

#line 74
            }

#line 74
            if(_bflag_2)
            {



                float _S846 = _S843.xyz_vs_0.x;

#line 79
                int _S847 = int(image_width_1);

#line 79
                float _S848 = _S843.xyz_vs_0.y;

#line 79
                int _S849 = int(image_height_1);
                rectangle_0 rect_tile_space_2 = get_rectangle_tile_space_0(make_float2 (s_primal_ctx_ndc2pix_0(_S846, _S847), s_primal_ctx_ndc2pix_0(_S848, _S849)), splat_radius_0(_S843.cov_vs_0, _S844), grid_height_2, grid_width_2, tile_height_2, tile_width_2);



                if((rect_tile_space_2.max_x_0 - rect_tile_space_2.min_x_0) * (rect_tile_space_2.max_y_0 - rect_tile_space_2.min_y_0) == int(0))
                {

#line 84
                    _bflag_3 = false;

#line 84
                }
                else
                {

#line 84
                    _bflag_3 = _bflag_2;

#line 84
                }

#line 84
                if(_bflag_3)
                {


                    Matrix<float, 2, 2>  _S850 = makeMatrix<float, 2, 2> (_S843.cov_vs_0.rows[int(1)].y, - _S843.cov_vs_0.rows[int(0)].y, - _S843.cov_vs_0.rows[int(1)].x, _S843.cov_vs_0.rows[int(0)].x);

#line 88
                    Matrix<float, 2, 2>  _S851 = makeMatrix<float, 2, 2> (_S844 * _S844);



                    uint2  _S852 = make_uint2 (g_idx_7, 0U);
                    uint2  _S853 = make_uint2 (g_idx_7, 1U);

#line 100
                    uint3  _S854 = make_uint3 (g_idx_7, 0U, 0U);
                    uint3  _S855 = make_uint3 (g_idx_7, 0U, 1U);
                    uint3  _S856 = make_uint3 (g_idx_7, 1U, 0U);
                    uint3  _S857 = make_uint3 (g_idx_7, 1U, 1U);

#line 103
                    _S826 = make_uint2 (g_idx_7, 2U);

#line 103
                    _S827 = _S853;

#line 103
                    _S828 = _S852;

#line 103
                    _S829 = _S857;

#line 103
                    _S830 = _S856;

#line 103
                    _S831 = _S855;

#line 103
                    _S832 = _S854;

#line 103
                    _S833 = _S851;

#line 103
                    _S834 = _S850;

#line 103
                }
                else
                {

#line 103
                    _S826 = _S823;

#line 103
                    _S827 = _S823;

#line 103
                    _S828 = _S823;

#line 103
                    _S829 = _S824;

#line 103
                    _S830 = _S824;

#line 103
                    _S831 = _S824;

#line 103
                    _S832 = _S824;

#line 103
                    _S833 = _S822;

#line 103
                    _S834 = _S822;

#line 103
                }

#line 103
                _S837 = _S848;

#line 103
                _S838 = _S846;

#line 103
                _S839 = _S849;

#line 103
                _S840 = _S847;

#line 103
            }
            else
            {

#line 103
                _bflag_3 = false;

#line 103
                _S826 = _S823;

#line 103
                _S827 = _S823;

#line 103
                _S828 = _S823;

#line 103
                _S829 = _S824;

#line 103
                _S830 = _S824;

#line 103
                _S831 = _S824;

#line 103
                _S832 = _S824;

#line 103
                _S837 = 0.0f;

#line 103
                _S838 = 0.0f;

#line 103
                _S833 = _S822;

#line 103
                _S834 = _S822;

#line 103
                _S839 = int(0);

#line 103
                _S840 = int(0);

#line 103
            }

#line 103
            _S835 = _S845;

#line 103
            _S836 = _S843.cov_vs_0;

#line 103
        }
        else
        {

#line 103
            _bflag_2 = false;

#line 103
            _bflag_3 = false;

#line 103
            _S826 = _S823;

#line 103
            _S827 = _S823;

#line 103
            _S828 = _S823;

#line 103
            _S829 = _S824;

#line 103
            _S830 = _S824;

#line 103
            _S831 = _S824;

#line 103
            _S832 = _S824;

#line 103
            _S837 = 0.0f;

#line 103
            _S838 = 0.0f;

#line 103
            _S833 = _S822;

#line 103
            _S834 = _S822;

#line 103
            _S835 = _S822;

#line 103
            _S839 = int(0);

#line 103
            _S840 = int(0);

#line 103
            _S836 = _S822;

#line 103
        }

#line 103
        _S841 = _S842;

#line 103
    }
    else
    {

#line 103
        _bflag_1 = false;

#line 103
        _bflag_2 = false;

#line 103
        _bflag_3 = false;

#line 103
        _S826 = _S823;

#line 103
        _S827 = _S823;

#line 103
        _S828 = _S823;

#line 103
        _S829 = _S824;

#line 103
        _S830 = _S824;

#line 103
        _S831 = _S824;

#line 103
        _S832 = _S824;

#line 103
        _S837 = 0.0f;

#line 103
        _S838 = 0.0f;

#line 103
        _S833 = _S822;

#line 103
        _S834 = _S822;

#line 103
        _S835 = _S822;

#line 103
        _S839 = int(0);

#line 103
        _S840 = int(0);

#line 103
        _S836 = _S822;

#line 103
        _S841 = int(0);

#line 103
    }

#line 1751 "core.meta.slang"
    float3  _S858 = make_float3 (0.0f);

#line 67 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/vertex_shader.slang"
    Splat_2D_Vertex_0 _S859 = Splat_2D_Vertex_x24_syn_dzero_0();

#line 67
    if(_S825)
    {

#line 67
        Splat_2D_Vertex_0 _S860;

#line 67
        float3  _S861;

#line 67
        if(_bflag_1)
        {

#line 67
            if(_bflag_2)
            {

#line 67
                float _S862;

#line 67
                float _S863;

#line 67
                float _S864;

#line 67
                if(_bflag_3)
                {

#line 67
                    float3  _S865 = make_float3 (AtomicAdd_storeOnce_backward_0(out_rgb_1.diff_1, _S828), AtomicAdd_storeOnce_backward_0(out_rgb_1.diff_1, _S827), AtomicAdd_storeOnce_backward_0(out_rgb_1.diff_1, _S826));

#line 103
                    float _S866 = AtomicAdd_storeOnce_backward_1(out_inv_cov_vs_1.diff_1, _S829);

#line 102
                    float _S867 = AtomicAdd_storeOnce_backward_1(out_inv_cov_vs_1.diff_1, _S830);

#line 1751 "core.meta.slang"
                    float2  _S868 = make_float2 (0.0f);

#line 1751
                    float2  _S869 = _S868;

#line 1751
                    *&((&_S869)->y) = _S866;

#line 1751
                    *&((&_S869)->x) = _S867;

#line 101 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/vertex_shader.slang"
                    float _S870 = AtomicAdd_storeOnce_backward_1(out_inv_cov_vs_1.diff_1, _S831);

#line 100
                    float _S871 = AtomicAdd_storeOnce_backward_1(out_inv_cov_vs_1.diff_1, _S832);

#line 100
                    float2  _S872 = _S868;

#line 100
                    *&((&_S872)->y) = _S870;

#line 100
                    *&((&_S872)->x) = _S871;

#line 99
                    float _S873 = AtomicAdd_storeOnce_backward_0(out_xyz_vs_1.diff_1, _S826);

#line 98
                    float _S874 = AtomicAdd_storeOnce_backward_0(out_xyz_vs_1.diff_1, _S827);

#line 97
                    float _S875 = AtomicAdd_storeOnce_backward_0(out_xyz_vs_1.diff_1, _S828);

#line 88
                    Matrix<float, 2, 2>  _S876 = _S822;

#line 88
                    _S876[int(1)] = _S869;

#line 88
                    _S876[int(0)] = _S872;

#line 88
                    Matrix<float, 2, 2>  _S877 = _S876 / _S833;

#line 88
                    Matrix<float, 2, 2>  _S878 = _S834 * - _S877;

#line 88
                    Matrix<float, 2, 2>  _S879 = _S835 * _S877;

#line 88
                    float _S880 = - _S879.rows[int(1)].x;

#line 88
                    float _S881 = - _S879.rows[int(0)].y;

#line 88
                    float2  _S882 = _S868;

#line 88
                    *&((&_S882)->x) = _S879.rows[int(1)].y;

#line 88
                    *&((&_S882)->y) = _S881;

#line 88
                    float2  _S883 = _S868;

#line 88
                    *&((&_S883)->x) = _S880;

#line 88
                    *&((&_S883)->y) = _S879.rows[int(0)].x;

#line 67
                    Splat_2D_Vertex_0 _S884 = _S859;

#line 67
                    (&_S884)->rgb_0 = _S865;

#line 67
                    Splat_2D_Vertex_0 _S885 = Splat_2D_Vertex_x24_syn_dadd_0(_S859, _S884);

#line 67
                    Matrix<float, 2, 2>  _S886 = _S822;

#line 67
                    _S886[int(0)] = _S882;

#line 67
                    _S886[int(1)] = _S883;

#line 67
                    _S862 = _S874;

#line 67
                    _S863 = _S875;

#line 67
                    _S833 = _S878;

#line 67
                    _S834 = _S886;

#line 67
                    _S860 = _S885;

#line 67
                    _S864 = _S873;

#line 67
                }
                else
                {

#line 67
                    _S862 = 0.0f;

#line 67
                    _S863 = 0.0f;

#line 67
                    _S833 = _S822;

#line 67
                    _S834 = _S822;

#line 67
                    _S860 = _S859;

#line 67
                    _S864 = 0.0f;

#line 67
                }

#line 79
                DiffPair_float_0 _S887;

#line 79
                (&_S887)->primal_1 = _S837;

#line 79
                (&_S887)->differential_0 = 0.0f;

#line 79
                s_bwd_prop_ndc2pix_0(&_S887, _S839, 0.0f);

#line 79
                float _S888 = _S887.differential_0 + _S862;

#line 79
                DiffPair_float_0 _S889;

#line 79
                (&_S889)->primal_1 = _S838;

#line 79
                (&_S889)->differential_0 = 0.0f;

#line 79
                s_bwd_prop_ndc2pix_0(&_S889, _S840, 0.0f);

#line 79
                float3  _S890 = make_float3 (_S889.differential_0 + _S863, _S888, 0.0f);

#line 79
                _S837 = _S864;

#line 79
                _S861 = _S890;

#line 79
            }
            else
            {

#line 79
                _S833 = _S822;

#line 79
                _S834 = _S822;

#line 79
                _S860 = _S859;

#line 79
                _S837 = 0.0f;

#line 79
                _S861 = _S858;

#line 79
            }

#line 72
            float _S891 = _S833.rows[int(0)].x + _S833.rows[int(0)].y + _S833.rows[int(1)].x + _S833.rows[int(1)].y;

#line 72
            DiffPair_matrixx3Cfloatx2C2x2C2x3E_0 _S892;

#line 72
            (&_S892)->primal_1 = _S836;

#line 72
            (&_S892)->differential_0 = _S822;

#line 72
            s_bwd_prop_compute_det_0(&_S892, _S891);

#line 72
            Matrix<float, 2, 2>  _S893 = _S892.differential_0 + _S834;

#line 67
            Splat_2D_Vertex_0 _S894 = _S859;

#line 67
            (&_S894)->cov_vs_0 = _S893;

#line 67
            _S860 = Splat_2D_Vertex_x24_syn_dadd_0(_S860, _S894);

#line 67
        }
        else
        {

#line 67
            _S837 = 0.0f;

#line 67
            _S861 = _S858;

#line 67
            _S860 = _S859;

#line 67
        }

#line 67
        float3  _S895 = _S861 + make_float3 (0.0f, 0.0f, _S837);

#line 67
        Splat_2D_Vertex_0 _S896 = _S859;

#line 67
        (&_S896)->xyz_vs_0 = _S895;

#line 67
        Splat_2D_Vertex_0 _S897 = Splat_2D_Vertex_x24_syn_dadd_0(_S860, _S896);

#line 67
        Gaussian_3D_0 _S898 = Gaussian_3D_x24_syn_dzero_0();

#line 67
        DiffPair_Gaussian_3D_0 _S899;

#line 67
        (&_S899)->primal_1 = _s_diff_ctx_1._S112;

#line 67
        (&_S899)->differential_0 = _S898;

#line 67
        Camera_Differential_0 _S900 = Camera_x24_syn_dzero_0();

#line 67
        DiffPair_Camera_0 _S901;

#line 67
        (&_S901)->primal_1 = _s_diff_ctx_1._S111;

#line 67
        (&_S901)->differential_0 = _S900;

#line 67
        s_bwd_prop_project_gaussian_to_camera_0(&_S899, &_S901, active_sh_13, _S897);

#line 67
        s_bwd_prop_load_gaussian_0(_S841, xyz_ws_7, sh_coeffs_8, rotations_5, scales_5, active_sh_13, _S899.differential_0);

#line 67
    }

#line 37
    return;
}


#line 37
__device__ void s_bwd_vertex_shader_0(DiffTensorView_0 _S902, DiffTensorView_0 _S903, DiffTensorView_0 _S904, DiffTensorView_0 _S905, uint _S906, TensorView _S907, TensorView _S908, TensorView _S909, TensorView _S910, TensorView _S911, TensorView _S912, DiffTensorView_0 _S913, DiffTensorView_0 _S914, DiffTensorView_0 _S915, float _S916, float _S917, uint _S918, uint _S919, uint _S920, uint _S921, uint _S922, uint _S923)
{

#line 58
    s_bwd_prop_vertex_shader_Intermediates_0 _S924;

#line 58
    s_primal_ctx_vertex_shader_0(_S902, _S903, _S904, _S905, _S906, _S907, _S908, _S909, _S910, _S911, _S912, _S913, _S914, _S915, _S916, _S917, _S918, _S919, _S920, _S921, _S922, _S923, &_S924);

#line 58
    s_bwd_prop_vertex_shader_0(_S902, _S903, _S904, _S905, _S906, _S907, _S908, _S909, _S910, _S911, _S912, _S913, _S914, _S915, _S916, _S917, _S918, _S919, _S920, _S921, _S922, _S923, _S924);

#line 58
    return;
}


#line 58
extern "C" {
__global__ void __kernel__vertex_shader_bwd_diff(DiffTensorView_0 xyz_ws_8, DiffTensorView_0 sh_coeffs_9, DiffTensorView_0 rotations_6, DiffTensorView_0 scales_6, uint active_sh_14, TensorView world_view_transform_5, TensorView proj_mat_5, TensorView cam_pos_3, TensorView out_tiles_touched_2, TensorView out_rect_tile_space_2, TensorView out_radii_2, DiffTensorView_0 out_xyz_vs_2, DiffTensorView_0 out_inv_cov_vs_2, DiffTensorView_0 out_rgb_2, float fovy_5, float fovx_5, uint image_height_2, uint image_width_2, uint grid_height_3, uint grid_width_3, uint tile_height_3, uint tile_width_3)
{

#line 58
    s_bwd_vertex_shader_0(xyz_ws_8, sh_coeffs_9, rotations_6, scales_6, active_sh_14, world_view_transform_5, proj_mat_5, cam_pos_3, out_tiles_touched_2, out_rect_tile_space_2, out_radii_2, out_xyz_vs_2, out_inv_cov_vs_2, out_rgb_2, fovy_5, fovx_5, image_height_2, image_width_2, grid_height_3, grid_width_3, tile_height_3, tile_width_3);

#line 58
    return;
}

}

#line 177 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
__device__ DiffPair_vectorx3Cfloatx2C3x3E_0 s_fwd_read_t3_float3_0(uint idx_6, DiffTensorView_0 t3_3)
{

#line 28
    uint2  _S925 = make_uint2 (idx_6, 0U);

#line 28
    float _S926 = ((t3_3.primal_0).load<float>((_S925)));

#line 28
    float _S927 = AtomicAdd_load_forward_0(t3_3.diff_1, _S925);
    uint2  _S928 = make_uint2 (idx_6, 1U);

#line 28
    float _S929 = ((t3_3.primal_0).load<float>((_S928)));

#line 28
    float _S930 = AtomicAdd_load_forward_0(t3_3.diff_1, _S928);

    uint2  _S931 = make_uint2 (idx_6, 2U);

#line 28
    float _S932 = ((t3_3.primal_0).load<float>((_S931)));

#line 28
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S933 = { make_float3 (_S926, _S929, _S932), make_float3 (_S927, _S930, AtomicAdd_load_forward_0(t3_3.diff_1, _S931)) };

#line 28
    return _S933;
}


#line 178
__device__ DiffPair_SpherHarmCoeffs_0 s_fwd_read_spherical_harmonics_coeffs_0(uint g_idx_8, DiffTensorView_0 sh_coeffs_10, uint active_sh_15)
{

#line 64 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/spherical_harmonics.slang"
    float3  _S934 = make_float3 (0.0f);
    uint3  _S935 = make_uint3 (g_idx_8, 0U, 0U);

#line 65
    float _S936 = ((sh_coeffs_10.primal_0).load<float>((_S935)));

#line 65
    float _S937 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S935);

#line 65
    uint3  _S938 = make_uint3 (g_idx_8, 0U, 1U);

#line 65
    float _S939 = ((sh_coeffs_10.primal_0).load<float>((_S938)));

#line 65
    float _S940 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S938);

#line 65
    uint3  _S941 = make_uint3 (g_idx_8, 0U, 2U);

#line 65
    float _S942 = ((sh_coeffs_10.primal_0).load<float>((_S941)));

#line 65
    float3  _S943 = make_float3 (_S936, _S939, _S942);

#line 65
    float3  _S944 = make_float3 (_S937, _S940, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S941));

#line 65
    SpherHarmCoeffs_0 g_sh_coeffs_2;

#line 65
    SpherHarmCoeffs_0 s_diff_g_sh_coeffs_0;

    if(active_sh_15 > 0U)
    {

#line 68
        uint3  _S945 = make_uint3 (g_idx_8, 1U, 0U);

#line 68
        float _S946 = ((sh_coeffs_10.primal_0).load<float>((_S945)));

#line 68
        float _S947 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S945);

#line 68
        uint3  _S948 = make_uint3 (g_idx_8, 1U, 1U);

#line 68
        float _S949 = ((sh_coeffs_10.primal_0).load<float>((_S948)));

#line 68
        float _S950 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S948);

#line 68
        uint3  _S951 = make_uint3 (g_idx_8, 1U, 2U);

#line 68
        float _S952 = ((sh_coeffs_10.primal_0).load<float>((_S951)));

#line 68
        float3  _S953 = make_float3 (_S946, _S949, _S952);

#line 68
        float3  _S954 = make_float3 (_S947, _S950, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S951));
        uint3  _S955 = make_uint3 (g_idx_8, 2U, 0U);

#line 69
        float _S956 = ((sh_coeffs_10.primal_0).load<float>((_S955)));

#line 69
        float _S957 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S955);

#line 69
        uint3  _S958 = make_uint3 (g_idx_8, 2U, 1U);

#line 69
        float _S959 = ((sh_coeffs_10.primal_0).load<float>((_S958)));

#line 69
        float _S960 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S958);

#line 69
        uint3  _S961 = make_uint3 (g_idx_8, 2U, 2U);

#line 69
        float _S962 = ((sh_coeffs_10.primal_0).load<float>((_S961)));

#line 69
        float3  _S963 = make_float3 (_S956, _S959, _S962);

#line 69
        float3  _S964 = make_float3 (_S957, _S960, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S961));
        uint3  _S965 = make_uint3 (g_idx_8, 3U, 0U);

#line 70
        float _S966 = ((sh_coeffs_10.primal_0).load<float>((_S965)));

#line 70
        float _S967 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S965);

#line 70
        uint3  _S968 = make_uint3 (g_idx_8, 3U, 1U);

#line 70
        float _S969 = ((sh_coeffs_10.primal_0).load<float>((_S968)));

#line 70
        float _S970 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S968);

#line 70
        uint3  _S971 = make_uint3 (g_idx_8, 3U, 2U);

#line 70
        float _S972 = ((sh_coeffs_10.primal_0).load<float>((_S971)));

#line 70
        float3  _S973 = make_float3 (_S966, _S969, _S972);

#line 70
        float3  _S974 = make_float3 (_S967, _S970, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S971));

        if(active_sh_15 > 1U)
        {

#line 73
            uint3  _S975 = make_uint3 (g_idx_8, 4U, 0U);

#line 73
            float _S976 = ((sh_coeffs_10.primal_0).load<float>((_S975)));

#line 73
            float _S977 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S975);

#line 73
            uint3  _S978 = make_uint3 (g_idx_8, 4U, 1U);

#line 73
            float _S979 = ((sh_coeffs_10.primal_0).load<float>((_S978)));

#line 73
            float _S980 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S978);

#line 73
            uint3  _S981 = make_uint3 (g_idx_8, 4U, 2U);

#line 73
            float _S982 = ((sh_coeffs_10.primal_0).load<float>((_S981)));

#line 73
            float3  _S983 = make_float3 (_S976, _S979, _S982);

#line 73
            float3  _S984 = make_float3 (_S977, _S980, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S981));
            uint3  _S985 = make_uint3 (g_idx_8, 5U, 0U);

#line 74
            float _S986 = ((sh_coeffs_10.primal_0).load<float>((_S985)));

#line 74
            float _S987 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S985);

#line 74
            uint3  _S988 = make_uint3 (g_idx_8, 5U, 1U);

#line 74
            float _S989 = ((sh_coeffs_10.primal_0).load<float>((_S988)));

#line 74
            float _S990 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S988);

#line 74
            uint3  _S991 = make_uint3 (g_idx_8, 5U, 2U);

#line 74
            float _S992 = ((sh_coeffs_10.primal_0).load<float>((_S991)));

#line 74
            float3  _S993 = make_float3 (_S986, _S989, _S992);

#line 74
            float3  _S994 = make_float3 (_S987, _S990, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S991));
            uint3  _S995 = make_uint3 (g_idx_8, 6U, 0U);

#line 75
            float _S996 = ((sh_coeffs_10.primal_0).load<float>((_S995)));

#line 75
            float _S997 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S995);

#line 75
            uint3  _S998 = make_uint3 (g_idx_8, 6U, 1U);

#line 75
            float _S999 = ((sh_coeffs_10.primal_0).load<float>((_S998)));

#line 75
            float _S1000 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S998);

#line 75
            uint3  _S1001 = make_uint3 (g_idx_8, 6U, 2U);

#line 75
            float _S1002 = ((sh_coeffs_10.primal_0).load<float>((_S1001)));

#line 75
            float3  _S1003 = make_float3 (_S996, _S999, _S1002);

#line 75
            float3  _S1004 = make_float3 (_S997, _S1000, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1001));
            uint3  _S1005 = make_uint3 (g_idx_8, 7U, 0U);

#line 76
            float _S1006 = ((sh_coeffs_10.primal_0).load<float>((_S1005)));

#line 76
            float _S1007 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1005);

#line 76
            uint3  _S1008 = make_uint3 (g_idx_8, 7U, 1U);

#line 76
            float _S1009 = ((sh_coeffs_10.primal_0).load<float>((_S1008)));

#line 76
            float _S1010 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1008);

#line 76
            uint3  _S1011 = make_uint3 (g_idx_8, 7U, 2U);

#line 76
            float _S1012 = ((sh_coeffs_10.primal_0).load<float>((_S1011)));

#line 76
            float3  _S1013 = make_float3 (_S1006, _S1009, _S1012);

#line 76
            float3  _S1014 = make_float3 (_S1007, _S1010, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1011));
            uint3  _S1015 = make_uint3 (g_idx_8, 8U, 0U);

#line 77
            float _S1016 = ((sh_coeffs_10.primal_0).load<float>((_S1015)));

#line 77
            float _S1017 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1015);

#line 77
            uint3  _S1018 = make_uint3 (g_idx_8, 8U, 1U);

#line 77
            float _S1019 = ((sh_coeffs_10.primal_0).load<float>((_S1018)));

#line 77
            float _S1020 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1018);

#line 77
            uint3  _S1021 = make_uint3 (g_idx_8, 8U, 2U);

#line 77
            float _S1022 = ((sh_coeffs_10.primal_0).load<float>((_S1021)));

#line 77
            float3  _S1023 = make_float3 (_S1016, _S1019, _S1022);

#line 77
            float3  _S1024 = make_float3 (_S1017, _S1020, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1021));

            if(active_sh_15 > 2U)
            {

#line 80
                uint3  _S1025 = make_uint3 (g_idx_8, 9U, 0U);

#line 80
                float _S1026 = ((sh_coeffs_10.primal_0).load<float>((_S1025)));

#line 80
                float _S1027 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1025);

#line 80
                uint3  _S1028 = make_uint3 (g_idx_8, 9U, 1U);

#line 80
                float _S1029 = ((sh_coeffs_10.primal_0).load<float>((_S1028)));

#line 80
                float _S1030 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1028);

#line 80
                uint3  _S1031 = make_uint3 (g_idx_8, 9U, 2U);

#line 80
                float _S1032 = ((sh_coeffs_10.primal_0).load<float>((_S1031)));

#line 80
                float3  _S1033 = make_float3 (_S1026, _S1029, _S1032);

#line 80
                float3  _S1034 = make_float3 (_S1027, _S1030, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1031));
                uint3  _S1035 = make_uint3 (g_idx_8, 10U, 0U);

#line 81
                float _S1036 = ((sh_coeffs_10.primal_0).load<float>((_S1035)));

#line 81
                float _S1037 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1035);

#line 81
                uint3  _S1038 = make_uint3 (g_idx_8, 10U, 1U);

#line 81
                float _S1039 = ((sh_coeffs_10.primal_0).load<float>((_S1038)));

#line 81
                float _S1040 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1038);

#line 81
                uint3  _S1041 = make_uint3 (g_idx_8, 10U, 2U);

#line 81
                float _S1042 = ((sh_coeffs_10.primal_0).load<float>((_S1041)));

#line 81
                float3  _S1043 = make_float3 (_S1036, _S1039, _S1042);

#line 81
                float3  _S1044 = make_float3 (_S1037, _S1040, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1041));
                uint3  _S1045 = make_uint3 (g_idx_8, 11U, 0U);

#line 82
                float _S1046 = ((sh_coeffs_10.primal_0).load<float>((_S1045)));

#line 82
                float _S1047 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1045);

#line 82
                uint3  _S1048 = make_uint3 (g_idx_8, 11U, 1U);

#line 82
                float _S1049 = ((sh_coeffs_10.primal_0).load<float>((_S1048)));

#line 82
                float _S1050 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1048);

#line 82
                uint3  _S1051 = make_uint3 (g_idx_8, 11U, 2U);

#line 82
                float _S1052 = ((sh_coeffs_10.primal_0).load<float>((_S1051)));

#line 82
                float3  _S1053 = make_float3 (_S1046, _S1049, _S1052);

#line 82
                float3  _S1054 = make_float3 (_S1047, _S1050, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1051));
                uint3  _S1055 = make_uint3 (g_idx_8, 12U, 0U);

#line 83
                float _S1056 = ((sh_coeffs_10.primal_0).load<float>((_S1055)));

#line 83
                float _S1057 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1055);

#line 83
                uint3  _S1058 = make_uint3 (g_idx_8, 12U, 1U);

#line 83
                float _S1059 = ((sh_coeffs_10.primal_0).load<float>((_S1058)));

#line 83
                float _S1060 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1058);

#line 83
                uint3  _S1061 = make_uint3 (g_idx_8, 12U, 2U);

#line 83
                float _S1062 = ((sh_coeffs_10.primal_0).load<float>((_S1061)));

#line 83
                float3  _S1063 = make_float3 (_S1056, _S1059, _S1062);

#line 83
                float3  _S1064 = make_float3 (_S1057, _S1060, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1061));
                uint3  _S1065 = make_uint3 (g_idx_8, 13U, 0U);

#line 84
                float _S1066 = ((sh_coeffs_10.primal_0).load<float>((_S1065)));

#line 84
                float _S1067 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1065);

#line 84
                uint3  _S1068 = make_uint3 (g_idx_8, 13U, 1U);

#line 84
                float _S1069 = ((sh_coeffs_10.primal_0).load<float>((_S1068)));

#line 84
                float _S1070 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1068);

#line 84
                uint3  _S1071 = make_uint3 (g_idx_8, 13U, 2U);

#line 84
                float _S1072 = ((sh_coeffs_10.primal_0).load<float>((_S1071)));

#line 84
                float3  _S1073 = make_float3 (_S1066, _S1069, _S1072);

#line 84
                float3  _S1074 = make_float3 (_S1067, _S1070, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1071));
                uint3  _S1075 = make_uint3 (g_idx_8, 14U, 0U);

#line 85
                float _S1076 = ((sh_coeffs_10.primal_0).load<float>((_S1075)));

#line 85
                float _S1077 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1075);

#line 85
                uint3  _S1078 = make_uint3 (g_idx_8, 14U, 1U);

#line 85
                float _S1079 = ((sh_coeffs_10.primal_0).load<float>((_S1078)));

#line 85
                float _S1080 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1078);

#line 85
                uint3  _S1081 = make_uint3 (g_idx_8, 14U, 2U);

#line 85
                float _S1082 = ((sh_coeffs_10.primal_0).load<float>((_S1081)));

#line 85
                float3  _S1083 = make_float3 (_S1076, _S1079, _S1082);

#line 85
                float3  _S1084 = make_float3 (_S1077, _S1080, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1081));
                uint3  _S1085 = make_uint3 (g_idx_8, 15U, 0U);

#line 86
                float _S1086 = ((sh_coeffs_10.primal_0).load<float>((_S1085)));

#line 86
                float _S1087 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1085);

#line 86
                uint3  _S1088 = make_uint3 (g_idx_8, 15U, 1U);

#line 86
                float _S1089 = ((sh_coeffs_10.primal_0).load<float>((_S1088)));

#line 86
                float _S1090 = AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1088);

#line 86
                uint3  _S1091 = make_uint3 (g_idx_8, 15U, 2U);

#line 86
                float _S1092 = ((sh_coeffs_10.primal_0).load<float>((_S1091)));

#line 86
                float3  _S1093 = make_float3 (_S1086, _S1089, _S1092);

#line 86
                float3  _S1094 = make_float3 (_S1087, _S1090, AtomicAdd_load_forward_1(sh_coeffs_10.diff_1, _S1091));

#line 86
                (&g_sh_coeffs_2)->coeff0_0 = _S943;

#line 86
                (&g_sh_coeffs_2)->coeff1_0 = _S953;

#line 86
                (&g_sh_coeffs_2)->coeff2_0 = _S963;

#line 86
                (&g_sh_coeffs_2)->coeff3_0 = _S973;

#line 86
                (&g_sh_coeffs_2)->coeff4_0 = _S983;

#line 86
                (&g_sh_coeffs_2)->coeff5_0 = _S993;

#line 86
                (&g_sh_coeffs_2)->coeff6_0 = _S1003;

#line 86
                (&g_sh_coeffs_2)->coeff7_0 = _S1013;

#line 86
                (&g_sh_coeffs_2)->coeff8_0 = _S1023;

#line 86
                (&g_sh_coeffs_2)->coeff9_0 = _S1033;

#line 86
                (&g_sh_coeffs_2)->coeff10_0 = _S1043;

#line 86
                (&g_sh_coeffs_2)->coeff11_0 = _S1053;

#line 86
                (&g_sh_coeffs_2)->coeff12_0 = _S1063;

#line 86
                (&g_sh_coeffs_2)->coeff13_0 = _S1073;

#line 86
                (&g_sh_coeffs_2)->coeff14_0 = _S1083;

#line 86
                (&g_sh_coeffs_2)->coeff15_0 = _S1093;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff0_0 = _S944;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff1_0 = _S954;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff2_0 = _S964;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff3_0 = _S974;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff4_0 = _S984;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff5_0 = _S994;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff6_0 = _S1004;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff7_0 = _S1014;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff8_0 = _S1024;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff9_0 = _S1034;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff10_0 = _S1044;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff11_0 = _S1054;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff12_0 = _S1064;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff13_0 = _S1074;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff14_0 = _S1084;

#line 86
                (&s_diff_g_sh_coeffs_0)->coeff15_0 = _S1094;

#line 79
            }
            else
            {

#line 79
                (&g_sh_coeffs_2)->coeff0_0 = _S943;

#line 79
                (&g_sh_coeffs_2)->coeff1_0 = _S953;

#line 79
                (&g_sh_coeffs_2)->coeff2_0 = _S963;

#line 79
                (&g_sh_coeffs_2)->coeff3_0 = _S973;

#line 79
                (&g_sh_coeffs_2)->coeff4_0 = _S983;

#line 79
                (&g_sh_coeffs_2)->coeff5_0 = _S993;

#line 79
                (&g_sh_coeffs_2)->coeff6_0 = _S1003;

#line 79
                (&g_sh_coeffs_2)->coeff7_0 = _S1013;

#line 79
                (&g_sh_coeffs_2)->coeff8_0 = _S1023;

#line 79
                (&g_sh_coeffs_2)->coeff9_0 = _S934;

#line 79
                (&g_sh_coeffs_2)->coeff10_0 = _S934;

#line 79
                (&g_sh_coeffs_2)->coeff11_0 = _S934;

#line 79
                (&g_sh_coeffs_2)->coeff12_0 = _S934;

#line 79
                (&g_sh_coeffs_2)->coeff13_0 = _S934;

#line 79
                (&g_sh_coeffs_2)->coeff14_0 = _S934;

#line 79
                (&g_sh_coeffs_2)->coeff15_0 = _S934;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff0_0 = _S944;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff1_0 = _S954;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff2_0 = _S964;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff3_0 = _S974;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff4_0 = _S984;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff5_0 = _S994;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff6_0 = _S1004;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff7_0 = _S1014;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff8_0 = _S1024;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff9_0 = _S934;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff10_0 = _S934;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff11_0 = _S934;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff12_0 = _S934;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff13_0 = _S934;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff14_0 = _S934;

#line 79
                (&s_diff_g_sh_coeffs_0)->coeff15_0 = _S934;

#line 79
            }

#line 72
        }
        else
        {

#line 72
            (&g_sh_coeffs_2)->coeff0_0 = _S943;

#line 72
            (&g_sh_coeffs_2)->coeff1_0 = _S953;

#line 72
            (&g_sh_coeffs_2)->coeff2_0 = _S963;

#line 72
            (&g_sh_coeffs_2)->coeff3_0 = _S973;

#line 72
            (&g_sh_coeffs_2)->coeff4_0 = _S934;

#line 72
            (&g_sh_coeffs_2)->coeff5_0 = _S934;

#line 72
            (&g_sh_coeffs_2)->coeff6_0 = _S934;

#line 72
            (&g_sh_coeffs_2)->coeff7_0 = _S934;

#line 72
            (&g_sh_coeffs_2)->coeff8_0 = _S934;

#line 72
            (&g_sh_coeffs_2)->coeff9_0 = _S934;

#line 72
            (&g_sh_coeffs_2)->coeff10_0 = _S934;

#line 72
            (&g_sh_coeffs_2)->coeff11_0 = _S934;

#line 72
            (&g_sh_coeffs_2)->coeff12_0 = _S934;

#line 72
            (&g_sh_coeffs_2)->coeff13_0 = _S934;

#line 72
            (&g_sh_coeffs_2)->coeff14_0 = _S934;

#line 72
            (&g_sh_coeffs_2)->coeff15_0 = _S934;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff0_0 = _S944;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff1_0 = _S954;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff2_0 = _S964;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff3_0 = _S974;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff4_0 = _S934;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff5_0 = _S934;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff6_0 = _S934;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff7_0 = _S934;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff8_0 = _S934;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff9_0 = _S934;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff10_0 = _S934;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff11_0 = _S934;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff12_0 = _S934;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff13_0 = _S934;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff14_0 = _S934;

#line 72
            (&s_diff_g_sh_coeffs_0)->coeff15_0 = _S934;

#line 72
        }

#line 67
    }
    else
    {

#line 67
        (&g_sh_coeffs_2)->coeff0_0 = _S943;

#line 67
        (&g_sh_coeffs_2)->coeff1_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff2_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff3_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff4_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff5_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff6_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff7_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff8_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff9_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff10_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff11_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff12_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff13_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff14_0 = _S934;

#line 67
        (&g_sh_coeffs_2)->coeff15_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff0_0 = _S944;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff1_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff2_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff3_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff4_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff5_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff6_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff7_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff8_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff9_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff10_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff11_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff12_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff13_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff14_0 = _S934;

#line 67
        (&s_diff_g_sh_coeffs_0)->coeff15_0 = _S934;

#line 67
    }

#line 67
    DiffPair_SpherHarmCoeffs_0 _S1095 = { g_sh_coeffs_2, s_diff_g_sh_coeffs_0 };

#line 90
    return _S1095;
}


#line 179 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
__device__ DiffPair_vectorx3Cfloatx2C4x3E_0 s_fwd_read_t4_float4_0(uint idx_7, DiffTensorView_0 t4_3)
{

#line 36
    uint2  _S1096 = make_uint2 (idx_7, 0U);

#line 36
    float _S1097 = ((t4_3.primal_0).load<float>((_S1096)));

#line 36
    float _S1098 = AtomicAdd_load_forward_0(t4_3.diff_1, _S1096);
    uint2  _S1099 = make_uint2 (idx_7, 1U);

#line 36
    float _S1100 = ((t4_3.primal_0).load<float>((_S1099)));

#line 36
    float _S1101 = AtomicAdd_load_forward_0(t4_3.diff_1, _S1099);

    uint2  _S1102 = make_uint2 (idx_7, 2U);

#line 36
    float _S1103 = ((t4_3.primal_0).load<float>((_S1102)));

#line 36
    float _S1104 = AtomicAdd_load_forward_0(t4_3.diff_1, _S1102);


    uint2  _S1105 = make_uint2 (idx_7, 3U);

#line 36
    float _S1106 = ((t4_3.primal_0).load<float>((_S1105)));

#line 36
    DiffPair_vectorx3Cfloatx2C4x3E_0 _S1107 = { make_float4 (_S1097, _S1100, _S1103, _S1106), make_float4 (_S1098, _S1101, _S1104, AtomicAdd_load_forward_0(t4_3.diff_1, _S1105)) };

#line 36
    return _S1107;
}


#line 36
__device__ DiffPair_Gaussian_3D_0 s_fwd_load_gaussian_0(int g_idx_9, DiffTensorView_0 xyz_ws_9, DiffTensorView_0 sh_coeffs_11, DiffTensorView_0 rotations_7, DiffTensorView_0 scales_7, uint active_sh_16)
{

#line 177
    uint _S1108 = uint(g_idx_9);

#line 177
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1109 = s_fwd_read_t3_float3_0(_S1108, xyz_ws_9);
    DiffPair_SpherHarmCoeffs_0 _S1110 = s_fwd_read_spherical_harmonics_coeffs_0(_S1108, sh_coeffs_11, active_sh_16);
    DiffPair_vectorx3Cfloatx2C4x3E_0 _S1111 = s_fwd_read_t4_float4_0(_S1108, rotations_7);
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1112 = s_fwd_read_t3_float3_0(_S1108, scales_7);

    Gaussian_3D_0 _S1113 = { _S1109.primal_1, _S1110.primal_1, _S1111.primal_1, _S1112.primal_1 };

#line 182
    Gaussian_3D_0 _S1114 = { _S1109.differential_0, _S1110.differential_0, _S1111.differential_0, _S1112.differential_0 };

#line 182
    DiffPair_Gaussian_3D_0 _S1115 = { _S1113, _S1114 };

#line 182
    return _S1115;
}


#line 182
struct DiffPair_Splat_2D_Vertex_0
{
    Splat_2D_Vertex_0 primal_1;
    Splat_2D_Vertex_0 differential_0;
};


#line 120
__device__ DiffPair_vectorx3Cfloatx2C3x3E_0 s_fwd_geom_transform_points_0(DiffPair_vectorx3Cfloatx2C3x3E_0 dppoint_6, DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 dptransf_matrix_4)
{

#line 107
    float4  _S1116 = make_float4 (dppoint_6.primal_1.x, dppoint_6.primal_1.y, dppoint_6.primal_1.z, 1.0f);

#line 107
    float4  _S1117 = mul_4(dptransf_matrix_4.primal_1, _S1116);

#line 107
    float4  _S1118 = mul_4(dptransf_matrix_4.differential_0, _S1116) + mul_4(dptransf_matrix_4.primal_1, make_float4 (dppoint_6.differential_0.x, dppoint_6.differential_0.y, dppoint_6.differential_0.z, 0.0f));
    float3  _S1119 = float3 {_S1117.x, _S1117.y, _S1117.z};

#line 108
    float _S1120 = _S1117.w + 1.00000001168609742e-07f;

#line 108
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1121 = { _S1119 / make_float3 (_S1120), (float3 {_S1118.x, _S1118.y, _S1118.z} * make_float3 (_S1120) - _S1119 * make_float3 (_S1118.w)) / make_float3 (_S1120 * _S1120) };

#line 108
    return _S1121;
}


#line 108
__device__ DiffPair_vectorx3Cfloatx2C3x3E_0 s_fwd_geom_transform_points2_0(DiffPair_vectorx3Cfloatx2C3x3E_0 dppoint_7, DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 dptransf_matrix_5)
{

#line 114
    float4  _S1122 = make_float4 (dppoint_7.primal_1.x, dppoint_7.primal_1.y, dppoint_7.primal_1.z, 1.0f);

#line 114
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1123 = { float3 {mul_4(dptransf_matrix_5.primal_1, _S1122).x, mul_4(dptransf_matrix_5.primal_1, _S1122).y, mul_4(dptransf_matrix_5.primal_1, _S1122).z}, float3 {(mul_4(dptransf_matrix_5.differential_0, _S1122) + mul_4(dptransf_matrix_5.primal_1, make_float4 (dppoint_7.differential_0.x, dppoint_7.differential_0.y, dppoint_7.differential_0.z, 0.0f))).x, (mul_4(dptransf_matrix_5.differential_0, _S1122) + mul_4(dptransf_matrix_5.primal_1, make_float4 (dppoint_7.differential_0.x, dppoint_7.differential_0.y, dppoint_7.differential_0.z, 0.0f))).y, (mul_4(dptransf_matrix_5.differential_0, _S1122) + mul_4(dptransf_matrix_5.primal_1, make_float4 (dppoint_7.differential_0.x, dppoint_7.differential_0.y, dppoint_7.differential_0.z, 0.0f))).z} };
    return _S1123;
}


#line 115
__device__ DiffPair_vectorx3Cfloatx2C3x3E_0 s_fwd_project_point_0(DiffPair_vectorx3Cfloatx2C3x3E_0 dppoint_8, DiffPair_Camera_0 dpcam_8)
{


    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1124 = { dppoint_8.primal_1, dppoint_8.differential_0 };

#line 119
    DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 _S1125 = { mul_2(dpcam_8.primal_1.proj_mat_1, dpcam_8.primal_1.world_view_transform_1), mul_2(dpcam_8.differential_0.proj_mat_0, dpcam_8.primal_1.world_view_transform_1) + mul_2(dpcam_8.primal_1.proj_mat_1, dpcam_8.differential_0.world_view_transform_0) };
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1126 = s_fwd_geom_transform_points_0(_S1124, _S1125);

#line 120
    DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 _S1127 = { dpcam_8.primal_1.world_view_transform_1, dpcam_8.differential_0.world_view_transform_0 };
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1128 = s_fwd_geom_transform_points2_0(_S1124, _S1127);
    float _S1129 = _S1128.primal_1.z;

#line 122
    float _S1130 = _S1128.differential_0.z;

#line 122
    float3  _S1131 = _S1126.primal_1;

#line 122
    *&((&_S1131)->z) = _S1129;

#line 122
    float3  _S1132 = _S1126.differential_0;

#line 122
    *&((&_S1132)->z) = _S1130;

#line 122
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1133 = { _S1131, _S1132 };
    return _S1133;
}


#line 2107 "diff.meta.slang"
__device__ DiffPair_float_0 s_fwd_length_impl_0(DiffPair_vectorx3Cfloatx2C3x3E_0 dpx_12)
{

#line 2092
    float _S1134 = dpx_12.primal_1.x;

#line 2092
    float _S1135 = dpx_12.differential_0.x * dpx_12.primal_1.x;

#line 2092
    float _S1136 = dpx_12.primal_1.y;

#line 2092
    float _S1137 = dpx_12.differential_0.y * dpx_12.primal_1.y;

#line 2092
    float _S1138 = dpx_12.primal_1.z;

#line 2092
    float _S1139 = dpx_12.differential_0.z * dpx_12.primal_1.z;

#line 2092
    DiffPair_float_0 _S1140 = { _S1134 * _S1134 + _S1136 * _S1136 + _S1138 * _S1138, _S1135 + _S1135 + (_S1137 + _S1137) + (_S1139 + _S1139) };

#line 2099
    DiffPair_float_0 _S1141 = _d_sqrt_1(_S1140);

#line 2099
    DiffPair_float_0 _S1142 = { _S1141.primal_1, _S1141.differential_0 };

#line 2099
    return _S1142;
}


#line 2099
__device__ DiffPair_vectorx3Cfloatx2C3x3E_0 s_fwd_normalize_impl_0(DiffPair_vectorx3Cfloatx2C3x3E_0 dpx_13)
{

#line 2154
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1143 = { dpx_13.primal_1, dpx_13.differential_0 };

    DiffPair_float_0 _S1144 = s_fwd_length_impl_0(_S1143);

#line 2156
    float _S1145 = 1.0f / _S1144.primal_1;

#line 2156
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1146 = { dpx_13.primal_1 * make_float3 (_S1145), dpx_13.differential_0 * make_float3 (_S1145) + make_float3 ((0.0f - _S1144.differential_0) / (_S1144.primal_1 * _S1144.primal_1)) * dpx_13.primal_1 };
    return _S1146;
}


#line 2157
__device__ DiffPair_vectorx3Cfloatx2C3x3E_0 s_fwd_compute_color_from_sh_coeffs_0(DiffPair_SpherHarmCoeffs_0 dpsh_2, DiffPair_vectorx3Cfloatx2C3x3E_0 dpg_xyz_ws_2, DiffPair_vectorx3Cfloatx2C3x3E_0 dpcam_pos_2, uint active_sh_17)
{

#line 94 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/spherical_harmonics.slang"
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1147 = { dpg_xyz_ws_2.primal_1 - dpcam_pos_2.primal_1, dpg_xyz_ws_2.differential_0 - dpcam_pos_2.differential_0 };

    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1148 = s_fwd_normalize_impl_0(_S1147);

    float3  rgb_14 = make_float3 (0.282094806432724f) * dpsh_2.primal_1.coeff0_0;

#line 98
    float3  _S1149 = dpsh_2.differential_0.coeff0_0 * make_float3 (0.282094806432724f);

#line 98
    float3  rgb_15;

#line 98
    float3  s_diff_rgb_0;
    if(active_sh_17 > 0U)
    {

#line 100
        float _S1150 = _S1148.primal_1.y;

#line 100
        float _S1151 = _S1148.differential_0.y;

#line 100
        float _S1152 = 0.48860251903533936f * _S1150;

#line 100
        float _S1153 = _S1148.primal_1.z;

#line 100
        float _S1154 = _S1148.differential_0.z;

#line 100
        float _S1155 = 0.48860251903533936f * _S1153;

#line 100
        float _S1156 = _S1148.primal_1.x;

#line 100
        float _S1157 = _S1148.differential_0.x;

#line 100
        float _S1158 = 0.48860251903533936f * _S1156;

#line 100
        float3  rgb_16 = rgb_14 - make_float3 (_S1152) * dpsh_2.primal_1.coeff1_0 + make_float3 (_S1155) * dpsh_2.primal_1.coeff2_0 - make_float3 (_S1158) * dpsh_2.primal_1.coeff3_0;

#line 100
        float3  s_diff_rgb_1 = _S1149 - (make_float3 (_S1151 * 0.48860251903533936f) * dpsh_2.primal_1.coeff1_0 + dpsh_2.differential_0.coeff1_0 * make_float3 (_S1152)) + (make_float3 (_S1154 * 0.48860251903533936f) * dpsh_2.primal_1.coeff2_0 + dpsh_2.differential_0.coeff2_0 * make_float3 (_S1155)) - (make_float3 (_S1157 * 0.48860251903533936f) * dpsh_2.primal_1.coeff3_0 + dpsh_2.differential_0.coeff3_0 * make_float3 (_S1158));
        if(active_sh_17 > 1U)
        {
            float xx_3 = _S1156 * _S1156;

#line 103
            float _S1159 = _S1157 * _S1156;

#line 103
            float s_diff_xx_0 = _S1159 + _S1159;

#line 103
            float yy_3 = _S1150 * _S1150;

#line 103
            float _S1160 = _S1151 * _S1150;

#line 103
            float s_diff_yy_0 = _S1160 + _S1160;

#line 103
            float zz_3 = _S1153 * _S1153;

#line 103
            float _S1161 = _S1154 * _S1153;

#line 103
            float s_diff_zz_0 = _S1161 + _S1161;
            float xy_3 = _S1156 * _S1150;

#line 104
            float s_diff_xy_0 = _S1157 * _S1150 + _S1151 * _S1156;

            float _S1162 = 1.09254848957061768f * xy_3;
            float _S1163 = -1.09254848957061768f * (_S1150 * _S1153);
            float _S1164 = 2.0f * zz_3;

#line 108
            float _S1165 = s_diff_zz_0 * 2.0f;

#line 108
            float _S1166 = 0.31539157032966614f * (_S1164 - xx_3 - yy_3);
            float _S1167 = -1.09254848957061768f * (_S1156 * _S1153);
            float _S1168 = xx_3 - yy_3;

#line 110
            float _S1169 = s_diff_xx_0 - s_diff_yy_0;

#line 110
            float _S1170 = 0.54627424478530884f * _S1168;

#line 109
            float3  rgb_17 = rgb_16 + make_float3 (_S1162) * dpsh_2.primal_1.coeff4_0 + make_float3 (_S1163) * dpsh_2.primal_1.coeff5_0 + make_float3 (_S1166) * dpsh_2.primal_1.coeff6_0 + make_float3 (_S1167) * dpsh_2.primal_1.coeff7_0 + make_float3 (_S1170) * dpsh_2.primal_1.coeff8_0;

#line 109
            float3  s_diff_rgb_2 = s_diff_rgb_1 + (make_float3 (s_diff_xy_0 * 1.09254848957061768f) * dpsh_2.primal_1.coeff4_0 + dpsh_2.differential_0.coeff4_0 * make_float3 (_S1162)) + (make_float3 ((_S1151 * _S1153 + _S1154 * _S1150) * -1.09254848957061768f) * dpsh_2.primal_1.coeff5_0 + dpsh_2.differential_0.coeff5_0 * make_float3 (_S1163)) + (make_float3 ((_S1165 - s_diff_xx_0 - s_diff_yy_0) * 0.31539157032966614f) * dpsh_2.primal_1.coeff6_0 + dpsh_2.differential_0.coeff6_0 * make_float3 (_S1166)) + (make_float3 ((_S1157 * _S1153 + _S1154 * _S1156) * -1.09254848957061768f) * dpsh_2.primal_1.coeff7_0 + dpsh_2.differential_0.coeff7_0 * make_float3 (_S1167)) + (make_float3 (_S1169 * 0.54627424478530884f) * dpsh_2.primal_1.coeff8_0 + dpsh_2.differential_0.coeff8_0 * make_float3 (_S1170));


            if(active_sh_17 > 2U)
            {

                float _S1171 = -0.59004360437393188f * _S1150;

#line 115
                float _S1172 = 3.0f * xx_3;

#line 115
                float _S1173 = s_diff_xx_0 * 3.0f;

#line 115
                float _S1174 = _S1172 - yy_3;

#line 115
                float _S1175 = _S1171 * _S1174;
                float _S1176 = 2.89061141014099121f * xy_3;

#line 116
                float _S1177 = _S1176 * _S1153;
                float _S1178 = -0.4570457935333252f * _S1150;

#line 117
                float _S1179 = 4.0f * zz_3 - xx_3 - yy_3;

#line 117
                float _S1180 = s_diff_zz_0 * 4.0f - s_diff_xx_0 - s_diff_yy_0;

#line 117
                float _S1181 = _S1178 * _S1179;
                float _S1182 = 0.37317633628845215f * _S1153;

#line 118
                float _S1183 = 3.0f * yy_3;

#line 118
                float _S1184 = s_diff_yy_0 * 3.0f;

#line 118
                float _S1185 = _S1164 - _S1172 - _S1183;

#line 118
                float _S1186 = _S1182 * _S1185;
                float _S1187 = -0.4570457935333252f * _S1156;

#line 119
                float _S1188 = _S1187 * _S1179;
                float _S1189 = 1.44530570507049561f * _S1153;

#line 120
                float _S1190 = _S1189 * _S1168;
                float _S1191 = -0.59004360437393188f * _S1156;

#line 121
                float _S1192 = xx_3 - _S1183;

#line 121
                float _S1193 = _S1191 * _S1192;

#line 120
                float3  _S1194 = s_diff_rgb_2 + (make_float3 (_S1151 * -0.59004360437393188f * _S1174 + (_S1173 - s_diff_yy_0) * _S1171) * dpsh_2.primal_1.coeff9_0 + dpsh_2.differential_0.coeff9_0 * make_float3 (_S1175)) + (make_float3 (s_diff_xy_0 * 2.89061141014099121f * _S1153 + _S1154 * _S1176) * dpsh_2.primal_1.coeff10_0 + dpsh_2.differential_0.coeff10_0 * make_float3 (_S1177)) + (make_float3 (_S1151 * -0.4570457935333252f * _S1179 + _S1180 * _S1178) * dpsh_2.primal_1.coeff11_0 + dpsh_2.differential_0.coeff11_0 * make_float3 (_S1181)) + (make_float3 (_S1154 * 0.37317633628845215f * _S1185 + (_S1165 - _S1173 - _S1184) * _S1182) * dpsh_2.primal_1.coeff12_0 + dpsh_2.differential_0.coeff12_0 * make_float3 (_S1186)) + (make_float3 (_S1157 * -0.4570457935333252f * _S1179 + _S1180 * _S1187) * dpsh_2.primal_1.coeff13_0 + dpsh_2.differential_0.coeff13_0 * make_float3 (_S1188)) + (make_float3 (_S1154 * 1.44530570507049561f * _S1168 + _S1169 * _S1189) * dpsh_2.primal_1.coeff14_0 + dpsh_2.differential_0.coeff14_0 * make_float3 (_S1190)) + (make_float3 (_S1157 * -0.59004360437393188f * _S1192 + (s_diff_xx_0 - _S1184) * _S1191) * dpsh_2.primal_1.coeff15_0 + dpsh_2.differential_0.coeff15_0 * make_float3 (_S1193));

#line 120
                rgb_15 = rgb_17 + make_float3 (_S1175) * dpsh_2.primal_1.coeff9_0 + make_float3 (_S1177) * dpsh_2.primal_1.coeff10_0 + make_float3 (_S1181) * dpsh_2.primal_1.coeff11_0 + make_float3 (_S1186) * dpsh_2.primal_1.coeff12_0 + make_float3 (_S1188) * dpsh_2.primal_1.coeff13_0 + make_float3 (_S1190) * dpsh_2.primal_1.coeff14_0 + make_float3 (_S1193) * dpsh_2.primal_1.coeff15_0;

#line 120
                s_diff_rgb_0 = _S1194;

#line 112
            }
            else
            {

#line 112
                rgb_15 = rgb_17;

#line 112
                s_diff_rgb_0 = s_diff_rgb_2;

#line 112
            }

#line 101
        }
        else
        {

#line 101
            rgb_15 = rgb_16;

#line 101
            s_diff_rgb_0 = s_diff_rgb_1;

#line 101
        }

#line 99
    }
    else
    {

#line 99
        rgb_15 = rgb_14;

#line 99
        s_diff_rgb_0 = _S1149;

#line 99
    }

#line 99
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1195 = { rgb_15 + make_float3 (0.5f), s_diff_rgb_0 };

#line 99
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1196 = { make_float3 (0.0f), make_float3 (0.0f) };

#line 128
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1197 = _d_max_vector_1(_S1195, _S1196);

#line 128
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1198 = { _S1197.primal_1, _S1197.differential_0 };

#line 128
    return _S1198;
}


#line 228 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
__device__ DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 s_fwd_get_covariance_from_quat_scales_0(DiffPair_vectorx3Cfloatx2C4x3E_0 dpq_2, DiffPair_vectorx3Cfloatx2C3x3E_0 dps_2)
{

#line 280
    float _S1199 = dpq_2.primal_1.z;



    float _S1200 = _S1199 * _S1199;

#line 284
    float _S1201 = dpq_2.differential_0.z * dpq_2.primal_1.z;

#line 284
    float _S1202 = _S1201 + _S1201;

#line 284
    float _S1203 = dpq_2.primal_1.w * dpq_2.primal_1.w;

#line 284
    float _S1204 = dpq_2.differential_0.w * dpq_2.primal_1.w;

#line 284
    float _S1205 = _S1204 + _S1204;

#line 284
    float _S1206 = dpq_2.primal_1.y * dpq_2.primal_1.z;

#line 284
    float _S1207 = dpq_2.differential_0.y * dpq_2.primal_1.z + dpq_2.differential_0.z * dpq_2.primal_1.y;

#line 284
    float _S1208 = dpq_2.primal_1.x * dpq_2.primal_1.w;

#line 284
    float _S1209 = dpq_2.differential_0.x * dpq_2.primal_1.w + dpq_2.differential_0.w * dpq_2.primal_1.x;

#line 284
    float _S1210 = dpq_2.primal_1.y * dpq_2.primal_1.w;

#line 284
    float _S1211 = dpq_2.differential_0.y * dpq_2.primal_1.w + dpq_2.differential_0.w * dpq_2.primal_1.y;

#line 284
    float _S1212 = dpq_2.primal_1.x * dpq_2.primal_1.z;

#line 284
    float _S1213 = dpq_2.differential_0.x * dpq_2.primal_1.z + dpq_2.differential_0.z * dpq_2.primal_1.x;
    float _S1214 = dpq_2.primal_1.y * dpq_2.primal_1.y;

#line 285
    float _S1215 = dpq_2.differential_0.y * dpq_2.primal_1.y;

#line 285
    float _S1216 = _S1215 + _S1215;

#line 285
    float _S1217 = dpq_2.primal_1.z * dpq_2.primal_1.w;

#line 285
    float _S1218 = dpq_2.differential_0.z * dpq_2.primal_1.w + dpq_2.differential_0.w * dpq_2.primal_1.z;

#line 285
    float _S1219 = dpq_2.primal_1.x * dpq_2.primal_1.y;

#line 285
    float _S1220 = dpq_2.differential_0.x * dpq_2.primal_1.y + dpq_2.differential_0.y * dpq_2.primal_1.x;

#line 283
    Matrix<float, 3, 3>  rotation_matrix_1 = makeMatrix<float, 3, 3> (1.0f - 2.0f * (_S1200 + _S1203), 2.0f * (_S1206 - _S1208), 2.0f * (_S1210 + _S1212), 2.0f * (_S1206 + _S1208), 1.0f - 2.0f * (_S1214 + _S1203), 2.0f * (_S1217 - _S1219), 2.0f * (_S1210 - _S1212), 2.0f * (_S1217 + _S1219), 1.0f - 2.0f * (_S1214 + _S1200));

#line 288
    Matrix<float, 3, 3>  scales_matrix_1 = makeMatrix<float, 3, 3> (dps_2.primal_1.x, 0.0f, 0.0f, 0.0f, dps_2.primal_1.y, 0.0f, 0.0f, 0.0f, dps_2.primal_1.z);



    Matrix<float, 3, 3>  _S1221 = mul_3(rotation_matrix_1, scales_matrix_1);

#line 292
    Matrix<float, 3, 3>  _S1222 = mul_3(makeMatrix<float, 3, 3> (0.0f - (_S1202 + _S1205) * 2.0f, (_S1207 - _S1209) * 2.0f, (_S1211 + _S1213) * 2.0f, (_S1207 + _S1209) * 2.0f, 0.0f - (_S1216 + _S1205) * 2.0f, (_S1218 - _S1220) * 2.0f, (_S1211 - _S1213) * 2.0f, (_S1218 + _S1220) * 2.0f, 0.0f - (_S1216 + _S1202) * 2.0f), scales_matrix_1) + mul_3(rotation_matrix_1, makeMatrix<float, 3, 3> (dps_2.differential_0.x, 0.0f, 0.0f, 0.0f, dps_2.differential_0.y, 0.0f, 0.0f, 0.0f, dps_2.differential_0.z));

    Matrix<float, 3, 3>  _S1223 = transpose_0(_S1221);

#line 294
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S1224 = { mul_3(_S1221, _S1223), mul_3(_S1222, _S1223) + mul_3(_S1221, transpose_0(_S1222)) };

#line 294
    return _S1224;
}


#line 153
__device__ DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 s_fwd_compute_jacobian_0(DiffPair_vectorx3Cfloatx2C3x3E_0 dpxyz_ws_4, DiffPair_Camera_0 dpcam_9)
{

#line 127
    DiffPair_float_0 _S1225 = { dpcam_9.primal_1.fovx_1 / 2.0f, dpcam_9.differential_0.fovx_0 * 0.5f };
    DiffPair_float_0 _S1226 = _d_tan_1(_S1225);

#line 128
    DiffPair_float_0 _S1227 = { dpcam_9.primal_1.fovy_1 / 2.0f, dpcam_9.differential_0.fovy_0 * 0.5f };
    DiffPair_float_0 _S1228 = _d_tan_1(_S1227);
    float _S1229 = float(dpcam_9.primal_1.W_0);

#line 130
    float _S1230 = 2.0f * _S1226.primal_1;

#line 130
    float h_x_3 = _S1229 / _S1230;

#line 130
    float s_diff_h_x_0 = (0.0f - _S1229 * (_S1226.differential_0 * 2.0f)) / (_S1230 * _S1230);
    float _S1231 = float(dpcam_9.primal_1.H_0);

#line 131
    float _S1232 = 2.0f * _S1228.primal_1;

#line 131
    float h_y_3 = _S1231 / _S1232;

#line 131
    float s_diff_h_y_0 = (0.0f - _S1231 * (_S1228.differential_0 * 2.0f)) / (_S1232 * _S1232);

#line 131
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1233 = { dpxyz_ws_4.primal_1, dpxyz_ws_4.differential_0 };

#line 131
    DiffPair_matrixx3Cfloatx2C4x2C4x3E_0 _S1234 = { dpcam_9.primal_1.world_view_transform_1, dpcam_9.differential_0.world_view_transform_0 };

    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1235 = s_fwd_geom_transform_points_0(_S1233, _S1234);


    float limx_3 = 1.29999995231628418f * _S1226.primal_1;

#line 136
    float _S1236 = _S1226.differential_0 * 1.29999995231628418f;
    float limy_3 = 1.29999995231628418f * _S1228.primal_1;

#line 137
    float _S1237 = _S1228.differential_0 * 1.29999995231628418f;
    float _S1238 = _S1235.primal_1.x;

#line 138
    float _S1239 = _S1235.primal_1.z;

#line 138
    float _S1240 = _S1235.differential_0.z;

#line 138
    float _S1241 = _S1239 * _S1239;
    float _S1242 = _S1235.primal_1.y;

#line 139
    float tytz_3 = _S1242 / _S1239;

#line 139
    float s_diff_tytz_0 = (_S1235.differential_0.y * _S1239 - _S1242 * _S1240) / _S1241;

#line 139
    DiffPair_float_0 _S1243 = { - limx_3, - _S1236 };

#line 139
    DiffPair_float_0 _S1244 = { _S1238 / _S1239, (_S1235.differential_0.x * _S1239 - _S1238 * _S1240) / _S1241 };
    DiffPair_float_0 _S1245 = _d_max_1(_S1243, _S1244);

#line 140
    DiffPair_float_0 _S1246 = { limx_3, _S1236 };

#line 140
    DiffPair_float_0 _S1247 = { _S1245.primal_1, _S1245.differential_0 };

#line 140
    DiffPair_float_0 _S1248 = _d_min_1(_S1246, _S1247);

#line 140
    float _S1249 = _S1248.primal_1 * _S1239;

#line 140
    float _S1250 = _S1248.differential_0 * _S1239 + _S1240 * _S1248.primal_1;

#line 140
    float3  _S1251 = _S1235.primal_1;

#line 140
    *&((&_S1251)->x) = _S1249;

#line 140
    float3  _S1252 = _S1235.differential_0;

#line 140
    *&((&_S1252)->x) = _S1250;

#line 140
    DiffPair_float_0 _S1253 = { - limy_3, - _S1237 };

#line 140
    DiffPair_float_0 _S1254 = { tytz_3, s_diff_tytz_0 };
    DiffPair_float_0 _S1255 = _d_max_1(_S1253, _S1254);

#line 141
    DiffPair_float_0 _S1256 = { limy_3, _S1237 };

#line 141
    DiffPair_float_0 _S1257 = { _S1255.primal_1, _S1255.differential_0 };

#line 141
    DiffPair_float_0 _S1258 = _d_min_1(_S1256, _S1257);

#line 141
    float _S1259 = _S1251.z;

#line 141
    float _S1260 = _S1258.differential_0 * _S1259 + _S1252.z * _S1258.primal_1;

#line 141
    *&((&_S1251)->y) = _S1258.primal_1 * _S1259;

#line 141
    *&((&_S1252)->y) = _S1260;

    float _S1261 = _S1251.z;

#line 143
    float _S1262 = _S1252.z;

#line 143
    float _S1263 = _S1261 * _S1261;

#line 143
    float _S1264 = _S1251.x;

#line 143
    float _S1265 = - (h_x_3 * _S1264);

#line 143
    float _S1266 = _S1262 * _S1261;

#line 143
    float _S1267 = _S1266 + _S1266;

#line 143
    float _S1268 = _S1263 * _S1263;
    float _S1269 = _S1251.y;

#line 144
    float _S1270 = - (h_y_3 * _S1269);

#line 144
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S1271 = { makeMatrix<float, 3, 3> (h_x_3 / _S1261, 0.0f, _S1265 / _S1263, 0.0f, h_y_3 / _S1261, _S1270 / _S1263, 0.0f, 0.0f, 0.0f), makeMatrix<float, 3, 3> ((s_diff_h_x_0 * _S1261 - h_x_3 * _S1262) / _S1263, 0.0f, (- (s_diff_h_x_0 * _S1264 + _S1252.x * h_x_3) * _S1263 - _S1265 * _S1267) / _S1268, 0.0f, (s_diff_h_y_0 * _S1261 - h_y_3 * _S1262) / _S1263, (- (s_diff_h_y_0 * _S1269 + _S1252.y * h_y_3) * _S1263 - _S1270 * _S1267) / _S1268, 0.0f, 0.0f, 0.0f) };


    return _S1271;
}


#line 147
__device__ DiffPair_matrixx3Cfloatx2C2x2C2x3E_0 s_fwd_covariance_3d_to_2d_0(DiffPair_Camera_0 dpcam_10, DiffPair_vectorx3Cfloatx2C3x3E_0 dpxyz_ws_5, DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 dpcov_ws_2)
{


    Matrix<float, 3, 3>  _S1272 = makeMatrix<float, 3, 3> (float3 {dpcam_10.primal_1.world_view_transform_1.rows[int(0)].x, dpcam_10.primal_1.world_view_transform_1.rows[int(0)].y, dpcam_10.primal_1.world_view_transform_1.rows[int(0)].z}, float3 {dpcam_10.primal_1.world_view_transform_1.rows[int(1)].x, dpcam_10.primal_1.world_view_transform_1.rows[int(1)].y, dpcam_10.primal_1.world_view_transform_1.rows[int(1)].z}, float3 {dpcam_10.primal_1.world_view_transform_1.rows[int(2)].x, dpcam_10.primal_1.world_view_transform_1.rows[int(2)].y, dpcam_10.primal_1.world_view_transform_1.rows[int(2)].z});

#line 151
    Matrix<float, 3, 3>  _S1273 = makeMatrix<float, 3, 3> (float3 {dpcam_10.differential_0.world_view_transform_0.rows[int(0)].x, dpcam_10.differential_0.world_view_transform_0.rows[int(0)].y, dpcam_10.differential_0.world_view_transform_0.rows[int(0)].z}, float3 {dpcam_10.differential_0.world_view_transform_0.rows[int(1)].x, dpcam_10.differential_0.world_view_transform_0.rows[int(1)].y, dpcam_10.differential_0.world_view_transform_0.rows[int(1)].z}, float3 {dpcam_10.differential_0.world_view_transform_0.rows[int(2)].x, dpcam_10.differential_0.world_view_transform_0.rows[int(2)].y, dpcam_10.differential_0.world_view_transform_0.rows[int(2)].z});

#line 151
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1274 = { dpxyz_ws_5.primal_1, dpxyz_ws_5.differential_0 };

#line 151
    DiffPair_Camera_0 _S1275 = { dpcam_10.primal_1, dpcam_10.differential_0 };

    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S1276 = s_fwd_compute_jacobian_0(_S1274, _S1275);
    Matrix<float, 3, 3>  _S1277 = transpose_0(_S1272);

#line 154
    Matrix<float, 3, 3>  _S1278 = transpose_0(_S1276.primal_1);

#line 154
    Matrix<float, 3, 3>  _S1279 = mul_3(_S1277, _S1278);

#line 154
    Matrix<float, 3, 3>  _S1280 = mul_3(dpcov_ws_2.primal_1, _S1279);

#line 154
    Matrix<float, 3, 3>  _S1281 = mul_3(_S1272, _S1280);

#line 154
    Matrix<float, 3, 3>  _S1282 = mul_3(_S1276.primal_1, _S1281);

#line 154
    Matrix<float, 3, 3>  _S1283 = mul_3(_S1276.differential_0, _S1281) + mul_3(_S1276.primal_1, mul_3(_S1273, _S1280) + mul_3(_S1272, mul_3(dpcov_ws_2.differential_0, _S1279) + mul_3(dpcov_ws_2.primal_1, mul_3(transpose_0(_S1273), _S1278) + mul_3(_S1277, transpose_0(_S1276.differential_0)))));
    float _S1284 = _S1282.rows[int(0)].x + 0.30000001192092896f;

#line 155
    Matrix<float, 3, 3>  _S1285 = _S1282;

#line 155
    *&(((&_S1285)->rows + (int(0)))->x) = _S1284;

#line 155
    Matrix<float, 3, 3>  _S1286 = _S1283;

#line 155
    *&(((&_S1286)->rows + (int(0)))->x) = _S1283.rows[int(0)].x;

#line 155
    *&(((&_S1285)->rows + (int(1)))->y) = _S1282.rows[int(1)].y + 0.30000001192092896f;

#line 155
    *&(((&_S1286)->rows + (int(1)))->y) = _S1283.rows[int(1)].y;

#line 155
    DiffPair_matrixx3Cfloatx2C2x2C2x3E_0 _S1287 = { makeMatrix<float, 2, 2> (float2 {_S1285.rows[int(0)].x, _S1285.rows[int(0)].y}, float2 {_S1285.rows[int(1)].x, _S1285.rows[int(1)].y}), makeMatrix<float, 2, 2> (float2 {_S1286.rows[int(0)].x, _S1286.rows[int(0)].y}, float2 {_S1286.rows[int(1)].x, _S1286.rows[int(1)].y}) };


    return _S1287;
}


#line 158
__device__ DiffPair_Splat_2D_Vertex_0 s_fwd_project_gaussian_to_camera_0(DiffPair_Gaussian_3D_0 dpg_2, DiffPair_Camera_0 dpcam_11, uint active_sh_18)
{

#line 222
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1288 = { dpg_2.primal_1.xyz_ws_0, dpg_2.differential_0.xyz_ws_0 };

#line 222
    DiffPair_Camera_0 _S1289 = { dpcam_11.primal_1, dpcam_11.differential_0 };
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1290 = s_fwd_project_point_0(_S1288, _S1289);
    if(_S1290.primal_1.z <= 0.20000000298023224f)
    {

#line 225
        float3  _S1291 = make_float3 (0.0f);

#line 225
        float3  _S1292 = make_float3 (0.0f);

#line 225
        Splat_2D_Vertex_0 _S1293 = { _S1291, _S1291, makeMatrix<float, 2, 2> (0.0f) };

#line 225
        Splat_2D_Vertex_0 _S1294 = { _S1292, _S1292, makeMatrix<float, 2, 2> (0.0f) };

#line 225
        DiffPair_Splat_2D_Vertex_0 _S1295 = { _S1293, _S1294 };

#line 225
        return _S1295;
    }

#line 225
    DiffPair_SpherHarmCoeffs_0 _S1296 = { dpg_2.primal_1.sh_coeffs_0, dpg_2.differential_0.sh_coeffs_0 };

#line 225
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1297 = { dpcam_11.primal_1.position_1, dpcam_11.differential_0.position_0 };

    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1298 = s_fwd_compute_color_from_sh_coeffs_0(_S1296, _S1288, _S1297, active_sh_18);

#line 227
    DiffPair_vectorx3Cfloatx2C4x3E_0 _S1299 = { dpg_2.primal_1.rotations_0, dpg_2.differential_0.rotations_0 };

#line 227
    DiffPair_vectorx3Cfloatx2C3x3E_0 _S1300 = { dpg_2.primal_1.scales_0, dpg_2.differential_0.scales_0 };
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S1301 = s_fwd_get_covariance_from_quat_scales_0(_S1299, _S1300);

#line 228
    DiffPair_matrixx3Cfloatx2C3x2C3x3E_0 _S1302 = { _S1301.primal_1, _S1301.differential_0 };
    DiffPair_matrixx3Cfloatx2C2x2C2x3E_0 _S1303 = s_fwd_covariance_3d_to_2d_0(_S1289, _S1288, _S1302);

    Splat_2D_Vertex_0 _S1304 = { _S1290.primal_1, _S1298.primal_1, _S1303.primal_1 };

#line 231
    Splat_2D_Vertex_0 _S1305 = { _S1290.differential_0, _S1298.differential_0, _S1303.differential_0 };

#line 231
    DiffPair_Splat_2D_Vertex_0 _S1306 = { _S1304, _S1305 };

#line 231
    return _S1306;
}


#line 72 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/vertex_shader.slang"
__device__ DiffPair_float_0 s_fwd_compute_det_0(DiffPair_matrixx3Cfloatx2C2x2C2x3E_0 dpM_2)
{

#line 203 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
    DiffPair_float_0 _S1307 = { dpM_2.primal_1.rows[int(0)].x * dpM_2.primal_1.rows[int(1)].y - dpM_2.primal_1.rows[int(0)].y * dpM_2.primal_1.rows[int(1)].x, dpM_2.differential_0.rows[int(0)].x * dpM_2.primal_1.rows[int(1)].y + dpM_2.differential_0.rows[int(1)].y * dpM_2.primal_1.rows[int(0)].x - (dpM_2.differential_0.rows[int(0)].y * dpM_2.primal_1.rows[int(1)].x + dpM_2.differential_0.rows[int(1)].x * dpM_2.primal_1.rows[int(0)].y) };
    return _S1307;
}


#line 79 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/vertex_shader.slang"
__device__ DiffPair_float_0 s_fwd_ndc2pix_0(DiffPair_float_0 dpv_2, int S_3)
{

#line 63 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/utils.slang"
    float _S1308 = float(S_3);

#line 63
    DiffPair_float_0 _S1309 = { ((dpv_2.primal_1 + 1.0f) * _S1308 - 1.0f) * 0.5f, dpv_2.differential_0 * _S1308 * 0.5f };

#line 63
    return _S1309;
}


#line 63
__device__ void s_fwd_vertex_shader_0(DiffTensorView_0 xyz_ws_10, DiffTensorView_0 sh_coeffs_12, DiffTensorView_0 rotations_8, DiffTensorView_0 scales_8, uint active_sh_19, TensorView world_view_transform_6, TensorView proj_mat_6, TensorView cam_pos_4, TensorView out_tiles_touched_3, TensorView out_rect_tile_space_3, TensorView out_radii_3, DiffTensorView_0 out_xyz_vs_3, DiffTensorView_0 out_inv_cov_vs_3, DiffTensorView_0 out_rgb_3, float fovy_6, float fovx_6, uint image_height_3, uint image_width_3, uint grid_height_4, uint grid_width_4, uint tile_height_4, uint tile_width_4)
{

#line 60 "d:/A_study/nerf3dgs/tiny-nerf/slangtorch3dgs/shader/vertex_shader.slang"
    uint g_idx_10 = ((blockIdx)).x * ((blockDim)).x + ((threadIdx)).x;

    if(g_idx_10 >= DiffTensorView_size_0(xyz_ws_10, 0U))
    {

#line 63
        return;
    }
    Camera_0 cam_5 = load_camera_0(world_view_transform_6, proj_mat_6, cam_pos_4, fovy_6, fovx_6, image_height_3, image_width_3);
    DiffPair_Gaussian_3D_0 _S1310 = s_fwd_load_gaussian_0(int(g_idx_10), xyz_ws_10, sh_coeffs_12, rotations_8, scales_8, active_sh_19);

#line 66
    DiffPair_Gaussian_3D_0 _S1311 = { _S1310.primal_1, _S1310.differential_0 };

#line 66
    DiffPair_Camera_0 _S1312 = { cam_5, Camera_x24_syn_dzero_0() };
    DiffPair_Splat_2D_Vertex_0 _S1313 = s_fwd_project_gaussian_to_camera_0(_S1311, _S1312, active_sh_19);
    float _S1314 = _S1313.primal_1.xyz_vs_0.z;

#line 68
    float _S1315 = _S1313.differential_0.xyz_vs_0.z;

#line 68
    if(_S1314 <= 0.20000000298023224f)
    {

#line 69
        return;
    }

#line 69
    DiffPair_matrixx3Cfloatx2C2x2C2x3E_0 _S1316 = { _S1313.primal_1.cov_vs_0, _S1313.differential_0.cov_vs_0 };


    DiffPair_float_0 _S1317 = s_fwd_compute_det_0(_S1316);

    if(_S1317.primal_1 == 0.0f)
    {

#line 75
        return;
    }

#line 76
    float radius_2 = splat_radius_0(_S1313.primal_1.cov_vs_0, _S1317.primal_1);

#line 76
    DiffPair_float_0 _S1318 = { _S1313.primal_1.xyz_vs_0.x, _S1313.differential_0.xyz_vs_0.x };

#line 76
    DiffPair_float_0 _S1319 = { _S1313.primal_1.xyz_vs_0.y, _S1313.differential_0.xyz_vs_0.y };



    rectangle_0 rect_tile_space_3 = get_rectangle_tile_space_0(make_float2 (s_fwd_ndc2pix_0(_S1318, int(image_width_3)).primal_1, s_fwd_ndc2pix_0(_S1319, int(image_height_3)).primal_1), radius_2, grid_height_4, grid_width_4, tile_height_4, tile_width_4);

    int n_tiles_1 = (rect_tile_space_3.max_x_0 - rect_tile_space_3.min_x_0) * (rect_tile_space_3.max_y_0 - rect_tile_space_3.min_y_0);

    if(n_tiles_1 == int(0))
    {

#line 85
        return;
    }

    Matrix<float, 2, 2>  _S1320 = makeMatrix<float, 2, 2> (_S1313.primal_1.cov_vs_0.rows[int(1)].y, - _S1313.primal_1.cov_vs_0.rows[int(0)].y, - _S1313.primal_1.cov_vs_0.rows[int(1)].x, _S1313.primal_1.cov_vs_0.rows[int(0)].x);

#line 88
    Matrix<float, 2, 2>  g_inv_cov_vs_1 = _S1320 / makeMatrix<float, 2, 2> (_S1317.primal_1);

#line 88
    Matrix<float, 2, 2>  s_diff_g_inv_cov_vs_0 = (makeMatrix<float, 2, 2> (_S1313.differential_0.cov_vs_0.rows[int(1)].y, - _S1313.differential_0.cov_vs_0.rows[int(0)].y, - _S1313.differential_0.cov_vs_0.rows[int(1)].x, _S1313.differential_0.cov_vs_0.rows[int(0)].x) * makeMatrix<float, 2, 2> (_S1317.primal_1) - _S1320 * makeMatrix<float, 2, 2> (_S1317.differential_0)) / makeMatrix<float, 2, 2> (_S1317.primal_1 * _S1317.primal_1);

    (out_radii_3).store<int>((g_idx_10), (int(uint(radius_2))));
    (out_tiles_touched_3).store<int>((g_idx_10), (n_tiles_1));
    uint2  _S1321 = make_uint2 (g_idx_10, 0U);

#line 92
    (out_rect_tile_space_3).store<int>((g_idx_10), (0U), (rect_tile_space_3.min_x_0));
    uint2  _S1322 = make_uint2 (g_idx_10, 1U);

#line 93
    (out_rect_tile_space_3).store<int>((g_idx_10), (1U), (rect_tile_space_3.min_y_0));
    uint2  _S1323 = make_uint2 (g_idx_10, 2U);

#line 94
    (out_rect_tile_space_3).store<int>((g_idx_10), (2U), (rect_tile_space_3.max_x_0));
    (out_rect_tile_space_3).store<int>((g_idx_10), (3U), (rect_tile_space_3.max_y_0));

    DiffTensorView_storeOnce_forward_0(out_xyz_vs_3, _S1321, _S1318);
    DiffTensorView_storeOnce_forward_0(out_xyz_vs_3, _S1322, _S1319);

#line 98
    DiffPair_float_0 _S1324 = { _S1314, _S1315 };
    DiffTensorView_storeOnce_forward_0(out_xyz_vs_3, _S1323, _S1324);

#line 99
    DiffPair_float_0 _S1325 = { g_inv_cov_vs_1.rows[int(0)].x, s_diff_g_inv_cov_vs_0.rows[int(0)].x };
    DiffTensorView_storeOnce_forward_1(out_inv_cov_vs_3, make_uint3 (g_idx_10, 0U, 0U), _S1325);

#line 100
    DiffPair_float_0 _S1326 = { g_inv_cov_vs_1.rows[int(0)].y, s_diff_g_inv_cov_vs_0.rows[int(0)].y };
    DiffTensorView_storeOnce_forward_1(out_inv_cov_vs_3, make_uint3 (g_idx_10, 0U, 1U), _S1326);

#line 101
    DiffPair_float_0 _S1327 = { g_inv_cov_vs_1.rows[int(1)].x, s_diff_g_inv_cov_vs_0.rows[int(1)].x };
    DiffTensorView_storeOnce_forward_1(out_inv_cov_vs_3, make_uint3 (g_idx_10, 1U, 0U), _S1327);

#line 102
    DiffPair_float_0 _S1328 = { g_inv_cov_vs_1.rows[int(1)].y, s_diff_g_inv_cov_vs_0.rows[int(1)].y };
    DiffTensorView_storeOnce_forward_1(out_inv_cov_vs_3, make_uint3 (g_idx_10, 1U, 1U), _S1328);

#line 103
    DiffPair_float_0 _S1329 = { _S1313.primal_1.rgb_0.x, _S1313.differential_0.rgb_0.x };
    DiffTensorView_storeOnce_forward_0(out_rgb_3, _S1321, _S1329);

#line 104
    DiffPair_float_0 _S1330 = { _S1313.primal_1.rgb_0.y, _S1313.differential_0.rgb_0.y };
    DiffTensorView_storeOnce_forward_0(out_rgb_3, _S1322, _S1330);

#line 105
    DiffPair_float_0 _S1331 = { _S1313.primal_1.rgb_0.z, _S1313.differential_0.rgb_0.z };
    DiffTensorView_storeOnce_forward_0(out_rgb_3, _S1323, _S1331);
    return;
}


#line 107
extern "C" {
__global__ void __kernel__vertex_shader_fwd_diff(DiffTensorView_0 xyz_ws_11, DiffTensorView_0 sh_coeffs_13, DiffTensorView_0 rotations_9, DiffTensorView_0 scales_9, uint active_sh_20, TensorView world_view_transform_7, TensorView proj_mat_7, TensorView cam_pos_5, TensorView out_tiles_touched_4, TensorView out_rect_tile_space_4, TensorView out_radii_4, DiffTensorView_0 out_xyz_vs_4, DiffTensorView_0 out_inv_cov_vs_4, DiffTensorView_0 out_rgb_4, float fovy_7, float fovx_7, uint image_height_4, uint image_width_4, uint grid_height_5, uint grid_width_5, uint tile_height_5, uint tile_width_5)
{

#line 107
    s_fwd_vertex_shader_0(xyz_ws_11, sh_coeffs_13, rotations_9, scales_9, active_sh_20, world_view_transform_7, proj_mat_7, cam_pos_5, out_tiles_touched_4, out_rect_tile_space_4, out_radii_4, out_xyz_vs_4, out_inv_cov_vs_4, out_rgb_4, fovy_7, fovx_7, image_height_4, image_width_4, grid_height_5, grid_width_5, tile_height_5, tile_width_5);

#line 107
    return;
}

}

#line 37
__global__ void __kernel__vertex_shader(DiffTensorView_0 xyz_ws_12, DiffTensorView_0 sh_coeffs_14, DiffTensorView_0 rotations_10, DiffTensorView_0 scales_10, uint active_sh_21, TensorView world_view_transform_8, TensorView proj_mat_8, TensorView cam_pos_6, TensorView out_tiles_touched_5, TensorView out_rect_tile_space_5, TensorView out_radii_5, DiffTensorView_0 out_xyz_vs_5, DiffTensorView_0 out_inv_cov_vs_5, DiffTensorView_0 out_rgb_5, float fovy_8, float fovx_8, uint image_height_5, uint image_width_5, uint grid_height_6, uint grid_width_6, uint tile_height_6, uint tile_width_6)
{

#line 60
    uint g_idx_11 = ((blockIdx)).x * ((blockDim)).x + ((threadIdx)).x;

    if(g_idx_11 >= DiffTensorView_size_0(xyz_ws_12, 0U))
    {

#line 63
        return;
    }
    Camera_0 cam_6 = load_camera_0(world_view_transform_8, proj_mat_8, cam_pos_6, fovy_8, fovx_8, image_height_5, image_width_5);

    Splat_2D_Vertex_0 splat_0 = project_gaussian_to_camera_0(load_gaussian_0(int(g_idx_11), xyz_ws_12, sh_coeffs_14, rotations_10, scales_10, active_sh_21), cam_6, active_sh_21);
    float _S1332 = splat_0.xyz_vs_0.z;

#line 68
    if(_S1332 <= 0.20000000298023224f)
    {

#line 69
        return;
    }

    float det_1 = compute_det_0(splat_0.cov_vs_0);

    if(det_1 == 0.0f)
    {

#line 75
        return;
    }

#line 76
    float radius_3 = splat_radius_0(splat_0.cov_vs_0, det_1);


    float _S1333 = splat_0.xyz_vs_0.x;

#line 79
    float _S1334 = splat_0.xyz_vs_0.y;
    rectangle_0 rect_tile_space_4 = get_rectangle_tile_space_0(make_float2 (ndc2pix_0(_S1333, int(image_width_5)), ndc2pix_0(_S1334, int(image_height_5))), radius_3, grid_height_6, grid_width_6, tile_height_6, tile_width_6);

    int n_tiles_2 = (rect_tile_space_4.max_x_0 - rect_tile_space_4.min_x_0) * (rect_tile_space_4.max_y_0 - rect_tile_space_4.min_y_0);

    if(n_tiles_2 == int(0))
    {

#line 85
        return;
    }

    Matrix<float, 2, 2>  g_inv_cov_vs_2 = makeMatrix<float, 2, 2> (splat_0.cov_vs_0.rows[int(1)].y, - splat_0.cov_vs_0.rows[int(0)].y, - splat_0.cov_vs_0.rows[int(1)].x, splat_0.cov_vs_0.rows[int(0)].x) / makeMatrix<float, 2, 2> (det_1);

    (out_radii_5).store<int>((g_idx_11), (int(uint(radius_3))));
    (out_tiles_touched_5).store<int>((g_idx_11), (n_tiles_2));
    (out_rect_tile_space_5).store<int>((g_idx_11), (0U), (rect_tile_space_4.min_x_0));
    (out_rect_tile_space_5).store<int>((g_idx_11), (1U), (rect_tile_space_4.min_y_0));
    (out_rect_tile_space_5).store<int>((g_idx_11), (2U), (rect_tile_space_4.max_x_0));
    (out_rect_tile_space_5).store<int>((g_idx_11), (3U), (rect_tile_space_4.max_y_0));

    uint2  _S1335 = make_uint2 (g_idx_11, 0U);

#line 97
    DiffTensorView_storeOnce_0(out_xyz_vs_5, _S1335, _S1333);
    uint2  _S1336 = make_uint2 (g_idx_11, 1U);

#line 98
    DiffTensorView_storeOnce_0(out_xyz_vs_5, _S1336, _S1334);
    uint2  _S1337 = make_uint2 (g_idx_11, 2U);

#line 99
    DiffTensorView_storeOnce_0(out_xyz_vs_5, _S1337, _S1332);
    DiffTensorView_storeOnce_1(out_inv_cov_vs_5, make_uint3 (g_idx_11, 0U, 0U), g_inv_cov_vs_2.rows[int(0)].x);
    DiffTensorView_storeOnce_1(out_inv_cov_vs_5, make_uint3 (g_idx_11, 0U, 1U), g_inv_cov_vs_2.rows[int(0)].y);
    DiffTensorView_storeOnce_1(out_inv_cov_vs_5, make_uint3 (g_idx_11, 1U, 0U), g_inv_cov_vs_2.rows[int(1)].x);
    DiffTensorView_storeOnce_1(out_inv_cov_vs_5, make_uint3 (g_idx_11, 1U, 1U), g_inv_cov_vs_2.rows[int(1)].y);
    DiffTensorView_storeOnce_0(out_rgb_5, _S1335, splat_0.rgb_0.x);
    DiffTensorView_storeOnce_0(out_rgb_5, _S1336, splat_0.rgb_0.y);
    DiffTensorView_storeOnce_0(out_rgb_5, _S1337, splat_0.rgb_0.z);
    return;
}

